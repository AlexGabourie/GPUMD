#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.h"
#include "lj1.h"


// best block size here: 128
#define BLOCK_SIZE_LJ1 128


// apply the mininum image convention
static __device__ void dev_apply_mic
(
    int pbc_x, int pbc_y, int pbc_z, real *x12, real *y12, real *z12,
    real lx, real ly, real lz
)
{
    if      (*x12 < - lx * HALF) {*x12 += lx * pbc_x;}
    else if (*x12 > + lx * HALF) {*x12 -= lx * pbc_x;}
    if      (*y12 < - ly * HALF) {*y12 += ly * pbc_y;}
    else if (*y12 > + ly * HALF) {*y12 -= ly * pbc_y;}
    if      (*z12 < - lz * HALF) {*z12 += lz * pbc_z;}
    else if (*z12 > + lz * HALF) {*z12 -= lz * pbc_z;}
}


// get U_ij and (d U_ij / d r_ij) / r_ij
static __device__ void find_p2_and_f2(LJ lj, real d12sq, real &p2, real &f2)
{
    real d12inv2 = ONE / d12sq;
    real d12inv6 = d12inv2 * d12inv2 * d12inv2;  
    f2 = (lj.s6e24 * d12inv6 - lj.s12e24 * TWO * d12inv6 * d12inv6) * d12inv2; 
    p2 = lj.s12e4 * d12inv6 * d12inv6 - lj.s6e4 * d12inv6;  
}


// force evaluation kernel for the LJ potential
template <int cal_p, int cal_j, int cal_q>
static __global__ void gpu_find_force
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z, LJ lj,
    int *g_neighbor_number, int *g_neighbor_list,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
#else
    real *g_x,  real *g_y,  real *g_z, real *g_vx, real *g_vy, real *g_vz,
#endif
    real *g_box, real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz, real *g_potential, 
    real *g_h, int *g_label, int *g_fv_index, real *g_fv 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x; // particle index

    __shared__ real s_fx[BLOCK_SIZE_LJ1];
    __shared__ real s_fy[BLOCK_SIZE_LJ1];
    __shared__ real s_fz[BLOCK_SIZE_LJ1];
    // if cal_p, then s1~s4 = px, py, pz, U; if cal_j, then s1~s5 = j1~j5
    __shared__ real s1[BLOCK_SIZE_LJ1];
    __shared__ real s2[BLOCK_SIZE_LJ1];
    __shared__ real s3[BLOCK_SIZE_LJ1];
    __shared__ real s4[BLOCK_SIZE_LJ1];
    __shared__ real s5[BLOCK_SIZE_LJ1];

    s_fx[threadIdx.x] = ZERO; 
    s_fy[threadIdx.x] = ZERO; 
    s_fz[threadIdx.x] = ZERO;  
    s1[threadIdx.x] = ZERO; 
    s2[threadIdx.x] = ZERO; 
    s3[threadIdx.x] = ZERO;
    s4[threadIdx.x] = ZERO;
    s5[threadIdx.x] = ZERO;

    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real vx1 = LDG(g_vx, n1); 
        real vy1 = LDG(g_vy, n1); 
        real vz1 = LDG(g_vz, n1);
        real lx = g_box[0]; 
        real ly = g_box[1]; 
        real lz = g_box[2];

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, &x12, &y12, &z12, lx, ly, lz);
            real d12sq = x12 * x12 + y12 * y12 + z12 * z12;
            if (d12sq >= lj.cutoff_square) {continue;}

            real p2, f2;
            find_p2_and_f2(lj, d12sq, p2, f2);

            // treat two-body potential in the same way as many-body potential
            real f12x = f2 * x12 * HALF; 
            real f12y = f2 * y12 * HALF; 
            real f12z = f2 * z12 * HALF; 
            real f21x = -f12x; 
            real f21y = -f12y; 
            real f21z = -f12z; 
       
            // accumulate force
            s_fx[threadIdx.x] += f12x - f21x; 
            s_fy[threadIdx.x] += f12y - f21y; 
            s_fz[threadIdx.x] += f12z - f21z; 
            
            // accumulate potential energy and virial
            if (cal_p) 
            {
                s4[threadIdx.x] += p2 * HALF; // two-body potential
                s1[threadIdx.x] -= x12 * (f12x - f21x) * HALF; 
                s2[threadIdx.x] -= y12 * (f12y - f21y) * HALF; 
                s3[threadIdx.x] -= z12 * (f12z - f21z) * HALF;
            }
            
            // heat current (EMD)
            if (cal_j) 
            {
                s1[threadIdx.x] += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s2[threadIdx.x] += (f21z * vz1) * x12;               // x-out
                s3[threadIdx.x] += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s4[threadIdx.x] += (f21z * vz1) * y12;               // y-out
                s5[threadIdx.x] += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            } 

            // heat across some section (NEMD)
            if (cal_q) 
            {
                int index_12 = g_fv_index[n1] * 12;
                if (index_12 >= 0 && g_fv_index[n1 + number_of_particles] == n2)
                {
                    g_fv[index_12 + 0]  = f12x;
                    g_fv[index_12 + 1]  = f12y;
                    g_fv[index_12 + 2]  = f12z;
                    g_fv[index_12 + 3]  = f21x;
                    g_fv[index_12 + 4]  = f21y;
                    g_fv[index_12 + 5]  = f21z;
                    g_fv[index_12 + 6]  = vx1;
                    g_fv[index_12 + 7]  = vy1;
                    g_fv[index_12 + 8]  = vz1;
                    g_fv[index_12 + 9]  = LDG(g_vx, n2);
                    g_fv[index_12 + 10] = LDG(g_vy, n2);
                    g_fv[index_12 + 11] = LDG(g_vz, n2);
                }  
            }
        }

        // save force
        g_fx[n1] = s_fx[threadIdx.x]; 
        g_fy[n1] = s_fy[threadIdx.x]; 
        g_fz[n1] = s_fz[threadIdx.x]; 

        // save stress and potential
        if (cal_p) 
        {
            g_sx[n1] = s1[threadIdx.x]; 
            g_sy[n1] = s2[threadIdx.x]; 
            g_sz[n1] = s3[threadIdx.x];
            g_potential[n1] = s4[threadIdx.x];
        }

        // save heat current
        if (cal_j) 
        {
            g_h[n1 + 0 * number_of_particles] = s1[threadIdx.x];
            g_h[n1 + 1 * number_of_particles] = s2[threadIdx.x];
            g_h[n1 + 2 * number_of_particles] = s3[threadIdx.x];
            g_h[n1 + 3 * number_of_particles] = s4[threadIdx.x];
            g_h[n1 + 4 * number_of_particles] = s5[threadIdx.x];
        }
    }
}    
 

// Find force and related quantities for the LJ1 potential (A wrapper)
void gpu_find_force_lj1(Parameters *para, LJ lj1, GPU_Data *gpu_data)
{
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE_LJ1 + 1;
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
#ifdef FIXED_NL
    int *NN = gpu_data->NN; 
    int *NL = gpu_data->NL;
#else
    int *NN = gpu_data->NN_local; 
    int *NL = gpu_data->NL_local;
#endif
    real *x = gpu_data->x; 
    real *y = gpu_data->y; 
    real *z = gpu_data->z;
    real *vx = gpu_data->vx; 
    real *vy = gpu_data->vy; 
    real *vz = gpu_data->vz;
    real *fx = gpu_data->fx; 
    real *fy = gpu_data->fy; 
    real *fz = gpu_data->fz;
    real *box = gpu_data->box_length;
    real *sx = gpu_data->virial_per_atom_x; 
    real *sy = gpu_data->virial_per_atom_y; 
    real *sz = gpu_data->virial_per_atom_z; 
    real *pe = gpu_data->potential_per_atom;
    real *h = gpu_data->heat_per_atom; 
    
    int *label = gpu_data->label;
    int *fv_index = gpu_data->fv_index;
    real *fv = gpu_data->fv;
           
    if (para->hac.compute)    
    {
        gpu_find_force<0, 1, 0><<<grid_size, BLOCK_SIZE_LJ1>>>
        (
            N, pbc_x, pbc_y, pbc_z, lj1, NN, NL, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }
    else if (para->shc.compute)
    {
        gpu_find_force<0, 0, 1><<<grid_size, BLOCK_SIZE_LJ1>>>
        (
            N, pbc_x, pbc_y, pbc_z, lj1, NN, NL, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }
    else
    {
        gpu_find_force<1, 0, 0><<<grid_size, BLOCK_SIZE_LJ1>>>
        (
            N, pbc_x, pbc_y, pbc_z, lj1, NN, NL, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }

    #ifdef DEGUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif
}


