#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The Berendsen thermostat:
[1] H. J. C. Berendsen et al. J. Chem. Phys. 81, 3684 (1984).
------------------------------------------------------------------------------*/


#include "ensemble_ber.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128


Ensemble_BER::Ensemble_BER(int t, real T, real Tc)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
}


Ensemble_BER::Ensemble_BER
(
    int t, real T, real Tc, real px, real py, real pz, real pc,
    int dx, int dy, int dz, real rate
)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
    pressure_x = px;
    pressure_y = py;
    pressure_z = pz;
    pressure_coupling = pc;
    deform_x = dx;
    deform_y = dy;
    deform_z = dz;
    deform_rate = rate;
}


Ensemble_BER::~Ensemble_BER(void)
{
    // nothing now
}


static __global__ void gpu_berendsen_temperature
(
    int N, real temperature, real coupling, real *g_prop, 
    real *g_vx, real *g_vy, real *g_vz
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {  
        real factor = sqrt(ONE + coupling * (temperature / g_prop[0] - ONE)); 
        g_vx[i] *= factor; 
        g_vy[i] *= factor; 
        g_vz[i] *= factor;
    }
}


static __global__ void gpu_berendsen_pressure
(
    int deform_x, int deform_y, int deform_z, real deform_rate,
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z,
    real p0x, real p0y, real p0z, real p_coupling, 
    real *g_prop, real *g_box_length, real *g_x, real *g_y, real *g_z
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < number_of_particles)
    {
        if (deform_x)
        {
            real scale_factor = g_box_length[0];
            scale_factor = (scale_factor + deform_rate) / scale_factor;
            g_x[i] *= scale_factor;
            if (i == 0) { g_box_length[0] *= scale_factor; }
        }
        else if (pbc_x == 1)
        {
            real scale_factor = ONE - p_coupling * (p0x - g_prop[2]);
            g_x[i] *= scale_factor;
            if (i == 0) { g_box_length[0] *= scale_factor; }
        }
        if (deform_y)
        {
            real scale_factor = g_box_length[1];
            scale_factor = (scale_factor + deform_rate) / scale_factor;
            g_y[i] *= scale_factor;
            if (i == 1) { g_box_length[1] *= scale_factor; }
        }
        else if (pbc_y == 1)
        {
            real scale_factor = ONE - p_coupling * (p0y - g_prop[3]);
            g_y[i] *= scale_factor;
            if (i == 1) { g_box_length[1] *= scale_factor; }
        }
        if (deform_z)
        {
            real scale_factor = g_box_length[2];
            scale_factor = (scale_factor + deform_rate) / scale_factor;
            g_z[i] *= scale_factor;
            if (i == 2) { g_box_length[2] *= scale_factor; }
        }
        else if (pbc_z == 1)
        {
            real scale_factor = ONE - p_coupling * (p0z - g_prop[4]);
            g_z[i] *= scale_factor;
            if (i == 2) { g_box_length[2] *= scale_factor; }
        }
    }
}


void Ensemble_BER::compute
(Atom *atom, Force *force, Measure* measure)
{
    int grid_size = (atom->N - 1) / BLOCK_SIZE + 1;
    velocity_verlet(atom, force, measure);
    find_thermo(atom);
    gpu_berendsen_temperature<<<grid_size, BLOCK_SIZE>>>
    (
        atom->N, temperature, temperature_coupling, atom->thermo,
        atom->vx, atom->vy, atom->vz
    );
    CUDA_CHECK_KERNEL
    if (type == 11)
    {
        gpu_berendsen_pressure<<<grid_size, BLOCK_SIZE>>>
        (
            deform_x, deform_y, deform_z, deform_rate,
            atom->N, atom->pbc_x, atom->pbc_y, atom->pbc_z, pressure_x, 
            pressure_y, pressure_z, pressure_coupling, atom->thermo,
            atom->box_length, atom->x, atom->y, atom->z
        );
        CUDA_CHECK_KERNEL
    }
}


