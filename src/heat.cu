#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "heat.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "memory.cuh"

#define FILE_NAME_LENGTH      200
#define DIM                   3




// allocate memory used for recording group temperatures 
// and energies of the heat source and sink
void Heat::preprocess_heat(Parameters *para, CPU_Data *cpu_data)
{
    if (sample)
    {
        // The last 2 data are the energy changes of the source and sink
        int num = (para->number_of_groups + 2) 
                * (para->number_of_steps / sample_interval);
        MY_MALLOC(cpu_data->group_temp, real, num);
    }
}




static __device__ void warp_reduce(volatile real *s, int t) 
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}




// sample block temperature (kernel)
static __global__ void find_group_temp
(
    int  *g_group_size,
    int  *g_group_size_sum,
    int  *g_group_contents,
    real *g_mass,
    real *g_vx,
    real *g_vy,
    real *g_vz,
    real *g_group_temp
)
{
    // <<<number_of_groups, 256>>> (one CUDA block for one group of atoms)

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int group_size = g_group_size[bid];
    int offset = g_group_size_sum[bid];
    int number_of_patches = (group_size - 1) / 256 + 1;
    __shared__ real s_ke[256];
    s_ke[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; patch++)
    {
        int k = tid + patch * 256;
        if (k < group_size)
        {
            int n = g_group_contents[offset + k]; // particle index
            real vx = g_vx[n];
            real vy = g_vy[n];
            real vz = g_vz[n];
            s_ke[tid] += g_mass[n] * (vx * vx + vy * vy + vz * vz);
        }
    }
    __syncthreads();

    if (tid <  128) { s_ke[tid] += s_ke[tid + 128]; }  __syncthreads();
    if (tid <   64) { s_ke[tid] += s_ke[tid + 64];  }  __syncthreads();
    if (tid <   32) { warp_reduce(s_ke, tid);       }  
    if (tid ==   0) {g_group_temp[bid] = s_ke[0] / (DIM * K_B * group_size);}
}




// sample block temperature (wrapper)
void Heat::sample_block_temperature
(
    int step, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, 
    Integrate *integrate
)
{
    if (sample)
    {
        if (step % sample_interval == 0)
        {
            int Ng = para->number_of_groups;
            int offset = (step / sample_interval) * (Ng + 2);
      
            // block temperatures
            real *temp_gpu;
            CHECK(hipMalloc((void**)&temp_gpu, sizeof(real) * Ng));
            int  *group_size = gpu_data->group_size;
            int  *group_size_sum = gpu_data->group_size_sum;
            int  *group_contents = gpu_data->group_contents;
            real *mass = gpu_data->mass;
            real *vx = gpu_data->vx;
            real *vy = gpu_data->vy;
            real *vz = gpu_data->vz;
            find_group_temp<<<Ng, 256>>>
            (
                group_size, group_size_sum, group_contents, 
                mass, vx, vy, vz, temp_gpu
            );
            #ifdef DEBUG
                CHECK(hipDeviceSynchronize());
                CHECK(hipGetLastError());
            #endif
            CHECK(hipMemcpy(cpu_data->group_temp+offset, temp_gpu, 
                sizeof(real)*Ng, hipMemcpyDeviceToHost));
            CHECK(hipFree(temp_gpu));

            // energies of the heat source and sink
            cpu_data->group_temp[offset + Ng]     
                = integrate->ensemble->energy_transferred[0];
            cpu_data->group_temp[offset + Ng + 1] 
                = integrate->ensemble->energy_transferred[1];
        }
    }
}




// Output block temperatures and energies of the heat source and sink; 
// free the used memory
void Heat::postprocess_heat
(char *input_dir, Parameters *para, CPU_Data *cpu_data, Integrate *integrate)
{
    if (sample)
    {
        int Nt = para->number_of_steps / sample_interval;
        int Ng = para->number_of_groups;
        char file_temperature[FILE_NAME_LENGTH];
        strcpy(file_temperature, input_dir);
        strcat(file_temperature, "/temperature.out");
        FILE *fid = fopen(file_temperature, "a");
        for (int nt = 0; nt < Nt; nt++)
        {
            int offset = nt * (Ng + 2);
            int number_of_data = Ng + 2;
            for (int k = 0; k < number_of_data; k++) 
            {
                fprintf(fid, "%15.6e", cpu_data->group_temp[offset + k]);
            }
            fprintf(fid, "\n");
        }
        fflush(fid);
        MY_FREE(cpu_data->group_temp); // allocated in preprocess_heat
    }
}




