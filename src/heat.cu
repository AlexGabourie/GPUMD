#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "heat.cuh"




// allocate memory used for recording group temperatures 
// and energies of the heat source and sink
void preprocess_heat(Parameters *para, CPU_Data *cpu_data)
{
    if (para->heat.sample)
    {
        // The last 2 data are the energy changes of the source and sink
        int num = (para->number_of_groups + 2) 
                * (para->number_of_steps / para->heat.sample_interval);
        MY_MALLOC(cpu_data->group_temp, real, num);
    }
}




// Sample block temperatures
static __device__ void warp_reduce(volatile real *s, int t) 
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}




// sample block temperature (kernel)
static __global__ void find_group_temp
(
    int  *g_group_size,
    int  *g_group_size_sum,
    real *g_mass,
    real *g_vx,
    real *g_vy,
    real *g_vz,
    real *g_group_temp
)
{
    // <<<number_of_groups, 256>>> (one CUDA block for one group of atoms)

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int group_size = g_group_size[bid];
    int offset = g_group_size_sum[bid];
    int number_of_patches = (group_size - 1) / 256 + 1;
    __shared__ real s_ke[256];
    s_ke[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; patch++)
    {
        int k = tid + patch * 256;
        if (k < group_size)
        {
            int n = offset + k; // particle index
            real vx = g_vx[n];
            real vy = g_vy[n];
            real vz = g_vz[n];
            s_ke[tid] += g_mass[n] * (vx * vx + vy * vy + vz * vz);
        }
    }
    __syncthreads();

    if (tid <  128) { s_ke[tid] += s_ke[tid + 128]; }  __syncthreads();
    if (tid <   64) { s_ke[tid] += s_ke[tid + 64];  }  __syncthreads();
    if (tid <   32) { warp_reduce(s_ke, tid);       }  
    if (tid ==   0) {g_group_temp[bid] = s_ke[0] / (DIM * K_B * group_size);}
}




// sample block temperature (wrapper)
void sample_block_temperature
(int step, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data)
{
    if (para->heat.sample)
    {
        if (step % para->heat.sample_interval == 0)
        {
            int Ng = para->number_of_groups;
            int offset = (step / para->heat.sample_interval) * (Ng + 2);
      
            // block temperatures
            real *temp_gpu;
            CHECK(hipMalloc((void**)&temp_gpu, sizeof(real) * Ng));
            int  *group_size = gpu_data->group_size;
            int  *group_size_sum = gpu_data->group_size_sum;
            real *mass = gpu_data->mass;
            real *vx = gpu_data->vx;
            real *vy = gpu_data->vy;
            real *vz = gpu_data->vz;
            find_group_temp<<<Ng, 256>>>
            (group_size, group_size_sum, mass, vx, vy, vz, temp_gpu);
            #ifdef DEBUG
                CHECK(hipDeviceSynchronize());
                CHECK(hipGetLastError());
            #endif
            CHECK(hipMemcpy(cpu_data->group_temp+offset, temp_gpu, 
                sizeof(real)*Ng, hipMemcpyDeviceToHost));
            CHECK(hipFree(temp_gpu));

            // energies of the heat source and sink
            real kT1 = K_B * (para->temperature + para->heat.delta_temperature); 
            real kT2 = K_B * (para->temperature - para->heat.delta_temperature); 
            real dN1 = (real) DIM * cpu_data->group_size[para->heat.source];
            real dN2 = (real) DIM * cpu_data->group_size[para->heat.sink];
            real energy_nhc1 = kT1 * dN1 * para->pos_nhc1[0];
            real energy_nhc2 = kT2 * dN2 * para->pos_nhc2[0];
            for (int m = 1; m < NOSE_HOOVER_CHAIN_LENGTH; m++)
            {
                energy_nhc1 += kT1 * para->pos_nhc1[m];
                energy_nhc2 += kT2 * para->pos_nhc2[m];
            }
            for (int m = 0; m < NOSE_HOOVER_CHAIN_LENGTH; m++)
            { 
                energy_nhc1 += HALF * para->vel_nhc1[m] * para->vel_nhc1[m] 
                             / para->mas_nhc1[m];
                energy_nhc2 += HALF * para->vel_nhc2[m] * para->vel_nhc2[m] 
                             / para->mas_nhc2[m];
            }
            cpu_data->group_temp[offset + Ng]     = energy_nhc1;
            cpu_data->group_temp[offset + Ng + 1] = energy_nhc2;
        }
    }
}


// Output block temperatures and energies of the heat source and sink; 
// free the used memory
void postprocess_heat(Files *files, Parameters *para, CPU_Data *cpu_data)
{
    if (para->heat.sample)
    {
        int Nt = para->number_of_steps / para->heat.sample_interval;
        int Ng = para->number_of_groups;
        FILE *fid = fopen(files->temperature, "a");
        for (int nt = 0; nt < Nt; nt++)
        {
            int offset = nt * (Ng + 2);
            int number_of_data = (para->ensemble == 4) ? (Ng + 2) : Ng;
            for (int k = 0; k < number_of_data; k++) 
            {
                fprintf(fid, "%15.6e", cpu_data->group_temp[offset + k]);
            }
            fprintf(fid, "\n");
        }
        fflush(fid);
        MY_FREE(cpu_data->group_temp); // allocated in preprocess_heat
    }
}

