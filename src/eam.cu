#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The EAM potential. Currently two analytical versions:
[1] X. W. Zhou et al. Phys. Rev. B 69, 144113 (2004).
[2] X. D. Dai et al. JPCM 18, 4527 (2006).
------------------------------------------------------------------------------*/


#include "eam.cuh"
#include "mic.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"
#define BLOCK_SIZE_FORCE 64
#define FIND_FORCE_EAM_STEP2(A)                                                \
    find_force_eam_step2<A><<<grid_size, BLOCK_SIZE_FORCE>>>                   \
    (                                                                          \
        eam2004zhou, eam2006dai, atom->N, N1, N2, atom->box,                   \
        atom->NN_local,                                                        \
        atom->NL_local, eam_data.Fp, atom->x, atom->y, atom->z, atom->vx,      \
        atom->vy, atom->vz, atom->fx, atom->fy, atom->fz,                      \
        atom->virial_per_atom, atom->potential_per_atom                        \
    ) 


EAM::EAM(FILE *fid, Atom* atom, char *name)
{

    if (strcmp(name, "eam_zhou_2004") == 0)  initialize_eam2004zhou(fid);
    if (strcmp(name, "eam_dai_2006") == 0)    initialize_eam2006dai(fid);

    // memory for the derivative of the density functional 
    CHECK(hipMalloc((void**)&eam_data.Fp, sizeof(double) * atom->N));
}


void EAM::initialize_eam2004zhou(FILE *fid)
{
    printf("Use the EAM-type potential in the following reference:\n");
    printf("    X. W. Zhou et al., PRB 69, 144113 (2004).\n");
    potential_model = 0;

    double x[21];
    for (int n = 0; n < 21; n++)
    {
        int count = fscanf(fid, "%lf", &x[n]);
        PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
    }
    eam2004zhou.re     = x[0];
    eam2004zhou.fe     = x[1];
    eam2004zhou.rho_e  = x[2];
    eam2004zhou.rho_s  = x[3];
    eam2004zhou.alpha  = x[4];
    eam2004zhou.beta   = x[5];
    eam2004zhou.A      = x[6];
    eam2004zhou.B      = x[7];
    eam2004zhou.kappa  = x[8];
    eam2004zhou.lambda = x[9];
    eam2004zhou.Fn0    = x[10];
    eam2004zhou.Fn1    = x[11];
    eam2004zhou.Fn2    = x[12];
    eam2004zhou.Fn3    = x[13];
    eam2004zhou.F0     = x[14];
    eam2004zhou.F1     = x[15];
    eam2004zhou.F2     = x[16];
    eam2004zhou.F3     = x[17];
    eam2004zhou.eta    = x[18];
    eam2004zhou.Fe     = x[19];
    eam2004zhou.rc     = x[20];
    eam2004zhou.rho_n  = eam2004zhou.rho_e * 0.85;
    eam2004zhou.rho_0  = eam2004zhou.rho_e * 1.15;
    rc                 = eam2004zhou.rc;
}


void EAM::initialize_eam2006dai(FILE *fid)
{
    printf("Use the EAM-type potential in the following reference:\n");
    printf("    X. D. Dai et al., JPCM 18, 4527 (2006).\n");
    potential_model = 1;

    double x[9];
    for (int n = 0; n < 9; n++)
    {
        int count = fscanf(fid, "%lf", &x[n]);
        PRINT_SCANF_ERROR(count, 1, "Reading error for EAM potential.");
    }
    eam2006dai.A  = x[0];
    eam2006dai.d  = x[1];
    eam2006dai.c  = x[2];
    eam2006dai.c0 = x[3];
    eam2006dai.c1 = x[4];
    eam2006dai.c2 = x[5];
    eam2006dai.c3 = x[6];
    eam2006dai.c4 = x[7];
    eam2006dai.B  = x[8];
    eam2006dai.rc = (eam2006dai.c>eam2006dai.d) ? eam2006dai.c : eam2006dai.d;
    rc            = eam2006dai.rc;
}


EAM::~EAM(void)
{
    CHECK(hipFree(eam_data.Fp));
}


// pair function (phi and phip have been intentionally halved here)
static __device__ void find_phi
(EAM2004Zhou eam, double d12, double &phi, double &phip)
{
    double r_ratio = d12 / eam.re;
    double tmp1 = (r_ratio - eam.kappa) * (r_ratio - eam.kappa); // 2
    tmp1 *= tmp1; // 4
    tmp1 *= tmp1 * tmp1 * tmp1 * tmp1; // 20
    double tmp2 = (r_ratio - eam.lambda) * (r_ratio - eam.lambda); // 2
    tmp2 *= tmp2; // 4
    tmp2 *= tmp2 * tmp2 * tmp2 * tmp2; // 20    
    double phi1 = HALF * eam.A * exp(-eam.alpha * (r_ratio - ONE)) / (ONE + tmp1);
    double phi2 = HALF * eam.B * exp( -eam.beta * (r_ratio - ONE)) / (ONE + tmp2);
    phi = phi1 - phi2;
    phip = (phi2/eam.re)*(eam.beta+20.0*tmp2/(r_ratio-eam.lambda)/(ONE+tmp2))
         - (phi1/eam.re)*(eam.alpha+20.0*tmp1/(r_ratio-eam.kappa)/(ONE+tmp1));
}


// density function f(r)
static __device__ void find_f(EAM2004Zhou eam, double d12, double &f)
{
    double r_ratio = d12 / eam.re;
    double tmp = (r_ratio - eam.lambda) * (r_ratio - eam.lambda); // 2
    tmp *= tmp; // 4
    tmp *= tmp * tmp * tmp * tmp; // 20  
    f = eam.fe * exp(-eam.beta * (r_ratio - ONE)) / (ONE + tmp);
}


// derivative of the density function f'(r)
static __device__ void find_fp(EAM2004Zhou eam, double d12, double &fp)
{
    double r_ratio = d12 / eam.re; 
    double tmp = (r_ratio - eam.lambda) * (r_ratio - eam.lambda); // 2
    tmp *= tmp; // 4
    tmp *= tmp * tmp * tmp * tmp; // 20  
    double f = eam.fe * exp(-eam.beta * (r_ratio - ONE)) / (ONE + tmp);
    fp = -(f/eam.re)*(eam.beta+20.0*tmp/(r_ratio-eam.lambda)/(ONE+tmp));
}


// embedding function
static __device__ void find_F(EAM2004Zhou eam, double rho, double &F, double &Fp)
{      
    if (rho < eam.rho_n)
    {
        double x = rho / eam.rho_n - ONE;
        F = ((eam.Fn3 * x + eam.Fn2) * x + eam.Fn1) * x + eam.Fn0;
        Fp = ((THREE * eam.Fn3 * x + TWO * eam.Fn2) * x + eam.Fn1) / eam.rho_n;
    }
    else if (rho < eam.rho_0)
    {
        double x = rho / eam.rho_e - ONE;
        F = ((eam.F3 * x + eam.F2) * x + eam.F1) * x + eam.F0;
        Fp = ((THREE * eam.F3 * x + TWO * eam.F2) * x + eam.F1) / eam.rho_e;
    }
    else
    {
        double x = rho / eam.rho_s;
        double x_eta = pow(x, eam.eta);
        F = eam.Fe * (ONE - eam.eta * log(x)) * x_eta;
        Fp = (eam.eta / rho) * (F - eam.Fe * x_eta);
    }
}


// pair function (phi and phip have been intentionally halved here)
static __device__ void find_phi(EAM2006Dai fs, double d12, double &phi, double &phip)
{
    if (d12 > fs.c)
    {
        phi = ZERO;
        phip = ZERO;
    }
    else
    {
        double tmp=((((fs.c4*d12 + fs.c3)*d12 + fs.c2)*d12 + fs.c1)*d12 + fs.c0);
        
        phi = HALF * (d12 - fs.c) * (d12 - fs.c) * tmp;
        
        phip = TWO * (d12 - fs.c) * tmp;
        phip += (((FOUR*fs.c4*d12 + THREE*fs.c3)*d12 + TWO*fs.c2)*d12 + fs.c1)
              * (d12 - fs.c) * (d12 - fs.c);
        phip *= HALF;
    }
}


// density function f(r)
static __device__ void find_f(EAM2006Dai fs, double d12, double &f)
{
    if (d12 > fs.d)
    {
        f = ZERO;
    }
    else
    {
        double tmp = (d12 - fs.d) * (d12 - fs.d);
        f = tmp  + fs.B * fs.B * tmp * tmp;
    }
}


// derivative of the density function f'(r)
static __device__ void find_fp(EAM2006Dai fs, double d12, double &fp)
{
    if (d12 > fs.d)
    {
        fp = ZERO;
    }
    else 
    {
        double tmp = TWO * (d12 - fs.d);
        fp = tmp * (ONE + fs.B * fs.B * tmp * (d12 - fs.d));
    }
}


// embedding function
static __device__ void find_F(EAM2006Dai fs, double rho, double &F, double &Fp)
{      
    double sqrt_rho = sqrt(rho);
    F = -fs.A * sqrt_rho;
    Fp = -fs.A * HALF / sqrt_rho;
}


// Calculate the embedding energy and its derivative
template <int potential_model>
static __global__ void find_force_eam_step1
(
    EAM2004Zhou  eam2004zhou, EAM2006Dai eam2006dai, 
    int N, int N1, int N2, Box box, 
    int* g_NN, int* g_NL,
    const double* __restrict__ g_x, 
    const double* __restrict__ g_y, 
    const double* __restrict__ g_z, 
    double* g_Fp, double* g_pe 
)
{ 
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
    
    if (n1 >= N1 && n1 < N2)
    {
        int NN = g_NN[n1];
           
        double x1 = LDG(g_x, n1); 
        double y1 = LDG(g_y, n1); 
        double z1 = LDG(g_z, n1);
          
        // Calculate the density
        double rho = ZERO;
        for (int i1 = 0; i1 < NN; ++i1)
        {      
            int n2 = g_NL[n1 + N * i1];
            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12); 
            double rho12 = ZERO;
            if (potential_model == 0) 
            {
                find_f(eam2004zhou, d12, rho12);
            }
            if (potential_model == 1) 
            {
                find_f(eam2006dai, d12, rho12);
            }
            rho += rho12;
        }
        
        // Calculate the embedding energy F and its derivative Fp
        double F, Fp;
        if (potential_model == 0) find_F(eam2004zhou, rho, F, Fp);
        if (potential_model == 1) find_F(eam2006dai, rho, F, Fp);

        g_pe[n1] += F; // many-body potential energy      
        g_Fp[n1] = Fp;   
    }
}


// Force evaluation kernel
template <int potential_model>
static __global__ void find_force_eam_step2
(
    EAM2004Zhou  eam2004zhou, EAM2006Dai eam2006dai,
    int N, int N1, int N2, Box box, 
    int *g_NN, int *g_NL,
    const double* __restrict__ g_Fp, 
    const double* __restrict__ g_x, 
    const double* __restrict__ g_y, 
    const double* __restrict__ g_z, 
    const double* __restrict__ g_vx, 
    const double* __restrict__ g_vy, 
    const double* __restrict__ g_vz,
    double *g_fx, double *g_fy, double *g_fz,
    double *g_virial, double *g_pe
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    double s_fx = ZERO; // force_x
    double s_fy = ZERO; // force_y
    double s_fz = ZERO; // force_z
    double s_pe = ZERO; // potential energy
    double s_sxx = ZERO; // virial_stress_xx
    double s_sxy = ZERO; // virial_stress_xy
    double s_sxz = ZERO; // virial_stress_xz
    double s_syx = ZERO; // virial_stress_yx
    double s_syy = ZERO; // virial_stress_yy
    double s_syz = ZERO; // virial_stress_yz
    double s_szx = ZERO; // virial_stress_zx
    double s_szy = ZERO; // virial_stress_zy
    double s_szz = ZERO; // virial_stress_zz

    if (n1 >= N1 && n1 < N2)
    {  
        int NN = g_NN[n1];        
        double x1 = LDG(g_x, n1); 
        double y1 = LDG(g_y, n1); 
        double z1 = LDG(g_z, n1);
        double Fp1 = LDG(g_Fp, n1);

        for (int i1 = 0; i1 < NN; ++i1)
        {   
            int n2 = g_NL[n1 + N * i1];
            double Fp2 = LDG(g_Fp, n2);
            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
        
            double phi, phip, fp;
            if (potential_model == 0) 
            {
                find_phi(eam2004zhou, d12, phi, phip);
                find_fp(eam2004zhou, d12, fp);
            }
            if (potential_model == 1) 
            {
                find_phi(eam2006dai, d12, phi, phip);
                find_fp(eam2006dai, d12, fp);
            }
            phip /= d12;
            fp   /= d12;
            double f12x =  x12 * (phip + Fp1 * fp); 
            double f12y =  y12 * (phip + Fp1 * fp); 
            double f12z =  z12 * (phip + Fp1 * fp); 
            double f21x = -x12 * (phip + Fp2 * fp); 
            double f21y = -y12 * (phip + Fp2 * fp); 
            double f21z = -z12 * (phip + Fp2 * fp); 
            
            // two-body potential energy
            s_pe += phi;
 
            // per atom force
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z;  

            // per-atom virial
            s_sxx += x12 * f21x;
            s_sxy += x12 * f21y;
            s_sxz += x12 * f21z;
            s_syx += y12 * f21x;
            s_syy += y12 * f21y;
            s_syz += y12 * f21z;
            s_szx += z12 * f21x;
            s_szy += z12 * f21y;
            s_szz += z12 * f21z;
        }

        // save force
        g_fx[n1] += s_fx; 
        g_fy[n1] += s_fy; 
        g_fz[n1] += s_fz;

        // save virial
        // xx xy xz    0 3 4
        // yx yy yz    6 1 5
        // zx zy zz    7 8 2
        g_virial[n1 + 0 * N] += s_sxx;
        g_virial[n1 + 1 * N] += s_syy;
        g_virial[n1 + 2 * N] += s_szz;
        g_virial[n1 + 3 * N] += s_sxy;
        g_virial[n1 + 4 * N] += s_sxz;
        g_virial[n1 + 5 * N] += s_syz;
        g_virial[n1 + 6 * N] += s_syx;
        g_virial[n1 + 7 * N] += s_szx;
        g_virial[n1 + 8 * N] += s_szy;

        // save potential energy
        g_pe[n1] += s_pe;
    }
}   


// Force evaluation wrapper
void EAM::compute(Atom *atom, Measure *measure, int potential_number)
{
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

    if (potential_model == 0)
    {
        find_force_eam_step1<0><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            eam2004zhou, eam2006dai, atom->N, N1, N2, atom->box, 
            atom->NN_local, atom->NL_local, atom->x, atom->y, atom->z, 
            eam_data.Fp, atom->potential_per_atom
        );
        CUDA_CHECK_KERNEL

        FIND_FORCE_EAM_STEP2(0);
        CUDA_CHECK_KERNEL
    }

    if (potential_model == 1)
    {
        find_force_eam_step1<1><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            eam2004zhou, eam2006dai, atom->N, N1, N2, atom->box, 
            atom->NN_local, atom->NL_local, atom->x, atom->y, atom->z, 
            eam_data.Fp, atom->potential_per_atom
        );
        CUDA_CHECK_KERNEL
        
        FIND_FORCE_EAM_STEP2(1);
        CUDA_CHECK_KERNEL
    }
}


