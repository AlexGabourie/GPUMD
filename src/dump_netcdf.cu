#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Dump atom positions in netcdf compatible format.
------------------------------------------------------------------------------*/

//#ifdef USE_NETCDF

#include <unistd.h>
#include "dump_netcdf.cuh"

/* Handle errors by printing an error message and exiting with a
 * non-zero status. */
#define ERRCODE 2
#define ERR(e) {printf("Error: %s\n", nc_strerror(e)); exit(ERRCODE);}
#define NC_CHECK(s) {if(s != NC_NOERR) ERR(s);}

const char SPATIAL_STR[] = "spatial";
const char FRAME_STR[] = "frame";
const char ATOM_STR[] = "atom";
const char CELL_SPATIAL_STR[] = "cell_spatial";
const char CELL_ANGULAR_STR[] = "cell_angular";
const char LABEL_STR[] = "label";
const char TIME_STR[] = "time";
const char COORDINATES_STR[] = "coordinates";
const char TYPE_STR[] = "type";
const char CELL_LENGTHS_STR[] = "cell_lengths";
const char CELL_ANGLES_STR[] = "cell_angles";
const char UNITS_STR[] = "units";

DUMP_NETCDF::DUMP_NETCDF(int N)
{
    this->N = N;
}

void DUMP_NETCDF::initialize(char *input_dir)
{
    strcpy(file_position, input_dir);
    strcat(file_position, "/movie.nc");

    // find appropriate file name
    //TODO  Append if same simulation, new file otherwise
    bool done = false;
    char filename[20];
    int filenum = 1;
    while (!done)
    {
        filenum++;
        if (access(file_position, F_OK) != -1)
        {
            strcpy(file_position, input_dir);
            sprintf(filename, "/movie_%d.nc", filenum);
            strcat(file_position, filename);
        }
        else
        {
            done = true;
        }
    }

    // create file (automatically placed in 'define' mode)
    NC_CHECK(nc_create(file_position, NC_64BIT_OFFSET, &ncid));

    // Global attributes
    NC_CHECK(nc_put_att_text(ncid, NC_GLOBAL, "program", 5, "GPUMD"));
    NC_CHECK(nc_put_att_text(ncid, NC_GLOBAL, "programVersion",
            strlen(GPUMD_VERSION), GPUMD_VERSION));
    NC_CHECK(nc_put_att_text(ncid, NC_GLOBAL, "Conventions", 5, "Amber"));
    NC_CHECK(nc_put_att_text(ncid, NC_GLOBAL, "ConventionVersion", 3, "1.0"));

    // dimensions
    NC_CHECK(nc_def_dim(ncid, FRAME_STR, NC_UNLIMITED, &frame_dim));
    NC_CHECK(nc_def_dim(ncid, SPATIAL_STR, 3, &spatial_dim));
    NC_CHECK(nc_def_dim(ncid, ATOM_STR, N, &atom_dim));
    NC_CHECK(nc_def_dim(ncid, CELL_SPATIAL_STR, 3, &cell_spatial_dim));
    NC_CHECK(nc_def_dim(ncid, CELL_ANGULAR_STR, 3, &cell_angular_dim));
    NC_CHECK(nc_def_dim(ncid, LABEL_STR, 10, &label_dim));

    // Label variables
    int dimids[3];
    dimids[0] = spatial_dim;
    NC_CHECK(nc_def_var(ncid, SPATIAL_STR, NC_CHAR, 1, dimids, &spatial_var));
    dimids[0] = cell_spatial_dim;
    NC_CHECK(nc_def_var(ncid, CELL_SPATIAL_STR, NC_CHAR, 1, dimids, &cell_spatial_var));
    dimids[0] = cell_angular_dim;
    dimids[1] = label_dim;
    NC_CHECK(nc_def_var(ncid, CELL_ANGULAR_STR, NC_CHAR, 2, dimids, &cell_angular_var));

    // Data variables
    dimids[0] = frame_dim;
    NC_CHECK(nc_def_var(ncid, TIME_STR, NC_DOUBLE, 1, dimids, &time_var));
    dimids[1] = cell_spatial_dim;
    NC_CHECK(nc_def_var(ncid, CELL_LENGTHS_STR, NC_DOUBLE, 2, dimids, &cell_lengths_var));
    dimids[1] = cell_angular_dim;
    NC_CHECK(nc_def_var(ncid, CELL_ANGLES_STR, NC_DOUBLE, 2, dimids, &cell_angles_var));

    // More extensive data variables (type, coordinates)
    dimids[0] = frame_dim;
    dimids[1] = atom_dim;
    dimids[2] = spatial_dim;

    // TODO implement option to choose precision
    NC_CHECK(nc_def_var(ncid, COORDINATES_STR, NC_DOUBLE, 3, dimids, &coordinates_var));
    NC_CHECK(nc_def_var(ncid, TYPE_STR, NC_INT, 2, dimids, &type_var));

    // Units
    NC_CHECK(nc_put_att_text(ncid, time_var, UNITS_STR, 10, "picosecond"));
    NC_CHECK(nc_put_att_text(ncid, cell_lengths_var, UNITS_STR, 8, "Angstrom"));
    NC_CHECK(nc_put_att_text(ncid, coordinates_var, UNITS_STR, 8, "Angstrom"));
    NC_CHECK(nc_put_att_text(ncid, cell_angles_var, UNITS_STR, 6, "degree"));

    // Definitions are complete -> leave define mode
    NC_CHECK(nc_enddef(ncid));

    // Write the Label Variables
    NC_CHECK(nc_put_var_text(ncid, spatial_var, "xyz"));
    NC_CHECK(nc_put_var_text(ncid, cell_spatial_var, "abc"));
    // 2D data, startp defines index for data to start
    //  countp determines the size of data in each dimension
    size_t startp[2] = {0, 0};
    size_t countp[2] = {1, 5};
    NC_CHECK(nc_put_vara_text(ncid, cell_angular_var, startp, countp, "alpha"));
    startp[0] = 1;
    countp[1] = 4;
    NC_CHECK(nc_put_vara_text(ncid, cell_angular_var, startp, countp, "beta"));
    startp[0] = 2;
    countp[1] = 5;
    NC_CHECK(nc_put_vara_text(ncid, cell_angular_var, startp, countp, "gamma"));

    // File not used until first dump. Close for now.
    NC_CHECK(nc_close(ncid));
}

void DUMP_NETCDF::open_file()
{
    if (access(file_position, F_OK) != -1)
    {
        NC_CHECK(nc_open(file_position, NC_WRITE, &ncid));
    }

    // TODO test is necessary (definitely the safe option)
    // get all dimension ids
    NC_CHECK(nc_inq_dimid(ncid, FRAME_STR, &frame_dim));
    NC_CHECK(nc_inq_dimid(ncid, SPATIAL_STR, &spatial_dim));
    NC_CHECK(nc_inq_dimid(ncid, ATOM_STR, &atom_dim));
    NC_CHECK(nc_inq_dimid(ncid, CELL_SPATIAL_STR, &cell_spatial_dim));
    NC_CHECK(nc_inq_dimid(ncid, CELL_ANGULAR_STR, &cell_angular_dim));
    NC_CHECK(nc_inq_dimid(ncid, LABEL_STR, &label_dim));

    // Label Variables
    NC_CHECK(nc_inq_varid(ncid, SPATIAL_STR, &spatial_var));
    NC_CHECK(nc_inq_varid(ncid, CELL_SPATIAL_STR, &cell_spatial_var));
    NC_CHECK(nc_inq_varid(ncid, CELL_ANGULAR_STR, &cell_angular_var));

    // Data Variables
    NC_CHECK(nc_inq_varid(ncid, TIME_STR, &time_var));
    NC_CHECK(nc_inq_varid(ncid, CELL_LENGTHS_STR, &cell_lengths_var));
    NC_CHECK(nc_inq_varid(ncid, CELL_ANGLES_STR, &cell_angles_var));

    NC_CHECK(nc_inq_varid(ncid, COORDINATES_STR, &coordinates_var));
    NC_CHECK(nc_inq_varid(ncid, TYPE_STR, &type_var));

    // get frame number
    NC_CHECK(nc_inq_dimlen(ncid, frame_dim, &lenp))

}

void DUMP_NETCDF::write(Atom *atom)
{

    //// Write Frame Header ////
     // Get cell lengths and angles
    real cell_lengths[3];
    real cell_angles[3];
    if (atom->box.triclinic)
    {
        real *t = atom->box.cpu_h;
        real cosgamma, cosbeta, cosalpha;
        cell_lengths[0] = sqrt(t[0]*t[0] + t[3]*t[3] + t[6]*t[6]); //a-side
        cell_lengths[1] = sqrt(t[1]*t[1] + t[4]*t[4] + t[7]*t[7]); //b-side
        cell_lengths[2] = sqrt(t[2]*t[2] + t[5]*t[5] + t[8]*t[8]); //c-side

        cosgamma = (t[0]*t[1]+t[3]*t[4]+t[6]*t[7])/
                (cell_lengths[0]*cell_lengths[1]);
        cosbeta = (t[0]*t[2]+t[3]*t[5]+t[6]*t[8])/
                (cell_lengths[0]*cell_lengths[2]);
        cosalpha = (t[1]*t[2]+t[4]*t[5]+t[7]*t[8])/
                (cell_lengths[1]*cell_lengths[2]);

        cell_angles[0] = acos(cosalpha)*180.0/PI;
        cell_angles[1] = acos(cosbeta)*180.0/PI;
        cell_angles[2] = acos(cosgamma)*180.0/PI;

    }
    else
    {
        cell_lengths[0] = atom->box.cpu_h[0];
        cell_lengths[1] = atom->box.cpu_h[1];
        cell_lengths[2] = atom->box.cpu_h[2];

        cell_angles[0] = 90;
        cell_angles[1] = 90;
        cell_angles[2] = 90;
    }

    // Set lengths to 0 if PBC is off
    if (atom->box.pbc_x) cell_lengths[0] = 0;
    if (atom->box.pbc_y) cell_lengths[1] = 0;
    if (atom->box.pbc_z) cell_lengths[2] = 0;

    size_t countp[3] = {1, 3, 0}; //3rd dimension unused until per-atom
    size_t startp[3] = {lenp, 0, 0};
    NC_CHECK(nc_put_var1_double(ncid, time_var, startp, &(atom->global_time)));
    NC_CHECK(nc_put_vara_double(ncid, cell_lengths_var, startp, countp, cell_lengths));
    NC_CHECK(nc_put_vara_double(ncid, cell_angles_var, startp, countp, cell_angles));

    //// Write Per-Atom Data ////
    countp[0] = 1;
    countp[1] = N;
    countp[2] = 1;
    NC_CHECK(nc_put_vara_int(ncid, type_var, startp, countp, atom->cpu_type));
    NC_CHECK(nc_put_vara_double(ncid, coordinates_var, startp, countp, atom->cpu_x));
    startp[2] = 1;
    NC_CHECK(nc_put_vara_double(ncid, coordinates_var, startp, countp, atom->cpu_y));
    startp[2] = 2;
    NC_CHECK(nc_put_vara_double(ncid, coordinates_var, startp, countp, atom->cpu_z));

}

void DUMP_NETCDF::finalize()
{
    // Do nothing. Needed to satisfy virtual dump_pos parent class function
}

void DUMP_NETCDF::dump(Atom *atom, int step)
{
    if ((step + 1) % interval != 0) return;
    open_file();
    write(atom);
    NC_CHECK(nc_close(ncid));
}

//#endif
