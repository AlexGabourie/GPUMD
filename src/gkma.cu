#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Green-Kubo Modal Analysis (GKMA)
- Currently only supports output of modal heat flux
 -> Green-Kubo integrals must be post-processed

GPUMD Contributing author: Alexander Gabourie (Stanford University)

Some code here and supporting code in 'potential.cu' is based on the LAMMPS
implementation provided by the Henry group at MIT. This code can be found:
https://drive.google.com/open?id=1IHJ7x-bLZISX3I090dW_Y_y-Mqkn07zg
------------------------------------------------------------------------------*/

#include "gkma.cuh"
#include "atom.cuh"
#include <fstream>
#include <string>
#include <iostream>

#define BLOCK_SIZE 128
#define ACCUM_BLOCK 1024
#define BIN_BLOCK 128


static __global__ void gpu_reset_data
(
        int num_elements, real* data
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < num_elements)
    {
        data[n] = ZERO;
    }
}

static __global__ void gpu_average_jm
(
        int num_elements, int samples_per_output, real* jm
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < num_elements)
    {
        jm[n]/=(float)samples_per_output;
    }
}

__global__ void gpu_gkma_reduce
(
        int N, int num_modes,
        const real* __restrict__ data_n,
        real* data
)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int number_of_patches = (N - 1) / ACCUM_BLOCK + 1;

    __shared__ real s_data_x[ACCUM_BLOCK];
    __shared__ real s_data_y[ACCUM_BLOCK];
    __shared__ real s_data_z[ACCUM_BLOCK];
    s_data_x[tid] = ZERO;
    s_data_y[tid] = ZERO;
    s_data_z[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; ++patch)
    {
        int n = tid + patch * ACCUM_BLOCK;
        if (n < N)
        {
            s_data_x[tid] += data_n[n + bid*N ];
            s_data_y[tid] += data_n[n + (bid + num_modes)*N];
            s_data_z[tid] += data_n[n + (bid + 2*num_modes)*N];
        }
    }

    __syncthreads();
    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_data_x[tid] += s_data_x[tid + offset];
            s_data_y[tid] += s_data_y[tid + offset];
            s_data_z[tid] += s_data_z[tid + offset];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        data[bid] = s_data_x[0];
        data[bid + num_modes] = s_data_y[0];
        data[bid + 2*num_modes] = s_data_z[0];
    }

}

__global__ void gpu_calc_xdotn
(
        int N, int N1, int N2, int num_modes,
        const real* __restrict__ g_vx,
        const real* __restrict__ g_vy,
        const real* __restrict__ g_vz,
        const real* __restrict__ g_mass,
        const real* __restrict__ g_eig,
        real* g_xdotn
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    if (n1 >= N1 && n1 < N2)
    {

        real vx1, vy1, vz1;
        vx1 = LDG(g_vx, n1);
        vy1 = LDG(g_vy, n1);
        vz1 = LDG(g_vz, n1);

        real sqrtmass = sqrt(LDG(g_mass, n1));
        for (int i = 0; i < num_modes; i++)
        {
            g_xdotn[n1 + i*N] = sqrtmass*g_eig[n1 + i*3*N]*vx1;
            g_xdotn[n1 + (i + num_modes)*N] =
                    sqrtmass*g_eig[n1 + (1 + i*3)*N]*vy1;
            g_xdotn[n1 + (i + 2*num_modes)*N] =
                    sqrtmass*g_eig[n1 + (2 + i*3)*N]*vz1;
        }
    }
}


static __global__ void gpu_bin_modes
(
       int num_modes, int bin_size, int num_bins,
       const real* __restrict__ g_jm,
       real* bin_out
)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int number_of_patches = (bin_size - 1) / BIN_BLOCK + 1;

    __shared__ real s_data_x[BIN_BLOCK];
    __shared__ real s_data_y[BIN_BLOCK];
    __shared__ real s_data_z[BIN_BLOCK];
    s_data_x[tid] = ZERO;
    s_data_y[tid] = ZERO;
    s_data_z[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; ++patch)
    {
        int n = tid + patch * BIN_BLOCK;
        if (n < bin_size)
        {
            s_data_x[tid] += g_jm[n + bid*bin_size];
            s_data_y[tid] += g_jm[n + bid*bin_size + num_modes];
            s_data_z[tid] += g_jm[n + bid*bin_size + 2*num_modes];
        }
    }

    __syncthreads();
    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_data_x[tid] += s_data_x[tid + offset];
            s_data_y[tid] += s_data_y[tid + offset];
            s_data_z[tid] += s_data_z[tid + offset];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        bin_out[bid] = s_data_x[0];
        bin_out[bid + num_bins] = s_data_y[0];
        bin_out[bid + 2*num_bins] = s_data_z[0];
    }

}

__global__ void gpu_find_gkma_jmn
(
    real fe_x, real fe_y, real fe_z,
    int N, int N1, int N2,
    int triclinic, int pbc_x, int pbc_y, int pbc_z,
    int *g_neighbor_number, int *g_neighbor_list,
    const real* __restrict__ g_f12x,
    const real* __restrict__ g_f12y,
    const real* __restrict__ g_f12z,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    const real* __restrict__ g_vx,
    const real* __restrict__ g_vy,
    const real* __restrict__ g_vz,
    const real* __restrict__ g_box,
    real *g_fx, real *g_fy, real *g_fz,
    const real* __restrict__ g_mass,
    const real* __restrict__ g_eig,
    const real* __restrict__ g_xdot,
    real* g_jmn,
    int num_modes
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    int nm = blockIdx.y * blockDim.y + threadIdx.y;

    if (n1 >= N1 && n1 < N2 && nm < num_modes)
    {
        int neighbor_number = g_neighbor_number[n1];
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        real vx_gk, vy_gk, vz_gk, j_common;
        real rsqrtmass = rsqrt(LDG(g_mass, n1));

        vx_gk=rsqrtmass*g_eig[n1 + nm*3*N]*g_xdot[nm];
        vy_gk=rsqrtmass*g_eig[n1 + (1 + nm*3)*N]*g_xdot[nm + num_modes];
        vz_gk=rsqrtmass*g_eig[n1 + (2 + nm*3)*N]*g_xdot[nm + 2*num_modes];

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * N + n1;
            int n2 = g_neighbor_list[index];
            int neighbor_number_2 = g_neighbor_number[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, g_box, x12, y12, z12);

            int offset = 0;
            for (int k = 0; k < neighbor_number_2; ++k)
            {
                if (n1 == g_neighbor_list[n2 + N * k])
                { offset = k; break; }
            }
            index = offset * N + n2;
            real f21x = LDG(g_f12x, index);
            real f21y = LDG(g_f12y, index);
            real f21z = LDG(g_f12z, index);

            j_common = (f21x*vx_gk + f21y*vy_gk + f21z*vz_gk);

            g_jmn[n1 + nm*N] += j_common*x12; // x-all
            g_jmn[n1 + (nm+num_modes)*N] += j_common*y12; // y-all
            g_jmn[n1 + (nm+2*num_modes)*N] += j_common*z12; // z-all
        }
    }
}


void GKMA::preprocess(char *input_dir, Atom *atom)
{
    if (!compute) return;
    num_modes = last_mode-first_mode+1;
    samples_per_output = output_interval/sample_interval;
    num_bins = num_modes/bin_size;

    strcpy(gkma_file_position, input_dir);
    strcat(gkma_file_position, "/heatmode.out");

    int N = atom->N;
    MY_MALLOC(cpu_eig, real, N * num_modes * 3);
    CHECK(hipMalloc(&eig, sizeof(real) * N * num_modes * 3));

    // initialize eigenvector data structures
    strcpy(eig_file_position, input_dir);
    strcat(eig_file_position, "/eigenvector.out");
    std::ifstream eigfile;
    eigfile.open(eig_file_position);
    if (!eigfile)
    {
        print_error("Cannot open eigenvector.out file.\n");
    }

    // GPU phonon code output format
    std::string val;
    double doubleval;
    // skips freq line and modes up to first_mode
    for (int i=0; i<first_mode; i++)
    {
      getline(eigfile,val);
    }
    for (int j=0; j<num_modes; j++) //modes
    {
        for (int i=0; i<3*N; i++) // xyz of eigvec
        {
            eigfile >> doubleval;
            cpu_eig[i + 3*N*j] = doubleval;
        }
    }
    eigfile.close();

    CHECK(hipMemcpy(eig, cpu_eig, sizeof(real) * N * num_modes * 3,
                            hipMemcpyHostToDevice));
    MY_FREE(cpu_eig);

    // Allocate modal variables
    MY_MALLOC(cpu_jm, real, num_modes * 3) //cpu
    MY_MALLOC(cpu_bin_out, real, num_bins*3);
    CHECK(hipMalloc(&xdot, sizeof(real) * num_modes * 3));
    CHECK(hipMalloc(&jm, sizeof(real) * num_modes * 3));
    CHECK(hipMalloc(&xdotn, sizeof(real) * num_modes * 3 * N));
    CHECK(hipMalloc(&jmn, sizeof(real) * num_modes * 3 * N));
    CHECK(hipMalloc(&bin_out, sizeof(real) * num_bins * 3))

    int num_elements = num_modes*3;
    gpu_reset_data<<<(num_elements-1)/BLOCK_SIZE+1, BLOCK_SIZE>>>
    (
            num_elements, jm
    );
    CUDA_CHECK_KERNEL

    gpu_reset_data<<<(num_elements*N-1)/BLOCK_SIZE+1, BLOCK_SIZE>>>
    (
            num_elements*N, jmn
    );
    CUDA_CHECK_KERNEL

    gpu_reset_data<<<(num_bins * 3 - 1)/BLOCK_SIZE+1, BLOCK_SIZE>>>
    (
            num_bins*3, bin_out
    );

}


void GKMA::process(int step, Atom *atom)
{
    if (!compute) return;
    if (!((step+1) % output_interval == 0)) return;

    int N = atom->N;
    gpu_gkma_reduce<<<num_modes, ACCUM_BLOCK>>>
    (
            N, num_modes, jmn, jm
    );
    CUDA_CHECK_KERNEL


    int num_elements = num_modes*3;
    gpu_average_jm<<<(num_elements-1)/BLOCK_SIZE+1, BLOCK_SIZE>>>
    (
            num_elements, samples_per_output, jm
    );
    CUDA_CHECK_KERNEL

    gpu_bin_modes<<<num_bins, BIN_BLOCK>>>
    (
           num_modes, bin_size, num_bins,
           jm, bin_out
    );
    CUDA_CHECK_KERNEL

    CHECK(hipMemcpy(cpu_bin_out, bin_out, sizeof(real) * num_bins * 3,
            hipMemcpyDeviceToHost));

    FILE *fid = fopen(gkma_file_position, "a");
    for (int i = 0; i < num_bins; i++)
    {
        fprintf(fid, "%25.15e %25.15e %25.15e\n",
         cpu_bin_out[i], cpu_bin_out[i+num_bins], cpu_bin_out[i+2*num_bins]);
    }
    fflush(fid);
    fclose(fid);

    gpu_reset_data<<<(num_elements*N-1)/BLOCK_SIZE+1, BLOCK_SIZE>>>
    (
            num_elements*N, jmn
    );
    CUDA_CHECK_KERNEL

}

void GKMA::postprocess()
{
    if (!compute) return;
    CHECK(hipFree(eig));
    CHECK(hipFree(xdot));
    CHECK(hipFree(xdotn));
    CHECK(hipFree(jm));
    CHECK(hipFree(jmn));
    CHECK(hipFree(bin_out));
    MY_FREE(cpu_jm);
    MY_FREE(cpu_bin_out);
}


