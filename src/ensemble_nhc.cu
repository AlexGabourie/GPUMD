#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "ensemble_nhc.cuh"

#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128
#define DIM 3




Ensemble_NHC::Ensemble_NHC(int t, int N, real T, real Tc, real dt)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
    // position and momentum variables for one NHC
    pos_nhc1[0] = pos_nhc1[1] = pos_nhc1[2] = pos_nhc1[3] = ZERO;
    vel_nhc1[0] = vel_nhc1[2] =  ONE;
    vel_nhc1[1] = vel_nhc1[3] = -ONE;

    real tau = dt * temperature_coupling; 
    real kT = K_B * temperature;
    real dN = DIM * N;
    for (int i = 0; i < NOSE_HOOVER_CHAIN_LENGTH; i++)
    {
        mas_nhc1[i] = kT * tau * tau;
    }
    mas_nhc1[0] *= dN;
}




Ensemble_NHC::Ensemble_NHC
(
    int t, int source_input, int sink_input, int N1, int N2, 
    real T, real Tc, real dT, real time_step
)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
    delta_temperature = dT;
    source = source_input;
    sink = sink_input;

    // position and momentum variables for NHC
    pos_nhc1[0] = pos_nhc1[1] = pos_nhc1[2] = pos_nhc1[3] =  ZERO;
    pos_nhc2[0] = pos_nhc2[1] = pos_nhc2[2] = pos_nhc2[3] =  ZERO;
    vel_nhc1[0] = vel_nhc1[2] = vel_nhc2[0] = vel_nhc2[2] =  ONE;
    vel_nhc1[1] = vel_nhc1[3] = vel_nhc2[1] = vel_nhc2[3] = -ONE;

    real tau = time_step * temperature_coupling;
    real kT1 = K_B * (temperature + delta_temperature);
    real kT2 = K_B * (temperature - delta_temperature);
    real dN1 = DIM * N1;
    real dN2 = DIM * N2;
    for (int i = 0; i < NOSE_HOOVER_CHAIN_LENGTH; i++)
    {
        mas_nhc1[i] = kT1 * tau * tau;
        mas_nhc2[i] = kT2 * tau * tau;
    }
    mas_nhc1[0] *= dN1;
    mas_nhc2[0] *= dN2;

    // initialize the energies transferred from the system to the baths
    energy_transferred[0] = 0.0;
    energy_transferred[1] = 0.0;
}




Ensemble_NHC::~Ensemble_NHC(void)
{
    // nothing now
}




//The Nose-Hover thermostat integrator
//Run it on the CPU, which requires copying the kinetic energy 
//from the GPU to the CPU
static real nhc
(
    int M, real* pos_eta, real *vel_eta, real *mas_eta,
    real Ek2, real kT, real dN, real dt2_particle
)
{
    // These constants are taken from Tuckerman's book
    int n_sy = 7;
    int n_respa = 4;
    const real w[7] = {
                             0.784513610477560,
                             0.235573213359357,
                             -1.17767998417887,
                              1.31518632068391,
                             -1.17767998417887,
                             0.235573213359357,
                             0.784513610477560
                        };
                            
    real factor = 1.0; // to be accumulated

    for (int n1 = 0; n1 < n_sy; n1++)
    {
        real dt2 = dt2_particle * w[n1] / n_respa;
        real dt4 = dt2 * 0.5;
        real dt8 = dt4 * 0.5;
        for (int n2 = 0; n2 < n_respa; n2++)
        {
        
            // update velocity of the last (M - 1) thermostat:
            real G = vel_eta[M - 2] * vel_eta[M - 2] / mas_eta[M - 2] - kT;
            vel_eta[M - 1] += dt4 * G;

            // update thermostat velocities from M - 2 to 0:
            for (int m = M - 2; m >= 0; m--)
            { 
                real tmp = exp(-dt8 * vel_eta[m + 1] / mas_eta[m + 1]);
                G = vel_eta[m - 1] * vel_eta[m - 1] / mas_eta[m - 1] - kT;
                if (m == 0) { G = Ek2 - dN  * kT; }
                vel_eta[m] = tmp * (tmp * vel_eta[m] + dt4 * G);   
            }

            // update thermostat positions from M - 1 to 0:
            for (int m = M - 1; m >= 0; m--)
            { 
                pos_eta[m] += dt2 * vel_eta[m] / mas_eta[m];  
            } 

            // compute the scale factor 
            real factor_local = exp(-dt2 * vel_eta[0] / mas_eta[0]); 
            Ek2 *= factor_local * factor_local;
            factor *= factor_local;

            // update thermostat velocities from 0 to M - 2:
            for (int m = 0; m < M - 1; m++)
            { 
                real tmp = exp(-dt8 * vel_eta[m + 1] / mas_eta[m + 1]);
                G = vel_eta[m - 1] * vel_eta[m - 1] / mas_eta[m - 1] - kT;
                if (m == 0) {G = Ek2 - dN * kT;}
                vel_eta[m] = tmp * (tmp * vel_eta[m] + dt4 * G);   
            }

            // update velocity of the last (M - 1) thermostat:
            G = vel_eta[M - 2] * vel_eta[M - 2] / mas_eta[M - 2] - kT;
            vel_eta[M - 1] += dt4 * G;
        }
    }
    return factor;
}




void Ensemble_NHC::integrate_nvt_nhc
(Atom *atom, Force *force, Measure* measure)
{
    int  N           = atom->N;
    real time_step   = atom->time_step;
    real *thermo             = atom->thermo;

    real kT = K_B * temperature;
    real dN = (real) DIM * N; 
    real dt2 = time_step * HALF;

    const int M = NOSE_HOOVER_CHAIN_LENGTH;
    find_thermo(atom);

    real *ek2;
    MY_MALLOC(ek2, real, sizeof(real) * 1);
    CHECK(hipMemcpy(ek2, thermo, sizeof(real) * 1, hipMemcpyDeviceToHost));
    ek2[0] *= DIM * N * K_B;
    real factor = nhc(M, pos_nhc1, vel_nhc1, mas_nhc1, ek2[0], kT, dN, dt2);
    scale_velocity_global(atom, factor);

    velocity_verlet_1(atom);
    force->compute(atom, measure);
    velocity_verlet_2(atom);
    find_thermo(atom);

    CHECK(hipMemcpy(ek2, thermo, sizeof(real) * 1, hipMemcpyDeviceToHost));
    ek2[0] *= DIM * N * K_B;
    factor = nhc(M, pos_nhc1, vel_nhc1, mas_nhc1, ek2[0], kT, dN, dt2);
    MY_FREE(ek2);
    scale_velocity_global(atom, factor);
}




// integrate by one step, with heating and cooling, 
// using Nose-Hoover chain method
void Ensemble_NHC::integrate_heat_nhc
(Atom *atom, Force *force, Measure* measure)
{
    real time_step   = atom->time_step;

    int label_1 = source;
    int label_2 = sink;

    int Ng = atom->number_of_groups;

    real kT1 = K_B * (temperature + delta_temperature); 
    real kT2 = K_B * (temperature - delta_temperature); 
    real dN1 = (real) DIM * atom->cpu_group_size[source];
    real dN2 = (real) DIM * atom->cpu_group_size[sink];
    real dt2 = time_step * HALF;

    // allocate some memory (to be improved)
    real *ek2;
    MY_MALLOC(ek2, real, sizeof(real) * Ng);
    real *vcx, *vcy, *vcz, *ke;
    CHECK(hipMalloc((void**)&vcx, sizeof(real) * Ng));
    CHECK(hipMalloc((void**)&vcy, sizeof(real) * Ng));
    CHECK(hipMalloc((void**)&vcz, sizeof(real) * Ng));
    CHECK(hipMalloc((void**)&ke, sizeof(real) * Ng));

    // NHC first
    find_vc_and_ke(atom, vcx, vcy, vcz, ke);
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));

    real factor_1 = nhc(NOSE_HOOVER_CHAIN_LENGTH, 
        pos_nhc1, vel_nhc1, mas_nhc1, ek2[label_1], kT1, dN1, dt2);
    real factor_2 = nhc(NOSE_HOOVER_CHAIN_LENGTH, 
        pos_nhc2, vel_nhc2, mas_nhc2, ek2[label_2], kT2, dN2, dt2);

    // accumulate the energies transferred from the system to the baths
    energy_transferred[0] += ek2[label_1] * 0.5 * (1.0 - factor_1 * factor_1);
    energy_transferred[1] += ek2[label_2] * 0.5 * (1.0 - factor_2 * factor_2);
    
    scale_velocity_local(atom, factor_1, factor_2, vcx, vcy, vcz, ke);

    // veloicty-Verlet
    velocity_verlet_1(atom);
    force->compute(atom, measure);
    velocity_verlet_2(atom);

    // NHC second
    find_vc_and_ke(atom, vcx, vcy, vcz, ke);
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));
    factor_1 = nhc(NOSE_HOOVER_CHAIN_LENGTH, 
        pos_nhc1, vel_nhc1, mas_nhc1, ek2[label_1], kT1, dN1, dt2);
    factor_2 = nhc(NOSE_HOOVER_CHAIN_LENGTH, 
        pos_nhc2, vel_nhc2, mas_nhc2, ek2[label_2], kT2, dN2, dt2);

    // accumulate the energies transferred from the system to the baths
    energy_transferred[0] += ek2[label_1] * 0.5 * (1.0 - factor_1 * factor_1);
    energy_transferred[1] += ek2[label_2] * 0.5 * (1.0 - factor_2 * factor_2);

    scale_velocity_local(atom, factor_1, factor_2, vcx, vcy, vcz, ke);

    // clean up
    MY_FREE(ek2);
    CHECK(hipFree(vcx));
    CHECK(hipFree(vcy));
    CHECK(hipFree(vcz));
    CHECK(hipFree(ke));
}




void Ensemble_NHC::compute
(Atom *atom, Force *force, Measure* measure)
{
    if (type == 2)
    {
        integrate_nvt_nhc(atom, force, measure);
    }
    else
    {
        integrate_heat_nhc(atom, force, measure);
    }
}




