#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The version of the Tersoff potential as described in
    [1] J. Tersoff, New empirical approach for the structure and energy
    of covalent systems, PRB 37, 6991 (1988).
------------------------------------------------------------------------------*/


#include "tersoff1988.cuh"

#include "mic.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE_FORCE 64 // 128 is also good

#ifndef USE_SP
    #define ONE_OVER_16   0.0625
    #define THREE_OVER_16 0.1875
    #define NINE_OVER_16  0.5625
#else
    #define ONE_OVER_16   0.0625f
    #define THREE_OVER_16 0.1875f
    #define NINE_OVER_16  0.5625f
#endif

#define EPSILON 1.0e-15

//Easy labels for indexing
#define A 0
#define B 1
#define LAMBDA 2
#define MU 3
#define BETA 4
#define EN 5 //special name for n to avoid conflict
#define C 6
#define D 7
#define H 8
#define R1 9
#define R2 10
#define M 11
#define ALPHA 12
#define GAMMA 13
#define C2 14
#define D2 15
#define ONE_PLUS_C2OVERD2 16
#define PI_FACTOR 17
#define MINUS_HALF_OVER_N 18

#define NUM_PARAMS 19


Tersoff1988::Tersoff1988(FILE *fid, Atom* atom, int num_of_types)
{
    num_types = num_of_types;
    printf("Use Tersoff-1988 (%d-element) potential.\n", num_types);
    int n_entries = num_types*num_types*num_types;
    // 14 parameters per entry of tersoff1988 + 5 pre-calculated values
    real *cpu_ters;
    MY_MALLOC(cpu_ters, real, n_entries*NUM_PARAMS);

    char err[50] = "Error: Illegal Tersoff parameter.";
    rc = 0;
    int count;
    double a, b,lambda, mu, beta, n, c, d, h, r1, r2, m, alpha, gamma;
    for (int i = 0; i < n_entries; i++)
    {
        count = fscanf
        (
            fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf",
            &a, &b, &lambda, &mu, &beta, &n, &c, &d, &h, &r1, &r2, &m, &alpha, &gamma
        );
        if (count!=14) {printf("Error: reading error for potential.in.\n");exit(1);}

        int m_int = round(m);
        // Parameter checking
        if (a < 0.0)
            {printf("%s A must be >= 0.\n",err); exit(1);}
        if (b < 0.0)
            {printf("%s B must be >= 0.\n",err); exit(1);}
        if (lambda < 0.0)
            {printf("%s Lambda must be >= 0.\n",err); exit(1);}
        if(mu < 0.0)
            {printf("%s Mu must be >= 0.\n",err); exit(1);}
        if(beta < 0.0)
            {printf("%s Beta must be >= 0.\n",err); exit(1);}
        if(n < 0.0)
            {printf("%s n must be >= 0.\n",err); exit(1);}
        if(c < 0.0)
            {printf("%s c must be >= 0.\n",err); exit(1);}
        if(d < 0.0)
            {printf("%s d must be >= 0.\n",err); exit(1);}
        if(r1 < 0.0)
            {printf("%s R must be >= 0.\n",err); exit(1);}
        if(r2 < 0.0)
            {printf("%s S must be >= 0.\n",err); exit(1);}
        if(r2 < r1)
            {printf("%s S-R must be >= 0.\n",err); exit(1);}
        if(m_int != 3 && m_int != 1)
            {printf("%s m must be 1 or 3.\n",err); exit(1);}
        if(gamma < 0.0)
            {printf("%s Gamma must be >= 0.\n",err); exit(1);}

        cpu_ters[i*NUM_PARAMS + A] = a;
        cpu_ters[i*NUM_PARAMS + B] = b;
        cpu_ters[i*NUM_PARAMS + LAMBDA] = lambda;
        cpu_ters[i*NUM_PARAMS + MU] = mu;
        cpu_ters[i*NUM_PARAMS + BETA] = beta;
        cpu_ters[i*NUM_PARAMS + EN] = n;
        cpu_ters[i*NUM_PARAMS + C] = c;
        cpu_ters[i*NUM_PARAMS + D] = d;
        cpu_ters[i*NUM_PARAMS + H] = h;
        cpu_ters[i*NUM_PARAMS + R1] = r1;
        cpu_ters[i*NUM_PARAMS + R2] = r2;
        cpu_ters[i*NUM_PARAMS + M] = m_int;
        if (alpha < EPSILON)
        {
            cpu_ters[i*NUM_PARAMS + ALPHA] = ZERO;
        }
        else
        {
            cpu_ters[i*NUM_PARAMS + ALPHA] = alpha;
        }
        cpu_ters[i*NUM_PARAMS + GAMMA] = gamma;
        cpu_ters[i*NUM_PARAMS + C2] = c * c;
        cpu_ters[i*NUM_PARAMS + D2] = d * d;
        cpu_ters[i*NUM_PARAMS + ONE_PLUS_C2OVERD2] = 1.0 +
        cpu_ters[i*NUM_PARAMS + C2] / cpu_ters[i*NUM_PARAMS + D2];
        cpu_ters[i*NUM_PARAMS + PI_FACTOR] = PI / (r2 - r1);
        cpu_ters[i*NUM_PARAMS + MINUS_HALF_OVER_N] = - 0.5 / n;
        rc = r2 > rc ? r2 : rc;
    }

    int num_of_neighbors = (atom->neighbor.MN < 20) ? atom->neighbor.MN : 20;
    int memory = sizeof(real)* atom->N * num_of_neighbors;
    CHECK(hipMalloc((void**)&tersoff_data.b,  memory));
    CHECK(hipMalloc((void**)&tersoff_data.bp, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12x, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12y, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12z, memory));
    CHECK(hipMalloc((void**)&ters, sizeof(real) * n_entries*NUM_PARAMS));
    CHECK(hipMemcpy(ters, cpu_ters,
        sizeof(real) * n_entries*NUM_PARAMS, hipMemcpyHostToDevice));

    MY_FREE(cpu_ters);
}


Tersoff1988::~Tersoff1988(void)
{
    CHECK(hipFree(tersoff_data.b));
    CHECK(hipFree(tersoff_data.bp));
    CHECK(hipFree(tersoff_data.f12x));
    CHECK(hipFree(tersoff_data.f12y));
    CHECK(hipFree(tersoff_data.f12z));
    CHECK(hipFree(ters));
}


static __device__ void find_fr_and_frp
(int i, const real* __restrict__ ters, real d12, real &fr, real &frp)
{
    fr  = LDG(ters,i + A) * exp(- LDG(ters,i + LAMBDA) * d12);
    frp = - LDG(ters,i + LAMBDA) * fr;
}


static __device__ void find_fa_and_fap
(int i, const real* __restrict__ ters, real d12, real &fa, real &fap)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
    fap = - LDG(ters, i + MU) * fa;
}


static __device__ void find_fa
(int i, const real* __restrict__ ters, real d12, real &fa)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
}


static __device__ void find_fc_and_fcp
(int i, const real* __restrict__ ters, real d12, real &fc, real &fcp)
{
    if (d12 < LDG(ters, i + R1)){fc = ONE; fcp = ZERO;}
    else if (d12 < LDG(ters, i + R2))
    {
        fc  =  cos(LDG(ters, i + PI_FACTOR) *
                (d12 - LDG(ters, i + R1))) * HALF + HALF;
        fcp = -sin(LDG(ters, i + PI_FACTOR) *
                (d12 - LDG(ters, i + R1)))*LDG(ters, i + PI_FACTOR)*HALF;
    }
    else {fc  = ZERO; fcp = ZERO;}
}


static __device__ void find_fc
(int i, const real* __restrict__ ters, real d12, real &fc)
{
    if (d12 < LDG(ters, i + R1)) {fc  = ONE;}
    else if (d12 < LDG(ters, i + R2))
    {
        fc = cos(LDG(ters, i + PI_FACTOR) *
                (d12 - LDG(ters, i + R1))) * HALF + HALF;
    }
    else {fc  = ZERO;}
}


static __device__ void find_g_and_gp
(int i, const real* __restrict__ ters, real cos, real &g, real &gp)
{
    real temp = LDG(ters, i + D2) + (cos - LDG(ters, i + H)) *
                (cos - LDG(ters, i + H));
    g  = LDG(ters, i + GAMMA) *
                (LDG(ters, i + ONE_PLUS_C2OVERD2) - LDG(ters, i + C2) / temp);
    gp = LDG(ters, i + GAMMA) *
            (TWO * LDG(ters, i + C2) * (cos - LDG(ters, i + H)) / (temp * temp));
}


static __device__ void find_g
(int i, const real* __restrict__ ters, real cos, real &g)
{
    real temp = LDG(ters, i + D2) + (cos - LDG(ters, i + H)) *
                (cos - LDG(ters, i + H));
    g  = LDG(ters, i + GAMMA) *
                (LDG(ters, i + ONE_PLUS_C2OVERD2) - LDG(ters, i + C2) / temp);
}


static __device__ void find_e_and_ep
(int i, const real* __restrict__ ters, real d12, real d13, real &e, real &ep)
{
    if (LDG(ters, i + ALPHA) < EPSILON){ e = ONE; ep = ZERO;}
    else
    {
        real r = d12 - d13;
        if (LDG(ters, i + M) > TWO) //if m == 3.0
        {
            e = exp(LDG(ters, i + ALPHA) * r * r * r);
            ep = LDG(ters, i + ALPHA) * THREE * r * r * e;
        }
        else
        {
            e = exp(LDG(ters, i + ALPHA) * r);
            ep = LDG(ters, i + ALPHA) * e;
        }
    }
}

static __device__ void find_e
(int i, const real* __restrict__ ters, real d12, real d13, real &e)
{
    if (LDG(ters, i + ALPHA) < EPSILON){ e = ONE;}
    else
    {
        real r = d12 - d13;
        if (LDG(ters, i + M) > TWO){ e = exp(LDG(ters, i + ALPHA) * r * r * r);}
        else{e = exp(LDG(ters, i + ALPHA) * r);}
    }
}


// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int N1, int N2, 
    int triclinic, int pbc_x, int pbc_y, int pbc_z,
    int num_types, int* g_neighbor_number, int* g_neighbor_list, int* g_type,
    const real* __restrict__ ters,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    const real* __restrict__ g_box,
    real* g_b, real* g_bp
)
{
    int num_types2 = num_types * num_types;
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int type2 = g_type[n2];
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, g_box, x12, y12, z12);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real zeta = ZERO;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2
                int type3 = g_type[n3];
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, g_box, 
                    x13, y13, z13);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
                real fc_ijk_13, g_ijk, e_ijk_12_13;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                if (d13 > LDG(ters, ijk*NUM_PARAMS + R2)) {continue;}
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_g(ijk*NUM_PARAMS, ters, cos123, g_ijk);
                find_e(ijk*NUM_PARAMS, ters, d12, d13, e_ijk_12_13);
                zeta += fc_ijk_13 * g_ijk * e_ijk_12_13;
            }
            real bzn, b_ijj;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            bzn = pow(LDG(ters, ijj*NUM_PARAMS + BETA) *
                zeta, LDG(ters, ijj*NUM_PARAMS + EN));
            b_ijj = pow(ONE + bzn, LDG(ters, ijj*NUM_PARAMS + MINUS_HALF_OVER_N));
            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = ONE;
                g_bp[i1 * number_of_particles + n1] = ZERO;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1]
                    = - b_ijj * bzn * HALF / ((ONE + bzn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int N1, int N2, 
    int triclinic, int pbc_x, int pbc_y, int pbc_z,
    int num_types, int *g_neighbor_number, int *g_neighbor_list, int *g_type,
    const real* __restrict__ ters,
    const real* __restrict__ g_b,
    const real* __restrict__ g_bp,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    const real* __restrict__ g_box,
    real *g_potential, real *g_f12x, real *g_f12y, real *g_f12z
)
{
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    int num_types2 = num_types * num_types;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        real pot_energy = ZERO;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, g_box, x12, y12, z12);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real d12inv = ONE / d12;
            real fc_ijj_12, fcp_ijj_12;
            real fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            find_fc_and_fcp(ijj*NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
            find_fa_and_fap(ijj*NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
            find_fr_and_frp(ijj*NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

            // (i,j) part
            real b12 = LDG(g_b, index);
            real factor3=(fcp_ijj_12*(fr_ijj_12-b12*fa_ijj_12)+
                          fc_ijj_12*(frp_ijj_12-b12*fap_ijj_12))*d12inv;
            real f12x = x12 * factor3 * HALF;
            real f12y = y12 * factor3 * HALF;
            real f12z = z12 * factor3 * HALF;

            // accumulate potential energy
            pot_energy += fc_ijj_12 * (fr_ijj_12 - b12 * fa_ijj_12) * HALF;

            // (i,j,k) part
            real bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }
                int type3 = g_type[n3];
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(triclinic, pbc_x, pbc_y, pbc_z, g_box, 
                    x13, y13, z13);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
                int ikj = type1 * num_types2 + type3 * num_types + type2;
                int ikk = type1 * num_types2 + type3 * num_types + type3;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ikk*NUM_PARAMS, ters, d13, fc_ikk_13);
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_fa(ikk*NUM_PARAMS, ters, d13, fa_ikk_13);
                find_fc_and_fcp(ikj*NUM_PARAMS, ters, d12,
                                	fc_ikj_12, fcp_ikj_12);
                real bp13 = LDG(g_bp, index_2);
                real one_over_d12d13 = ONE / (d12 * d13);
                real cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                real cos123_over_d12d12 = cos123*d12inv*d12inv;
                real g_ijk, gp_ijk;
                find_g_and_gp(ijk*NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

                real g_ikj, gp_ikj;
                find_g_and_gp(ikj*NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

                // exp with d12 - d13
                real e_ijk_12_13, ep_ijk_12_13;
                find_e_and_ep(ijk*NUM_PARAMS, ters, d12, d13,
                                	e_ijk_12_13, ep_ijk_12_13);

                // exp with d13 - d12
                real e_ikj_13_12, ep_ikj_13_12;
                find_e_and_ep(ikj*NUM_PARAMS, ters, d13, d12,
                                	e_ikj_13_12, ep_ikj_13_12);

                // derivatives with cosine
                real dc=-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*gp_ijk*e_ijk_12_13+
                        -fc_ikj_12*bp13*fa_ikk_13*fc_ikk_13*gp_ikj*e_ikj_13_12;
                // derivatives with rij
                real dr=(-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*g_ijk*ep_ijk_12_13 +
                  (-fcp_ikj_12*bp13*fa_ikk_13*g_ikj*e_ikj_13_12 +
                  fc_ikj_12*bp13*fa_ikk_13*g_ikj*ep_ikj_13_12)*fc_ikk_13)*d12inv;
                real cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*HALF;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*HALF;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*HALF;
            }
            g_f12x[index] = f12x; g_f12y[index] = f12y; g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += pot_energy;
    }
}


// Wrapper of force evaluation for the Tersoff potential
void Tersoff1988::compute(Atom *atom, Measure *measure)
{
    int N = atom->N;
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;
    int triclinic = atom->box.triclinic;
    int pbc_x = atom->box.pbc_x;
    int pbc_y = atom->box.pbc_y;
    int pbc_z = atom->box.pbc_z;
    int *NN = atom->NN_local;
    int *NL = atom->NL_local;
    int *type = atom->type_local;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real *box = atom->box.h;
    real *pe = atom->potential_per_atom;

    // special data for Tersoff potential
    real *f12x = tersoff_data.f12x;
    real *f12y = tersoff_data.f12y;
    real *f12z = tersoff_data.f12z;
    real *b    = tersoff_data.b;
    real *bp   = tersoff_data.bp;

    // pre-compute the bond order functions and their derivatives
    find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, triclinic, pbc_x, pbc_y, pbc_z, num_types,
        NN, NL, type, ters, x, y, z, box, b, bp
    );
    CUDA_CHECK_KERNEL

    // pre-compute the partial forces
    find_force_tersoff_step2<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, triclinic, pbc_x, pbc_y, pbc_z, num_types,
        NN, NL, type, ters, b, bp, x, y, z, box, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL

    // the final step: calculate force and related quantities
    find_properties_many_body(atom, measure, NN, NL, f12x, f12y, f12z);
}
