#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




/*----------------------------------------------------------------------------80
The version of the Tersoff potential as described in
    [1] J. Tersoff, New empirical approach for the structure and energy
    		of covalent systems, PRB 37, 6991 (1988).
------------------------------------------------------------------------------*/




#include "tersoff1988.cuh"

#include "ldg.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE_FORCE 64 // 128 is also good

#ifndef USE_SP
    #define ONE_OVER_16   0.0625
    #define THREE_OVER_16 0.1875
    #define NINE_OVER_16  0.5625
#else
    #define ONE_OVER_16   0.0625f
    #define THREE_OVER_16 0.1875f
    #define NINE_OVER_16  0.5625f
#endif

//Easy labels for indexing
#define A 0
#define B 1
#define LAMBDA 2
#define MU 3
#define BETA 4
#define EN 5 //special name for n to avoid conflict
#define C 6
#define D 7
#define H 8
#define R1 9
#define R2 10
#define M 11
#define ALPHA 12
#define GAMMA 13
#define C2 14
#define D2 15
#define ONE_PLUS_C2OVERD2 16
#define PI_FACTOR 17
#define MINUS_HALF_OVER_N 18

#define NUM_PARAMS 19


Tersoff1988::Tersoff1988(FILE *fid, Atom* atom, int num_of_types)
{
	num_types = num_of_types;
    printf("Use Tersoff-1988 (%d-element) potential.\n", num_types);
    int n_entries = num_types*num_types*num_types;
    // 14 parameters per entry of tersoff1988 + 5 pre-calculated values
    real *cpu_ters;
    MY_MALLOC(cpu_ters, real, n_entries*NUM_PARAMS);

    rc = 0;
    int count;
	double a, b,lambda, mu, beta, n, c, d, h, r1, r2, m, alpha, gamma;
    for (int i = 0; i < n_entries; i++)
    {
        count = fscanf
        (
            fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf",
            &a, &b, &lambda, &mu, &beta, &n, &c, &d, &h, &r1, &r2, &m, &alpha, &gamma
        );
        if (count!=14) {printf("Error: reading error for potential.in.\n");exit(1);}
        cpu_ters[i*NUM_PARAMS + A] = a;
        cpu_ters[i*NUM_PARAMS + B] = b;
        cpu_ters[i*NUM_PARAMS + LAMBDA] = lambda;
        cpu_ters[i*NUM_PARAMS + MU] = mu;
        cpu_ters[i*NUM_PARAMS + BETA] = beta;
        cpu_ters[i*NUM_PARAMS + EN] = n;
        cpu_ters[i*NUM_PARAMS + C] = c;
        cpu_ters[i*NUM_PARAMS + D] = d;
        cpu_ters[i*NUM_PARAMS + H] = h;
        cpu_ters[i*NUM_PARAMS + R1] = r1;
        cpu_ters[i*NUM_PARAMS + R2] = r2;
        cpu_ters[i*NUM_PARAMS + M] = m;
        cpu_ters[i*NUM_PARAMS + ALPHA] = alpha;
        cpu_ters[i*NUM_PARAMS + GAMMA] = gamma;
        cpu_ters[i*NUM_PARAMS + C2] = c * c;
        cpu_ters[i*NUM_PARAMS + D2] = d * d;
        cpu_ters[i*NUM_PARAMS + ONE_PLUS_C2OVERD2] = 1.0 +
        		cpu_ters[i*NUM_PARAMS + C2] / cpu_ters[i*NUM_PARAMS + D2];
        cpu_ters[i*NUM_PARAMS + PI_FACTOR] = PI / (r2 - r1);
        cpu_ters[i*NUM_PARAMS + MINUS_HALF_OVER_N] = - 0.5 / n;
        rc = r2 > rc ? r2 : rc;
    }

    int num_of_neighbors = (atom->neighbor.MN < 20) ? atom->neighbor.MN : 20;
    int memory = sizeof(real)* atom->N * num_of_neighbors;
    CHECK(hipMalloc((void**)&tersoff_data.b,  memory));
    CHECK(hipMalloc((void**)&tersoff_data.bp, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12x, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12y, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12z, memory));
    CHECK(hipMalloc((void**)&ters, sizeof(real) * n_entries*NUM_PARAMS));
    CHECK(hipMemcpy(ters, cpu_ters,
    		sizeof(real) * n_entries*NUM_PARAMS, hipMemcpyHostToDevice));

    MY_FREE(cpu_ters);
}


Tersoff1988::~Tersoff1988(void)
{
    CHECK(hipFree(tersoff_data.b));
    CHECK(hipFree(tersoff_data.bp));
    CHECK(hipFree(tersoff_data.f12x));
    CHECK(hipFree(tersoff_data.f12y));
    CHECK(hipFree(tersoff_data.f12z));
    CHECK(hipFree(ters));
}


static __device__ void find_fr_and_frp
(
    int i, real *ters, real d12, real &fr, real &frp
)
{
	fr  = ters[i + A] * exp(- ters[i + LAMBDA] * d12);
	frp = - ters[i + LAMBDA] * fr;
}


static __device__ void find_fa_and_fap
(
    int i, real *ters, real d12, real &fa, real &fap
)
{
	fa  = ters[i + B] * exp(- ters[i + MU] * d12);
	fap = - ters[i + MU] * fa;
}


static __device__ void find_fa
(
	int i, real *ters,
    real d12, real &fa
)
{
	fa  = ters[i + B] * exp(- ters[i + MU] * d12);
}


static __device__ void find_fc_and_fcp
(
	int i, real *ters, real d12, real &fc, real &fcp
)
{
	if (d12 < ters[i + R1]) {fc = ONE; fcp = ZERO;}
	else if (d12 < ters[i + R2])
	{
		fc  =  cos(ters[i + PI_FACTOR] * (d12 - ters[i + R1])) * HALF + HALF;
		fcp = -sin(ters[i + PI_FACTOR] *
				(d12 - ters[i + R1]))*ters[i + PI_FACTOR]*HALF;
	}
	else {fc  = ZERO; fcp = ZERO;}
}


static __device__ void find_fc
(
	int i, real *ters, real d12, real &fc
)
{
	if (d12 < ters[i + R1]) {fc  = ONE;}
	else if (d12 < ters[i + R2])
	{fc = cos(ters[i + PI_FACTOR] * (d12 - ters[i + R1])) * HALF + HALF;}
	else {fc  = ZERO;}
}


static __device__ void find_g_and_gp
(
	int i, real *ters, real cos, real &g, real &gp
)
{
	real temp = ters[i + D2] + (cos - ters[i + H]) * (cos - ters[i + H]);
	g  = ters[i + GAMMA] * (ters[i + ONE_PLUS_C2OVERD2] - ters[i + C2] / temp);
	gp = ters[i + GAMMA] *
			(TWO * ters[i + C2] * (cos - ters[i + H]) / (temp * temp));
}


static __device__ void find_g
(
	int i, real *ters, real cos, real &g
)
{
	real temp = ters[i + D2] + (cos - ters[i + H]) * (cos - ters[i + H]);
	g  = ters[i + GAMMA] * (ters[i + ONE_PLUS_C2OVERD2] - ters[i + C2] / temp);
}


static __device__ void find_e_and_ep
(
  int i, real *ters, real d12, real d13, real &e, real &ep
)
{
	e = exp(ters[i + ALPHA] * pow(d12 - d13, ters[i + M]));
	ep = ters[i + ALPHA] * ters[i + M] * pow(d12 - d13, ters[i + M] - ONE)*e;
}

static __device__ void find_e
(
  int i, real *ters, real d12, real d13, real &e
)
{
	e = exp(ters[i + ALPHA] * pow(d12 - d13, ters[i + M]));
}


// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int N1, int N2, int pbc_x, int pbc_y, int pbc_z,
    real *ters, int num_types,
    int* g_neighbor_number, int* g_neighbor_list, int* g_type,
#ifdef USE_LDG
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    const real* __restrict__ g_box_length,
#else
    real* g_x, real* g_y, real* g_z, real* g_box_length,
#endif
    real* g_b, real* g_bp
)
{
	int num_types2 = num_types * num_types;
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        real lx = LDG(g_box_length, 0);
        real ly = LDG(g_box_length, 1);
        real lz = LDG(g_box_length, 2);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int type2 = g_type[n2];
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real zeta = ZERO;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2
                int type3 = g_type[n3];
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(pbc_x, pbc_y, pbc_z, x13, y13, z13, lx, ly, lz);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
                real fc_ijk_13, g_ijk, e_ijk_12_13;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_g(ijk*NUM_PARAMS, ters, cos123, g_ijk);
                find_e(ijk*NUM_PARAMS, ters, d12, d13, e_ijk_12_13);
                zeta += fc_ijk_13 * g_ijk * e_ijk_12_13;
            }
            real bzn, b_ijj;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
			bzn = pow(ters[ijj*NUM_PARAMS + BETA] *
					zeta, ters[ijj*NUM_PARAMS + EN]);
			b_ijj = pow(ONE + bzn, ters[ijj*NUM_PARAMS + MINUS_HALF_OVER_N]);
            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = ONE;
                g_bp[i1 * number_of_particles + n1] = ZERO;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1]
                    = - b_ijj * bzn * HALF / ((ONE + bzn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int N1, int N2, int pbc_x, int pbc_y, int pbc_z,
    real *ters, int num_types,
    int *g_neighbor_number, int *g_neighbor_list, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_b,
    const real* __restrict__ g_bp,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    const real* __restrict__ g_box_length,
#else
    real* g_b, real* g_bp, real* g_x, real* g_y, real* g_z, real* g_box_length,
#endif
    real *g_potential, real *g_f12x, real *g_f12y, real *g_f12z
)
{
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    int num_types2 = num_types * num_types;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        real lx = LDG(g_box_length, 0);
        real ly = LDG(g_box_length, 1);
        real lz = LDG(g_box_length, 2);
        real pot_energy = ZERO;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real d12inv = ONE / d12;
            real fc_ijj_12, fcp_ijj_12;
            real fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            find_fc_and_fcp(ijj*NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
            find_fa_and_fap(ijj*NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
            find_fr_and_frp(ijj*NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

            // (i,j) part
            real b12 = LDG(g_b, index);
            real factor3=(fcp_ijj_12*(fr_ijj_12-b12*fa_ijj_12)+
            			  fc_ijj_12*(frp_ijj_12-b12*fap_ijj_12))*d12inv;
            real f12x = x12 * factor3 * HALF;
            real f12y = y12 * factor3 * HALF;
            real f12z = z12 * factor3 * HALF;

            // accumulate potential energy
            pot_energy += fc_ijj_12 * (fr_ijj_12 - b12 * fa_ijj_12) * HALF;

            // (i,j,k) part
            real bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }
                int type3 = g_type[n3];
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(pbc_x, pbc_y, pbc_z, x13, y13, z13, lx, ly, lz);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
                int ikj = type1 * num_types2 + type3 * num_types + type2;
                int ikk = type1 * num_types2 + type3 * num_types + type3;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ikk*NUM_PARAMS, ters, d13, fc_ikk_13);
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_fa(ikk*NUM_PARAMS, ters, d13, fa_ikk_13);
                find_fc_and_fcp(ikj*NUM_PARAMS, ters, d12,
                					fc_ikj_12, fcp_ikj_12);
                real bp13 = LDG(g_bp, index_2);
                real one_over_d12d13 = ONE / (d12 * d13);
                real cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                real cos123_over_d12d12 = cos123*d12inv*d12inv;
                real g_ijk, gp_ijk;
                find_g_and_gp(ijk*NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

                real g_ikj, gp_ikj;
                find_g_and_gp(ikj*NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

                // exp with d12 - d13
				real e_ijk_12_13, ep_ijk_12_13;
				find_e_and_ep(ijk*NUM_PARAMS, ters, d12, d13,
									e_ijk_12_13, ep_ijk_12_13);

				// exp with d13 - d12
				real e_ikj_13_12, ep_ikj_13_12;
				find_e_and_ep(ikj*NUM_PARAMS, ters, d13, d12,
									e_ikj_13_12, ep_ikj_13_12);

				// derivatives with cosine
                real dc=-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*gp_ijk*e_ijk_12_13+
						-fc_ikj_12*bp13*fa_ikk_13*fc_ikk_13*gp_ikj*e_ikj_13_12;
                // derivatives with rij
                real dr=(-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*g_ijk*ep_ijk_12_13 +
				  (-fcp_ikj_12*bp13*fa_ikk_13*g_ikj*e_ikj_13_12 +
				  fc_ikj_12*bp13*fa_ikk_13*g_ikj*ep_ikj_13_12)*fc_ikk_13)*d12inv;
                real cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*HALF;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*HALF;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*HALF;
            }
            g_f12x[index] = f12x; g_f12y[index] = f12y; g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += pot_energy;
    }
}


// Wrapper of force evaluation for the Tersoff potential
void Tersoff1988::compute(Atom *atom, Measure *measure)
{
    int N = atom->N;
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;
    int pbc_x = atom->pbc_x;
    int pbc_y = atom->pbc_y;
    int pbc_z = atom->pbc_z;
    int *NN = atom->NN_local;
    int *NL = atom->NL_local;
    int *type = atom->type_local;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real *box_length = atom->box_length;
    real *pe = atom->potential_per_atom;

    // special data for Tersoff potential
    real *f12x = tersoff_data.f12x;
    real *f12y = tersoff_data.f12y;
    real *f12z = tersoff_data.f12z;
    real *b    = tersoff_data.b;
    real *bp   = tersoff_data.bp;

    // pre-compute the bond order functions and their derivatives
    find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, pbc_x, pbc_y, pbc_z, ters, num_types,
        NN, NL, type, x, y, z, box_length, b, bp
    );
    CUDA_CHECK_KERNEL

    // pre-compute the partial forces
    find_force_tersoff_step2<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, pbc_x, pbc_y, pbc_z, ters, num_types,
        NN, NL, type, b, bp, x, y, z, box_length, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL

    // the final step: calculate force and related quantities
    find_properties_many_body(atom, measure, NN, NL, f12x, f12y, f12z);
}
