#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "gpumd.cuh"




int main(int argc, char *argv[])
{
    printf("\n");
    printf("***************************************************************\n");
    printf("*                 Welcome to use GPUMD                        *\n");
    printf("*     (Graphics Processing Units Molecular Dynamics)          *\n");
    printf("*       (Author:  Zheyong Fan <brucenju@gmail.com>)           *\n");
    printf("***************************************************************\n");
    printf("\n");
    
    printf("\n");
    printf("===============================================================\n");
    printf("INFO: Compiled with the following options:\n");

#ifdef DEBUG
    printf("\n");
    printf("* Debug mode is activated.\n");
    printf("  -- There is no randomness in the calculations.\n");
    printf("  -- Always use the O(N^2) algorithm to build neighbor list.\n");
    printf("\n");
#else
    srand(time(NULL));
    printf("\n");
    printf("* Debug mode is not activated.\n");
    printf("  -- There are some randomnesses in the calculations.\n");
    printf("\n");
#endif

#ifdef USE_DP
    printf("\n");
    printf("* Use double precision. Slower but more accurate.\n");
    printf("\n");
#else
    printf("\n");
    printf("* Use single precision. Faster but less accurate.\n");
    printf("\n");
#endif

#ifdef USE_LDG
    printf("\n");
    printf("* Use the __ldg() function in the force evalulation kernels.\n");
    printf("  -- This is not supported for compute capability < 3.5.\n");
    printf("\n");
#else
    printf("\n");
    printf("* Not use the __ldg() function.\n");
    printf("\n");
#endif

#ifdef FORCE
    printf("\n");
    printf("* Will calculate and output the initial forces.\n");
    printf("  -- This can be used for lattice dynamics calculations.\n");
    printf("\n");
#endif

#ifdef TRICLINIC
    printf("\n");
    printf("* Use triclinic box.\n");
    printf("  -- Currently only for the REBO potential of Mo-S systems.\n");
    printf("  -- Currently only for NVE and NVT ensembles.\n");
    printf("\n");
#else
    printf("\n");
    printf("* Use rectangular box.\n");
    printf("\n");
#endif

    printf("===============================================================\n");
    printf("\n");
    
    // get the number of input directories
    int number_of_inputs;
    char input_directory[100];

    int count = scanf("%d", &number_of_inputs); 
    if (count != 1)
    {
        printf("Error: reading error for number of inputs.\n");
        exit(1);
    }

    // Run GPUMD for the input directories one by one
    for (int n = 0; n < number_of_inputs; ++n)
    {
        count = scanf("%s", input_directory);
        if (count != 1)
        {
            printf("Error: reading error for input directory.\n");
            exit(1);
        }

        printf("\n");
        printf("===========================================================\n");
        printf("Run simulation for '%s'.\n", input_directory); 
        printf("===========================================================\n");
        printf("\n");

        clock_t time_begin = clock();

        //  Run GPUMD for "input_directory"
        GPUMD gpumd(input_directory);

        clock_t time_finish = clock();

        double time_used = (time_finish - time_begin) / double(CLOCKS_PER_SEC);

        printf("\n");
        printf("===========================================================\n");
        printf("Time used for '%s' = %f s.\n", input_directory, time_used); 
        printf("===========================================================\n");
        printf("\n");
    }

    return EXIT_SUCCESS;
}




