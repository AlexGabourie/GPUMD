#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Calculate the velocity autocorrelation function (VAC)
[1] J. M. Dickey and A. Paskin, 
Computer Simulation of the Lattice Dynamics of Solids, 
Phys. Rev. 188, 1407 (1969).
------------------------------------------------------------------------------*/


#include "vac.cuh"
#include "group.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128


// Allocate memory for recording velocity data
void VAC::preprocess(Atom *atom)
{
    if (!compute_dos && !compute_sdc) return;
    if (compute_dos == compute_sdc)
    {
        PRINT_INPUT_ERROR("DOS and SDC commands cannot be used simultaneously.");
    }
    Group sel_group;  //selected group
    if (grouping_method == -1) { N = atom->N; }
    else
    {
    	sel_group = atom->group[grouping_method];
    	N = sel_group.cpu_size[group];

    	// initialize array that stores atom indices for the group
		int *gindex;
		MY_MALLOC(gindex, int, N);
		int group_index = sel_group.cpu_size_sum[group];
		for (int i = 0; i < N; i++)
		{
			gindex[i] = sel_group.cpu_contents[group_index];
			group_index++;
		}
	    // Copy indices to GPU
	    CHECK(hipMalloc((void**)&g_gindex, sizeof(int) * N));
	    CHECK(hipMemcpy(g_gindex, gindex, sizeof(int) * N, hipMemcpyHostToDevice));
	    MY_FREE(gindex);
    }
    int num = N * (atom->number_of_steps / sample_interval);
    CHECK(hipMalloc((void**)&vx_all, sizeof(real) * num));
    CHECK(hipMalloc((void**)&vy_all, sizeof(real) * num));
    CHECK(hipMalloc((void**)&vz_all, sizeof(real) * num));

    if (compute_dos)
    {
        // set default number of DOS points
        if (num_dos_points == -1) {num_dos_points = Nc;}
        float sample_frequency = 1000.0/(atom->time_step * sample_interval); // THz
        if (sample_frequency < omega_max/PI)
        {
            printf("WARNING: VAC sampling rate is less than Nyquist frequency.\n");
        }
    }
}


// Record velocity data (kernel)
static __global__ void gpu_copy_velocity
(
    int N, int nd, int grouped,
    real *g_in_x, real *g_in_y, real *g_in_z,
    real *g_out_x, real *g_out_y, real *g_out_z,
    const int* __restrict__ g_gindex
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x; // atom index
    if (n < N)
    {
        int m = nd * N + n;
        if (grouped)
        {
        	g_out_x[m] = g_in_x[LDG(g_gindex, n)];
			g_out_y[m] = g_in_y[LDG(g_gindex, n)];
			g_out_z[m] = g_in_z[LDG(g_gindex, n)];
        }
        else
        {
        	g_out_x[m] = g_in_x[n];
			g_out_y[m] = g_in_y[n];
			g_out_z[m] = g_in_z[n];
        }
    }
}


// Record velocity data (wrapper)
void VAC::process(int step, Atom *atom)
{
    if (!(compute_dos || compute_sdc)) return;
    if (step % sample_interval != 0) return;
    int nd = step / sample_interval;  
    int grid_size = (N - 1) / BLOCK_SIZE + 1;
    int grouped = (grouping_method != -1);
    gpu_copy_velocity<<<grid_size, BLOCK_SIZE>>>
    (N, nd, grouped,
    		atom->vx, atom->vy, atom->vz, vx_all, vy_all, vz_all, g_gindex);
    CUDA_CHECK_KERNEL
}


static __global__ void gpu_find_vac
(
    int N, int M, int compute_dos,
    const real* __restrict__ g_mass,
    const real* __restrict__ g_vx,
    const real* __restrict__ g_vy,
    const real* __restrict__ g_vz,
    real *g_vac_x, real *g_vac_y, real *g_vac_z,
    const int* __restrict__ g_gindex,
    int grouping_method
)
{
    //<<<Nc, 128>>>

    __shared__ real s_vac_x[128];
    __shared__ real s_vac_y[128];
    __shared__ real s_vac_z[128];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int number_of_patches = (N - 1) / 128 + 1;  

    s_vac_x[tid] = 0.0;  
    s_vac_y[tid] = 0.0;
    s_vac_z[tid] = 0.0;

    for (int m = 0; m < M; ++m)
    {
        int index_1 = m * N;
        int index_2 = (m + bid) * N;
        for (int patch = 0; patch < number_of_patches; ++patch)
        { 
            int n = tid + patch * 128;
            if (n < N)
            {
            	if (compute_dos)
            	{
            		real mass;
            		if (grouping_method != -1){ mass = LDG(g_mass, LDG(g_gindex,n));}
            		else {mass = LDG(g_mass, n);}
					s_vac_x[tid] += mass * LDG(g_vx, index_1 + n) *
							LDG(g_vx, index_2 + n);
					s_vac_y[tid] += mass * LDG(g_vy, index_1 + n) *
							LDG(g_vy, index_2 + n);
					s_vac_z[tid] += mass * LDG(g_vz, index_1 + n) *
							LDG(g_vz, index_2 + n);
            	}
            	else
            	{
            		s_vac_x[tid] += LDG(g_vx, index_1 + n) * LDG(g_vx, index_2 + n);
					s_vac_y[tid] += LDG(g_vy, index_1 + n) * LDG(g_vy, index_2 + n);
					s_vac_z[tid] += LDG(g_vz, index_1 + n) * LDG(g_vz, index_2 + n);
            	}
            }
        }
    }
    __syncthreads();

    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_vac_x[tid] += s_vac_x[tid + offset];
            s_vac_y[tid] += s_vac_y[tid + offset];
            s_vac_z[tid] += s_vac_z[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        int number_of_data = M * N;
        g_vac_x[bid] = s_vac_x[0] / number_of_data;
        g_vac_y[bid] = s_vac_y[0] / number_of_data;
        g_vac_z[bid] = s_vac_z[0] / number_of_data;
    }
}

// Calculate VAC
void VAC::find_vac(char *input_dir, Atom *atom)
{
    // rename variables
    int number_of_steps = atom->number_of_steps;
    real time_step = atom->time_step;
    real *mass = atom->mass;

    // other parameters
    int Nd = number_of_steps / sample_interval;
    int M = Nd - Nc; // number of time origins
    real dt = time_step * sample_interval;
    real dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps

    // major data
    MY_MALLOC(vac_x, real, Nc);
    MY_MALLOC(vac_y, real, Nc);
    MY_MALLOC(vac_z, real, Nc);
    MY_MALLOC(vac_x_normalized, real, Nc);
    MY_MALLOC(vac_y_normalized, real, Nc);
    MY_MALLOC(vac_z_normalized, real, Nc);

    for (int nc = 0; nc < Nc; nc++) {vac_x[nc] = vac_y[nc] = vac_z[nc] = 0.0;}

    real *g_vac_x, *g_vac_y, *g_vac_z;
    CHECK(hipMalloc((void**)&g_vac_x, sizeof(real) * Nc));
    CHECK(hipMalloc((void**)&g_vac_y, sizeof(real) * Nc));
    CHECK(hipMalloc((void**)&g_vac_z, sizeof(real) * Nc));

    // Here, the block size is fixed to 128, which is a good choice
    gpu_find_vac<<<Nc, 128>>>
    (
        N, M, compute_dos, mass,
        vx_all, vy_all, vz_all,
        g_vac_x, g_vac_y, g_vac_z,
        g_gindex, grouping_method
    );
    CUDA_CHECK_KERNEL

    CHECK(hipMemcpy(vac_x, g_vac_x, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(vac_y, g_vac_y, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(vac_z, g_vac_z, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipFree(g_vac_x));
    CHECK(hipFree(g_vac_y));
    CHECK(hipFree(g_vac_z));

    real vac_x_0 = vac_x[0];
    real vac_y_0 = vac_y[0];
    real vac_z_0 = vac_z[0];
    for (int nc = 0; nc < Nc; nc++)
    {
        vac_x_normalized[nc] = vac_x[nc] / vac_x_0;
        vac_y_normalized[nc] = vac_y[nc] / vac_y_0;
        vac_z_normalized[nc] = vac_z[nc] / vac_z_0;
    }

    if (compute_dos)
    {
		char file_vac[FILE_NAME_LENGTH];
		strcpy(file_vac, input_dir);
		strcat(file_vac, "/mvac.out");
		FILE *fid = fopen(file_vac, "a");
		for (int nc = 0; nc < Nc; nc++)
		{
			real t = nc * dt_in_ps;

			// change to A^2/ps^2
			vac_x[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;
			vac_y[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;
			vac_z[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;

			fprintf(fid, "%25.15e",                                             t);
			fprintf(fid, "%25.15e%25.15e%25.15e", vac_x[nc], vac_y[nc], vac_z[nc]);
			fprintf(fid, "\n");
		}
		fflush(fid);
		fclose(fid);
    }

}

// Calculate phonon density of states (DOS)
// using the method by Dickey and Paskin
static void perform_dft
(
    int N, int Nc, int num_dos_points,
    real delta_t, real omega_0, real d_omega,
    real *vac_x_normalized, real *vac_y_normalized, real *vac_z_normalized,
    real *dos_x, real *dos_y, real *dos_z
)
{
    // Apply Hann window and normalize by the correct factor
    for (int nc = 0; nc < Nc; nc++)
    {
        real hann_window = (cos((PI * nc) / Nc) + 1.0) * 0.5;

        real multiply_factor = 2.0 * hann_window;
        if (nc == 0)
        {
            multiply_factor = 1.0 * hann_window;
        }

        vac_x_normalized[nc] *= multiply_factor;
        vac_y_normalized[nc] *= multiply_factor;
        vac_z_normalized[nc] *= multiply_factor;
    }

    // Calculate DOS by discrete Fourier transform
    for (int nw = 0; nw < num_dos_points; nw++)
    {
        real omega = omega_0 + nw * d_omega;
        for (int nc = 0; nc < Nc; nc++)
        {
            real cos_factor = cos(omega * nc * delta_t);
            dos_x[nw] += vac_x_normalized[nc] * cos_factor;
            dos_y[nw] += vac_y_normalized[nc] * cos_factor;
            dos_z[nw] += vac_z_normalized[nc] * cos_factor;
        }
        dos_x[nw] *= delta_t*2.0*N;
        dos_y[nw] *= delta_t*2.0*N;
        dos_z[nw] *= delta_t*2.0*N;
    }
}


// Calculate phonon density of states
void VAC::find_dos(char *input_dir, Atom *atom)
{
    // rename variables
    real time_step = atom->time_step;

    // other parameters
    real dt = time_step * sample_interval;
    real dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps
    real d_omega = omega_max / num_dos_points;
    real omega_0 = d_omega;

    // major data
    real *dos_x, *dos_y, *dos_z;
    MY_MALLOC(dos_x, real, num_dos_points);
    MY_MALLOC(dos_y, real, num_dos_points);
    MY_MALLOC(dos_z, real, num_dos_points);

    for (int nw = 0; nw < num_dos_points; nw++)
    {
    	dos_x[nw] = dos_y[nw] = dos_z[nw] = 0.0;
    }
    perform_dft
    (
        N, Nc, num_dos_points, dt_in_ps, omega_0, d_omega,
        vac_x_normalized, vac_y_normalized, vac_z_normalized,
        dos_x, dos_y, dos_z
    );

    char file_dos[FILE_NAME_LENGTH];
    strcpy(file_dos, input_dir);
    strcat(file_dos, "/dos.out");
    FILE *fid = fopen(file_dos, "a");
    for (int nw = 0; nw < num_dos_points; nw++)
    {
        real omega = omega_0 + d_omega * nw;
        fprintf(fid, "%25.15e",                                         omega);
        fprintf(fid, "%25.15e%25.15e%25.15e", dos_x[nw], dos_y[nw], dos_z[nw]);
        fprintf(fid, "\n");
    }
    fflush(fid);
    fclose(fid);
    MY_FREE(dos_x); MY_FREE(dos_y); MY_FREE(dos_z);
}


// Calculate the Self Diffusion Coefficient (SDC)
// from the VAC using the Green-Kubo formula
static void integrate_vac
(
    int Nc, real dt, real *vac_x, real *vac_y, real *vac_z,
    real *sdc_x, real *sdc_y, real *sdc_z
)
{
    real dt2 = dt * 0.5;
    for (int nc = 1; nc < Nc; nc++)
    {
        sdc_x[nc] = sdc_x[nc - 1] + (vac_x[nc - 1] + vac_x[nc]) * dt2;
        sdc_y[nc] = sdc_y[nc - 1] + (vac_y[nc - 1] + vac_y[nc]) * dt2;
        sdc_z[nc] = sdc_z[nc - 1] + (vac_z[nc - 1] + vac_z[nc]) * dt2;
    }
}


void VAC::find_sdc(char *input_dir, Atom *atom)
{
    // rename variables
    real time_step = atom->time_step;

    // other parameters
    real dt = time_step * sample_interval;
    real dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps

    // major data
    real *sdc_x, *sdc_y, *sdc_z;
    MY_MALLOC(sdc_x, real, Nc);
    MY_MALLOC(sdc_y, real, Nc);
    MY_MALLOC(sdc_z, real, Nc);

    for (int nc = 0; nc < Nc; nc++) {sdc_x[nc] = sdc_y[nc] = sdc_z[nc] = 0.0;}

    integrate_vac(Nc, dt, vac_x, vac_y, vac_z, sdc_x, sdc_y, sdc_z);

    char file_sdc[FILE_NAME_LENGTH];
    strcpy(file_sdc, input_dir);
    strcat(file_sdc, "/sdc.out");
    FILE *fid = fopen(file_sdc, "a");
    for (int nc = 0; nc < Nc; nc++)
    {
        real t = nc * dt_in_ps;

        // change to A^2/ps^2
        vac_x[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;
        vac_y[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;
        vac_z[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION;

        sdc_x[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps
        sdc_y[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps
        sdc_z[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps

        fprintf(fid, "%25.15e",                                             t);
        fprintf(fid, "%25.15e%25.15e%25.15e", vac_x[nc], vac_y[nc], vac_z[nc]);
        fprintf(fid, "%25.15e%25.15e%25.15e", sdc_x[nc], sdc_y[nc], sdc_z[nc]);
        fprintf(fid, "\n");
    }
    fflush(fid);
    fclose(fid);

    MY_FREE(sdc_x); MY_FREE(sdc_y); MY_FREE(sdc_z);
}


// postprocess VAC and related quantities.
void VAC::postprocess(char *input_dir, Atom *atom)
{
    if (!(compute_dos || compute_sdc)) return;
    print_line_1();
    printf("Start to calculate VAC and related quantities.\n");
    find_vac(input_dir, atom);

    if (compute_dos)
    {
        find_dos(input_dir, atom);
    }
    else
    {
        find_sdc(input_dir, atom);
    }

    MY_FREE(vac_x);
    MY_FREE(vac_y);
    MY_FREE(vac_z);
    MY_FREE(vac_x_normalized);
    MY_FREE(vac_y_normalized);
    MY_FREE(vac_z_normalized);
    CHECK(hipFree(vx_all));
    CHECK(hipFree(vy_all));
    CHECK(hipFree(vz_all));
    if (grouping_method != -1)
    {
        CHECK(hipFree(g_gindex));
    }
    printf("VAC and related quantities are calculated.\n");
    print_line_2();
}


