#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Calculate the (1) velocity autocorrelation function (VAC)
              (2) running diffusion coefficient (RDC)
              (3) phonon (vibrational) density of states (DOS)
[1] J. M. Dickey and A. Paskin, 
Computer Simulation of the Lattice Dynamics of Solids, 
Phys. Rev. 188, 1407 (1969).
------------------------------------------------------------------------------*/


#include "vac.cuh"
#include "atom.cuh"
#include "warp_reduce.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128
#define FILE_NAME_LENGTH      200


// Allocate memory for recording velocity data
void VAC::preprocess(Atom *atom)
{
    if (!compute) return;
    int num = atom->N * (atom->number_of_steps / sample_interval);
    CHECK(hipMalloc((void**)&vx_all, sizeof(real) * num));
    CHECK(hipMalloc((void**)&vy_all, sizeof(real) * num));
    CHECK(hipMalloc((void**)&vz_all, sizeof(real) * num));
}


// Record velocity data (kernel)
static __global__ void gpu_copy_velocity
(
    int N, int nd, real *g_in_x, real *g_in_y, real *g_in_z, 
    real *g_out_x, real *g_out_y, real *g_out_z
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x; // atom index
    if (n < N)
    {
        int m = nd * N + n;
        g_out_x[m] = g_in_x[n];
        g_out_y[m] = g_in_y[n];
        g_out_z[m] = g_in_z[n];
    }
}


// Record velocity data (wrapper)
void VAC::process(int step, Atom *atom)
{
    if (!compute) return;
    if (step % sample_interval != 0) return;
    int N = atom->N;
    int nd = step / sample_interval;  
    int grid_size = (N - 1) / BLOCK_SIZE + 1;
    gpu_copy_velocity<<<grid_size, BLOCK_SIZE>>>
    (N, nd, atom->vx, atom->vy, atom->vz, vx_all, vy_all, vz_all);
    CUDA_CHECK_KERNEL
}


static __global__ void gpu_find_vac
(
    int N, int M, real *g_vx, real *g_vy, real *g_vz, 
    real *g_vac_x, real *g_vac_y, real *g_vac_z
)
{
    //<<<Nc, 128>>>

    __shared__ real s_vac_x[128];
    __shared__ real s_vac_y[128];
    __shared__ real s_vac_z[128];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int number_of_patches = (N - 1) / 128 + 1;  

    s_vac_x[tid] = 0.0;  
    s_vac_y[tid] = 0.0;
    s_vac_z[tid] = 0.0;

    for (int m = 0; m < M; ++m)
    {
        int index_1 = m * N;
        int index_2 = (m + bid) * N;
        for (int patch = 0; patch < number_of_patches; ++patch)
        { 
            int n = tid + patch * 128;
            if (n < N)
            {
                s_vac_x[tid] += g_vx[index_1 + n] * g_vx[index_2 + n];
                s_vac_y[tid] += g_vy[index_1 + n] * g_vy[index_2 + n];
                s_vac_z[tid] += g_vz[index_1 + n] * g_vz[index_2 + n];  
            }
        }
    }
    __syncthreads();

    if (tid < 64)
    {
        s_vac_x[tid] += s_vac_x[tid + 64];
        s_vac_y[tid] += s_vac_y[tid + 64];
        s_vac_z[tid] += s_vac_z[tid + 64];
    }
    __syncthreads();
 
    if (tid < 32)
    {
        warp_reduce(s_vac_x, tid);
        warp_reduce(s_vac_y, tid);
        warp_reduce(s_vac_z, tid); 
    }
   
    if (tid == 0)
    {
        int number_of_data = M * N;
        g_vac_x[bid] = s_vac_x[0] / number_of_data;
        g_vac_y[bid] = s_vac_y[0] / number_of_data;
        g_vac_z[bid] = s_vac_z[0] / number_of_data;
    }
}


// Calculate the Running Diffusion Constant (RDC) 
// from the VAC using the Green-Kubo formula
static void find_rdc
(
    int Nc, real dt, real *vac_x, real *vac_y, real *vac_z, 
    real *rdc_x, real *rdc_y, real *rdc_z
)
{
    real dt2 = dt * 0.5;
    for (int nc = 1; nc < Nc; nc++)  
    {
        rdc_x[nc] = rdc_x[nc - 1] + (vac_x[nc - 1] + vac_x[nc]) * dt2;
        rdc_y[nc] = rdc_y[nc - 1] + (vac_y[nc - 1] + vac_y[nc]) * dt2;
        rdc_z[nc] = rdc_z[nc - 1] + (vac_z[nc - 1] + vac_z[nc]) * dt2;
    }
}


// Calculate phonon density of states (DOS) 
// using the method by Dickey and Paskin
static void find_dos
(
    int Nc, real delta_t, real omega_0, real d_omega,
    real *vac_x_normalized, real *vac_y_normalized, real *vac_z_normalized,
    real *dos_x, real *dos_y, real *dos_z
)
{
    // Apply Hann window and normalize by the correct factor
    for (int nc = 0; nc < Nc; nc++)
    {
        real hann_window = (cos((PI * nc) / Nc) + 1.0) * 0.5;

        real multiply_factor = 2.0 * hann_window;
        if (nc == 0) 
        { 
            multiply_factor = 1.0 * hann_window; 
        }

        vac_x_normalized[nc] *= multiply_factor;
        vac_y_normalized[nc] *= multiply_factor;
        vac_z_normalized[nc] *= multiply_factor;
    }

    // Calculate DOS by discrete Fourier transform
    for (int nw = 0; nw < Nc; nw++)
    {
        real omega = omega_0 + nw * d_omega;
        for (int nc = 0; nc < Nc; nc++)
        {
            real cos_factor = cos(omega * nc * delta_t);
            dos_x[nw] += vac_x_normalized[nc] * cos_factor;
            dos_y[nw] += vac_y_normalized[nc] * cos_factor;
            dos_z[nw] += vac_z_normalized[nc] * cos_factor;
        }
        dos_x[nw] *= delta_t;
        dos_y[nw] *= delta_t;
        dos_z[nw] *= delta_t;
    }
}


// Calculate (1) VAC, (2) RDC, and (3) DOS = phonon density of states
void VAC::find_vac_rdc_dos(char *input_dir, Atom *atom)
{
    // rename variables
    int N = atom->N;
    int number_of_steps = atom->number_of_steps;
    real time_step = atom->time_step;

    // other parameters
    int Nd = number_of_steps / sample_interval;
    int M = Nd - Nc; // number of time origins
    real dt = time_step * sample_interval;
    real dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps
    real d_omega = omega_max / Nc;
    real omega_0 = d_omega;

    // major data
    real *vac_x, *vac_y, *vac_z;
    real *dos_x, *dos_y, *dos_z;
    real *rdc_x, *rdc_y, *rdc_z;
    real *vac_x_normalized, *vac_y_normalized, *vac_z_normalized;
    MY_MALLOC(vac_x, real, Nc);
    MY_MALLOC(vac_y, real, Nc);
    MY_MALLOC(vac_z, real, Nc);
    MY_MALLOC(rdc_x, real, Nc);
    MY_MALLOC(rdc_y, real, Nc);
    MY_MALLOC(rdc_z, real, Nc);
    MY_MALLOC(dos_x, real, Nc);
    MY_MALLOC(dos_y, real, Nc);
    MY_MALLOC(dos_z, real, Nc);
    MY_MALLOC(vac_x_normalized, real, Nc);
    MY_MALLOC(vac_y_normalized, real, Nc);
    MY_MALLOC(vac_z_normalized, real, Nc);

    for (int nc = 0; nc < Nc; nc++) {vac_x[nc] = vac_y[nc] = vac_z[nc] = 0.0;}
    for (int nc = 0; nc < Nc; nc++) {rdc_x[nc] = rdc_y[nc] = rdc_z[nc] = 0.0;}
    for (int nw = 0; nw < Nc; nw++) {dos_x[nw] = dos_y[nw] = dos_z[nw] = 0.0;}

    real *g_vac_x, *g_vac_y, *g_vac_z;
    CHECK(hipMalloc((void**)&g_vac_x, sizeof(real) * Nc));
    CHECK(hipMalloc((void**)&g_vac_y, sizeof(real) * Nc));
    CHECK(hipMalloc((void**)&g_vac_z, sizeof(real) * Nc));

    // Here, the block size is fixed to 128, which is a good choice
    gpu_find_vac<<<Nc, 128>>>
    (
        N, M, vx_all, vy_all, vz_all, 
        g_vac_x, g_vac_y, g_vac_z
    );
    CUDA_CHECK_KERNEL

    CHECK(hipMemcpy(vac_x, g_vac_x, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(vac_y, g_vac_y, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(vac_z, g_vac_z, sizeof(real)*Nc, hipMemcpyDeviceToHost));
    CHECK(hipFree(g_vac_x));
    CHECK(hipFree(g_vac_y));
    CHECK(hipFree(g_vac_z));

    find_rdc(Nc, dt, vac_x, vac_y, vac_z, rdc_x, rdc_y, rdc_z);
    real vac_x_0 = vac_x[0];
    real vac_y_0 = vac_y[0];
    real vac_z_0 = vac_z[0];
    for (int nc = 0; nc < Nc; nc++)
    {
        vac_x_normalized[nc] = vac_x[nc] / vac_x_0;
        vac_y_normalized[nc] = vac_y[nc] / vac_y_0;
        vac_z_normalized[nc] = vac_z[nc] / vac_z_0;
    }

    find_dos
    (
        Nc, dt_in_ps, omega_0, d_omega, 
        vac_x_normalized, vac_y_normalized, vac_z_normalized, 
        dos_x, dos_y, dos_z
    );

    char file_vac[FILE_NAME_LENGTH];
    strcpy(file_vac, input_dir);
    strcat(file_vac, "/vac.out");
    FILE *fid = fopen(file_vac, "a");
    for (int nc = 0; nc < Nc; nc++)
    {
        real t = nc * dt_in_ps;
        real omega = omega_0 + d_omega * nc;

        // change to A^2/ps^2
        vac_x[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION; 
        vac_y[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION; 
        vac_z[nc] *= 1000000.0 / TIME_UNIT_CONVERSION / TIME_UNIT_CONVERSION; 

        rdc_x[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps
        rdc_y[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps
        rdc_z[nc] *= 1000.0 / TIME_UNIT_CONVERSION; // change to A^2/ps

        fprintf(fid, "%25.15e",                                             t);
        fprintf(fid, "%25.15e%25.15e%25.15e", vac_x[nc], vac_y[nc], vac_z[nc]);
        fprintf(fid, "%25.15e%25.15e%25.15e", rdc_x[nc], rdc_y[nc], rdc_z[nc]);
        fprintf(fid, "%25.15e",                                         omega);
        fprintf(fid, "%25.15e%25.15e%25.15e", dos_x[nc], dos_y[nc], dos_z[nc]);
        fprintf(fid, "\n");
    }
    fflush(fid);
    fclose(fid);

    MY_FREE(vac_x); MY_FREE(vac_y); MY_FREE(vac_z);
    MY_FREE(vac_x_normalized); 
    MY_FREE(vac_y_normalized); 
    MY_FREE(vac_z_normalized);
    MY_FREE(rdc_x); MY_FREE(rdc_y); MY_FREE(rdc_z);
    MY_FREE(dos_x); MY_FREE(dos_y); MY_FREE(dos_z);
}


// postprocess VAC and related quantities.
void VAC::postprocess(char *input_dir, Atom *atom)
{
    if (!compute) return;
    print_line_1();
    printf("Start to calculate VAC and related quantities.\n");
    find_vac_rdc_dos(input_dir, atom);
    CHECK(hipFree(vx_all));
    CHECK(hipFree(vy_all));
    CHECK(hipFree(vz_all));
    printf("VAC and related quantities are calculated.\n");
    print_line_2();
}


