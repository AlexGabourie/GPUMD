/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Some wrappers for the cuSOLVER library
------------------------------------------------------------------------------*/


#include "cusolver_wrapper.cuh"
#include "error.cuh"
#include <hipsolver.h>


void eig_hermitian_QR(int N, double* AR, double* AI, double* W_cpu)
{
    // get A
    int N2 = N * N;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, N2);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2));
    for (int n = 0; n < N2; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // get work
    int lwork = 0;
    hipsolverDnZheevd_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));
    hipsolverDnZheevd(handle, jobz, uplo, N, A, N, W, work, lwork, info);
    hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost);

    // free
    hipsolverDnDestroy(handle);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


void eig_hermitian_Jacobi(int N, double* AR, double* AI, double* W_cpu)
{
    // get A
    int N2 = N * N;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, N2);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2));
    for (int n = 0; n < N2; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // some parameters for the Jacobi method
    hipsolverSyevjInfo_t para = NULL;
    hipsolverDnCreateSyevjInfo(&para);

    // get work
    int lwork = 0;
    hipsolverDnZheevj_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork, para);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));
    hipsolverDnZheevj(handle, jobz, uplo, N, A, N, W, work, lwork, info, para);
    hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost);

    // free
    hipsolverDnDestroy(handle);
    hipsolverDnDestroySyevjInfo(para);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


void eig_hermitian_Jacobi_batch
(int N, int batch_size, double* AR, double* AI, double* W_cpu)
{
    // get A
    int M = N * N * batch_size;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, M);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * M));
    for (int n = 0; n < M; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * M, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N * batch_size));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // some parameters for the Jacobi method
    hipsolverSyevjInfo_t para = NULL;
    hipsolverDnCreateSyevjInfo(&para);

    // get work
    int lwork = 0;
    hipsolverDnZheevjBatched_bufferSize
    (handle, jobz, uplo, N, A, N, W, &lwork, para, batch_size);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int) * batch_size));
    hipsolverDnZheevjBatched
    (handle, jobz, uplo, N, A, N, W, work, lwork, info, para, batch_size);
    hipMemcpy(W_cpu, W, sizeof(double)*N*batch_size, hipMemcpyDeviceToHost);

    // free
    hipsolverDnDestroy(handle);
    hipsolverDnDestroySyevjInfo(para);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


