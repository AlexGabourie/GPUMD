/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Some wrappers for the cuSOLVER library
------------------------------------------------------------------------------*/


#include "cusolver_wrapper.cuh"
#include "error.cuh"
#include <hipsolver.h>


void eig_hermitian_QR(size_t N, double* AR, double* AI, double* W_cpu)
{
    // get A
    size_t N2 = N * N;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, N2);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2));
    for (size_t n = 0; n < N2; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // get work
    int lwork = 0;
    hipsolverDnZheevd_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));
    hipsolverDnZheevd(handle, jobz, uplo, N, A, N, W, work, lwork, info);
    hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost);

    // free
    hipsolverDnDestroy(handle);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


void eig_hermitian_Jacobi(size_t N, double* AR, double* AI, double* W_cpu)
{
    // get A
    size_t N2 = N * N;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, N2);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * N2));
    for (size_t n = 0; n < N2; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * N2, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // some parameters for the Jacobi method
    hipsolverSyevjInfo_t para = NULL;
    hipsolverDnCreateSyevjInfo(&para);

    // get work
    int lwork = 0;
    hipsolverDnZheevj_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork, para);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));
    hipsolverDnZheevj(handle, jobz, uplo, N, A, N, W, work, lwork, info, para);
    CHECK(hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost));

    // free
    hipsolverDnDestroy(handle);
    hipsolverDnDestroySyevjInfo(para);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


void eigenvectors_symmetric_Jacobi
(size_t N, double* A_cpu, double* W_cpu, double* eigenvectors_cpu)
{
    // get A
    size_t N2 = N * N;
    double *A; 
    CHECK(hipMalloc((void**)&A, sizeof(double) * N2));
    CHECK(hipMemcpy(A, A_cpu, sizeof(double) * N2, hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // some parameters for the Jacobi method
    hipsolverSyevjInfo_t para = NULL;
    hipsolverDnCreateSyevjInfo(&para);

    // get work
    int lwork = 0;
    hipsolverDnDsyevj_bufferSize(handle, jobz, uplo, N, A, N, W, &lwork, para);
    double* work;
    CHECK(hipMalloc((void**)&work, sizeof(double) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int)));
    hipsolverDnDsyevj(handle, jobz, uplo, N, A, N, W, work, lwork, info, para);
    CHECK(hipMemcpy(W_cpu, W, sizeof(double) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(eigenvectors_cpu, A, sizeof(double)*N*N, 
        hipMemcpyDeviceToHost));

    // free
    hipsolverDnDestroy(handle);
    hipsolverDnDestroySyevjInfo(para);
    CHECK(hipFree(W));
    CHECK(hipFree(A));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


void eig_hermitian_Jacobi_batch
(size_t N, size_t batch_size, double* AR, double* AI, double* W_cpu)
{
    // get A
    size_t M = N * N * batch_size;
    hipDoubleComplex *A, *A_cpu; 
    MY_MALLOC(A_cpu, hipDoubleComplex, M);
    CHECK(hipMalloc((void**)&A, sizeof(hipDoubleComplex) * M));
    for (size_t n = 0; n < M; ++n) { A_cpu[n].x = AR[n]; A_cpu[n].y = AI[n]; }
    CHECK(hipMemcpy(A, A_cpu, sizeof(hipDoubleComplex) * M, 
        hipMemcpyHostToDevice));

    // define W
    double* W; CHECK(hipMalloc((void**)&W, sizeof(double) * N * batch_size));

    // get handle
    hipsolverHandle_t handle = NULL;
    hipsolverDnCreate(&handle);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    // some parameters for the Jacobi method
    hipsolverSyevjInfo_t para = NULL;
    hipsolverDnCreateSyevjInfo(&para);

    // get work
    int lwork = 0;
    hipsolverDnZheevjBatched_bufferSize
    (handle, jobz, uplo, N, A, N, W, &lwork, para, batch_size);
    hipDoubleComplex* work;
    CHECK(hipMalloc((void**)&work, sizeof(hipDoubleComplex) * lwork));

    // get W
    int* info;
    CHECK(hipMalloc((void**)&info, sizeof(int) * batch_size));
    hipsolverDnZheevjBatched
    (handle, jobz, uplo, N, A, N, W, work, lwork, info, para, batch_size);
    hipMemcpy(W_cpu, W, sizeof(double)*N*batch_size, hipMemcpyDeviceToHost);

    // free
    hipsolverDnDestroy(handle);
    hipsolverDnDestroySyevjInfo(para);
    MY_FREE(A_cpu);
    CHECK(hipFree(A));
    CHECK(hipFree(W));
    CHECK(hipFree(work));
    CHECK(hipFree(info));
}


