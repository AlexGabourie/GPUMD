#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Calculate the heat current autocorrelation (HAC) function.
------------------------------------------------------------------------------*/


#include "hac.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "atom.cuh"
#include "error.cuh"

#define NUM_OF_HEAT_COMPONENTS 5
#define FILE_NAME_LENGTH 200
#define DIM 3


//Allocate memory for recording heat current data
void HAC::preprocess(Atom *atom)
{
    if (compute)
    {
        int number_of_frames = atom->number_of_steps / sample_interval;
        heat_all.resize(NUM_OF_HEAT_COMPONENTS * number_of_frames);
    }
}


// calculate the per-atom heat current 
static __global__ void gpu_get_peratom_heat
(
    int N, double *sxx, double *sxy, double *sxz, double *syx, double *syy, double *syz,
    double *szx, double *szy, double *szz, double *vx, double *vy, double *vz, 
    double *jx_in, double *jx_out, double *jy_in, double *jy_out, double *jz
)
{
    int n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n < N)
    {
        jx_in[n] = sxx[n] * vx[n] + sxy[n] * vy[n];
        jx_out[n] = sxz[n] * vz[n];
        jy_in[n] = syx[n] * vx[n] + syy[n] * vy[n];
        jy_out[n] = syz[n] * vz[n];
        jz[n] = szx[n] * vx[n] + szy[n] * vy[n] + szz[n] * vz[n];
    }
}


// sum up the per-atom heat current to get the total heat current
static __global__ void gpu_sum_heat
(int N, int Nd, int nd, double *g_heat, double *g_heat_all)
{
    // <<<NUM_OF_HEAT_COMPONENTS, 1024>>> 
    int tid = threadIdx.x; 
    int number_of_patches = (N - 1) / 1024 + 1;

    __shared__ double s_data[1024];  
    s_data[tid] = ZERO;
 
    for (int patch = 0; patch < number_of_patches; ++patch)
    {
        int n = tid + patch * 1024; 
        if (n < N) { s_data[tid] += g_heat[n + N * blockIdx.x]; }
    }

    __syncthreads();
    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_data[tid] += s_data[tid + offset]; }
        __syncthreads();
    }
    if (tid ==  0) { g_heat_all[nd + Nd * blockIdx.x] = s_data[0]; }
}


// sample heat current data for HAC calculations.
void HAC::process(int step, char *input_dir, Atom *atom)
{
    if (!compute) return; 
    if ((++step) % sample_interval != 0) return;

    // the virial tensor:
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    gpu_get_peratom_heat<<<(atom->N - 1) / 128 + 1, 128>>>
    (
        atom->N, 
        atom->virial_per_atom, 
        atom->virial_per_atom + atom->N * 3,
        atom->virial_per_atom + atom->N * 4,
        atom->virial_per_atom + atom->N * 6,
        atom->virial_per_atom + atom->N * 1,
        atom->virial_per_atom + atom->N * 5,
        atom->virial_per_atom + atom->N * 7,
        atom->virial_per_atom + atom->N * 8,
        atom->virial_per_atom + atom->N * 2,
        atom->vx, atom->vy, atom->vz, 
        atom->heat_per_atom, 
        atom->heat_per_atom + atom->N,
        atom->heat_per_atom + atom->N * 2,
        atom->heat_per_atom + atom->N * 3,
        atom->heat_per_atom + atom->N * 4
    );
    CUDA_CHECK_KERNEL
 
    int nd = step / sample_interval - 1;
    int Nd = atom->number_of_steps / sample_interval;
    gpu_sum_heat<<<NUM_OF_HEAT_COMPONENTS, 1024>>>(atom->N, Nd, nd,
        atom->heat_per_atom, heat_all.data());
    CUDA_CHECK_KERNEL
}


// Calculate the Heat current Auto-Correlation function (HAC) 
__global__ void gpu_find_hac(int Nc, int Nd, double *g_heat, double *g_hac)
{
    //<<<Nc, 128>>>

    __shared__ double s_hac_xi[128];
    __shared__ double s_hac_xo[128];
    __shared__ double s_hac_yi[128];
    __shared__ double s_hac_yo[128];
    __shared__ double s_hac_z[128];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int number_of_patches = (Nd - 1) / 128 + 1;
    int number_of_data = Nd - bid;

    s_hac_xi[tid] = ZERO;
    s_hac_xo[tid] = ZERO;
    s_hac_yi[tid] = ZERO;
    s_hac_yo[tid] = ZERO;
    s_hac_z[tid]  = ZERO;

    for (int patch = 0; patch < number_of_patches; ++patch)
    { 
        int index = tid + patch * 128;
        if (index + bid < Nd)
        {
            s_hac_xi[tid] += g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 0]
                           + g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 1];
            s_hac_xo[tid] += g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 1]
                           + g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 0];
            s_hac_yi[tid] += g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 2]
                           + g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 3];
            s_hac_yo[tid] += g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 3]
                           + g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 2];
            s_hac_z[tid]  += g_heat[index + Nd * 4] * g_heat[index + bid + Nd * 4];
        }
    }
    __syncthreads();

    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_hac_xi[tid] += s_hac_xi[tid + offset];
            s_hac_xo[tid] += s_hac_xo[tid + offset];
            s_hac_yi[tid] += s_hac_yi[tid + offset];
            s_hac_yo[tid] += s_hac_yo[tid + offset];
            s_hac_z[tid]  += s_hac_z[tid  + offset];
        }
        __syncthreads();
    }
   
    if (tid == 0)
    {
        g_hac[bid + Nc * 0] = s_hac_xi[0] / number_of_data;
        g_hac[bid + Nc * 1] = s_hac_xo[0] / number_of_data;
        g_hac[bid + Nc * 2] = s_hac_yi[0] / number_of_data;
        g_hac[bid + Nc * 3] = s_hac_yo[0] / number_of_data;
        g_hac[bid + Nc * 4] = s_hac_z[0]  / number_of_data;
    }
}


// Calculate the Running Thermal Conductivity (RTC) from the HAC
static void find_rtc(int Nc, double factor, double *hac, double *rtc)
{
    for (int k = 0; k < NUM_OF_HEAT_COMPONENTS; k++)
    {
        for (int nc = 1; nc < Nc; nc++)  
        {
            int index = Nc * k + nc;
            rtc[index] = rtc[index - 1] + (hac[index - 1] + hac[index])*factor;
        }
    }
}


// Calculate 
// (1) HAC = Heat current Auto-Correlation and 
// (2) RTC = Running Thermal Conductivity
void HAC::find_hac_kappa(char *input_dir, Atom *atom, Integrate *integrate)
{
    // rename variables
    int number_of_steps = atom->number_of_steps;
    double temperature = integrate->temperature2;
    double time_step = atom->time_step;

    // other parameters
    int Nd = number_of_steps / sample_interval;
    double dt = time_step * sample_interval;
    double dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps

    // major data
    std::vector<double> hac(Nc * NUM_OF_HEAT_COMPONENTS, 0.0);
    std::vector<double> rtc(Nc * NUM_OF_HEAT_COMPONENTS, 0.0);

    double *g_hac;
    CHECK
    (hipMalloc((void**)&g_hac, sizeof(double) * Nc * NUM_OF_HEAT_COMPONENTS));

    // Here, the block size is fixed to 128, which is a good choice
    gpu_find_hac<<<Nc, 128>>>(Nc, Nd, heat_all.data(), g_hac);
    CUDA_CHECK_KERNEL

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(hac.data(), g_hac, sizeof(double) * Nc * NUM_OF_HEAT_COMPONENTS, 
        hipMemcpyDeviceToHost));
    CHECK(hipFree(g_hac));

    double volume = atom->box.get_volume();
    double factor = dt * 0.5 / (K_B * temperature * temperature * volume);
    factor *= KAPPA_UNIT_CONVERSION;
 
    find_rtc(Nc, factor, hac.data(), rtc.data());

    char file_hac[FILE_NAME_LENGTH];
    strcpy(file_hac, input_dir);
    strcat(file_hac, "/hac.out");
    FILE *fid = fopen(file_hac, "a");
    int number_of_output_data = Nc / output_interval;
    for (int nd = 0; nd < number_of_output_data; nd++)
    {
        int nc = nd * output_interval;
        double hac_ave[NUM_OF_HEAT_COMPONENTS] = {ZERO};
        double rtc_ave[NUM_OF_HEAT_COMPONENTS] = {ZERO};
        for (int k = 0; k < NUM_OF_HEAT_COMPONENTS; k++)
        {
            for (int m = 0; m < output_interval; m++)
            {
                int count = Nc * k + nc + m;
                hac_ave[k] += hac[count];
                rtc_ave[k] += rtc[count];
            }
        }
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++)
        {
            hac_ave[m] /= output_interval;
            rtc_ave[m] /= output_interval;
        }
        fprintf
        (fid, "%25.15e", (nc + output_interval * 0.5) * dt_in_ps);
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++) 
        { fprintf(fid, "%25.15e", hac_ave[m]); }
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++) 
        { fprintf(fid, "%25.15e", rtc_ave[m]); }
        fprintf(fid, "\n");
    }  
    fflush(fid);  
    fclose(fid);   
}


// Calculate HAC (heat currant auto-correlation function) 
// and RTC (running thermal conductivity)
void HAC::postprocess(char *input_dir, Atom *atom, Integrate *integrate)
{
    if (!compute) return;
    print_line_1();
    printf("Start to calculate HAC and related quantities.\n");
    find_hac_kappa(input_dir, atom, integrate);
    printf("HAC and related quantities are calculated.\n");
    print_line_2();
}


