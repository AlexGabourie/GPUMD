#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "hac.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"




static __device__ void warp_reduce(volatile real *s, int t) 
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}




//Allocate memory for recording heat current data
void preprocess_hac(Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data)
{
    if (para->hac.compute)
    {
        int num = NUM_OF_HEAT_COMPONENTS * para->number_of_steps 
                / para->hac.sample_interval;
        CHECK(hipMalloc((void**)&gpu_data->heat_all, sizeof(real) * num));
    }
}




// sum up the per-atom heat current to get the total heat current
static __global__ void gpu_sum_heat
(
    int N, int Nd, int nd, real *g_vx, real *g_vy, real *g_vz, 
    real *g_mass, real *g_potential, real *g_heat, 
    real *g_heat_all, real *g_heat_tmp
)
{
    // <<<8, 1024>>> 

    int tid = threadIdx.x; 
    int number_of_patches = (N - 1) / 1024 + 1;

    __shared__ real s_data[1024];  
    s_data[tid] = ZERO;
 
    for (int patch = 0; patch < number_of_patches; ++patch)
    {
        int n = tid + patch * 1024; 
        if (n < N) 
        { 
            if (blockIdx.x < NUM_OF_HEAT_COMPONENTS)
            {
                s_data[tid] += g_heat[n + N * blockIdx.x];
            }
            else
            {
                real vx = g_vx[n];
                real vy = g_vy[n];
                real vz = g_vz[n];
                real v_square = vx * vx + vy * vy + vz * vz;
                real energy = g_mass[n] * v_square * HALF + g_potential[n];
                if (blockIdx.x == 5) s_data[tid] += vx * energy;
                if (blockIdx.x == 6) s_data[tid] += vy * energy;
                if (blockIdx.x == 7) s_data[tid] += vz * energy;
            }
        }
    }

    __syncthreads();
    if (tid < 512) { s_data[tid] += s_data[tid + 512]; } __syncthreads();
    if (tid < 256) { s_data[tid] += s_data[tid + 256]; } __syncthreads();
    if (tid < 128) { s_data[tid] += s_data[tid + 128]; } __syncthreads();
    if (tid <  64) { s_data[tid] += s_data[tid +  64]; } __syncthreads();
    if (tid <  32) { warp_reduce(s_data, tid);         } 
    if (tid ==  0) 
    { 
        g_heat_tmp[blockIdx.x] = s_data[0];
        if (blockIdx.x < NUM_OF_HEAT_COMPONENTS)
            g_heat_all[nd + Nd * blockIdx.x] = s_data[0];
    }
}




// sample heat current data for HAC calculations.
void sample_hac
(
    int step, char *input_dir, Parameters *para, 
    CPU_Data *cpu_data, GPU_Data *gpu_data
)
{
    if (para->hac.compute)
    { 
        if (step % para->hac.sample_interval == 0)
        {   
            // get the total heat current from the per-atom heat current
            int nd = step / para->hac.sample_interval;
            int Nd = para->number_of_steps / para->hac.sample_interval;
            int M = NUM_OF_HEAT_COMPONENTS + DIM;
            real *gpu_heat;
            CHECK(hipMalloc((void**)&gpu_heat, sizeof(real) * M));
            gpu_sum_heat<<<M, 1024>>>
            (
                para->N, Nd, nd, gpu_data->vx, gpu_data->vy, gpu_data->vz,
                gpu_data->mass, gpu_data->potential_per_atom,
                gpu_data->heat_per_atom, gpu_data->heat_all, gpu_heat
            );
#ifdef HEAT_CURRENT
            // dump the heat current components
            char file_heat[FILE_NAME_LENGTH];
            strcpy(file_heat, input_dir);
            strcat(file_heat, "/heat_current.out");
            FILE *fid = fopen(file_heat, "a");
            real *cpu_heat;
            MY_MALLOC(cpu_heat, real, M);
            CHECK(hipMemcpy(cpu_heat, gpu_heat, sizeof(real) * M, 
                hipMemcpyDeviceToHost));
            for (int m = 0; m < M; ++m)
                fprintf(fid, "%25.15e", cpu_heat[m]);
            fprintf(fid, "\n");
            fflush(fid);  
            fclose(fid);
            MY_FREE(cpu_heat);
#endif
            hipFree(gpu_heat);
        }
    }
}




// Calculate the Heat current Auto-Correlation function (HAC) 
__global__ void gpu_find_hac(int Nc, int Nd, real *g_heat, real *g_hac)
{
    //<<<Nc, 128>>>

    __shared__ real s_hac_xi[128];
    __shared__ real s_hac_xo[128];
    __shared__ real s_hac_yi[128];
    __shared__ real s_hac_yo[128];
    __shared__ real s_hac_z[128];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int M = Nd - Nc;
    int number_of_patches = M / 128; 
    int number_of_data = number_of_patches * 128; 

    s_hac_xi[tid] = ZERO;
    s_hac_xo[tid] = ZERO;
    s_hac_yi[tid] = ZERO;
    s_hac_yo[tid] = ZERO;
    s_hac_z[tid]  = ZERO;

    for (int patch = 0; patch < number_of_patches; ++patch)
    { 
        int index = tid + patch * 128;
        s_hac_xi[tid] += g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 0]
                       + g_heat[index + Nd * 0] * g_heat[index + bid + Nd * 1];
        s_hac_xo[tid] += g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 1]
                       + g_heat[index + Nd * 1] * g_heat[index + bid + Nd * 0];
        s_hac_yi[tid] += g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 2]
                       + g_heat[index + Nd * 2] * g_heat[index + bid + Nd * 3];
        s_hac_yo[tid] += g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 3]
                       + g_heat[index + Nd * 3] * g_heat[index + bid + Nd * 2];
        s_hac_z[tid]  += g_heat[index + Nd * 4] * g_heat[index + bid + Nd * 4];
    }
    __syncthreads();

    if (tid < 64)
    {
        s_hac_xi[tid] += s_hac_xi[tid + 64];
        s_hac_xo[tid] += s_hac_xo[tid + 64];
        s_hac_yi[tid] += s_hac_yi[tid + 64];
        s_hac_yo[tid] += s_hac_yo[tid + 64];
        s_hac_z[tid]  += s_hac_z[tid  + 64];
    }
    __syncthreads();
 
    if (tid < 32)
    {
        warp_reduce(s_hac_xi, tid); 
        warp_reduce(s_hac_xo, tid);  
        warp_reduce(s_hac_yi, tid); 
        warp_reduce(s_hac_yo, tid);  
        warp_reduce(s_hac_z,  tid);
    }
   
    if (tid == 0)
    {
        g_hac[bid + Nc * 0] = s_hac_xi[0] / number_of_data;
        g_hac[bid + Nc * 1] = s_hac_xo[0] / number_of_data;
        g_hac[bid + Nc * 2] = s_hac_yi[0] / number_of_data;
        g_hac[bid + Nc * 3] = s_hac_yo[0] / number_of_data;
        g_hac[bid + Nc * 4] = s_hac_z[0]  / number_of_data;
    }
}




// Calculate the Running Thermal Conductivity (RTC) from the HAC
static void find_rtc(int Nc, real factor, real *hac, real *rtc)
{
    for (int k = 0; k < NUM_OF_HEAT_COMPONENTS; k++)
    {
        for (int nc = 1; nc < Nc; nc++)  
        {
            int index = Nc * k + nc;
            rtc[index] = rtc[index - 1] + (hac[index - 1] + hac[index])*factor;
        }
    }
}



static real get_volume(real *box_gpu)
{
    real *box_cpu;
    MY_MALLOC(box_cpu, real, 3);
    hipMemcpy(box_cpu, box_gpu, sizeof(real) * 3, hipMemcpyDeviceToHost);
    real volume = box_cpu[0] * box_cpu[1] * box_cpu[2];
    MY_FREE(box_cpu);
    return volume;
}




// Calculate 
// (1) HAC = Heat current Auto-Correlation and 
// (2) RTC = Running Thermal Conductivity
static void find_hac_kappa
(
    char *input_dir, Parameters *para, CPU_Data *cpu_data, 
    GPU_Data *gpu_data, Integrate *integrate
)
{
    // rename variables
    int number_of_steps = para->number_of_steps;
    int sample_interval = para->hac.sample_interval;
    int Nc = para->hac.Nc;
    real temperature = para->temperature2;
    real time_step = para->time_step;

    // other parameters
    int Nd = number_of_steps / sample_interval;
    real dt = time_step * sample_interval;
    real dt_in_ps = dt * TIME_UNIT_CONVERSION / 1000.0; // ps

    // major data
    real *hac;
    real *rtc;
    MY_MALLOC(hac, real, Nc * NUM_OF_HEAT_COMPONENTS);
    MY_MALLOC(rtc, real, Nc * NUM_OF_HEAT_COMPONENTS);
    
    for (int nc = 0; nc < Nc * NUM_OF_HEAT_COMPONENTS; nc++) 
    { hac[nc] = rtc[nc] = 0.0; }

    real *g_hac;
    CHECK
    (hipMalloc((void**)&g_hac, sizeof(real) * Nc * NUM_OF_HEAT_COMPONENTS));

    // Here, the block size is fixed to 128, which is a good choice
    gpu_find_hac<<<Nc, 128>>>(Nc, Nd, gpu_data->heat_all, g_hac);

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(hac, g_hac, sizeof(real) * Nc * NUM_OF_HEAT_COMPONENTS, 
        hipMemcpyDeviceToHost));
    CHECK(hipFree(g_hac));

    real volume = get_volume(gpu_data->box_length);
    real factor = dt * 0.5 / (K_B * temperature * temperature * volume);
    factor *= KAPPA_UNIT_CONVERSION;
 
    find_rtc(Nc, factor, hac, rtc);

    char file_hac[FILE_NAME_LENGTH];
    strcpy(file_hac, input_dir);
    strcat(file_hac, "/hac.out");
    FILE *fid = fopen(file_hac, "a");
    int number_of_output_data = Nc / para->hac.output_interval;
    for (int nd = 0; nd < number_of_output_data; nd++)
    {
        int nc = nd * para->hac.output_interval;
        real hac_ave[NUM_OF_HEAT_COMPONENTS] = {ZERO};
        real rtc_ave[NUM_OF_HEAT_COMPONENTS] = {ZERO};
        for (int k = 0; k < NUM_OF_HEAT_COMPONENTS; k++)
        {
            for (int m = 0; m < para->hac.output_interval; m++)
            {
                int count = Nc * k + nc + m;
                hac_ave[k] += hac[count];
                rtc_ave[k] += rtc[count];
            }
        }
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++)
        {
            hac_ave[m] /= para->hac.output_interval;
            rtc_ave[m] /= para->hac.output_interval;
        }
        fprintf
        (fid, "%25.15e", (nc + para->hac.output_interval * 0.5) * dt_in_ps);
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++) 
        { fprintf(fid, "%25.15e", hac_ave[m]); }
        for (int m = 0; m < NUM_OF_HEAT_COMPONENTS; m++) 
        { fprintf(fid, "%25.15e", rtc_ave[m]); }
        fprintf(fid, "\n");
    }  
    fflush(fid);  
    fclose(fid);
    MY_FREE(hac);
    MY_FREE(rtc);    
}




// Calculate HAC (heat currant auto-correlation function) 
// and RTC (running thermal conductivity)
void postprocess_hac
(
    char *input_dir, Parameters *para, CPU_Data *cpu_data,
    GPU_Data *gpu_data, Integrate *integrate
)
{
    if (para->hac.compute) 
    {
        printf("INFO:  start to calculate HAC and related quantities.\n");
        find_hac_kappa(input_dir, para, cpu_data, gpu_data, integrate);
        CHECK(hipFree(gpu_data->heat_all));
        printf("INFO:  HAC and related quantities are calculated.\n\n");
    }
}




