#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.h"
#include "mic.cu" // static __device__ dev_apply_mic(...)
#include "tersoff_1989_1.h"


/*----------------------------------------------------------------------------80
The single-element version of the Tersoff potential as described in  
    [1] J. Tersoff, Modeling solid-state chemistry: Interatomic potentials 
        for multicomponent systems, PRB 39, 5566 (1989).
------------------------------------------------------------------------------*/


// best block size here: 64 or 128
#define BLOCK_SIZE_FORCE 64


/*------------------------------------------------------------------------------
    Some simple functions and their derivatives
------------------------------------------------------------------------------*/


static __device__ void find_fr_and_frp
(Tersoff ters0, real d12, real &fr, real &frp)
{   
    fr  = ters0.a * exp(- ters0.lambda * d12);    
    frp = - ters0.lambda * fr;
}


static __device__ void find_fa_and_fap
(Tersoff ters0, real d12, real &fa, real &fap)
{    
    fa  = ters0.b * exp(- ters0.mu * d12);    
    fap = - ters0.mu * fa; 
}


static __device__ void find_fa(Tersoff ters0, real d12, real &fa)
{   
    fa  = ters0.b * exp(- ters0.mu * d12);   
}


static __device__ void find_fc_and_fcp
(
    Tersoff ters0, 
    real d12, real &fc, real &fcp
)
{
    if (d12 < ters0.r1) {fc = ONE; fcp = ZERO;}
    else if (d12 < ters0.r2)
    {              
        fc  =  cos(ters0.pi_factor * (d12 - ters0.r1)) * HALF + HALF;
        fcp = -sin(ters0.pi_factor * (d12 - ters0.r1))*ters0.pi_factor*HALF;
    }
    else {fc  = ZERO; fcp = ZERO;}
}


static __device__ void find_fc(Tersoff ters0, real d12, real &fc)
{
    if (d12 < ters0.r1) {fc  = ONE;}
    else if (d12 < ters0.r2) 
    {fc = cos(ters0.pi_factor * (d12 - ters0.r1)) * HALF + HALF;}
    else {fc  = ZERO;}
}


static __device__ void find_g_and_gp(Tersoff ters0, real cos, real &g, real &gp)
{  
    real temp = ters0.d2 + (cos - ters0.h) * (cos - ters0.h);
    g  = ters0.one_plus_c2overd2 - ters0.c2 / temp;    
    gp = TWO * ters0.c2 * (cos - ters0.h) / (temp * temp); 
}


static __device__ void find_g(Tersoff ters0, real cos, real &g)
{ 
    real temp = ters0.d2 + (cos - ters0.h) * (cos - ters0.h);
    g  = ters0.one_plus_c2overd2 - ters0.c2 / temp;  
}
 

/*------------------------------------------------------------------------------
    Find the bond-order functions and their derivatives first.
    This is an efficient approach.
------------------------------------------------------------------------------*/
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z,
    Tersoff ters0, 
    int* g_neighbor_number, int* g_neighbor_list,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z,
    const real* __restrict__ g_box_length, 
#else
    real* g_x, real* g_y, real* g_z, real* g_box_length,
#endif
    real* g_b, real* g_bp
)
{
    //<<<(number_of_particles - 1) / MAX_THREAD + 1, MAX_THREAD>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];

        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {      
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic
            (
                pbc_x, pbc_y, pbc_z, x12, y12, z12, LDG(g_box_length, 0), 
                LDG(g_box_length, 1), LDG(g_box_length, 2)
            );
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real zeta = ZERO;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];  
                if (n3 == n2) { continue; } // ensure that n3 != n2

                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;         
                dev_apply_mic
                (
                    pbc_x, pbc_y, pbc_z, x13, y13, z13, LDG(g_box_length, 0), 
                    LDG(g_box_length, 1), LDG(g_box_length, 2)
                );
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
                real fc13, g123; 

                find_fc(ters0, d13, fc13);
                find_g(ters0, cos123, g123);

                zeta += fc13 * g123;
            } 
            real bzn, b12;

            bzn = pow(ters0.beta * zeta, ters0.n);
            b12 = pow(ONE + bzn, ters0.minus_half_over_n);

            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = ONE;
                g_bp[i1 * number_of_particles + n1] = ZERO; 
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b12;
                g_bp[i1 * number_of_particles + n1] 
                    = - b12 * bzn * HALF / ((ONE + bzn) * zeta); 
            }
        }
    }
}




// second step: calculate the partial forces dU_i/dr_ij
template <int cal_p>
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z,
    Tersoff ters0, int *g_neighbor_number, int *g_neighbor_list,
#ifdef USE_LDG
    const real* __restrict__ g_b, 
    const real* __restrict__ g_bp,
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_box_length,
#else
    real* g_b, real* g_bp, real* g_x, real* g_y, real* g_z, real* g_box_length,
#endif
    real *g_potential, real *g_f12x, real *g_f12y, real *g_f12z 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];

        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real potential_energy = ZERO;

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        { 

            int index = i1 * number_of_particles + n1;   
            int n2 = g_neighbor_list[index];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic
            (
                pbc_x, pbc_y, pbc_z, x12, y12, z12, LDG(g_box_length, 0), 
                LDG(g_box_length, 1), LDG(g_box_length, 2)
            );
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real d12inv = ONE / d12;
            real fc12, fcp12, fa12, fap12, fr12, frp12;

            find_fc_and_fcp(ters0, d12, fc12, fcp12);
            find_fa_and_fap(ters0, d12, fa12, fap12);
            find_fr_and_frp(ters0, d12, fr12, frp12);
   
            // accumulate_force_12 
            real b12 = LDG(g_b, index);    
            real factor3=(fcp12*(fr12-b12*fa12)+fc12*(frp12-b12*fap12))*d12inv;   
            real f12x = x12 * factor3 * HALF; 
            real f12y = y12 * factor3 * HALF;
            real f12z = z12 * factor3 * HALF;

            if (cal_p) // accumulate potential energy
            {
                potential_energy += fc12 * (fr12 - b12 * fa12) * HALF;
            }

            // accumulate_force_123
            real bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;        
                int n3 = g_neighbor_list[index_2];   
                if (n3 == n2) { continue; } 
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic
                (
                    pbc_x, pbc_y, pbc_z, x13, y13, z13, LDG(g_box_length, 0), 
                    LDG(g_box_length, 1), LDG(g_box_length, 2)
                );
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);   
                real fc13, fa13;
                find_fc(ters0, d13, fc13);
                find_fa(ters0, d13, fa13); 

                real bp13 = LDG(g_bp, index_2);
                real one_over_d12d13 = ONE / (d12 * d13);   
                real cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                real cos123_over_d12d12 = cos123*d12inv*d12inv; 
                real g123, gp123; 
                find_g_and_gp(ters0, cos123, g123, gp123);

                real temp123a=(-bp12*fc12*fa12*fc13-bp13*fc13*fa13*fc12)*gp123;
                real temp123b= - bp13 * fc13 * fa13 * fcp12 * g123 * d12inv;
                real cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * temp123b + temp123a * cos_d)*HALF; 
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * temp123b + temp123a * cos_d)*HALF;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * temp123b + temp123a * cos_d)*HALF;
            }
            g_f12x[index] = f12x;
            g_f12y[index] = f12y;
            g_f12z[index] = f12z;
        }
        if (cal_p) // save potential
        {
            g_potential[n1] = potential_energy;
        }
    }
}   


            
/*----------------------------------------------------------------------------80
    Calculate forces, potential energy, and virial stress
------------------------------------------------------------------------------*/
template <int cal_p, int cal_j, int cal_q>
static __global__ void find_force_tersoff_step3
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z,
    Tersoff ters0, 
    int *g_neighbor_number, int *g_neighbor_list,
#ifdef USE_LDG
    const real* __restrict__ g_f12x, 
    const real* __restrict__ g_f12y,
    const real* __restrict__ g_f12z,
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
    const real* __restrict__ g_box_length,
#else
    real* g_f12x, real* g_f12y, real* g_f12z, real* g_x, real* g_y, real* g_z, 
    real* g_vx, real* g_vy, real* g_vz, real* g_box_length,
#endif
    real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz,
    real *g_h, int *g_label, int *g_fv_index, real *g_fv 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    real s_fx = ZERO;
    real s_fy = ZERO;
    real s_fz = ZERO;

    // if cal_p, then s1~s4 = px, py, pz, U; if cal_j, then s1~s5 = j1~j5
    __shared__ real s1[BLOCK_SIZE_FORCE];
    __shared__ real s2[BLOCK_SIZE_FORCE];
    __shared__ real s3[BLOCK_SIZE_FORCE];
    __shared__ real s4[BLOCK_SIZE_FORCE];
    __shared__ real s5[BLOCK_SIZE_FORCE];
    s1[threadIdx.x] = ZERO; 
    s2[threadIdx.x] = ZERO; 
    s3[threadIdx.x] = ZERO;
    s4[threadIdx.x] = ZERO;
    s5[threadIdx.x] = ZERO;

    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];

        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);

        real vx1, vy1, vz1;
        if (cal_j || cal_q)
        {
            vx1 = LDG(g_vx, n1);
            vy1 = LDG(g_vy, n1); 
            vz1 = LDG(g_vz, n1);
        }

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        { 

            int index = i1 * number_of_particles + n1;   
            int n2 = g_neighbor_list[index];
            int neighbor_number_2 = g_neighbor_number[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic
            (
                pbc_x, pbc_y, pbc_z, x12, y12, z12, LDG(g_box_length, 0), 
                LDG(g_box_length, 1), LDG(g_box_length, 2)
            );

            real f12x = LDG(g_f12x, index); 
            real f12y = LDG(g_f12y, index);
            real f12z = LDG(g_f12z, index);
            int offset = 0;
            for (int k = 0; k < neighbor_number_2; ++k)
            {
                if (n1 == g_neighbor_list[n2 + number_of_particles * k]) 
                { 
                    offset = k; break; 
                }
            }
            index = offset * number_of_particles + n2; 
            real f21x = LDG(g_f12x, index);
            real f21y = LDG(g_f12y, index);
            real f21z = LDG(g_f12z, index);
    
            // per atom force 
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z; 

            // per-atom stress
            if (cal_p)
            {
                s1[threadIdx.x] -= x12 * (f12x - f21x) * HALF; 
                s2[threadIdx.x] -= y12 * (f12y - f21y) * HALF; 
                s3[threadIdx.x] -= z12 * (f12z - f21z) * HALF;
            }

            // per-atom heat current
            if (cal_j)
            {
                s1[threadIdx.x] += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s2[threadIdx.x] += (f21z * vz1) * x12;               // x-out
                s3[threadIdx.x] += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s4[threadIdx.x] += (f21z * vz1) * y12;               // y-out
                s5[threadIdx.x] += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            }
 
            // accumulate heat across some sections (for NEMD)
            if (cal_q)
            {
                int index_12 = g_fv_index[n1] * 12;
                if (index_12 >= 0 && g_fv_index[n1 + number_of_particles] == n2)
                {
                    g_fv[index_12 + 0]  = f12x;
                    g_fv[index_12 + 1]  = f12y;
                    g_fv[index_12 + 2]  = f12z;
                    g_fv[index_12 + 3]  = f21x;
                    g_fv[index_12 + 4]  = f21y;
                    g_fv[index_12 + 5]  = f21z;
                    g_fv[index_12 + 6]  = vx1;
                    g_fv[index_12 + 7]  = vy1;
                    g_fv[index_12 + 8]  = vz1;
                    g_fv[index_12 + 9]  = LDG(g_vx, n2);
                    g_fv[index_12 + 10] = LDG(g_vy, n2);
                    g_fv[index_12 + 11] = LDG(g_vz, n2);
                }  
            }
        }

        // save force
        g_fx[n1] = s_fx; 
        g_fy[n1] = s_fy; 
        g_fz[n1] = s_fz;

        if (cal_p) // save stress and potential
        {
            g_sx[n1] = s1[threadIdx.x]; 
            g_sy[n1] = s2[threadIdx.x]; 
            g_sz[n1] = s3[threadIdx.x];
        }

        if (cal_j) // save heat current
        {
            g_h[n1 + 0 * number_of_particles] = s1[threadIdx.x];
            g_h[n1 + 1 * number_of_particles] = s2[threadIdx.x];
            g_h[n1 + 2 * number_of_particles] = s3[threadIdx.x];
            g_h[n1 + 3 * number_of_particles] = s4[threadIdx.x];
            g_h[n1 + 4 * number_of_particles] = s5[threadIdx.x];
        }
    }
}   


            

/*
    Force evaluation for the Tersoff potential (a wrapper)
*/
void gpu_find_force_tersoff1
(Parameters *para, Force_Model *force_model, GPU_Data *gpu_data)
{
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE_FORCE + 1;
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    int *NN = gpu_data->NN_local; 
    int *NL = gpu_data->NL_local;
    real *x = gpu_data->x; 
    real *y = gpu_data->y; 
    real *z = gpu_data->z;
    real *vx = gpu_data->vx; 
    real *vy = gpu_data->vy; 
    real *vz = gpu_data->vz;
    real *fx = gpu_data->fx; 
    real *fy = gpu_data->fy; 
    real *fz = gpu_data->fz;
    real *b = gpu_data->b; 
    real *bp = gpu_data->bp; 
    real *box_length = gpu_data->box_length;
    real *sx = gpu_data->virial_per_atom_x; 
    real *sy = gpu_data->virial_per_atom_y; 
    real *sz = gpu_data->virial_per_atom_z; 
    real *pe = gpu_data->potential_per_atom;
    real *h = gpu_data->heat_per_atom;   
    
    int *label = gpu_data->label;
    int *fv_index = gpu_data->fv_index;
    real *fv = gpu_data->fv;

    real *f12x = gpu_data->f12x; 
    real *f12y = gpu_data->f12y; 
    real *f12z = gpu_data->f12z; 
    
    find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>
    (       
        N, pbc_x, pbc_y, pbc_z, force_model->ters0, 
        NN, NL, x, y, z, box_length, b, bp
    );

    if (para->hac.compute)
    {
        find_force_tersoff_step2<0><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z, force_model->ters0, 
            NN, NL, b, bp, x, y, z, box_length, pe, f12x, f12y, f12z
        );
        find_force_tersoff_step3<0, 1, 0><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z,
            force_model->ters0, NN, NL, 
            f12x, f12y, f12z, x, y, z, vx, vy, vz, box_length, fx, fy, fz, 
            sx, sy, sz, h, label, fv_index, fv
        );
    }
    else if (para->shc.compute)
    {
        find_force_tersoff_step2<0><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z, force_model->ters0, 
            NN, NL, b, bp, x, y, z, box_length, pe, f12x, f12y, f12z
        );
        find_force_tersoff_step3<0, 0, 1><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z,
            force_model->ters0, NN, NL, 
            f12x, f12y, f12z, x, y, z, vx, vy, vz, box_length, fx, fy, fz, 
            sx, sy, sz, h, label, fv_index, fv
        );
    }
    else
    {
        find_force_tersoff_step2<1><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z, force_model->ters0, 
            NN, NL, b, bp, x, y, z, box_length, pe, f12x, f12y, f12z
        );
        find_force_tersoff_step3<1, 0, 0><<<grid_size, BLOCK_SIZE_FORCE>>>
        (
            N, para->pbc_x, para->pbc_y, para->pbc_z,
            force_model->ters0, NN, NL, 
            f12x, f12y, f12z, x, y, z, vx, vy, vz, box_length, fx, fy, fz, 
            sx, sy, sz, h, label, fv_index, fv
        );
    }

}



