#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "vashishta.cuh"

#include "ldg.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE_VASHISHTA 64 
#ifndef USE_SP
    #define HALF  0.5
    #define ZERO  0.0
    #define ONE   1.0
    #define TWO   2.0
    #define FOUR  4.0
    #define SIX   6.0
    #define K_C   1.441959e+1
#else
    #define HALF  0.5f
    #define ZERO  0.0f
    #define ONE   1.0f
    #define TWO   2.0f
    #define FOUR  4.0f
    #define SIX   6.0f
    #define K_C   1.441959e+1f
#endif




/*----------------------------------------------------------------------------80
    Reference: 
        P. Vashishta et al., J. Appl. Phys. 101, 103515 (2007).
*-----------------------------------------------------------------------------*/




void Vashishta::initialize_0(FILE *fid)
{
    printf("Use Vashishta potential.\n");
    int count;

    double B_0, B_1, cos0_0, cos0_1, C, r0, cut;
    count = fscanf
    (fid, "%lf%lf%lf%lf%lf%lf%lf", &B_0, &B_1, &cos0_0, &cos0_1, &C, &r0, &cut);
    if (count != 7) print_error("reading error for Vashishta potential.\n");
    vashishta_para.B[0] = B_0;
    vashishta_para.B[1] = B_1;
    vashishta_para.cos0[0] = cos0_0;
    vashishta_para.cos0[1] = cos0_1;
    vashishta_para.C = C;
    vashishta_para.r0 = r0;
    vashishta_para.rc = cut;
    rc = cut;

    double H[3], qq[3], lambda_inv[3], D[3], xi_inv[3], W[3];
    int eta[3];
    for (int n = 0; n < 3; n++)
    {
        count = fscanf
        (
            fid, "%lf%d%lf%lf%lf%lf%lf", 
            &H[n], &eta[n], &qq[n], &lambda_inv[n], &D[n], &xi_inv[n], &W[n]
        );
        if (count != 7) print_error("reading error for Vashishta potential.\n");
        qq[n] *= K_C;         // Gauss -> SI
        D[n] *= (K_C * HALF); // Gauss -> SI and D -> D/2
        lambda_inv[n] = ONE / lambda_inv[n];
        xi_inv[n] = ONE / xi_inv[n];

        vashishta_para.H[n] = H[n];
        vashishta_para.eta[n] = eta[n];
        vashishta_para.qq[n] = qq[n];
        vashishta_para.lambda_inv[n] = lambda_inv[n];
        vashishta_para.D[n] = D[n];
        vashishta_para.xi_inv[n] = xi_inv[n];
        vashishta_para.W[n] = W[n];

        real rci = ONE / rc;
        real rci4 = rci * rci * rci * rci;
        real rci6 = rci4 * rci * rci;
        real p2_steric = H[n] * pow(rci, real(eta[n]));
        real p2_charge = qq[n] * rci * exp(-rc*lambda_inv[n]);
        real p2_dipole = D[n] * rci4 * exp(-rc*xi_inv[n]);
        real p2_vander = W[n] * rci6;
        vashishta_para.v_rc[n] = p2_steric+p2_charge-p2_dipole-p2_vander;
        vashishta_para.dv_rc[n] = p2_dipole * (xi_inv[n] + FOUR * rci) 
                                + p2_vander * (SIX * rci)
                                - p2_charge * (lambda_inv[n] + rci)
                                - p2_steric * (eta[n] * rci);
    }
}




// get U_ij and (d U_ij / d r_ij) / r_ij for the 2-body part
static void find_p2_and_f2_host
(
    real H, int eta, real qq, real lambda_inv, real D, real xi_inv, real W, 
    real v_rc, real dv_rc, real rc, real d12, real &p2, real &f2
)
{
    real d12inv = ONE / d12;
    real d12inv2 = d12inv * d12inv;
    real p2_steric = eta; p2_steric = H * pow(d12inv, eta);
    real p2_charge = qq * d12inv * exp(-d12 * lambda_inv);
    real p2_dipole = D * (d12inv2 * d12inv2) * exp(-d12 * xi_inv);
    real p2_vander = W * (d12inv2 * d12inv2 * d12inv2);
    p2 = p2_steric + p2_charge - p2_dipole - p2_vander; 
    p2 -= v_rc + (d12 - rc) * dv_rc; // shifted potential
    f2 = p2_dipole * (xi_inv + FOUR*d12inv) + p2_vander * (SIX * d12inv);
    f2 -= p2_charge * (lambda_inv + d12inv) + p2_steric * (eta * d12inv);
    f2 = (f2 - dv_rc) * d12inv;      // shifted force
}




void Vashishta::initialize_1(FILE *fid)
{
    printf("Use tabulated Vashishta potential.\n");
    int count;

    int N; double rmin;
    count = fscanf(fid, "%d%lf", &N, &rmin);
    if (count != 2) 
    {
        print_error("reading error for Vashishta potential.\n");
        exit(1);
    }
    vashishta_para.N = N;
    vashishta_para.rmin = rmin;

    real *table;
    MY_MALLOC(table, real, N * 6);
    
    double B_0, B_1, cos0_0, cos0_1, C, r0, cut;
    count = fscanf
    (fid, "%lf%lf%lf%lf%lf%lf%lf", &B_0, &B_1, &cos0_0, &cos0_1, &C, &r0, &cut);
    if (count != 7) print_error("reading error for Vashishta potential.\n");

    vashishta_para.B[0] = B_0;
    vashishta_para.B[1] = B_1;
    vashishta_para.cos0[0] = cos0_0;
    vashishta_para.cos0[1] = cos0_1;
    vashishta_para.C = C;
    vashishta_para.r0 = r0;
    vashishta_para.rc = cut;
    vashishta_para.scale = (N-ONE)/(cut-rmin);
    rc = cut;

    double H[3], qq[3], lambda_inv[3], D[3], xi_inv[3], W[3];
    int eta[3];
    for (int n = 0; n < 3; n++)
    {
        count = fscanf
        (
            fid, "%lf%d%lf%lf%lf%lf%lf", 
            &H[n], &eta[n], &qq[n], &lambda_inv[n], &D[n], &xi_inv[n], &W[n]
        );
        if (count != 7) print_error("reading error for Vashishta potential.\n");

        qq[n] *= K_C;         // Gauss -> SI
        D[n] *= (K_C * HALF); // Gauss -> SI and D -> D/2
        lambda_inv[n] = ONE / lambda_inv[n];
        xi_inv[n] = ONE / xi_inv[n];

        vashishta_para.H[n] = H[n];
        vashishta_para.eta[n] = eta[n];
        vashishta_para.qq[n] = qq[n];
        vashishta_para.lambda_inv[n] = lambda_inv[n];
        vashishta_para.D[n] = D[n];
        vashishta_para.xi_inv[n] = xi_inv[n];
        vashishta_para.W[n] = W[n];

        real rci = ONE / rc;
        real rci4 = rci * rci * rci * rci;
        real rci6 = rci4 * rci * rci;
        real p2_steric = H[n] * pow(rci, real(eta[n]));
        real p2_charge = qq[n] * rci * exp(-rc*lambda_inv[n]);
        real p2_dipole = D[n] * rci4 * exp(-rc*xi_inv[n]);
        real p2_vander = W[n] * rci6;
        vashishta_para.v_rc[n] = p2_steric+p2_charge-p2_dipole-p2_vander;
        vashishta_para.dv_rc[n] = p2_dipole * (xi_inv[n] + FOUR * rci) 
                                + p2_vander * (SIX * rci)
                                - p2_charge * (lambda_inv[n] + rci)
                                - p2_steric * (eta[n] * rci);

        // build the table
        for (int m = 0; m < N; m++) 
        {
            real d12 = rmin + m * (cut - rmin) / (N-ONE);
            real p2, f2;
            find_p2_and_f2_host
            (
                H[n], eta[n], qq[n], lambda_inv[n], D[n], xi_inv[n], W[n], 
                vashishta_para.v_rc[n], 
                vashishta_para.dv_rc[n], 
                rc, d12, p2, f2
            );
            int index_p = m + N * n;
            int index_f = m + N * (n + 3);
            table[index_p] = p2;
            table[index_f] = f2;
        }
    }

    int memory = sizeof(real) * N * 6;
    CHECK(hipMalloc((void**)&vashishta_data.table, memory));
    CHECK(hipMemcpy(vashishta_data.table, table, memory,
        hipMemcpyHostToDevice));
    MY_FREE(table);
}




Vashishta::Vashishta(FILE *fid, Atom* atom, int use_table_input)
{
    use_table = use_table_input;
    if (use_table == 0) initialize_0(fid);
    if (use_table == 1) initialize_1(fid);

    int num = ((atom->neighbor.MN<20) ? atom->neighbor.MN : 20);
    int memory = sizeof(real) * atom->N * num;
    CHECK(hipMalloc((void**)&vashishta_data.f12x, memory));
    CHECK(hipMalloc((void**)&vashishta_data.f12y, memory));
    CHECK(hipMalloc((void**)&vashishta_data.f12z, memory));
    memory = sizeof(int) * atom->N;
    CHECK(hipMalloc((void**)&vashishta_data.NN_short, memory));
    memory = sizeof(int) * atom->N * num;
    CHECK(hipMalloc((void**)&vashishta_data.NL_short, memory));
}




Vashishta::~Vashishta(void)
{
    if (use_table) { CHECK(hipFree(vashishta_data.table)); }
    CHECK(hipFree(vashishta_data.f12x));
    CHECK(hipFree(vashishta_data.f12y));
    CHECK(hipFree(vashishta_data.f12z));
    CHECK(hipFree(vashishta_data.NN_short));
    CHECK(hipFree(vashishta_data.NL_short));
}




// eta is always an integer and we don't need the very slow pow()
static __device__ real my_pow(real x, int n) 
{
    if (n == 7) 
    { 
        real y = x;
        x *= x;
        y *= x; // x^3
        x *= x; // x^4
        return y * x;
    }
    else if (n == 9) 
    { 
        real y = x;
        x *= x; // x^2
        x *= x; // x^4
        y *= x; // x^5
        return y * x; 
    }
    else // n = 11
    { 
        real y = x;
        x *= x; // x^2
        y *= x; // x^3
        x *= x; // x^4
        x *= x; // x^8
        return y * x; 
    }
}




// get U_ij and (d U_ij / d r_ij) / r_ij for the 2-body part
static __device__ void find_p2_and_f2
(
    real H, int eta, real qq, real lambda_inv, real D, real xi_inv, real W, 
    real v_rc, real dv_rc, real rc, real d12, real &p2, real &f2
)
{
    real d12inv = ONE / d12;
    real d12inv2 = d12inv * d12inv;
    real p2_steric = H * my_pow(d12inv, eta);
    real p2_charge = qq * d12inv * exp(-d12 * lambda_inv);
    real p2_dipole = D * (d12inv2 * d12inv2) * exp(-d12 * xi_inv);
    real p2_vander = W * (d12inv2 * d12inv2 * d12inv2);
    p2 = p2_steric + p2_charge - p2_dipole - p2_vander; 
    p2 -= v_rc + (d12 - rc) * dv_rc; // shifted potential
    f2 = p2_dipole * (xi_inv + FOUR*d12inv) + p2_vander * (SIX * d12inv);
    f2 -= p2_charge * (lambda_inv + d12inv) + p2_steric * (eta * d12inv);
    f2 = (f2 - dv_rc) * d12inv;      // shifted force
}




static __device__ void dev_apply_mic
(
    int pbc_x, int pbc_y, int pbc_z, real &x12, real &y12, real &z12, 
    real lx, real ly, real lz
)
{
    if      (pbc_x == 1 && x12 < - lx * HALF) {x12 += lx;}
    else if (pbc_x == 1 && x12 > + lx * HALF) {x12 -= lx;}
    if      (pbc_y == 1 && y12 < - ly * HALF) {y12 += ly;}
    else if (pbc_y == 1 && y12 > + ly * HALF) {y12 -= ly;}
    if      (pbc_z == 1 && z12 < - lz * HALF) {z12 += lz;}
    else if (pbc_z == 1 && z12 > + lz * HALF) {z12 -= lz;}
}




// 2-body part of the Vashishta potential (kernel)
template <int use_table, int cal_j, int cal_q, int cal_k>
static __global__ void gpu_find_force_vashishta_2body
(
    real fe_x, real fe_y, real fe_z,
    int number_of_particles, int N1, int N2, int pbc_x, int pbc_y, int pbc_z, 
    Vashishta_Para vas,
    int *g_NN, int *g_NL, int *g_NN_local, int *g_NL_local, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_table,
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
#else
    real *g_table, 
    real *g_x,  real *g_y,  real *g_z, real *g_vx, real *g_vy, real *g_vz,
#endif
    real *g_box_length, real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz, real *g_potential, 
    real *g_h, int *g_label, int *g_fv_index, real *g_fv,
    int *g_a_map, int *g_b_map, int g_count_b
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
    real s_fx = ZERO; // force_x
    real s_fy = ZERO; // force_y
    real s_fz = ZERO; // force_z
    real s_pe = ZERO; // potential energy
    real s_sx = ZERO; // virial_stress_x
    real s_sy = ZERO; // virial_stress_y
    real s_sz = ZERO; // virial_stress_z
    real s_h1 = ZERO; // heat_x_in
    real s_h2 = ZERO; // heat_x_out
    real s_h3 = ZERO; // heat_y_in
    real s_h4 = ZERO; // heat_y_out
    real s_h5 = ZERO; // heat_z

    // driving force 
    real fx_driving = ZERO;
    real fy_driving = ZERO;
    real fz_driving = ZERO;

    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_NN[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real vx1, vy1, vz1;
        if (cal_j || cal_q || cal_k)
        {
            vx1 = LDG(g_vx, n1);
            vy1 = LDG(g_vy, n1); 
            vz1 = LDG(g_vz, n1);
        }
        real lx = g_box_length[0]; 
        real ly = g_box_length[1]; 
        real lz = g_box_length[2];
        
        int count = 0; // initialize g_NN_local[n1] to 0

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = g_NL[n1 + number_of_particles * i1];
            
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            if (d12 >= vas.rc) { continue; }
            if (d12 < vas.r0) // r0 is much smaller than rc
            {                    
                g_NL_local[n1 + number_of_particles * (count++)] = n2;
            }
            int type2 = g_type[n2];
            int type12 = type1 + type2; // 0 = AA; 1 = AB or BA; 2 = BB
            real p2, f2;

            if (use_table == 1)
            {
                if (d12 > vas.rmin)
                {
                    real tmp = (d12 - vas.rmin) * vas.scale;
                    int index = tmp; // 0 <= index < N-1
                    real x = tmp - index; // 0 <= x < 1
                    index += type12 * vas.N;
                    p2 = (ONE-x)*LDG(g_table, index) + x*LDG(g_table, index+1);
                    index += vas.N * 3;    
                    f2 = (ONE-x)*LDG(g_table, index) + x*LDG(g_table, index+1);
                }
                else
                {
                    find_p2_and_f2
                    (
                        vas.H[type12], vas.eta[type12], vas.qq[type12], 
                        vas.lambda_inv[type12], vas.D[type12], 
                        vas.xi_inv[type12], vas.W[type12], vas.v_rc[type12], 
                        vas.dv_rc[type12], vas.rc, d12, p2, f2
                    );
                }
            }

            if (use_table == 0)
            {
                find_p2_and_f2
                (
                    vas.H[type12], vas.eta[type12], vas.qq[type12], 
                    vas.lambda_inv[type12], vas.D[type12], vas.xi_inv[type12],
                    vas.W[type12], vas.v_rc[type12], vas.dv_rc[type12], 
                    vas.rc, d12, p2, f2
                );
            }

            // treat two-body potential in the same way as many-body potential
            real f12x = f2 * x12 * HALF; 
            real f12y = f2 * y12 * HALF; 
            real f12z = f2 * z12 * HALF; 
            real f21x = -f12x; 
            real f21y = -f12y; 
            real f21z = -f12z; 
       
            // accumulate force
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z; 

            // driving force
            if (cal_k)
            { 
                fx_driving += f21x * (x12 * fe_x + y12 * fe_y + z12 * fe_z);
                fy_driving += f21y * (x12 * fe_x + y12 * fe_y + z12 * fe_z);
                fz_driving += f21z * (x12 * fe_x + y12 * fe_y + z12 * fe_z);
            } 
            
            // accumulate potential energy and virial
            s_pe += p2 * HALF; // two-body potential
            s_sx -= x12 * (f12x - f21x) * HALF; 
            s_sy -= y12 * (f12y - f21y) * HALF; 
            s_sz -= z12 * (f12z - f21z) * HALF;
            
            if (cal_j || cal_k) // heat current (EMD)
            {
                s_h1 += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s_h2 += (f21z * vz1) * x12;               // x-out
                s_h3 += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s_h4 += (f21z * vz1) * y12;               // y-out
                s_h5 += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            }

            // accumulate heat across some sections (for NEMD)
            // check if AB pair possible & exists
            if (cal_q && g_a_map[n1] != -1 && g_b_map[n2] != -1 &&
                g_fv_index[g_a_map[n1] * g_count_b + g_b_map[n2]] != -1)
            {
                int index_12 = 
                    g_fv_index[g_a_map[n1] * g_count_b + g_b_map[n2]] * 12;
                g_fv[index_12 + 0]  += f12x;
                g_fv[index_12 + 1]  += f12y;
                g_fv[index_12 + 2]  += f12z;
                g_fv[index_12 + 3]  += f21x;
                g_fv[index_12 + 4]  += f21y;
                g_fv[index_12 + 5]  += f21z;
                g_fv[index_12 + 6]  = vx1;
                g_fv[index_12 + 7]  = vy1;
                g_fv[index_12 + 8]  = vz1;
                g_fv[index_12 + 9]  = LDG(g_vx, n2);
                g_fv[index_12 + 10] = LDG(g_vy, n2);
                g_fv[index_12 + 11] = LDG(g_vz, n2);
            }
        }

        g_NN_local[n1] = count; // now the local neighbor list has been built

        // add driving force
        if (cal_k)
        {
            s_fx += fx_driving;
            s_fy += fy_driving;
            s_fz += fz_driving;
        }

        g_fx[n1] += s_fx; // save force
        g_fy[n1] += s_fy;
        g_fz[n1] += s_fz;

        // save stress and potential
        g_sx[n1] += s_sx;
        g_sy[n1] += s_sy;
        g_sz[n1] += s_sz;
        g_potential[n1] += s_pe;
        if (cal_j || cal_k) // save heat current
        {
            g_h[n1 + 0 * number_of_particles] += s_h1;
            g_h[n1 + 1 * number_of_particles] += s_h2;
            g_h[n1 + 2 * number_of_particles] += s_h3;
            g_h[n1 + 3 * number_of_particles] += s_h4;
            g_h[n1 + 4 * number_of_particles] += s_h5;
        }
    }
}




// calculate the partial forces dU_i/dr_ij
static __global__ void gpu_find_force_vashishta_partial
(
    int number_of_particles, int N1, int N2, int pbc_x, int pbc_y, int pbc_z, 
    Vashishta_Para vas,
    int *g_neighbor_number, int *g_neighbor_list, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
#else
    real *g_x,  real *g_y,  real *g_z,
#endif
    real *g_box_length,
    real *g_potential, real *g_f12x, real *g_f12y, real *g_f12z  
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real lx = g_box_length[0]; 
        real ly = g_box_length[1]; 
        real lz = g_box_length[2];
        real potential_energy = ZERO;

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real d12inv = ONE / d12;
          
            real f12x = ZERO; real f12y = ZERO; real f12z = ZERO;
            real gamma2 = ONE / ((d12 - vas.r0) * (d12 - vas.r0)); // gamma=1
             
            // accumulate_force_123
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];  
                if (n3 == n2) { continue; }
                int type3 = g_type[n3];           // only consider ABB and BAA
                if (type3 != type2) { continue; } // exclude AAB, BBA, ABA, BAB
                if (type3 == type1) { continue; } // exclude AAA, BBB

                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(pbc_x, pbc_y, pbc_z, x13, y13, z13, lx, ly, lz);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);

                real exp123 = exp(ONE / (d12 - vas.r0) + ONE / (d13 - vas.r0));
                real one_over_d12d13 = ONE / (d12 * d13);
                real cos123 = (x12*x13 + y12*y13 + z12*z13) * one_over_d12d13;
                real cos123_over_d12d12 = cos123*d12inv*d12inv;
                real cos_inv = cos123 - vas.cos0[type1];
                cos_inv = ONE / (ONE + vas.C * cos_inv * cos_inv);

                // accumulate potential energy
                potential_energy += (cos123 - vas.cos0[type1])
                                  * (cos123 - vas.cos0[type1])
                                  * cos_inv*HALF*vas.B[type1]*exp123;

                real tmp1=vas.B[type1]*exp123*cos_inv*(cos123-vas.cos0[type1]);
                real tmp2=gamma2 * (cos123 - vas.cos0[type1]) * d12inv;

                real cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += tmp1*(TWO*cos_d*cos_inv-tmp2*x12);
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += tmp1*(TWO*cos_d*cos_inv-tmp2*y12);
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += tmp1*(TWO*cos_d*cos_inv-tmp2*z12);
            }
            g_f12x[index] = f12x;
            g_f12y[index] = f12y;
            g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += potential_energy;
    }
}




// Find force and related quantities for the Vashishta potential (A wrapper)
void Vashishta::compute(Atom *atom, Measure *measure)
{
    int N = atom->N;
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_VASHISHTA + 1;
    int pbc_x = atom->pbc_x;
    int pbc_y = atom->pbc_y;
    int pbc_z = atom->pbc_z;
    int *NN = atom->NN_local;             // for 2-body
    int *NL = atom->NL_local;             // for 2-body
    int *NN_local = vashishta_data.NN_short;  // for 3-body
    int *NL_local = vashishta_data.NL_short;  // for 3-body
    int *type = atom->type_local;
    real *x = atom->x; 
    real *y = atom->y; 
    real *z = atom->z;
    real *vx = atom->vx; 
    real *vy = atom->vy; 
    real *vz = atom->vz;
    real *fx = atom->fx; 
    real *fy = atom->fy; 
    real *fz = atom->fz;
    real *box_length = atom->box_length;
    real *sx = atom->virial_per_atom_x; 
    real *sy = atom->virial_per_atom_y; 
    real *sz = atom->virial_per_atom_z; 
    real *pe = atom->potential_per_atom;
    real *h = atom->heat_per_atom; 
    
    int *label = atom->label;
    int *fv_index = measure->shc.fv_index;
    int *a_map = measure->shc.a_map;
    int *b_map = measure->shc.b_map;
    int count_b = measure->shc.count_b;
    real *fv = measure->shc.fv;

    real *table = vashishta_data.table;
    real *f12x  = vashishta_data.f12x; 
    real *f12y  = vashishta_data.f12y; 
    real *f12z  = vashishta_data.f12z;

    real fe_x = measure->hnemd.fe_x;
    real fe_y = measure->hnemd.fe_y;
    real fe_z = measure->hnemd.fe_z;

    if (measure->hac.compute)
    {
        if (use_table == 0)
        {
            gpu_find_force_vashishta_2body<0, 1, 0, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
        else
        {
            gpu_find_force_vashishta_2body<1, 1, 0, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
    }
    else if (measure->shc.compute && !measure->hnemd.compute)
    {
        if (use_table == 0)
        {
            gpu_find_force_vashishta_2body<0, 0, 1, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
        else
        {
            gpu_find_force_vashishta_2body<1, 0, 1, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
    }
    else if (measure->hnemd.compute && !measure->shc.compute)
    {
        if (use_table == 0)
        {
            gpu_find_force_vashishta_2body<0, 0, 0, 1>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
        else
        {
            gpu_find_force_vashishta_2body<1, 0, 0, 1>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
    }
    else if (measure->hnemd.compute && measure->shc.compute)
    {
        if (use_table == 0)
        {
            gpu_find_force_vashishta_2body<0, 0, 1, 1>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z,
                vashishta_para, NN, NL, NN_local,
                NL_local, type, table, x, y, z, vx, vy, vz, box_length,
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
        else
        {
            gpu_find_force_vashishta_2body<1, 0, 1, 1>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z,
                vashishta_para, NN, NL, NN_local,
                NL_local, type, table, x, y, z, vx, vy, vz, box_length,
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
    }
    else
    {
        if (use_table == 0)
        {
            gpu_find_force_vashishta_2body<0, 0, 0, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
        else
        {
            gpu_find_force_vashishta_2body<1, 0, 0, 0>
            <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
            (
                fe_x, fe_y, fe_z, N, N1, N2, pbc_x, pbc_y, pbc_z, 
                vashishta_para, NN, NL, NN_local, 
                NL_local, type, table, x, y, z, vx, vy, vz, box_length, 
                fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv,
                a_map, b_map, count_b
            );
            CUDA_CHECK_KERNEL
        }
    }

    gpu_find_force_vashishta_partial
    <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
    (
        N, N1, N2, pbc_x, pbc_y, pbc_z, 
        vashishta_para, NN_local, NL_local, type,
        x, y, z, box_length, pe, f12x, f12y, f12z 
    );
    CUDA_CHECK_KERNEL

    // 3-body part
    find_properties_many_body
    (atom, measure, NN_local, NL_local, f12x, f12y, f12z);
}




