#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.h"
#include "mic.cu"
#include "vashishta.h"




// best block size here: 64 or 128
#define BLOCK_SIZE_VASHISHTA 64




/*----------------------------------------------------------------------------80
    Reference: 
        P. Vashishta et al., J. Appl. Phys. 101, 103515 (2007).
*-----------------------------------------------------------------------------*/




// eta is always an integer and we don't need the very slow pow()
static __device__ real my_pow(real x, int n) 
{
    if (n == 7) 
    { 
        real y = x;
        x *= x;
        y *= x; // x^3
        x *= x; // x^4
        return y * x;
    }
    else if (n == 9) 
    { 
        real y = x;
        x *= x; // x^2
        x *= x; // x^4
        y *= x; // x^5
        return y * x; 
    }
    else // n = 11
    { 
        real y = x;
        x *= x; // x^2
        y *= x; // x^3
        x *= x; // x^4
        x *= x; // x^8
        return y * x; 
    }
}




// get U_ij and (d U_ij / d r_ij) / r_ij for the 2-body part
static __device__ void find_p2_and_f2
(
    real H, int eta, real qq, real lambda_inv, real D, real xi_inv, real W, 
    real v_rc, real dv_rc, real rc, real d12, real &p2, real &f2
)
{
    real d12inv = ONE / d12;
    real d12inv2 = d12inv * d12inv;
    // real p2_steric = eta; p2_steric = H * pow(d12inv, p2_steric); // slow
    real p2_steric = H * my_pow(d12inv, eta); // super fast
    real p2_charge = qq * d12inv * exp(-d12 * lambda_inv);
    real p2_dipole = D * (d12inv2 * d12inv2) * exp(-d12 * xi_inv);
    real p2_vander = W * (d12inv2 * d12inv2 * d12inv2);
    p2 = p2_steric + p2_charge - p2_dipole - p2_vander; 
    p2 -= v_rc + (d12 - rc) * dv_rc; // shifted potential
    f2 = p2_dipole * (xi_inv + FOUR*d12inv) + p2_vander * (SIX * d12inv);
    f2 -= p2_charge * (lambda_inv + d12inv) + p2_steric * (eta * d12inv);
    f2 = (f2 - dv_rc) * d12inv;      // shifted force
}




// 2-body part of the Vashishta potential (kernel)
template <int cal_p, int cal_j, int cal_q>
static __global__ void gpu_find_force_vashishta_2body
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z, Vashishta vas,
    int *g_NN, int *g_NL, int *g_NN_local, int *g_NL_local, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
#else
    real *g_x,  real *g_y,  real *g_z, real *g_vx, real *g_vy, real *g_vz,
#endif
    real *g_box_length, real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz, real *g_potential, 
    real *g_h, int *g_label, int *g_fv_index, real *g_fv 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x; // particle index

    __shared__ real s_fx[BLOCK_SIZE_VASHISHTA];
    __shared__ real s_fy[BLOCK_SIZE_VASHISHTA];
    __shared__ real s_fz[BLOCK_SIZE_VASHISHTA];
    // if cal_p, then s1~s4 = px, py, pz, U; if cal_j, then s1~s5 = j1~j5
    __shared__ real s1[BLOCK_SIZE_VASHISHTA];
    __shared__ real s2[BLOCK_SIZE_VASHISHTA];
    __shared__ real s3[BLOCK_SIZE_VASHISHTA];
    __shared__ real s4[BLOCK_SIZE_VASHISHTA];
    __shared__ real s5[BLOCK_SIZE_VASHISHTA];

    s_fx[threadIdx.x] = ZERO; 
    s_fy[threadIdx.x] = ZERO; 
    s_fz[threadIdx.x] = ZERO;  
    s1[threadIdx.x] = ZERO; 
    s2[threadIdx.x] = ZERO; 
    s3[threadIdx.x] = ZERO;
    s4[threadIdx.x] = ZERO;
    s5[threadIdx.x] = ZERO;

    if (n1 < number_of_particles)
    {
        int neighbor_number = g_NN[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real vx1 = LDG(g_vx, n1); 
        real vy1 = LDG(g_vy, n1); 
        real vz1 = LDG(g_vz, n1);
        real lx = g_box_length[0]; 
        real ly = g_box_length[1]; 
        real lz = g_box_length[2];
        
        int count = 0; // initialize g_NN_local[n1] to 0

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = g_NL[n1 + number_of_particles * i1];
            
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            if (d12 >= vas.rc) { continue; }
            if (d12 < vas.r0) // r0 is much smaller than rc
            {                    
                g_NL_local[n1 + number_of_particles * (count++)] = n2;
            }
            int type2 = g_type[n2];
            int type12 = type1 + type2; // 0 = AA; 1 = AB or BA; 2 = BB
            real p2, f2;
            find_p2_and_f2
            (
                vas.H[type12], vas.eta[type12], vas.qq[type12], 
                vas.lambda_inv[type12], vas.D[type12], vas.xi_inv[type12],
                vas.W[type12], vas.v_rc[type12], vas.dv_rc[type12], 
                vas.rc, d12, p2, f2
            );	    

            // treat two-body potential in the same way as many-body potential
            real f12x = f2 * x12 * HALF; 
            real f12y = f2 * y12 * HALF; 
            real f12z = f2 * z12 * HALF; 
            real f21x = -f12x; 
            real f21y = -f12y; 
            real f21z = -f12z; 
       
            // accumulate force
            s_fx[threadIdx.x] += f12x - f21x; 
            s_fy[threadIdx.x] += f12y - f21y; 
            s_fz[threadIdx.x] += f12z - f21z; 
            
            // accumulate potential energy and virial
            if (cal_p) 
            {
                s4[threadIdx.x] += p2 * HALF; // two-body potential
                s1[threadIdx.x] -= x12 * (f12x - f21x) * HALF; 
                s2[threadIdx.x] -= y12 * (f12y - f21y) * HALF; 
                s3[threadIdx.x] -= z12 * (f12z - f21z) * HALF;
            }
            
            if (cal_j) // heat current (EMD)
            {
                s1[threadIdx.x] += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s2[threadIdx.x] += (f21z * vz1) * x12;               // x-out
                s3[threadIdx.x] += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s4[threadIdx.x] += (f21z * vz1) * y12;               // y-out
                s5[threadIdx.x] += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            } 

            if (cal_q) // heat across some section (NEMD)
            {
                int index_12 = g_fv_index[n1] * 12;
                if (index_12 >= 0 && g_fv_index[n1 + number_of_particles] == n2)
                {
                    g_fv[index_12 + 0]  = f12x;
                    g_fv[index_12 + 1]  = f12y;
                    g_fv[index_12 + 2]  = f12z;
                    g_fv[index_12 + 3]  = f21x;
                    g_fv[index_12 + 4]  = f21y;
                    g_fv[index_12 + 5]  = f21z;
                    g_fv[index_12 + 6]  = vx1;
                    g_fv[index_12 + 7]  = vy1;
                    g_fv[index_12 + 8]  = vz1;
                    g_fv[index_12 + 9]  = LDG(g_vx, n2);
                    g_fv[index_12 + 10] = LDG(g_vy, n2);
                    g_fv[index_12 + 11] = LDG(g_vz, n2);
                }  
            }
        }

        g_NN_local[n1] = count; // now the local neighbor list has been built

        g_fx[n1] = s_fx[threadIdx.x]; // save force
        g_fy[n1] = s_fy[threadIdx.x]; 
        g_fz[n1] = s_fz[threadIdx.x];  
        if (cal_p) // save stress and potential
        {
            g_sx[n1] = s1[threadIdx.x]; 
            g_sy[n1] = s2[threadIdx.x]; 
            g_sz[n1] = s3[threadIdx.x];
            g_potential[n1] = s4[threadIdx.x];
        }
        if (cal_j) // save heat current
        {
            g_h[n1 + 0 * number_of_particles] = s1[threadIdx.x];
            g_h[n1 + 1 * number_of_particles] = s2[threadIdx.x];
            g_h[n1 + 2 * number_of_particles] = s3[threadIdx.x];
            g_h[n1 + 3 * number_of_particles] = s4[threadIdx.x];
            g_h[n1 + 4 * number_of_particles] = s5[threadIdx.x];
        }
    }
}    




// 3-body part of the Vashishta potential (kernel)
template <int cal_p, int cal_j, int cal_q>
static __global__ void gpu_find_force_vashishta_3body
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z, Vashishta vas,
    int *g_neighbor_number, int *g_neighbor_list, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
#else
    real *g_x,  real *g_y,  real *g_z, real *g_vx, real *g_vy, real *g_vz,
#endif
    real *g_box_length, real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz, real *g_potential, 
    real *g_h, int *g_label, int *g_fv_index, real *g_fv 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x; // particle index

    __shared__ real s_fx[BLOCK_SIZE_VASHISHTA];
    __shared__ real s_fy[BLOCK_SIZE_VASHISHTA];
    __shared__ real s_fz[BLOCK_SIZE_VASHISHTA];
    // if cal_p, then s1~s4 = px, py, pz, U; if cal_j, then s1~s5 = j1~j5
    __shared__ real s1[BLOCK_SIZE_VASHISHTA];
    __shared__ real s2[BLOCK_SIZE_VASHISHTA];
    __shared__ real s3[BLOCK_SIZE_VASHISHTA];
    __shared__ real s4[BLOCK_SIZE_VASHISHTA];
    __shared__ real s5[BLOCK_SIZE_VASHISHTA];

    s_fx[threadIdx.x] = ZERO; 
    s_fy[threadIdx.x] = ZERO; 
    s_fz[threadIdx.x] = ZERO;  
    s1[threadIdx.x] = ZERO; 
    s2[threadIdx.x] = ZERO; 
    s3[threadIdx.x] = ZERO;
    s4[threadIdx.x] = ZERO;
    s5[threadIdx.x] = ZERO;

    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real vx1 = LDG(g_vx, n1); 
        real vy1 = LDG(g_vy, n1); 
        real vz1 = LDG(g_vz, n1);
        real lx = g_box_length[0]; 
        real ly = g_box_length[1]; 
        real lz = g_box_length[2];

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int neighbor_number_2 = g_neighbor_number[n2];
            int type2 = g_type[n2];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
          
            real f12x = ZERO; real f12y = ZERO; real f12z = ZERO; 
            real f21x = ZERO; real f21y = ZERO; real f21z = ZERO; 
            real gamma2 = ONE / ((d12 - vas.r0) * (d12 - vas.r0)); // gamma=1
             
            // accumulate_force_123
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {       
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];  
                if (n3 == n2) { continue; }
                int type3 = g_type[n3];           // only consider ABB and BAA
                if (type3 != type2) { continue; } // exclude AAB, BBA, ABA, BAB
                if (type3 == type1) { continue; } // exclude AAA, BBB

                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(pbc_x, pbc_y, pbc_z, x13, y13, z13, lx, ly, lz);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);

                real exp123 = exp(ONE / (d12 - vas.r0) + ONE / (d13 - vas.r0));
                real cos123 = (x12*x13 + y12*y13 + z12*z13) / (d12*d13);
                real cos_inv = cos123 - vas.cos0[type1];
                cos_inv = ONE / (ONE + vas.C * cos_inv * cos_inv); 
				    
                if (cal_p) // accumulate potential energy
                {
                    s4[threadIdx.x] += (cos123 - vas.cos0[type1])
		                     * (cos123 - vas.cos0[type1])
                                     * cos_inv*HALF*vas.B[type1]*exp123;
                }
 
                real cos_d = x13 / (d12 * d13) - x12 * cos123 / (d12 * d12); 	 
                f12x += vas.B[type1]*exp123*cos_inv*(cos123-vas.cos0[type1])*
                    (TWO*cos_d*cos_inv-gamma2*(cos123-vas.cos0[type1])*x12/d12);
                cos_d = y13 / (d12 * d13) - y12 * cos123 / (d12 * d12);
                f12y += vas.B[type1]*exp123*cos_inv*(cos123-vas.cos0[type1])*
                    (TWO*cos_d*cos_inv-gamma2*(cos123-vas.cos0[type1])*y12/d12);
                cos_d = z13 / (d12 * d13) - z12 * cos123 / (d12 * d12);
                f12z += vas.B[type1]*exp123*cos_inv*(cos123-vas.cos0[type1])*
                    (TWO*cos_d*cos_inv-gamma2*(cos123-vas.cos0[type1])*z12/d12);
            }

            // accumulate_force_213
            for (int i2 = 0; i2 < neighbor_number_2; ++i2)
            {
                int n3 = g_neighbor_list[n2 + number_of_particles * i2];        
                if (n3 == n1) { continue; } 
                int type3 = g_type[n3];
                if (type3 != type1) { continue; } // exclude AAB, BBA, ABA, BAB
                if (type3 == type2) { continue; } // exclude AAA, BBB

                real x23 = LDG(g_x, n3) - LDG(g_x, n2);
                real y23 = LDG(g_y, n3) - LDG(g_y, n2);
                real z23 = LDG(g_z, n3) - LDG(g_z, n2);
                dev_apply_mic(pbc_x, pbc_y, pbc_z, x23, y23, z23, lx, ly, lz);
                real d23 = sqrt(x23 * x23 + y23 * y23 + z23 * z23); 
				       
                real exp213 = exp(ONE / (d12 - vas.r0) + ONE / (d23 - vas.r0));
                real cos213 = -(x12*x23 + y12*y23 + z12*z23) / (d12*d23); 
                real cos_inv = cos213 - vas.cos0[type2];
                cos_inv = ONE / (ONE + vas.C * cos_inv * cos_inv);               
                
                real cos_d = x23 / (d12 * d23) + x12 * cos213 / (d12 * d12);
                f21x += vas.B[type2]*exp213*cos_inv*(cos213-vas.cos0[type2])*
                    (TWO*cos_d*cos_inv+gamma2*(cos213-vas.cos0[type2])*x12/d12);
                cos_d = y23 / (d12 * d23) + y12 * cos213 / (d12 * d12);
                f21y += vas.B[type2]*exp213*cos_inv*(cos213-vas.cos0[type2])*
                    (TWO*cos_d*cos_inv+gamma2*(cos213-vas.cos0[type2])*y12/d12);
                cos_d = z23 / (d12 * d23) + z12 * cos213 / (d12 * d12);
                f21z += vas.B[type2]*exp213*cos_inv*(cos213-vas.cos0[type2])*
                    (TWO*cos_d*cos_inv+gamma2*(cos213-vas.cos0[type2])*z12/d12);
            }  
               
            
            s_fx[threadIdx.x] += f12x - f21x; // accumulate force
            s_fy[threadIdx.x] += f12y - f21y; 
            s_fz[threadIdx.x] += f12z - f21z; 
            
            if (cal_p) // accumulate virial
            {
                s1[threadIdx.x] -= x12 * (f12x - f21x) * HALF; 
                s2[threadIdx.x] -= y12 * (f12y - f21y) * HALF; 
                s3[threadIdx.x] -= z12 * (f12z - f21z) * HALF;
            }
            
            if (cal_j) // heat current (EMD)
            {
                s1[threadIdx.x] += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s2[threadIdx.x] += (f21z * vz1) * x12;               // x-out
                s3[threadIdx.x] += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s4[threadIdx.x] += (f21z * vz1) * y12;               // y-out
                s5[threadIdx.x] += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            } 

            if (cal_q) // heat current (NEMD)
            {
                int index_12 = g_fv_index[n1] * 12;
                if (index_12 >= 0 && g_fv_index[n1 + number_of_particles] == n2)
                {
                    g_fv[index_12 + 0]  = f12x;
                    g_fv[index_12 + 1]  = f12y;
                    g_fv[index_12 + 2]  = f12z;
                    g_fv[index_12 + 3]  = f21x;
                    g_fv[index_12 + 4]  = f21y;
                    g_fv[index_12 + 5]  = f21z;
                    g_fv[index_12 + 6]  = vx1;
                    g_fv[index_12 + 7]  = vy1;
                    g_fv[index_12 + 8]  = vz1;
                    g_fv[index_12 + 9]  = LDG(g_vx, n2);
                    g_fv[index_12 + 10] = LDG(g_vy, n2);
                    g_fv[index_12 + 11] = LDG(g_vz, n2);
                }  
            }
        }

        // accumulate on top of the 2-body part (hence += instead of =)
        g_fx[n1] += s_fx[threadIdx.x]; // accumulate force
        g_fy[n1] += s_fy[threadIdx.x]; 
        g_fz[n1] += s_fz[threadIdx.x];  
        if (cal_p) // accumulate stress and potential
        {
            g_sx[n1] += s1[threadIdx.x]; 
            g_sy[n1] += s2[threadIdx.x]; 
            g_sz[n1] += s3[threadIdx.x];
            g_potential[n1] += s4[threadIdx.x];
        }
        if (cal_j) // accumulate heat current
        {
            g_h[n1 + 0 * number_of_particles] += s1[threadIdx.x];
            g_h[n1 + 1 * number_of_particles] += s2[threadIdx.x];
            g_h[n1 + 2 * number_of_particles] += s3[threadIdx.x];
            g_h[n1 + 3 * number_of_particles] += s4[threadIdx.x];
            g_h[n1 + 4 * number_of_particles] += s5[threadIdx.x];
        }
    }
}    
 



// Find force and related quantities for the Vashishta potential (A wrapper)
void gpu_find_force_vashishta
(Parameters *para, Vashishta vas, GPU_Data *gpu_data)
{
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE_VASHISHTA + 1;
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    int *NN = gpu_data->NN;             // for 2-body
    int *NL = gpu_data->NL;             // for 2-body
    int *NN_local = gpu_data->NN_local; // for 3-body
    int *NL_local = gpu_data->NL_local; // for 3-body
    int *type = gpu_data->type;
    real *x = gpu_data->x; 
    real *y = gpu_data->y; 
    real *z = gpu_data->z;
    real *vx = gpu_data->vx; 
    real *vy = gpu_data->vy; 
    real *vz = gpu_data->vz;
    real *fx = gpu_data->fx; 
    real *fy = gpu_data->fy; 
    real *fz = gpu_data->fz;
    real *box_length = gpu_data->box_length;
    real *sx = gpu_data->virial_per_atom_x; 
    real *sy = gpu_data->virial_per_atom_y; 
    real *sz = gpu_data->virial_per_atom_z; 
    real *pe = gpu_data->potential_per_atom;
    real *h = gpu_data->heat_per_atom; 
    
    int *label = gpu_data->label;
    int *fv_index = gpu_data->fv_index;
    real *fv = gpu_data->fv;
           
    if (para->hac.compute)    
    {
        gpu_find_force_vashishta_2body<0, 1, 0>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN, NL, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );

        gpu_find_force_vashishta_3body<0, 1, 0>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );
    }
    else if (para->shc.compute)
    {
        gpu_find_force_vashishta_2body<0, 0, 1>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN, NL, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );

        gpu_find_force_vashishta_3body<0, 0, 1>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );
    }
    else
    {
        gpu_find_force_vashishta_2body<1, 0, 0>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN, NL, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );

        gpu_find_force_vashishta_3body<1, 0, 0>
        <<<grid_size, BLOCK_SIZE_VASHISHTA>>>
        (
            N, pbc_x, pbc_y, pbc_z, vas, NN_local, NL_local, type, 
            x, y, z, vx, vy, vz, box_length, fx, fy, fz, sx, sy, sz, pe, h, 
            label, fv_index, fv
        );
         
    }

    #ifdef DEBUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif
}




