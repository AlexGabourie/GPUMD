/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "measure.cuh"
#include "heat.cuh"
#include "vac.cuh"
#include "hac.cuh"
#include "shc.cuh"
#include "hnemd_kappa.cuh"

#define DIM 3
#define NUM_OF_HEAT_COMPONENTS 5




static FILE *my_fopen(const char *filename, const char *mode)
{
    FILE *fid = fopen(filename, mode);
    if (fid == NULL) 
    {
        printf ("Failed to open %s!\n", filename);
        printf ("%s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    return fid;
}




Measure::Measure(char *input_dir)
{
    dump_thermo = 0;
    dump_position = 0;
    dump_velocity = 0;
    dump_force = 0;
    dump_potential = 0;
    dump_virial = 0;
    dump_heat = 0;

    strcpy(file_thermo, input_dir);
    strcpy(file_position, input_dir);
    strcpy(file_velocity, input_dir);
    strcpy(file_force, input_dir);
    strcpy(file_potential, input_dir);
    strcpy(file_virial, input_dir);
    strcpy(file_heat, input_dir);

    strcat(file_thermo, "/thermo.out");
    strcat(file_position, "/xyz.out");
    strcat(file_velocity, "/v.out");
    strcat(file_force, "/f.out");
    strcat(file_potential, "/potential.out");
    strcat(file_virial, "/virial.out");
    strcat(file_heat, "/heat.out");
}




Measure::~Measure(void)
{
    // nothing
}




void Measure::initialize
(Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data)
{
    if (dump_thermo)    {fid_thermo   = my_fopen(file_thermo,   "a");}
    if (dump_position)  {fid_position = my_fopen(file_position, "a");}
    if (dump_velocity)  {fid_velocity = my_fopen(file_velocity, "a");}
    if (dump_force)     {fid_force    = my_fopen(file_force,    "a");}
    if (dump_potential) {fid_potential= my_fopen(file_potential,"a");}
    if (dump_virial)    {fid_virial   = my_fopen(file_virial,   "a");}
    if (dump_heat)      {fid_heat     = my_fopen(file_heat,     "a");}

    preprocess_vac(para,  cpu_data, gpu_data);
    preprocess_hac(para,  cpu_data, gpu_data);  
    preprocess_shc(para,  cpu_data, gpu_data); 
    preprocess_heat(para, cpu_data);      
    preprocess_hnemd_kappa(para, cpu_data, gpu_data);  
}




void Measure::finalize
(
    char *input_dir, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, 
    Integrate *integrate
)
{
    if (dump_thermo)    {fclose(fid_thermo);    dump_thermo    = 0;}
    if (dump_position)  {fclose(fid_position);  dump_position  = 0;}
    if (dump_velocity)  {fclose(fid_velocity);  dump_velocity  = 0;}
    if (dump_force)     {fclose(fid_force);     dump_force     = 0;}
    if (dump_potential) {fclose(fid_potential); dump_potential = 0;}
    if (dump_virial)    {fclose(fid_virial);    dump_virial    = 0;}
    if (dump_heat)      {fclose(fid_heat);      dump_heat      = 0;}

    postprocess_vac(input_dir, para, cpu_data, gpu_data);
    postprocess_hac(input_dir, para, cpu_data, gpu_data, integrate);
    postprocess_shc(para, cpu_data, gpu_data);
    postprocess_heat(input_dir, para, cpu_data, integrate);
    postprocess_hnemd_kappa(para, cpu_data, gpu_data);
}




// dump thermodynamic properties
static void gpu_sample_thermo
(
    FILE *fid, Parameters *para, CPU_Data *cpu_data, 
    real *gpu_thermo, real *gpu_box_length, Ensemble *ensemble
)
{

    // copy data from GPU to CPU
    real *thermo = cpu_data->thermo;
    real *box_length = cpu_data->box_length;
    int m1 = sizeof(real) * 6;
    int m2 = sizeof(real) * DIM;
    CHECK(hipMemcpy(thermo, gpu_thermo, m1, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(box_length, gpu_box_length, m2, hipMemcpyDeviceToHost));

    // system energies
    real energy_system_kin = (HALF * DIM) * para->N * K_B * thermo[0];
    real energy_system_pot = thermo[1];
    real energy_system_total = energy_system_kin + energy_system_pot; 

    if (ensemble->type == 2)
    {
        // energy of the Nose-Hoover chain thermostat
        real kT = K_B * ensemble->temperature; 
        real energy_nhc = kT * (DIM * para->N) * ensemble->pos_nhc1[0];
        for (int m = 1; m < NOSE_HOOVER_CHAIN_LENGTH; m++)
        {
            energy_nhc += kT * ensemble->pos_nhc1[m];
        }
        for (int m = 0; m < NOSE_HOOVER_CHAIN_LENGTH; m++)
        { 
            energy_nhc += 0.5 * ensemble->vel_nhc1[m] 
                        * ensemble->vel_nhc1[m] / ensemble->mas_nhc1[m];
        }
        fprintf
        (
            fid, "%20.10e%20.10e%20.10e", thermo[0], 
            energy_system_total, energy_nhc
        );
    }
    else
    {
        fprintf
        (
            fid, "%20.10e%20.10e%20.10e", thermo[0], 
            energy_system_kin, energy_system_pot
        );
    }    

    fprintf // presure (x, y, z)
    (
        fid, "%20.10e%20.10e%20.10e", 
        thermo[2] * PRESSURE_UNIT_CONVERSION, 
        thermo[3] * PRESSURE_UNIT_CONVERSION, 
        thermo[4] * PRESSURE_UNIT_CONVERSION
    ); 

    // box length (x, y, z)
    fprintf
    (
        fid, "%20.10e%20.10e%20.10e\n", 
        box_length[0], box_length[1], box_length[2]
    ); 

    fflush(fid);
}




// dump thermodynamic properties (A wrapper function)
void Measure::dump_thermos
(
    FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, 
    Integrate *integrate, int step
)
{
    if (dump_thermo)
    {
        if ((step + 1) % sample_interval_thermo == 0)
        {
            gpu_sample_thermo
            (
                fid, para, cpu_data, gpu_data->thermo, gpu_data->box_length, 
                integrate->ensemble
            );
        }
    }
}




static void gpu_dump_3(int N, FILE *fid, real *a, real *b, real *c)
{
    real *cpu_a, *cpu_b, *cpu_c;
    MY_MALLOC(cpu_a, real, N);
    MY_MALLOC(cpu_b, real, N);
    MY_MALLOC(cpu_c, real, N);
    CHECK(hipMemcpy(cpu_a, a, sizeof(real) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_b, b, sizeof(real) * N, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_c, c, sizeof(real) * N, hipMemcpyDeviceToHost));

    for (int n = 0; n < N; n++)
    {
        fprintf(fid, "%20.10e%20.10e%20.10e\n", cpu_a[n], cpu_b[n], cpu_c[n]);
    }
    fflush(fid);

    MY_FREE(cpu_a);
    MY_FREE(cpu_b);
    MY_FREE(cpu_c);
}




void Measure::dump_positions
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_position)
    {
        if ((step + 1) % sample_interval_position == 0)
        {
            gpu_dump_3(para->N, fid, gpu_data->x, gpu_data->y, gpu_data->z);
        }
    }
}




void Measure::dump_velocities
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_velocity)
    {
        if ((step + 1) % sample_interval_velocity == 0)
        {
            gpu_dump_3(para->N, fid, gpu_data->vx, gpu_data->vy, gpu_data->vz);
        }
    }
}




void Measure::dump_forces
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_force)
    {
        if ((step + 1) % sample_interval_force == 0)
        {
            gpu_dump_3(para->N, fid, gpu_data->fx, gpu_data->fy, gpu_data->fz);
        }
    }
}




void Measure::dump_virials
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_virial)
    {
        if ((step + 1) % sample_interval_virial == 0)
        {
            gpu_dump_3
            (
                para->N, fid, gpu_data->virial_per_atom_x, 
                gpu_data->virial_per_atom_y, gpu_data->virial_per_atom_z
            );
        }
    }
}




static real get_volume(real *box_gpu)
{
    real *box_cpu;
    MY_MALLOC(box_cpu, real, 3);
    hipMemcpy(box_cpu, box_gpu, sizeof(real) * 3, hipMemcpyDeviceToHost);
    real volume = box_cpu[0] * box_cpu[1] * box_cpu[2];
    MY_FREE(box_cpu);
    return volume;
}




void Measure::dump_heats
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_heat)
    {
        if (step == 0)
        {
            for (int n = 0; n < para->N * NUM_OF_HEAT_COMPONENTS; n++)
            {
                cpu_data->heat_per_atom[n] = ZERO;
            }
        }

        if ((step + 1) % sample_interval_heat == 0)
        {
            real *heat_cpu;
            MY_MALLOC(heat_cpu, real, para->N * NUM_OF_HEAT_COMPONENTS);
            hipMemcpy
            (
                heat_cpu, gpu_data->heat_per_atom, 
                sizeof(real) * para->N * NUM_OF_HEAT_COMPONENTS, 
                hipMemcpyDeviceToHost
            );
            for (int n = 0; n < para->N * NUM_OF_HEAT_COMPONENTS; n++)
            {
                cpu_data->heat_per_atom[n] += heat_cpu[n];
            }

            if ((step + 1) == para->number_of_steps)
            {
                int num = para->number_of_steps / sample_interval_heat;
                real volume = get_volume(gpu_data->box_length) / para->N;
                real factor = 10 * KAPPA_UNIT_CONVERSION / (num * volume);
                for (int n = 0; n < para->N; n++)
                {
                    for (int k = 0; k < NUM_OF_HEAT_COMPONENTS; k++)
                    {
                        // output per-atom heat flux in units of GW/m^2
                        real tmp = cpu_data->heat_per_atom[k * para->N + n];
                        fprintf(fid, "%25.15f", tmp * factor);
                    }
                    fprintf(fid, "\n");
                }
                fflush(fid);
            }
            MY_FREE(heat_cpu);
        }
    }
}




static void gpu_dump_1(int N, FILE *fid, real *a)
{
    real *cpu_a;
    MY_MALLOC(cpu_a, real, N);
    CHECK(hipMemcpy(cpu_a, a, sizeof(real) * N, hipMemcpyDeviceToHost));
    for (int n = 0; n < N; n++)
    {
        fprintf(fid, "%20.10e\n", cpu_a[n]);
    }
    fflush(fid);
    MY_FREE(cpu_a);
}




void Measure::dump_potentials
(FILE *fid, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, int step)
{
    if (dump_potential)
    {
        if ((step + 1) % sample_interval_potential == 0)
        {
            gpu_dump_1(para->N, fid, gpu_data->potential_per_atom);
        }
    }
}




void Measure::compute
(
    char *input_dir, Parameters *para, CPU_Data *cpu_data, GPU_Data *gpu_data, 
    Integrate *integrate, int step
)
{
    dump_thermos(fid_thermo, para, cpu_data, gpu_data, integrate, step);
    dump_positions(fid_position, para, cpu_data, gpu_data, step);
    dump_velocities(fid_velocity, para, cpu_data, gpu_data, step);
    dump_forces(fid_force, para, cpu_data, gpu_data, step);
    dump_potentials(fid_potential, para, cpu_data, gpu_data, step);
    dump_virials(fid_virial, para, cpu_data, gpu_data, step);
    dump_heats(fid_heat, para, cpu_data, gpu_data, step);

    sample_vac(step, para, cpu_data, gpu_data);
    sample_hac(step, input_dir, para, cpu_data, gpu_data);
    sample_block_temperature(step, para, cpu_data, gpu_data, integrate);
    process_shc(step, input_dir, para, cpu_data, gpu_data);
    process_hnemd_kappa(step, input_dir, para, cpu_data, gpu_data, integrate); 
}




