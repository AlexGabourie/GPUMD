#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.cuh"
#include "mic.cuh" // static __device__ dev_apply_mic(...)
#include "ri.cuh"

// References: 
// [1] Wolf
// [2] Fennell

#define USE_MY_ERFC // 20%-30% faster

// best block size here: 128
#define BLOCK_SIZE_RI 128

#ifdef USE_DP
    #define RI_ALPHA     0.2
    #define RI_ALPHA_SQ  0.04
    #define RI_PI_FACTOR 0.225675833419103 // ALPHA * 2 / SQRT(PI)
    #define RI_a1        0.254829592
    #define RI_a2        0.284496736
    #define RI_a3        1.421413741
    #define RI_a4        1.453152027
    #define RI_a5        1.061405429
    #define RI_p         0.3275911  
#else
    #define RI_ALPHA     0.2f
    #define RI_ALPHA_SQ  0.04f
    #define RI_PI_FACTOR 0.225675833419103f // ALPHA * 2 / SQRT(PI)
    #define RI_a1        0.254829592f
    #define RI_a2        0.284496736f
    #define RI_a3        1.421413741f
    #define RI_a4        1.453152027f
    #define RI_a5        1.061405429f
    #define RI_p         0.3275911f 
#endif


// TODO: there is plenty space for improving the performance
// get U_ij and (d U_ij / d r_ij) / r_ij
static __device__ void find_p2_and_f2
(int type1, int type2, RI ri, real d12sq, real &p2, real &f2)
{
    real a, b, c, qq; 
    if (type1 == 0 && type2 == 0)
    {
        a  = ri.a11; 
        b  = ri.b11; 
        c  = ri.c11; 
        qq = ri.qq11;
    }
    else if (type1 == 1 && type2 == 1)
    {
        a  = ri.a22; 
        b  = ri.b22; 
        c  = ri.c22; 
        qq = ri.qq22;
    }  
    else 
    {
        a  = ri.a12; 
        b  = ri.b12; 
        c  = ri.c12; 
        qq = ri.qq12;
    }  

    real d12         = sqrt(d12sq);     
    real exponential = exp(-d12 * b);     // b = 1/rho
    p2 = a * exponential - c / (d12sq * d12sq * d12sq);
    f2 = SIX * c / (d12sq * d12sq * d12sq * d12sq);
    c = ONE / d12; // reuse c
    f2 -= a * exponential * b * c;
    a = RI_ALPHA * ri.cutoff; // reuse a
    b = ONE / ri.cutoff; // reuse b
    
#ifndef USE_MY_ERFC // use the erfc function in CUDA
    real erfc_r = erfc(RI_ALPHA * d12) * c;
    real erfc_R = erfc(a) * b; 
    real exp_r  = RI_PI_FACTOR * c * exp(-RI_ALPHA_SQ * d12sq);
    real exp_R  = RI_PI_FACTOR * b * exp(-a * a);
#else // use my own erfc function 
    real exp_r = exp(-RI_ALPHA_SQ * d12sq) * c;
    real exp_R = exp(-a * a) * b;
    real t = ONE / (RI_p * RI_ALPHA * d12 + ONE);
    real erfc_r = ((((RI_a5*t - RI_a4)*t + RI_a3)*t - RI_a2)*t + RI_a1)*t*exp_r;
    t = ONE / (RI_p * a + ONE);
    real erfc_R = ((((RI_a5*t - RI_a4)*t + RI_a3)*t - RI_a2)*t + RI_a1)*t*exp_R;
    exp_r = RI_PI_FACTOR * exp_r;
    exp_R = RI_PI_FACTOR * exp_R;
#endif
    
    p2 += qq * ( erfc_r - erfc_R + (erfc_R * b + exp_R) * (d12 - ri.cutoff) );
    f2 += (erfc_R * b - erfc_r * c + exp_R - exp_r) * (qq * c);
}


// force evaluation kernel for the RI potential
template <int cal_p, int cal_j, int cal_q>
static __global__ void gpu_find_force
(
    int number_of_particles, int pbc_x, int pbc_y, int pbc_z, RI ri,
    int *g_neighbor_number, int *g_neighbor_list, int *g_type,
#ifdef USE_LDG
    const real* __restrict__ g_x, 
    const real* __restrict__ g_y, 
    const real* __restrict__ g_z, 
    const real* __restrict__ g_vx, 
    const real* __restrict__ g_vy, 
    const real* __restrict__ g_vz,
#else
    real *g_x,  real *g_y,  real *g_z, real *g_vx, real *g_vy, real *g_vz,
#endif
    real *g_box, real *g_fx, real *g_fy, real *g_fz,
    real *g_sx, real *g_sy, real *g_sz, real *g_potential, 
    real *g_h, int *g_label, int *g_fv_index, real *g_fv 
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x; // particle index

    __shared__ real s_fx[BLOCK_SIZE_RI];
    __shared__ real s_fy[BLOCK_SIZE_RI];
    __shared__ real s_fz[BLOCK_SIZE_RI];
    // if cal_p, then s1~s4 = px, py, pz, U; if cal_j, then s1~s5 = j1~j5
    __shared__ real s1[BLOCK_SIZE_RI];
    __shared__ real s2[BLOCK_SIZE_RI];
    __shared__ real s3[BLOCK_SIZE_RI];
    __shared__ real s4[BLOCK_SIZE_RI];
    __shared__ real s5[BLOCK_SIZE_RI];

    s_fx[threadIdx.x] = ZERO; 
    s_fy[threadIdx.x] = ZERO; 
    s_fz[threadIdx.x] = ZERO;  
    s1[threadIdx.x] = ZERO; 
    s2[threadIdx.x] = ZERO; 
    s3[threadIdx.x] = ZERO;
    s4[threadIdx.x] = ZERO;
    s5[threadIdx.x] = ZERO;

    if (n1 < number_of_particles)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1];
        real x1 = LDG(g_x, n1); 
        real y1 = LDG(g_y, n1); 
        real z1 = LDG(g_z, n1);
        real vx1 = LDG(g_vx, n1); 
        real vy1 = LDG(g_vy, n1); 
        real vz1 = LDG(g_vz, n1);
        real lx = g_box[0]; 
        real ly = g_box[1]; 
        real lz = g_box[2];

        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real d12sq = x12 * x12 + y12 * y12 + z12 * z12;
            if (d12sq >= ri.cutoff * ri.cutoff) {continue;}
            int type2 = g_type[n2];

            real p2, f2;
            find_p2_and_f2(type1, type2, ri, d12sq, p2, f2);

            // treat two-body potential in the same way as many-body potential
            real f12x = f2 * x12 * HALF; 
            real f12y = f2 * y12 * HALF; 
            real f12z = f2 * z12 * HALF; 
            real f21x = -f12x; 
            real f21y = -f12y; 
            real f21z = -f12z; 
       
            // accumulate force
            s_fx[threadIdx.x] += f12x - f21x; 
            s_fy[threadIdx.x] += f12y - f21y; 
            s_fz[threadIdx.x] += f12z - f21z; 
            
            // accumulate potential energy and virial
            if (cal_p) 
            {
                s4[threadIdx.x] += p2 * HALF; // two-body potential
                s1[threadIdx.x] -= x12 * (f12x - f21x) * HALF; 
                s2[threadIdx.x] -= y12 * (f12y - f21y) * HALF; 
                s3[threadIdx.x] -= z12 * (f12z - f21z) * HALF;
            }
            
            // heat current (EMD)
            if (cal_j) 
            {
                s1[threadIdx.x] += (f21x * vx1 + f21y * vy1) * x12;  // x-in
                s2[threadIdx.x] += (f21z * vz1) * x12;               // x-out
                s3[threadIdx.x] += (f21x * vx1 + f21y * vy1) * y12;  // y-in
                s4[threadIdx.x] += (f21z * vz1) * y12;               // y-out
                s5[threadIdx.x] += (f21x*vx1+f21y*vy1+f21z*vz1)*z12; // z-all
            } 

            // heat across some section (NEMD)
            if (cal_q) 
            {
                int index_12 = g_fv_index[n1] * 12;
                if (index_12 >= 0 && g_fv_index[n1 + number_of_particles] == n2)
                {
                    g_fv[index_12 + 0]  = f12x;
                    g_fv[index_12 + 1]  = f12y;
                    g_fv[index_12 + 2]  = f12z;
                    g_fv[index_12 + 3]  = f21x;
                    g_fv[index_12 + 4]  = f21y;
                    g_fv[index_12 + 5]  = f21z;
                    g_fv[index_12 + 6]  = vx1;
                    g_fv[index_12 + 7]  = vy1;
                    g_fv[index_12 + 8]  = vz1;
                    g_fv[index_12 + 9]  = LDG(g_vx, n2);
                    g_fv[index_12 + 10] = LDG(g_vy, n2);
                    g_fv[index_12 + 11] = LDG(g_vz, n2);
                }  
            }
        }

        // save force
        g_fx[n1] = s_fx[threadIdx.x]; 
        g_fy[n1] = s_fy[threadIdx.x]; 
        g_fz[n1] = s_fz[threadIdx.x]; 

        // save stress and potential
        if (cal_p) 
        {
            g_sx[n1] = s1[threadIdx.x]; 
            g_sy[n1] = s2[threadIdx.x]; 
            g_sz[n1] = s3[threadIdx.x];
            g_potential[n1] = s4[threadIdx.x];
        }

        // save heat current
        if (cal_j) 
        {
            g_h[n1 + 0 * number_of_particles] = s1[threadIdx.x];
            g_h[n1 + 1 * number_of_particles] = s2[threadIdx.x];
            g_h[n1 + 2 * number_of_particles] = s3[threadIdx.x];
            g_h[n1 + 3 * number_of_particles] = s4[threadIdx.x];
            g_h[n1 + 4 * number_of_particles] = s5[threadIdx.x];
        }
    }
}    
 

// Find force and related quantities for the LJ1 potential (A wrapper)
void gpu_find_force_ri(Parameters *para, RI ri, GPU_Data *gpu_data)
{
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE_RI + 1;
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    int *NN = gpu_data->NN; 
    int *NL = gpu_data->NL;
    int *type = gpu_data->type;
    real *x = gpu_data->x; 
    real *y = gpu_data->y; 
    real *z = gpu_data->z;
    real *vx = gpu_data->vx; 
    real *vy = gpu_data->vy; 
    real *vz = gpu_data->vz;
    real *fx = gpu_data->fx; 
    real *fy = gpu_data->fy; 
    real *fz = gpu_data->fz;
    real *box = gpu_data->box_length;
    real *sx = gpu_data->virial_per_atom_x; 
    real *sy = gpu_data->virial_per_atom_y; 
    real *sz = gpu_data->virial_per_atom_z; 
    real *pe = gpu_data->potential_per_atom;
    real *h = gpu_data->heat_per_atom; 
    
    int *label = gpu_data->label;
    int *fv_index = gpu_data->fv_index;
    real *fv = gpu_data->fv;
           
    if (para->hac.compute)    
    {
        gpu_find_force<0, 1, 0><<<grid_size, BLOCK_SIZE_RI>>>
        (
            N, pbc_x, pbc_y, pbc_z, ri, NN, NL, type, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }
    else if (para->shc.compute)
    {
        gpu_find_force<0, 0, 1><<<grid_size, BLOCK_SIZE_RI>>>
        (
            N, pbc_x, pbc_y, pbc_z, ri, NN, NL, type, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }
    else
    {
        gpu_find_force<1, 0, 0><<<grid_size, BLOCK_SIZE_RI>>>
        (
            N, pbc_x, pbc_y, pbc_z, ri, NN, NL, type, x, y, z, vx, vy, vz, box,
            fx, fy, fz, sx, sy, sz, pe, h, label, fv_index, fv
        );
    }

    #ifdef DDEGUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif
}


