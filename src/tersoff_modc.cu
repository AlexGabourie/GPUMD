#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The modified Tersoff potentials as described in
    [1] T. Kumagai et al., Comput. Mater. Sci. 39, 457 (2007);
    [2] G. P. Purja Pun and Y. Mishin, Phys. Rev. B 95, 224103 (2017).
------------------------------------------------------------------------------*/


#include "tersoff_modc.cuh"
#include "mic.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"
#define BLOCK_SIZE_FORCE 64 // 128 is also good
#define ONE_OVER_16      0.0625
#define NINE_OVER_16     0.5625
//Easy labels for indexing
#define A          0
#define B          1
#define LAMBDA     2
#define MU         3
#define ETA        4
#define DELTA      5
#define ALPHA      6
#define BETA       7
#define C0         8
#define C1         9
#define C2         10
#define C3         11
#define C4         12
#define C5         13
#define H          14
#define R1         15
#define R2         16
#define PI_FACTOR1 17
#define PI_FACTOR3 18
#define NUM_PARAMS 19


Tersoff_modc::Tersoff_modc(FILE *fid, Atom* atom, int num_of_types)
{
    num_types = num_of_types;
    printf("Use Tersoff-modc (%d-element) potential.\n", num_types);
    int n_entries = num_types*num_types*num_types;
    real *cpu_ters;
    MY_MALLOC(cpu_ters, real, n_entries*NUM_PARAMS);

    rc = 0;
    int count;
    double a, b, lambda, mu, eta, delta, alpha, beta;
    double c0, c1, c2, c3, c4, c5, h, r1, r2;
    for (int i = 0; i < n_entries; i++)
    {
        count = fscanf
        (
            fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf",
            &a, &b, &lambda, &mu, &eta, &delta, &alpha, &beta, 
            &c0, &c1, &c2, &c3, &c4, &c5, &h, &r1, &r2
        );
        if (count!=17) {printf("Error: reading error for Tersoff-modc.\n"); exit(1);}

        cpu_ters[i*NUM_PARAMS + A] = a;
        cpu_ters[i*NUM_PARAMS + B] = b;
        cpu_ters[i*NUM_PARAMS + LAMBDA] = lambda;
        cpu_ters[i*NUM_PARAMS + MU] = mu;
        cpu_ters[i*NUM_PARAMS + ETA] = eta;
        cpu_ters[i*NUM_PARAMS + DELTA] = delta;
        cpu_ters[i*NUM_PARAMS + ALPHA] = alpha;
        cpu_ters[i*NUM_PARAMS + BETA] = beta;
        cpu_ters[i*NUM_PARAMS + C0] = c0;
        cpu_ters[i*NUM_PARAMS + C1] = c1;
        cpu_ters[i*NUM_PARAMS + C2] = c2;
        cpu_ters[i*NUM_PARAMS + C3] = c3;
        cpu_ters[i*NUM_PARAMS + C4] = c4;
        cpu_ters[i*NUM_PARAMS + C5] = c5;
        cpu_ters[i*NUM_PARAMS + H] = h;
        cpu_ters[i*NUM_PARAMS + R1] = r1;
        cpu_ters[i*NUM_PARAMS + R2] = r2;
        cpu_ters[i*NUM_PARAMS + PI_FACTOR1] = PI / (r2 - r1);
        cpu_ters[i*NUM_PARAMS + PI_FACTOR3] = 3.0 * PI / (r2 - r1);
        rc = r2 > rc ? r2 : rc;
    }

    int num_of_neighbors = (atom->neighbor.MN < 20) ? atom->neighbor.MN : 20;
    int memory = sizeof(real)* atom->N * num_of_neighbors;
    CHECK(hipMalloc((void**)&tersoff_data.b,  memory));
    CHECK(hipMalloc((void**)&tersoff_data.bp, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12x, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12y, memory));
    CHECK(hipMalloc((void**)&tersoff_data.f12z, memory));
    CHECK(hipMalloc((void**)&ters, sizeof(real) * n_entries*NUM_PARAMS));
    CHECK(hipMemcpy(ters, cpu_ters,
        sizeof(real) * n_entries*NUM_PARAMS, hipMemcpyHostToDevice));
    MY_FREE(cpu_ters);
}


Tersoff_modc::~Tersoff_modc(void)
{
    CHECK(hipFree(tersoff_data.b));
    CHECK(hipFree(tersoff_data.bp));
    CHECK(hipFree(tersoff_data.f12x));
    CHECK(hipFree(tersoff_data.f12y));
    CHECK(hipFree(tersoff_data.f12z));
    CHECK(hipFree(ters));
}


static __device__ void find_fr_and_frp
(int i, const real* __restrict__ ters, real d12, real &fr, real &frp)
{
    fr  = LDG(ters,i + A) * exp(- LDG(ters,i + LAMBDA) * d12);
    frp = - LDG(ters,i + LAMBDA) * fr;
}


static __device__ void find_fa_and_fap
(int i, const real* __restrict__ ters, real d12, real &fa, real &fap)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
    fap = - LDG(ters, i + MU) * fa;
}


static __device__ void find_fa
(int i, const real* __restrict__ ters, real d12, real &fa)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
}


static __device__ void find_fc_and_fcp
(int i, const real* __restrict__ ters, real d12, real &fc, real &fcp)
{
    if (d12 < LDG(ters, i + R1)) {fc = 1.0; fcp = 0.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        double tmp = d12 - LDG(ters, i + R1);
        double pi_factor1 = LDG(ters, i + PI_FACTOR1);
        double pi_factor3 = LDG(ters, i + PI_FACTOR3);

        fc = NINE_OVER_16 * cos(pi_factor1 * tmp)
           - ONE_OVER_16  * cos(pi_factor3 * tmp)
           + 0.5;

        fcp = sin(pi_factor3 * tmp) * pi_factor3 * ONE_OVER_16
            - sin(pi_factor1 * tmp) * pi_factor1 * NINE_OVER_16;
    }
    else {fc  = 0.0; fcp = 0.0;}
}


static __device__ void find_fc
(int i, const real* __restrict__ ters, real d12, real &fc)
{
    if (d12 < LDG(ters, i + R1)) {fc  = 1.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        double tmp = d12 - LDG(ters, i + R1);
        fc = NINE_OVER_16 * cos(LDG(ters, i + PI_FACTOR1) * tmp)
           - ONE_OVER_16  * cos(LDG(ters, i + PI_FACTOR3) * tmp)
           + 0.5;
    }
    else {fc  = 0.0;}
}


static __device__ void find_g_and_gp
(int i, const real* __restrict__ ters, real cos, real &g, real &gp)
{
    real x = (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
    real exp_factor = exp(-LDG(ters, i + C5) * x);
    real c2c3_factor = LDG(ters, i + C2) * x / (LDG(ters, i + C3) + x);
    g  = (1.0 + LDG(ters, i + C4) * exp_factor) * c2c3_factor
       + LDG(ters, i + C1);
    gp = LDG(ters, i + C2) * LDG(ters, i + C3) 
       / ( (LDG(ters, i + C3) + x) * (LDG(ters, i + C3) + x) )
       * (1.0 + LDG(ters, i + C4) * exp_factor)
       - LDG(ters, i + C4) * LDG(ters, i + C5) * c2c3_factor 
       * exp_factor;
    gp *= 2.0 * (cos - LDG(ters, i + H));
}


static __device__ void find_g
(int i, const real* __restrict__ ters, real cos, real &g)
{
    real x = (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
    g  = (1.0 + LDG(ters, i + C4) * exp(-LDG(ters, i + C5) * x))
       * LDG(ters, i + C2) * x / (LDG(ters, i + C3) + x)
       + LDG(ters, i + C1);
}


static __device__ void find_e_and_ep
(int i, const real* __restrict__ ters, real d12, real d13, real &e, real &ep)
{
    real r = d12 - d13;
    if (LDG(ters, i + BETA) > TWO) //if beta == 3
    {
        e = exp(LDG(ters, i + ALPHA) * r * r * r);
        ep = LDG(ters, i + ALPHA) * THREE * r * r * e;
    }
    else // beta = 1
    {
        e = exp(LDG(ters, i + ALPHA) * r);
        ep = LDG(ters, i + ALPHA) * e;
    }
}

static __device__ void find_e
(int i, const real* __restrict__ ters, real d12, real d13, real &e)
{
    real r = d12 - d13;
    if (LDG(ters, i + BETA) > TWO) { e = exp(LDG(ters, i + ALPHA) * r * r * r);}
    else {e = exp(LDG(ters, i + ALPHA) * r);}
}


// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int N1, int N2, Box box,
    int num_types, int* g_neighbor_number, int* g_neighbor_list,
    int* g_type, int shift,
    const real* __restrict__ ters,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    real* g_b, real* g_bp
)
{
    int num_types2 = num_types * num_types;
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1] - shift;
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int type2 = g_type[n2] - shift;
            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real zeta = ZERO;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2
                int type3 = g_type[n3] - shift;
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(box, x13, y13, z13);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
                real fc_ijk_13, g_ijk, e_ijk_12_13;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                if (d13 > LDG(ters, ijk*NUM_PARAMS + R2)) {continue;}
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_g(ijk*NUM_PARAMS, ters, cos123, g_ijk);
                find_e(ijk*NUM_PARAMS, ters, d12, d13, e_ijk_12_13);
                zeta += fc_ijk_13 * g_ijk * e_ijk_12_13;
            }
            real zn, b_ijj;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            zn = pow(zeta, LDG(ters, ijj*NUM_PARAMS + ETA));
            b_ijj = pow(ONE + zn, -LDG(ters, ijj*NUM_PARAMS + DELTA));
            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = ONE;
                g_bp[i1 * number_of_particles + n1] = ZERO;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1]
                    = - b_ijj * zn * LDG(ters, ijj*NUM_PARAMS + ETA) 
                    * LDG(ters, ijj*NUM_PARAMS + DELTA) / ((ONE + zn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int N1, int N2, Box box,
    int num_types, int *g_neighbor_number, int *g_neighbor_list,
    int *g_type, int shift,
    const real* __restrict__ ters,
    const real* __restrict__ g_b,
    const real* __restrict__ g_bp,
    const real* __restrict__ g_x,
    const real* __restrict__ g_y,
    const real* __restrict__ g_z,
    real *g_potential, real *g_f12x, real *g_f12y, real *g_f12z
)
{
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    int num_types2 = num_types * num_types;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1] - shift;
        real x1 = LDG(g_x, n1); real y1 = LDG(g_y, n1); real z1 = LDG(g_z, n1);
        real pot_energy = ZERO;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2] - shift;

            real x12  = LDG(g_x, n2) - x1;
            real y12  = LDG(g_y, n2) - y1;
            real z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            real d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            real d12inv = ONE / d12;
            real fc_ijj_12, fcp_ijj_12;
            real fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            find_fc_and_fcp(ijj*NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
            find_fa_and_fap(ijj*NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
            find_fr_and_frp(ijj*NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

            // (i,j) part
            real b12 = LDG(g_b, index);
            real factor3 = d12inv *
            (
                fcp_ijj_12 * 
                (fr_ijj_12 - b12 * fa_ijj_12 + LDG(ters, ijj*NUM_PARAMS + C0))
                + fc_ijj_12 * (frp_ijj_12 - b12 * fap_ijj_12)
            );
            real f12x = x12 * factor3 * HALF;
            real f12y = y12 * factor3 * HALF;
            real f12z = z12 * factor3 * HALF;

            // accumulate potential energy
            pot_energy += fc_ijj_12 * HALF *
            (
                fr_ijj_12 - b12 * fa_ijj_12 + LDG(ters, ijj*NUM_PARAMS + C0)
            );

            // (i,j,k) part
            real bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }
                int type3 = g_type[n3] - shift;
                real x13 = LDG(g_x, n3) - x1;
                real y13 = LDG(g_y, n3) - y1;
                real z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(box, x13, y13, z13);
                real d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                real fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
                int ikj = type1 * num_types2 + type3 * num_types + type2;
                int ikk = type1 * num_types2 + type3 * num_types + type3;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ikk*NUM_PARAMS, ters, d13, fc_ikk_13);
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_fa(ikk*NUM_PARAMS, ters, d13, fa_ikk_13);
                find_fc_and_fcp(ikj*NUM_PARAMS, ters, d12,
                                	fc_ikj_12, fcp_ikj_12);
                real bp13 = LDG(g_bp, index_2);
                real one_over_d12d13 = ONE / (d12 * d13);
                real cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                real cos123_over_d12d12 = cos123*d12inv*d12inv;
                real g_ijk, gp_ijk;
                find_g_and_gp(ijk*NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

                real g_ikj, gp_ikj;
                find_g_and_gp(ikj*NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

                // exp with d12 - d13
                real e_ijk_12_13, ep_ijk_12_13;
                find_e_and_ep(ijk*NUM_PARAMS, ters, d12, d13,
                                	e_ijk_12_13, ep_ijk_12_13);

                // exp with d13 - d12
                real e_ikj_13_12, ep_ikj_13_12;
                find_e_and_ep(ikj*NUM_PARAMS, ters, d13, d12,
                                	e_ikj_13_12, ep_ikj_13_12);

                // derivatives with cosine
                real dc=-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*gp_ijk*e_ijk_12_13+
                        -fc_ikj_12*bp13*fa_ikk_13*fc_ikk_13*gp_ikj*e_ikj_13_12;
                // derivatives with rij
                real dr=(-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*g_ijk*ep_ijk_12_13 +
                  (-fcp_ikj_12*bp13*fa_ikk_13*g_ikj*e_ikj_13_12 +
                  fc_ikj_12*bp13*fa_ikk_13*g_ikj*ep_ikj_13_12)*fc_ikk_13)*d12inv;
                real cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*HALF;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*HALF;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*HALF;
            }
            g_f12x[index] = f12x; g_f12y[index] = f12y; g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += pot_energy;
    }
}


// Wrapper of force evaluation for the Tersoff potential
void Tersoff_modc::compute(Atom *atom, Measure *measure, int potential_number)
{
    int N = atom->N;
    int shift = atom->shift[potential_number];
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;
    int *NN = atom->NN_local;
    int *NL = atom->NL_local;
    int *type = atom->type;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real *pe = atom->potential_per_atom;

    // special data for Tersoff potential
    real *f12x = tersoff_data.f12x;
    real *f12y = tersoff_data.f12y;
    real *f12z = tersoff_data.f12z;
    real *b    = tersoff_data.b;
    real *bp   = tersoff_data.bp;

    // pre-compute the bond order functions and their derivatives
    find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, atom->box, num_types,
        NN, NL, type, shift, ters, x, y, z, b, bp
    );
    CUDA_CHECK_KERNEL

    // pre-compute the partial forces
    find_force_tersoff_step2<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, atom->box, num_types,
        NN, NL, type, shift, ters, b, bp, x, y, z, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL

    // the final step: calculate force and related quantities
    find_properties_many_body(atom, measure, NN, NL, f12x, f12y, f12z);
}
