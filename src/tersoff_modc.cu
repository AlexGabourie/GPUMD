#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The modified Tersoff potentials as described in
    [1] T. Kumagai et al., Comput. Mater. Sci. 39, 457 (2007);
    [2] G. P. Purja Pun and Y. Mishin, Phys. Rev. B 95, 224103 (2017).
------------------------------------------------------------------------------*/


#include "tersoff_modc.cuh"
#include "mic.cuh"
#include "atom.cuh"
#include "error.cuh"
#define BLOCK_SIZE_FORCE 64 // 128 is also good
#define ONE_OVER_16      0.0625
#define NINE_OVER_16     0.5625
//Easy labels for indexing
#define A          0
#define B          1
#define LAMBDA     2
#define MU         3
#define ETA        4
#define DELTA      5
#define ALPHA      6
#define BETA       7
#define C0         8
#define C1         9
#define C2         10
#define C3         11
#define C4         12
#define C5         13
#define H          14
#define R1         15
#define R2         16
#define PI_FACTOR1 17
#define PI_FACTOR3 18
#define NUM_PARAMS 19


Tersoff_modc::Tersoff_modc(FILE *fid, Atom* atom, int num_of_types)
{
    num_types = num_of_types;
    printf("Use Tersoff-modc (%d-element) potential.\n", num_types);
    int n_entries = num_types * num_types * num_types;
    std::vector<double> cpu_ters(n_entries * NUM_PARAMS);

    rc = 0;
    int count;
    double a, b, lambda, mu, eta, delta, alpha, beta;
    double c0, c1, c2, c3, c4, c5, h, r1, r2;
    for (int i = 0; i < n_entries; i++)
    {
        count = fscanf
        (
            fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf",
            &a, &b, &lambda, &mu, &eta, &delta, &alpha, &beta, 
            &c0, &c1, &c2, &c3, &c4, &c5, &h, &r1, &r2
        );
        if (count!=17) {printf("Error: reading error for Tersoff-modc.\n"); exit(1);}

        cpu_ters[i*NUM_PARAMS + A] = a;
        cpu_ters[i*NUM_PARAMS + B] = b;
        cpu_ters[i*NUM_PARAMS + LAMBDA] = lambda;
        cpu_ters[i*NUM_PARAMS + MU] = mu;
        cpu_ters[i*NUM_PARAMS + ETA] = eta;
        cpu_ters[i*NUM_PARAMS + DELTA] = delta;
        cpu_ters[i*NUM_PARAMS + ALPHA] = alpha;
        cpu_ters[i*NUM_PARAMS + BETA] = beta;
        cpu_ters[i*NUM_PARAMS + C0] = c0;
        cpu_ters[i*NUM_PARAMS + C1] = c1;
        cpu_ters[i*NUM_PARAMS + C2] = c2;
        cpu_ters[i*NUM_PARAMS + C3] = c3;
        cpu_ters[i*NUM_PARAMS + C4] = c4;
        cpu_ters[i*NUM_PARAMS + C5] = c5;
        cpu_ters[i*NUM_PARAMS + H] = h;
        cpu_ters[i*NUM_PARAMS + R1] = r1;
        cpu_ters[i*NUM_PARAMS + R2] = r2;
        cpu_ters[i*NUM_PARAMS + PI_FACTOR1] = PI / (r2 - r1);
        cpu_ters[i*NUM_PARAMS + PI_FACTOR3] = 3.0 * PI / (r2 - r1);
        rc = r2 > rc ? r2 : rc;
    }

    int num_of_neighbors = (atom->neighbor.MN < 50) ? atom->neighbor.MN : 50;
    tersoff_data.b.resize(atom->N * num_of_neighbors);
    tersoff_data.bp.resize(atom->N * num_of_neighbors);
    tersoff_data.f12x.resize(atom->N * num_of_neighbors);
    tersoff_data.f12y.resize(atom->N * num_of_neighbors);
    tersoff_data.f12z.resize(atom->N * num_of_neighbors);
    ters.resize(n_entries * NUM_PARAMS);
    ters.copy_from_host(cpu_ters.data());
}


Tersoff_modc::~Tersoff_modc(void)
{
    // nothing
}


static __device__ void find_fr_and_frp
(int i, const double* __restrict__ ters, double d12, double &fr, double &frp)
{
    fr  = LDG(ters,i + A) * exp(- LDG(ters,i + LAMBDA) * d12);
    frp = - LDG(ters,i + LAMBDA) * fr;
}


static __device__ void find_fa_and_fap
(int i, const double* __restrict__ ters, double d12, double &fa, double &fap)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
    fap = - LDG(ters, i + MU) * fa;
}


static __device__ void find_fa
(int i, const double* __restrict__ ters, double d12, double &fa)
{
    fa  = LDG(ters, i + B) * exp(- LDG(ters, i + MU) * d12);
}


static __device__ void find_fc_and_fcp
(int i, const double* __restrict__ ters, double d12, double &fc, double &fcp)
{
    if (d12 < LDG(ters, i + R1)) {fc = 1.0; fcp = 0.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        double tmp = d12 - LDG(ters, i + R1);
        double pi_factor1 = LDG(ters, i + PI_FACTOR1);
        double pi_factor3 = LDG(ters, i + PI_FACTOR3);

        fc = NINE_OVER_16 * cos(pi_factor1 * tmp)
           - ONE_OVER_16  * cos(pi_factor3 * tmp)
           + 0.5;

        fcp = sin(pi_factor3 * tmp) * pi_factor3 * ONE_OVER_16
            - sin(pi_factor1 * tmp) * pi_factor1 * NINE_OVER_16;
    }
    else {fc  = 0.0; fcp = 0.0;}
}


static __device__ void find_fc
(int i, const double* __restrict__ ters, double d12, double &fc)
{
    if (d12 < LDG(ters, i + R1)) {fc  = 1.0;}
    else if (d12 < LDG(ters, i + R2))
    {
        double tmp = d12 - LDG(ters, i + R1);
        fc = NINE_OVER_16 * cos(LDG(ters, i + PI_FACTOR1) * tmp)
           - ONE_OVER_16  * cos(LDG(ters, i + PI_FACTOR3) * tmp)
           + 0.5;
    }
    else {fc  = 0.0;}
}


static __device__ void find_g_and_gp
(int i, const double* __restrict__ ters, double cos, double &g, double &gp)
{
    double x = (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
    double exp_factor = exp(-LDG(ters, i + C5) * x);
    double c2c3_factor = LDG(ters, i + C2) * x / (LDG(ters, i + C3) + x);
    g  = (1.0 + LDG(ters, i + C4) * exp_factor) * c2c3_factor
       + LDG(ters, i + C1);
    gp = LDG(ters, i + C2) * LDG(ters, i + C3) 
       / ( (LDG(ters, i + C3) + x) * (LDG(ters, i + C3) + x) )
       * (1.0 + LDG(ters, i + C4) * exp_factor)
       - LDG(ters, i + C4) * LDG(ters, i + C5) * c2c3_factor 
       * exp_factor;
    gp *= 2.0 * (cos - LDG(ters, i + H));
}


static __device__ void find_g
(int i, const double* __restrict__ ters, double cos, double &g)
{
    double x = (cos - LDG(ters, i + H)) * (cos - LDG(ters, i + H));
    g  = (1.0 + LDG(ters, i + C4) * exp(-LDG(ters, i + C5) * x))
       * LDG(ters, i + C2) * x / (LDG(ters, i + C3) + x)
       + LDG(ters, i + C1);
}


static __device__ void find_e_and_ep
(int i, const double* __restrict__ ters, double d12, double d13, double &e, double &ep)
{
    double r = d12 - d13;
    if (LDG(ters, i + BETA) > TWO) //if beta == 3
    {
        e = exp(LDG(ters, i + ALPHA) * r * r * r);
        ep = LDG(ters, i + ALPHA) * THREE * r * r * e;
    }
    else // beta = 1
    {
        e = exp(LDG(ters, i + ALPHA) * r);
        ep = LDG(ters, i + ALPHA) * e;
    }
}

static __device__ void find_e
(int i, const double* __restrict__ ters, double d12, double d13, double &e)
{
    double r = d12 - d13;
    if (LDG(ters, i + BETA) > TWO) { e = exp(LDG(ters, i + ALPHA) * r * r * r);}
    else {e = exp(LDG(ters, i + ALPHA) * r);}
}


// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int N1, int N2, Box box,
    int num_types, int* g_neighbor_number, int* g_neighbor_list,
    int* g_type, int shift,
    const double* __restrict__ ters,
    const double* __restrict__ g_x,
    const double* __restrict__ g_y,
    const double* __restrict__ g_z,
    double* g_b, double* g_bp
)
{
    int num_types2 = num_types * num_types;
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1] - shift;
        double x1 = LDG(g_x, n1); double y1 = LDG(g_y, n1); double z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];
            int type2 = g_type[n2] - shift;
            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            double zeta = ZERO;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2
                int type3 = g_type[n3] - shift;
                double x13 = LDG(g_x, n3) - x1;
                double y13 = LDG(g_y, n3) - y1;
                double z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(box, x13, y13, z13);
                double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
                double fc_ijk_13, g_ijk, e_ijk_12_13;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                if (d13 > LDG(ters, ijk*NUM_PARAMS + R2)) {continue;}
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_g(ijk*NUM_PARAMS, ters, cos123, g_ijk);
                find_e(ijk*NUM_PARAMS, ters, d12, d13, e_ijk_12_13);
                zeta += fc_ijk_13 * g_ijk * e_ijk_12_13;
            }
            double zn, b_ijj;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            zn = pow(zeta, LDG(ters, ijj*NUM_PARAMS + ETA));
            b_ijj = pow(ONE + zn, -LDG(ters, ijj*NUM_PARAMS + DELTA));
            if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = ONE;
                g_bp[i1 * number_of_particles + n1] = ZERO;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1]
                    = - b_ijj * zn * LDG(ters, ijj*NUM_PARAMS + ETA) 
                    * LDG(ters, ijj*NUM_PARAMS + DELTA) / ((ONE + zn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int N1, int N2, Box box,
    int num_types, int *g_neighbor_number, int *g_neighbor_list,
    int *g_type, int shift,
    const double* __restrict__ ters,
    const double* __restrict__ g_b,
    const double* __restrict__ g_bp,
    const double* __restrict__ g_x,
    const double* __restrict__ g_y,
    const double* __restrict__ g_z,
    double *g_potential, double *g_f12x, double *g_f12y, double *g_f12z
)
{
    // start from the N1-th atom
    int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
    int num_types2 = num_types * num_types;
    // to the (N2-1)-th atom
    if (n1 >= N1 && n1 < N2)
    {
        int neighbor_number = g_neighbor_number[n1];
        int type1 = g_type[n1] - shift;
        double x1 = LDG(g_x, n1); double y1 = LDG(g_y, n1); double z1 = LDG(g_z, n1);
        double pot_energy = ZERO;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int type2 = g_type[n2] - shift;

            double x12  = LDG(g_x, n2) - x1;
            double y12  = LDG(g_y, n2) - y1;
            double z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(box, x12, y12, z12);
            double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            double d12inv = ONE / d12;
            double fc_ijj_12, fcp_ijj_12;
            double fa_ijj_12, fap_ijj_12, fr_ijj_12, frp_ijj_12;
            int ijj = type1 * num_types2 + type2 * num_types + type2;
            find_fc_and_fcp(ijj*NUM_PARAMS, ters, d12, fc_ijj_12, fcp_ijj_12);
            find_fa_and_fap(ijj*NUM_PARAMS, ters, d12, fa_ijj_12, fap_ijj_12);
            find_fr_and_frp(ijj*NUM_PARAMS, ters, d12, fr_ijj_12, frp_ijj_12);

            // (i,j) part
            double b12 = LDG(g_b, index);
            double factor3 = d12inv *
            (
                fcp_ijj_12 * 
                (fr_ijj_12 - b12 * fa_ijj_12 + LDG(ters, ijj*NUM_PARAMS + C0))
                + fc_ijj_12 * (frp_ijj_12 - b12 * fap_ijj_12)
            );
            double f12x = x12 * factor3 * HALF;
            double f12y = y12 * factor3 * HALF;
            double f12z = z12 * factor3 * HALF;

            // accumulate potential energy
            pot_energy += fc_ijj_12 * HALF *
            (
                fr_ijj_12 - b12 * fa_ijj_12 + LDG(ters, ijj*NUM_PARAMS + C0)
            );

            // (i,j,k) part
            double bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }
                int type3 = g_type[n3] - shift;
                double x13 = LDG(g_x, n3) - x1;
                double y13 = LDG(g_y, n3) - y1;
                double z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(box, x13, y13, z13);
                double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                double fc_ikk_13, fc_ijk_13, fa_ikk_13, fc_ikj_12, fcp_ikj_12;
                int ikj = type1 * num_types2 + type3 * num_types + type2;
                int ikk = type1 * num_types2 + type3 * num_types + type3;
                int ijk = type1 * num_types2 + type2 * num_types + type3;
                find_fc(ikk*NUM_PARAMS, ters, d13, fc_ikk_13);
                find_fc(ijk*NUM_PARAMS, ters, d13, fc_ijk_13);
                find_fa(ikk*NUM_PARAMS, ters, d13, fa_ikk_13);
                find_fc_and_fcp(ikj*NUM_PARAMS, ters, d12,
                                	fc_ikj_12, fcp_ikj_12);
                double bp13 = LDG(g_bp, index_2);
                double one_over_d12d13 = ONE / (d12 * d13);
                double cos123 = (x12*x13 + y12*y13 + z12*z13)*one_over_d12d13;
                double cos123_over_d12d12 = cos123*d12inv*d12inv;
                double g_ijk, gp_ijk;
                find_g_and_gp(ijk*NUM_PARAMS, ters, cos123, g_ijk, gp_ijk);

                double g_ikj, gp_ikj;
                find_g_and_gp(ikj*NUM_PARAMS, ters, cos123, g_ikj, gp_ikj);

                // exp with d12 - d13
                double e_ijk_12_13, ep_ijk_12_13;
                find_e_and_ep(ijk*NUM_PARAMS, ters, d12, d13,
                                	e_ijk_12_13, ep_ijk_12_13);

                // exp with d13 - d12
                double e_ikj_13_12, ep_ikj_13_12;
                find_e_and_ep(ikj*NUM_PARAMS, ters, d13, d12,
                                	e_ikj_13_12, ep_ikj_13_12);

                // derivatives with cosine
                double dc=-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*gp_ijk*e_ijk_12_13+
                        -fc_ikj_12*bp13*fa_ikk_13*fc_ikk_13*gp_ikj*e_ikj_13_12;
                // derivatives with rij
                double dr=(-fc_ijj_12*bp12*fa_ijj_12*fc_ijk_13*g_ijk*ep_ijk_12_13 +
                  (-fcp_ikj_12*bp13*fa_ikk_13*g_ikj*e_ikj_13_12 +
                  fc_ikj_12*bp13*fa_ikk_13*g_ikj*ep_ikj_13_12)*fc_ikk_13)*d12inv;
                double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*HALF;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*HALF;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*HALF;
            }
            g_f12x[index] = f12x; g_f12y[index] = f12y; g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += pot_energy;
    }
}


// Wrapper of force evaluation for the Tersoff potential
void Tersoff_modc::compute(Atom *atom, int potential_number)
{
    int N = atom->N;
    int shift = atom->shift[potential_number];
    int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;
    int *NN = atom->NN_local;
    int *NL = atom->NL_local;
    int *type = atom->type;
    double *x = atom->x;
    double *y = atom->y;
    double *z = atom->z;
    double *pe = atom->potential_per_atom;

    // special data for Tersoff potential
    double *f12x = tersoff_data.f12x.data();
    double *f12y = tersoff_data.f12y.data();
    double *f12z = tersoff_data.f12z.data();
    double *b    = tersoff_data.b.data();
    double *bp   = tersoff_data.bp.data();

    // pre-compute the bond order functions and their derivatives
    find_force_tersoff_step1<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, atom->box, num_types,
        NN, NL, type, shift, ters.data(), x, y, z, b, bp
    );
    CUDA_CHECK_KERNEL

    // pre-compute the partial forces
    find_force_tersoff_step2<<<grid_size, BLOCK_SIZE_FORCE>>>
    (
        N, N1, N2, atom->box, num_types,
        NN, NL, type, shift, ters.data(), b, bp, x, y, z, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL

    // the final step: calculate force and related quantities
    find_properties_many_body(atom, NN, NL, f12x, f12y, f12z);
}
