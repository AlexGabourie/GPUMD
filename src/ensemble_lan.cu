#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




/*----------------------------------------------------------------------------80
The Bussi-Parrinello integrator of the Langevin thermostat:
[1] G. Bussi and M. Parrinello, Phys. Rev. E 75, 056707 (2007).
------------------------------------------------------------------------------*/




#include "ensemble_lan.cuh"

#include "force.cuh"
#include <hiprand/hiprand_kernel.h>
#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128

#ifndef USE_SP
    #define CURAND_NORMAL(a) hiprand_normal_double(a)
#else
    #define CURAND_NORMAL(a) hiprand_normal(a)
#endif




// initialize hiprand states
static __global__ void initialize_curand_states(hiprandState *state, int N)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    // We can use a fixed seed here.
    if (n < N) { hiprand_init(12345678, n, 0, &state[n]); }
}




Ensemble_LAN::Ensemble_LAN(int t, int N, real T, real Tc)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
    c1 = exp(-HALF/temperature_coupling);
    c2 = sqrt((1 - c1 * c1) * K_B * T);
    CHECK(hipMalloc((void**)&curand_states, sizeof(hiprandState) * N));
    int grid_size = (N - 1) / BLOCK_SIZE + 1;
    initialize_curand_states<<<grid_size, BLOCK_SIZE>>>(curand_states, N);
    CUDA_CHECK_KERNEL
}




Ensemble_LAN::Ensemble_LAN
(
    int t, int source_input, int sink_input, int source_size, int sink_size, 
    int source_offset, int sink_offset, real T, real Tc, real dT
)
{
    type = t;
    temperature = T;
    temperature_coupling = Tc;
    delta_temperature = dT;
    source = source_input;
    sink   = sink_input;
    N_source = source_size;
    N_sink = sink_size;
    offset_source = source_offset; 
    offset_sink = sink_offset;
    c1 = exp(-HALF/temperature_coupling);
    c2_source = sqrt((1 - c1 * c1) * K_B * (T + dT));
    c2_sink   = sqrt((1 - c1 * c1) * K_B * (T - dT));

    CHECK(hipMalloc((void**)&curand_states_source,
        sizeof(hiprandState) * N_source));
    CHECK(hipMalloc((void**)&curand_states_sink,
        sizeof(hiprandState) * N_sink));

    int grid_size_source = (N_source - 1) / BLOCK_SIZE + 1;
    int grid_size_sink   = (N_sink - 1)   / BLOCK_SIZE + 1;
    initialize_curand_states<<<grid_size_source, BLOCK_SIZE>>>
    (curand_states_source, N_source);
    CUDA_CHECK_KERNEL
    initialize_curand_states<<<grid_size_sink, BLOCK_SIZE>>>
    (curand_states_sink,   N_sink);
    CUDA_CHECK_KERNEL

    energy_transferred[0] = 0.0;
    energy_transferred[1] = 0.0;
}




Ensemble_LAN::~Ensemble_LAN(void)
{
    if (type == 3)
    {
        CHECK(hipFree(curand_states));
    }
    else
    {
        CHECK(hipFree(curand_states_source));
        CHECK(hipFree(curand_states_sink));
    }
}




// global Langevin thermostatting
static __global__ void gpu_langevin
(
    hiprandState *g_state, int N, real c1, real c2, real *g_mass, 
    real *g_vx, real *g_vy, real *g_vz
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < N)
    {
        // get the hiprand state
        hiprandState state = g_state[n];

        real c2m = c2 * sqrt(ONE / g_mass[n]);
        g_vx[n] = c1 * g_vx[n] + c2m * CURAND_NORMAL(&state);
        g_vy[n] = c1 * g_vy[n] + c2m * CURAND_NORMAL(&state);
        g_vz[n] = c1 * g_vz[n] + c2m * CURAND_NORMAL(&state);

        // save the hiprand state
        g_state[n] = state;
    }
}




// local Langevin thermostatting 
static __global__ void gpu_langevin
(
    hiprandState *g_state, int N, int offset, int *g_group_contents,
    real c1, real c2, real *g_mass, real *g_vx, real *g_vy, real *g_vz
)
{
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    if (m < N)
    {
        // get the hiprand state
        hiprandState state = g_state[m];

        int n = g_group_contents[offset + m];
        real c2m = c2 * sqrt(ONE / g_mass[n]);
        g_vx[n] = c1 * g_vx[n] + c2m * CURAND_NORMAL(&state);
        g_vy[n] = c1 * g_vy[n] + c2m * CURAND_NORMAL(&state);
        g_vz[n] = c1 * g_vz[n] + c2m * CURAND_NORMAL(&state);

        // save the hiprand state
        g_state[m] = state;
    }
}




static __device__ void warp_reduce(volatile real *s, int t) 
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}




// group kinetic energy
static __global__ void find_ke
(
    int  *g_group_size,
    int  *g_group_size_sum,
    int  *g_group_contents,
    real *g_mass,
    real *g_vx, 
    real *g_vy, 
    real *g_vz,
    real *g_ke
)
{
    //<<<number_of_groups, 512>>>

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int group_size = g_group_size[bid];
    int offset = g_group_size_sum[bid];
    int number_of_patches = (group_size - 1) / 512 + 1; 
    __shared__ real s_ke[512]; // relative kinetic energy
    s_ke[tid] = ZERO;
    
    for (int patch = 0; patch < number_of_patches; ++patch)
    { 
        int n = tid + patch * 512;
        if (n < group_size)
        {  
            int index = g_group_contents[offset + n];     
            real mass = g_mass[index];
            real vx = g_vx[index];
            real vy = g_vy[index];
            real vz = g_vz[index];
            s_ke[tid] += (vx * vx + vy * vy + vz * vz) * mass;
        }
    }
    __syncthreads();

    if (tid < 256) {s_ke[tid] += s_ke[tid + 256];} __syncthreads();
    if (tid < 128) {s_ke[tid] += s_ke[tid + 128];} __syncthreads();
    if (tid <  64) {s_ke[tid] += s_ke[tid + 64];}  __syncthreads();
    if (tid <  32) {warp_reduce(s_ke, tid);}  
    if (tid == 0)  {g_ke[bid] = s_ke[0];} // kinetic energy times 2
}




void Ensemble_LAN::integrate_nvt_lan
(Atom *atom, Force *force, Measure* measure)
{
    int  N           = atom->N;
    int  grid_size   = (N - 1) / BLOCK_SIZE + 1;
    real *mass = atom->mass;
    real *vx   = atom->vx;
    real *vy   = atom->vy;
    real *vz   = atom->vz;

    // the first half of Langevin, before velocity-Verlet
    gpu_langevin<<<grid_size, BLOCK_SIZE>>>
    (curand_states, N, c1, c2, mass, vx, vy, vz);
    CUDA_CHECK_KERNEL

    // the standard velocity-Verlet
    velocity_verlet_1(atom);
    force->compute(atom, measure);
    velocity_verlet_2(atom);

    // the second half of Langevin, after velocity-Verlet
    gpu_langevin<<<grid_size, BLOCK_SIZE>>>
    (curand_states, N, c1, c2, mass, vx, vy, vz);
    CUDA_CHECK_KERNEL

    // thermo
    find_thermo(atom);
}




// integrate by one step, with heating and cooling
void Ensemble_LAN::integrate_heat_lan
(Atom *atom, Force *force, Measure* measure)
{
    int grid_size_source = (N_source - 1) / BLOCK_SIZE + 1;
    int grid_size_sink   = (N_sink - 1)   / BLOCK_SIZE + 1;
    int *group_size      = atom->group[0].size;
    int *group_size_sum  = atom->group[0].size_sum;
    int *group_contents  = atom->group[0].contents;
    real *mass = atom->mass;
    real *vx   = atom->vx;
    real *vy   = atom->vy;
    real *vz   = atom->vz;
    int label_1 = source;
    int label_2 = sink;
    int Ng = atom->group[0].number;

    // allocate some memory
    real *ek2;
    MY_MALLOC(ek2, real, sizeof(real) * Ng);
    real *ke;
    CHECK(hipMalloc((void**)&ke, sizeof(real) * Ng));

    // the first half of Langevin, before velocity-Verlet
    find_ke<<<Ng, 512>>>
    (group_size, group_size_sum, group_contents, mass, vx, vy, vz, ke);
    CUDA_CHECK_KERNEL
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));
    energy_transferred[0] += ek2[label_1] * 0.5;
    energy_transferred[1] += ek2[label_2] * 0.5;

    gpu_langevin<<<grid_size_source, BLOCK_SIZE>>>
    (
        curand_states_source, N_source, offset_source, group_contents, 
        c1, c2_source, mass, vx, vy, vz
    );
    CUDA_CHECK_KERNEL
    gpu_langevin<<<grid_size_sink, BLOCK_SIZE>>>
    (
        curand_states_sink, N_sink, offset_sink, group_contents, 
        c1, c2_sink, mass, vx, vy, vz
    );
    CUDA_CHECK_KERNEL

    find_ke<<<Ng, 512>>>
    (group_size, group_size_sum, group_contents, mass, vx, vy, vz, ke);
    CUDA_CHECK_KERNEL
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));
    energy_transferred[0] -= ek2[label_1] * 0.5;
    energy_transferred[1] -= ek2[label_2] * 0.5;

    // the standard veloicty-Verlet
    velocity_verlet_1(atom);
    force->compute(atom, measure);
    velocity_verlet_2(atom);

    // the second half of Langevin, after velocity-Verlet
    find_ke<<<Ng, 512>>>
    (group_size, group_size_sum, group_contents, mass, vx, vy, vz, ke);
    CUDA_CHECK_KERNEL
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));
    energy_transferred[0] += ek2[label_1] * 0.5;
    energy_transferred[1] += ek2[label_2] * 0.5;

    gpu_langevin<<<grid_size_source, BLOCK_SIZE>>>
    (
        curand_states_source, N_source, offset_source, group_contents, 
        c1, c2_source, mass, vx, vy, vz
    );
    CUDA_CHECK_KERNEL
    gpu_langevin<<<grid_size_sink, BLOCK_SIZE>>>
    (
        curand_states_sink, N_sink, offset_sink, group_contents, 
        c1, c2_sink, mass, vx, vy, vz
    );
    CUDA_CHECK_KERNEL

    find_ke<<<Ng, 512>>>
    (group_size, group_size_sum, group_contents, mass, vx, vy, vz, ke);
    CUDA_CHECK_KERNEL
    CHECK(hipMemcpy(ek2, ke, sizeof(real) * Ng, hipMemcpyDeviceToHost));
    energy_transferred[0] -= ek2[label_1] * 0.5;
    energy_transferred[1] -= ek2[label_2] * 0.5;

    // clean up
    MY_FREE(ek2); CHECK(hipFree(ke));
}




void Ensemble_LAN::compute
(Atom *atom, Force *force, Measure* measure)
{
    if (type == 3)
    {
        integrate_nvt_lan(atom, force, measure);
    }
    else
    {
        integrate_heat_lan(atom, force, measure);
    }
}




