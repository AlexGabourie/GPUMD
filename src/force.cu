#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "potential.cuh"
#include "force.cuh"
#include "tersoff.cuh"
#include "rebo_mos2.cuh"
#include "vashishta.cuh"
#include "sw.cuh"
#include "pair.cuh"
#include "eam.cuh"
#include "mic.cuh"




Force::Force(void)
{
    for (int m = 0; m < MAX_NUM_OF_POTENTIALS; m++)
    {
        potential[m] = NULL;
    }
    num_of_potentials = 1;
    rc_max = ZERO;
}




Force::~Force(void)
{
    for (int m = 0; m < num_of_potentials; m++)
    {
        delete potential[m];
        potential[m] = NULL;
    }
}




void Force::initialize_one_potential(Parameters *para, int m)
{
    printf("INFO:  read in potential parameters.\n");
    FILE *fid_potential = my_fopen(file_potential[m], "r");
    char potential_name[20];
    int count = fscanf(fid_potential, "%s", potential_name);
    if (count != 1) 
    {
        print_error("reading error for potential.in.\n");
        exit(1);
    }
    
    // determine the potential
    if (strcmp(potential_name, "tersoff_1989_1") == 0) 
    { 
         potential[m] = new Tersoff2(fid_potential, para, 1);
    }
    else if (strcmp(potential_name, "tersoff_1989_2") == 0) 
    { 
         potential[m] = new Tersoff2(fid_potential, para, 2);
    }
    else if (strcmp(potential_name, "sw_1985") == 0) 
    { 
         potential[m] = new SW2(fid_potential, para, 1);
    }
    else if (strcmp(potential_name, "sw_1985_2") == 0) 
    { 
         potential[m] = new SW2(fid_potential, para, 2);
    }
    else if (strcmp(potential_name, "sw_1985_3") == 0) 
    { 
         potential[m] = new SW2(fid_potential, para, 3);
    }
    else if (strcmp(potential_name, "rebo_mos2") == 0) 
    { 
         potential[m] = new REBO_MOS(para);
    }
    else if (strcmp(potential_name, "lj1") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 1);
    }
    else if (strcmp(potential_name, "lj2") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 2);
    }
    else if (strcmp(potential_name, "lj3") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 3);
    }
    else if (strcmp(potential_name, "lj4") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 4);
    }
    else if (strcmp(potential_name, "lj5") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 5);
    }
    else if (strcmp(potential_name, "ri") == 0)
    { 
         potential[m] = new Pair(fid_potential, para, 0);
    }
    else if (strcmp(potential_name, "eam_zhou_2004_1") == 0) 
    { 
         potential[m] = new EAM_Analytical(fid_potential, para, potential_name);
    }
    else if (strcmp(potential_name, "eam_dai_2006") == 0) 
    { 
         potential[m] = new EAM_Analytical(fid_potential, para, potential_name);
    }
    else if (strcmp(potential_name, "vashishta") == 0) 
    { 
         potential[m] = new Vashishta(fid_potential, para, 0);
    }
    else if (strcmp(potential_name, "vashishta_table") == 0) 
    { 
         potential[m] = new Vashishta(fid_potential, para, 1);
    }
    else    
    { 
        print_error("illegal potential model.\n"); 
        exit(1); 
    }

    fclose(fid_potential);
    printf("INFO:  potential parameters initialized.\n\n");
}




void Force::initialize(Parameters *para)
{
    for (int m = 0; m < num_of_potentials; m++)
    {
        initialize_one_potential(para, m);
        if (rc_max < potential[m]->rc) rc_max = potential[m]->rc;
    }
}




// Construct the local neighbor list from the global one (Kernel)
static __global__ void gpu_find_neighbor_local
(
    int pbc_x, int pbc_y, int pbc_z, 
    int N, real cutoff_square, real *box_length,
    int *NN, int *NL, int *NN_local, int *NL_local, 
#ifdef USE_LDG
    const real* __restrict__ x, 
    const real* __restrict__ y, 
    const real* __restrict__ z
#else
    real *x, real *y, real *z
#endif
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    int count = 0;
    real lx = box_length[0];
    real ly = box_length[1];
    real lz = box_length[2];
    if (n1 < N)
    {  
        int neighbor_number = NN[n1];
        real x1 = LDG(x, n1);   
        real y1 = LDG(y, n1);
        real z1 = LDG(z, n1);  
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {   
            int n2 = NL[n1 + N * i1];
            real x12  = LDG(x, n2) - x1;
            real y12  = LDG(y, n2) - y1;
            real z12  = LDG(z, n2) - z1;
            dev_apply_mic(pbc_x, pbc_y, pbc_z, x12, y12, z12, lx, ly, lz);
            real distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < cutoff_square)
            {        
                NL_local[count * N + n1] = n2;
                ++count;
            }
        }
        NN_local[n1] = count;
    }
}




// Construct the local neighbor list from the global one (Wrapper)
static void find_neighbor_local
(Parameters *para, GPU_Data *gpu_data, real rc2)
{  
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE + 1; 
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    int *NN = gpu_data->NN;
    int *NL = gpu_data->NL;
    int *NN_local = gpu_data->NN_local;
    int *NL_local = gpu_data->NL_local;
    real *x = gpu_data->x;
    real *y = gpu_data->y;
    real *z = gpu_data->z;
    real *box = gpu_data->box_length;
      
    gpu_find_neighbor_local<<<grid_size, BLOCK_SIZE>>>
    (pbc_x, pbc_y, pbc_z, N, rc2, box, NN, NL, NN_local, NL_local, x, y, z);
}




static __global__ void initialize_properties
(
    int N, int M, real *g_fx, real *g_fy, real *g_fz, real *g_pe,
    real *g_sx, real *g_sy, real *g_sz, real *g_h, real *g_fv
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {  
        g_fx[n1] = ZERO;
        g_fy[n1] = ZERO;
        g_fz[n1] = ZERO;
        g_sx[n1] = ZERO;
        g_sy[n1] = ZERO;
        g_sz[n1] = ZERO;
        g_pe[n1] = ZERO;
        g_h[n1 + 0 * N] = ZERO;
        g_h[n1 + 1 * N] = ZERO;
        g_h[n1 + 2 * N] = ZERO;
        g_h[n1 + 3 * N] = ZERO;
        g_h[n1 + 4 * N] = ZERO;
    }
    if (n1 < M)
    {  
        g_fv[n1] = ZERO;
    }
}




void Force::compute(Parameters *para, GPU_Data *gpu_data)
{
    initialize_properties<<<(para->N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    (
        para->N, para->shc.number_of_pairs * 12,
        gpu_data->fx, gpu_data->fy, gpu_data->fz, 
        gpu_data->potential_per_atom,  
        gpu_data->virial_per_atom_x,  
        gpu_data->virial_per_atom_y,
        gpu_data->virial_per_atom_z,
        gpu_data->heat_per_atom, gpu_data->fv
    );

    for (int m = 0; m < num_of_potentials; m++)
    {
        real cutoff_square = potential[m]->rc * potential[m]->rc;
        find_neighbor_local(para, gpu_data, cutoff_square); 
        potential[m]->compute(para, gpu_data);
    }

}




