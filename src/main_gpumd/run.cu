#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Run simulation according to the inputs in the run.in file.
------------------------------------------------------------------------------*/


#include "run.cuh"
#include "velocity.cuh"
#include "force/force.cuh"
#include "integrate/integrate.cuh"
#include "integrate/ensemble.cuh"
#include "measure/measure.cuh"
#include "model/atom.cuh"
#include "model/neighbor.cuh"
#include "model/box.cuh"
#include "utilities/read_file.cuh"
#include "utilities/error.cuh"


Run::Run
(
    char* input_dir, Atom* atom, Force* force,
    Integrate* integrate, Measure* measure
)
{
    run(input_dir, atom, force, integrate, measure, 1);
    force->initialize_participation_and_shift
    (
        atom->group,
        atom->number_of_types
    );
    run(input_dir, atom, force, integrate, measure, 0);
}


Run::~Run(void)
{
    // nothing
}


// set some default values after each run
void Run::initialize_run(Atom* atom, Integrate* integrate, Measure* measure)
{
    atom->neighbor.update = 0;
    atom->neighbor.number_of_updates = 0;
    integrate->fixed_group = -1; // no group has an index of -1
    integrate->deform_x = 0;
    integrate->deform_y = 0;
    integrate->deform_z = 0;
    measure->compute.compute_temperature  = 0;
    measure->compute.compute_potential    = 0;
    measure->compute.compute_force        = 0;
    measure->compute.compute_virial       = 0;
    measure->compute.compute_jp           = 0;
    measure->compute.compute_jk           = 0;
    measure->shc.compute    = 0;
    measure->vac.compute_dos= 0;
    measure->vac.compute_sdc= 0;
    measure->modal_analysis.compute   = 0;
    measure->modal_analysis.method   = NO_METHOD;
    measure->vac.grouping_method = -1;
    measure->vac.group = -1;
    measure->vac.num_dos_points = -1;
    measure->hac.compute    = 0;
    measure->hnemd.compute  = 0;
    measure->dump_thermo    = 0;
    measure->dump_velocity  = 0;
    measure->dump_restart   = 0;

    /*
     * Delete dump_pos if it exists. Ensure that dump_pos is NULL in case
     * it isn't set in parse. If we don't set to NULL, then we may end up
     * deleting some random address, corrupting memory.
     */
    if (measure->dump_pos)
    {
    	delete measure->dump_pos;
    }
    measure->dump_pos = NULL;

}


void Run::print_velocity_and_potential_error(void)
{
    if (0 == number_of_times_potential)
    {
        PRINT_INPUT_ERROR("No 'potential' keyword before run.");
    }

    if (0 == number_of_times_velocity)
    {
        PRINT_INPUT_ERROR("No 'velocity' keyword before run.");
    }
}


void Run::print_velocity_error(void)
{
    if (1 < number_of_times_velocity)
    {
        PRINT_INPUT_ERROR("Multiple 'velocity' keywords.");
    }
}


static void check_run_parameters
(Atom *atom, Integrate *integrate, Measure *measure)
{
    if (atom->box.triclinic == 1 && integrate->type == 11)
    {
        PRINT_INPUT_ERROR("Cannot use triclinic box with NPT ensemble.");
    }
}


static void print_finished_steps(int step, int number_of_steps)
{
    int base = (10 <= number_of_steps) ? (number_of_steps / 10) : 1;
    if (0 == (step + 1) % base)
    {
        printf("    %d steps completed.\n", step + 1);
    }
}


static void print_time_and_speed(clock_t time_begin, Atom* atom)
{
    print_line_1();
    clock_t time_finish = clock();
    double time_used = (time_finish - time_begin) / (double) CLOCKS_PER_SEC;
    printf("Number of neighbor list updates = %d.\n",
        atom->neighbor.number_of_updates);
    printf("Time used for this run = %g s.\n", time_used);
    double run_speed = atom->N * (atom->number_of_steps / time_used);
    printf("Speed of this run = %g atom*step/second.\n", run_speed);
    print_line_2();
}


// run a number of steps for a given set of inputs
static void process_run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    integrate->initialize(atom->N, atom->time_step, atom->group);

    measure->initialize
    (
        input_dir,
        atom->number_of_steps,
        atom->time_step,
        atom->group,
        atom->cpu_type_size,
        atom->mass
    );

    clock_t time_begin = clock();

    for (int step = 0; step < atom->number_of_steps; ++step)
    {
        atom->step = step;
        atom->global_time += atom->time_step;
		
#ifndef USE_FCP // the FCP does not use a neighbor list at all
        if (atom->neighbor.update)
        {
            atom->neighbor.find_neighbor
            (
                0,
                atom->box,
                atom->position_per_atom
            );
        }
#endif

        integrate->compute1
        (
            atom->time_step,
            double(step) / atom->number_of_steps,
            atom->group,
            atom->mass,
            atom->potential_per_atom,
            atom->force_per_atom,
            atom->virial_per_atom,
            atom->box,
            atom->position_per_atom,
            atom->velocity_per_atom,
            atom->thermo
        );

        force->compute
        (
            atom->box,
            atom->position_per_atom,
            atom->type,
            atom->group,
            atom->neighbor,
            atom->potential_per_atom,
            atom->force_per_atom,
            atom->virial_per_atom
        );

        integrate->compute2
        (
            atom->time_step,
            double(step) / atom->number_of_steps,
            atom->group,
            atom->mass,
            atom->potential_per_atom,
            atom->force_per_atom,
            atom->virial_per_atom,
            atom->box,
            atom->position_per_atom,
            atom->velocity_per_atom,
            atom->thermo
        );

        measure->process
        (
            input_dir,
            atom->number_of_steps,
            step,
            integrate->fixed_group,
            atom->global_time,
            integrate->temperature2,
            integrate->ensemble->energy_transferred,
            atom->cpu_type,
            atom->box,
            atom->neighbor,
            atom->group,
            atom->thermo,
            atom->mass,
            atom->cpu_mass,
            atom->position_per_atom,
            atom->cpu_position_per_atom,
            atom->velocity_per_atom,
            atom->cpu_velocity_per_atom,
            atom->potential_per_atom,
            atom->force_per_atom,
            atom->virial_per_atom,
            atom->heat_per_atom
        );
        print_finished_steps(step, atom->number_of_steps);
    }

    print_time_and_speed(time_begin, atom);

    measure->finalize
    (
        input_dir,
        atom->number_of_steps,
        atom->time_step,
        integrate->temperature2,
        atom->box.get_volume()
    );

    integrate->finalize();
}


static void print_start(int check)
{
    print_line_1();
    if (check) { printf("Started checking the inputs in run.in.\n"); }
    else { printf("Started executing the commands in run.in.\n"); }
    print_line_2();
}


static void print_finish(int check)
{
    print_line_1();
    if (check) { printf("Finished checking the inputs in run.in.\n"); }
    else { printf("Finished executing the commands in run.in.\n"); }
    print_line_2();
}


// do something when the keyword is "potential"
void Run::add_potential
(
    char* input_dir, int check, Atom* atom, Force* force, Measure* measure
)
{
    if (!is_potential) { return; }
    if (check) { number_of_times_potential++; }
    else
    {
        force->add_potential
        (
            input_dir,
            atom->box,
            atom->neighbor,
            atom->group,
            atom->cpu_type,
            atom->cpu_type_size
        );
    }
}


// do something when the keyword is "velocity"
void Run::check_velocity(int check, Atom* atom)
{
    if (!is_velocity) { return; }
    if (check)
    {
        number_of_times_velocity++;
    }
    else
    {
        Velocity velocity;
        velocity.initialize
        (
            atom->has_velocity_in_xyz,
            atom->initial_temperature,
            atom->cpu_mass,
            atom->cpu_position_per_atom,
            atom->cpu_velocity_per_atom,
            atom->velocity_per_atom
        );
    }
}


// do something when the keyword is "run"
void Run::check_run
(
    char* input_dir, int check, Atom* atom,
    Force* force, Integrate* integrate, Measure* measure
)
{
    if (!is_run) { return; }
    if (check)
    {
        print_velocity_and_potential_error();
        check_run_parameters(atom, integrate, measure);
    }
    else
    {
        force->valdiate_potential_definitions();
        bool compute_hnemd = measure->hnemd.compute ||
            (
                measure->modal_analysis.compute &&
                measure->modal_analysis.method == HNEMA_METHOD
            );
        force->set_hnemd_parameters
        (
            compute_hnemd, measure->hnemd.fe_x, measure->hnemd.fe_y, 
            measure->hnemd.fe_z
        );
        process_run(input_dir, atom, force, integrate, measure);
    }
    initialize_run(atom, integrate, measure);
}


// Read and process the inputs from the "run.in" file
void Run::run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure, int check
)
{
    char file_run[200];
    strcpy(file_run, input_dir);
    strcat(file_run, "/run.in");
    char *input = get_file_contents(file_run);
    char *input_ptr = input; // Keep the pointer in order to free later
    const int max_num_param = 10; // never use more than 9 parameters
    int num_param;
    char *param[max_num_param];

    force->num_of_potentials = 0;

    initialize_run(atom, integrate, measure); // set some default values

    print_start(check);

    while (input_ptr)
    {
        input_ptr = row_find_param(input_ptr, param, &num_param);

        if (num_param == 0) { continue; } 

        is_potential = false;
        is_velocity = false;
        is_run = false;

        parse(param, num_param, atom, force, integrate, measure);
        add_potential(input_dir, check, atom, force, measure);
        check_velocity(check, atom);
        check_run(input_dir, check, atom, force, integrate, measure);
    }

    print_velocity_error();

    print_finish(check);

    free(input); // Free the input file contents
}


void Run::parse
(
    char **param, int num_param, Atom* atom,
    Force *force, Integrate *integrate, Measure *measure
)
{
    if (strcmp(param[0], "potential_definition") == 0)
    {
        force->parse_potential_definition(param, num_param);
    }
    else if (strcmp(param[0], "potential") == 0)
    {
        is_potential = true;
        force->parse_potential(param, num_param);
    }
    else if (strcmp(param[0], "velocity") == 0)
    {
        is_velocity = true;
        atom->parse_velocity(param, num_param);
    }
    else if (strcmp(param[0], "ensemble") == 0)
    {
        integrate->parse_ensemble(param, num_param, atom->group);
    }
    else if (strcmp(param[0], "time_step") == 0)
    {
        atom->parse_time_step(param, num_param);
    }
    else if (strcmp(param[0], "neighbor") == 0)
    {
        atom->parse_neighbor(param, num_param, force->rc_max);
    }
    else if (strcmp(param[0], "dump_thermo") == 0)
    {
        measure->parse_dump_thermo(param, num_param);
    }
    else if (strcmp(param[0], "dump_position") == 0)
    {
        measure->parse_dump_position(param, num_param);
    }
    else if (strcmp(param[0], "dump_restart") == 0)
    {
        measure->parse_dump_restart(param, num_param);
    }
    else if (strcmp(param[0], "dump_velocity") == 0)
    {
        measure->parse_dump_velocity(param, num_param);
    }
    else if (strcmp(param[0], "compute_dos") == 0)
    {
        measure->parse_compute_dos(param, num_param, atom->group.data());
    }
    else if (strcmp(param[0], "compute_sdc") == 0)
    {
        measure->parse_compute_sdc(param, num_param, atom->group.data());
    }
    else if (strcmp(param[0], "compute_hac") == 0)
    {
        measure->parse_compute_hac(param, num_param);
    }
    else if (strcmp(param[0], "compute_hnemd") == 0)
    {
        measure->parse_compute_hnemd(param, num_param);
    }
    else if (strcmp(param[0], "compute_shc") == 0)
    {
        measure->parse_compute_shc(param, num_param, atom->group);
    }
    else if (strcmp(param[0], "compute_gkma") == 0)
    {
        measure->parse_compute_gkma(param, num_param, atom->number_of_types);
    }
    else if (strcmp(param[0], "compute_hnema") == 0)
    {
        measure->parse_compute_hnema(param, num_param, atom->number_of_types);
    }
    else if (strcmp(param[0], "deform") == 0)
    {
        integrate->parse_deform(param, num_param);
    }
    else if (strcmp(param[0], "compute") == 0)
    {
        measure->parse_compute(param, num_param, atom->group);
    }
    else if (strcmp(param[0], "fix") == 0)
    {
        integrate->parse_fix(param, num_param, atom->group);
    }
    else if (strcmp(param[0], "run") == 0)
    {
        is_run = true;
        atom->parse_run(param, num_param);
    }
    else
    {
        PRINT_KEYWORD_ERROR(param[0]);
    }
}


