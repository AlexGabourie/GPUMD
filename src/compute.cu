#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Compute block (space) averages of various per-atom quantities.
------------------------------------------------------------------------------*/


#include "compute.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "atom.cuh"
#include "error.cuh"

#define DIM 3


void Compute::preprocess(char* input_dir, Atom* atom)
{
    number_of_scalars = 0;
    if (compute_temperature) number_of_scalars += 1;
    if (compute_potential) number_of_scalars += 1;
    if (compute_force) number_of_scalars += 3;
    if (compute_virial) number_of_scalars += 3;
    if (compute_jp) number_of_scalars += 3;
    if (compute_jk) number_of_scalars += 3;
    if (number_of_scalars == 0) return;

    int number_of_columns = 
        atom->group[grouping_method].number * number_of_scalars;
    MY_MALLOC(cpu_group_sum, double, number_of_columns);
    MY_MALLOC(cpu_group_sum_ave, double, number_of_columns);
    for (int n = 0; n < number_of_columns; ++n) cpu_group_sum_ave[n] = 0.0;

    CHECK(hipMalloc((void**)&gpu_group_sum, sizeof(double) * number_of_columns));
    CHECK(hipMalloc((void**)&gpu_per_atom_x, sizeof(double) * atom->N));
    CHECK(hipMalloc((void**)&gpu_per_atom_y, sizeof(double) * atom->N));
    CHECK(hipMalloc((void**)&gpu_per_atom_z, sizeof(double) * atom->N));

    char filename[200];
    strcpy(filename, input_dir);
    strcat(filename, "/compute.out");
    fid = my_fopen(filename, "a");
}


void Compute::postprocess(Atom* atom, Integrate *integrate)
{
    if (number_of_scalars == 0) return;
    MY_FREE(cpu_group_sum);
    MY_FREE(cpu_group_sum_ave);
    CHECK(hipFree(gpu_group_sum));
    CHECK(hipFree(gpu_per_atom_x));
    CHECK(hipFree(gpu_per_atom_y));
    CHECK(hipFree(gpu_per_atom_z));
    fclose(fid);
}


static __global__ void find_per_atom_temperature
(int N, double *g_mass, double *g_vx, double *g_vy, double *g_vz, double *g_temperature)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < N)
    {
        double vx = g_vx[n]; double vy = g_vy[n]; double vz = g_vz[n];
        double ek2 = g_mass[n] * (vx * vx + vy * vy + vz * vz);
        g_temperature[n] = ek2 / (DIM * K_B);
    }
}


static __global__ void find_per_atom_jp
(
    int N, double *sxx, double *sxy, double *sxz, double *syx, double *syy, double *syz,
    double *szx, double *szy, double *szz, double *vx, double *vy, double *vz, 
    double *jx, double *jy, double *jz
)
{
    int n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n < N)
    {
        jx[n] = sxx[n] * vx[n] + sxy[n] * vy[n] + sxz[n] * vz[n];
        jy[n] = syx[n] * vx[n] + syy[n] * vy[n] + syz[n] * vz[n];
        jz[n] = szx[n] * vx[n] + szy[n] * vy[n] + szz[n] * vz[n];
    }
}


static __global__ void find_per_atom_jk
(
    int N, double* g_potential, double *g_mass, double *g_vx, double* g_vy, double* g_vz,
    double* g_jx, double* g_jy, double* g_jz
)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < N)
    {
        double potential = g_potential[n]; double mass = g_mass[n];
        double vx = g_vx[n]; double vy = g_vy[n]; double vz = g_vz[n];
        double energy = mass * (vx * vx + vy * vy + vz * vz) * HALF + potential;
        g_jx[n] = vx * energy; g_jy[n] = vy * energy; g_jz[n] = vz * energy;
    }
}


static __global__ void find_group_sum_1
(
    int  *g_group_size, int  *g_group_size_sum, int  *g_group_contents,
    double *g_in, double *g_out
)
{
    // <<<number_of_groups, 256>>> (one CUDA block for one group of atoms)
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int group_size = g_group_size[bid];
    int offset = g_group_size_sum[bid];
    int number_of_patches = (group_size - 1) / 256 + 1;
    __shared__ double s_data[256];
    s_data[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; patch++)
    {
        int k = tid + patch * 256;
        if (k < group_size)
        {
            int n = g_group_contents[offset + k]; // particle index
            s_data[tid] += g_in[n];
        }
    }
    __syncthreads();

    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_data[tid] += s_data[tid + offset]; }
        __syncthreads();
    }

    if (tid ==  0) { g_out[bid] = s_data[0]; }
}


static __global__ void find_group_sum_3
(
    int *g_group_size, int *g_group_size_sum, int *g_group_contents,
    double *g_fx, double *g_fy, double *g_fz, double *g_out
)
{
    // <<<number_of_groups, 256>>> (one CUDA block for one group of atoms)
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int group_size = g_group_size[bid];
    int offset = g_group_size_sum[bid];
    int number_of_patches = (group_size - 1) / 256 + 1;
    __shared__ double s_fx[256];
    __shared__ double s_fy[256];
    __shared__ double s_fz[256];
    s_fx[tid] = ZERO; s_fy[tid] = ZERO; s_fz[tid] = ZERO;

    for (int patch = 0; patch < number_of_patches; patch++)
    {
        int k = tid + patch * 256;
        if (k < group_size)
        {
            int n = g_group_contents[offset + k]; // particle index
            s_fx[tid] += g_fx[n]; s_fy[tid] += g_fy[n]; s_fz[tid] += g_fz[n];
        }
    }
    __syncthreads();

    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_fx[tid] += s_fx[tid + offset];
            s_fy[tid] += s_fy[tid + offset];
            s_fz[tid] += s_fz[tid + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_out[bid] = s_fx[0];
        g_out[bid + gridDim.x] = s_fy[0];
        g_out[bid + gridDim.x * 2] = s_fz[0];
    }
}


void Compute::process(int step, Atom *atom, Integrate *integrate)
{
    if (number_of_scalars == 0) return;
    if ((++step) % sample_interval != 0) return;

    int output_flag = ((step/sample_interval) % output_interval == 0);
    
    int Ng = atom->group[grouping_method].number;
    int N = atom->N;

    int offset = 0;
    if (compute_temperature)
    {
        find_per_atom_temperature<<<(N - 1) / 256 + 1, 256>>>(N, atom->mass,
            atom->vx, atom->vy, atom->vz, gpu_per_atom_x);
        CUDA_CHECK_KERNEL
        find_group_sum_1<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            gpu_per_atom_x, gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng;
    }
    if (compute_potential)
    {
        find_group_sum_1<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            atom->potential_per_atom, gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng;
    }
    if (compute_force)
    {
        find_group_sum_3<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            atom->fx, atom->fy, atom->fz,
            gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng * 3;
    }
    if (compute_virial)
    {
        find_group_sum_3<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            atom->virial_per_atom, atom->virial_per_atom + N,
            atom->virial_per_atom + N * 2, gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng * 3;
    }
    if (compute_jp)
    {
        // the virial tensor:
        // xx xy xz    0 3 4
        // yx yy yz    6 1 5
        // zx zy zz    7 8 2
        find_per_atom_jp<<<(N - 1) / 128 + 1, 128>>>
        (
            N, 
            atom->virial_per_atom, 
            atom->virial_per_atom + N * 3,
            atom->virial_per_atom + N * 4,
            atom->virial_per_atom + N * 6,
            atom->virial_per_atom + N * 1,
            atom->virial_per_atom + N * 5,
            atom->virial_per_atom + N * 7,
            atom->virial_per_atom + N * 8,
            atom->virial_per_atom + N * 2,
            atom->vx, atom->vy, atom->vz, 
            gpu_per_atom_x, gpu_per_atom_y, gpu_per_atom_z
        );
        CUDA_CHECK_KERNEL

        find_group_sum_3<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            gpu_per_atom_x, gpu_per_atom_y,
            gpu_per_atom_z, gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng * 3;
    }
    if (compute_jk)
    {
        find_per_atom_jk<<<(N-1)/256+1, 256>>>(N, atom->potential_per_atom,
            atom->mass, atom->vx, atom->vy, atom->vz, gpu_per_atom_x,
            gpu_per_atom_y, gpu_per_atom_z);
        CUDA_CHECK_KERNEL

        find_group_sum_3<<<Ng, 256>>>(atom->group[grouping_method].size,
            atom->group[grouping_method].size_sum,
            atom->group[grouping_method].contents,
            gpu_per_atom_x, gpu_per_atom_y,
            gpu_per_atom_z, gpu_group_sum + offset);
        CUDA_CHECK_KERNEL
        offset += Ng * 3;
    }

    CHECK(hipMemcpy(cpu_group_sum, gpu_group_sum, 
        sizeof(double) * Ng * number_of_scalars, hipMemcpyDeviceToHost));

    for (int n = 0; n < Ng * number_of_scalars; ++n)
        cpu_group_sum_ave[n] += cpu_group_sum[n];

    if (output_flag) 
    { 
        output_results(atom, integrate);
        for (int n = 0; n < Ng * number_of_scalars; ++n)
            cpu_group_sum_ave[n] = 0.0;
    }
}


void Compute::output_results(Atom *atom, Integrate *integrate)
{
    int Ng = atom->group[grouping_method].number;
    for (int n = 0; n < number_of_scalars; ++n)
    {
        int offset = n * Ng;
        for (int k = 0; k < Ng; k++)
        {
            double tmp = cpu_group_sum_ave[k + offset] / output_interval;
            if (compute_temperature && n == 0) 
            {
                tmp /= atom->group[grouping_method].cpu_size[k];
            }
            fprintf(fid, "%15.6e", tmp);
        }     
    }

    if (compute_temperature)
    {
        fprintf(fid, "%15.6e", integrate->ensemble->energy_transferred[0]);
        fprintf(fid, "%15.6e", integrate->ensemble->energy_transferred[1]);
    }

    fprintf(fid, "\n");
    fflush(fid);
}


