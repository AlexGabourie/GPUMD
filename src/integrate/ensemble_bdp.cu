#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
The Bussi-Donadio-Parrinello thermostat:
[1] G. Bussi et al. J. Chem. Phys. 126, 014101 (2007).
------------------------------------------------------------------------------*/


#include "ensemble_bdp.cuh"
#include "utilities/common.cuh"
#define DIM 3


// These functions are from  Bussi's website
// https://sites.google.com/site/giovannibussi/Research/algorithms
// See the end of this file for the function definitions
static double resamplekin(double kk, double sigma, int ndeg, double taut);
static double resamplekin_sumnoises(int nn);
static double ran1();
static double gasdev();
static double gamdev(const int ia);


Ensemble_BDP::Ensemble_BDP(int t, int fg, double T, double Tc)
{
    type = t;
    fixed_group = fg;
    temperature = T;
    temperature_coupling = Tc;
}


Ensemble_BDP::Ensemble_BDP
(
    int t,
    int fg,
    int source_input,
    int sink_input,
    double T,
    double Tc,
    double dT
)
{
    type = t;
    fixed_group = fg;
    temperature = T;
    temperature_coupling = Tc;
    delta_temperature = dT;
    source = source_input;
    sink = sink_input;
    // initialize the energies transferred from the system to the baths
    energy_transferred[0] = 0.0;
    energy_transferred[1] = 0.0;
}


Ensemble_BDP::~Ensemble_BDP(void)
{
    // nothing now
}


void Ensemble_BDP::integrate_nvt_bdp_2
(
    const double time_step,
    const double volume,
    const std::vector<Group>& group,
    const GPU_Vector<double>& mass,
    const GPU_Vector<double>& potential_per_atom,
    const GPU_Vector<double>& force_per_atom,
    const GPU_Vector<double>& virial_per_atom,
    GPU_Vector<double>& position_per_atom,
    GPU_Vector<double>& velocity_per_atom,
    GPU_Vector<double>& thermo
)
{
    const int number_of_atoms = mass.size();

    velocity_verlet
    (
        false,
        time_step,
        group,
        mass,
        force_per_atom,
        position_per_atom,
        velocity_per_atom
     );

    // get thermo
    int N_fixed = (fixed_group == -1) ? 0 : group[0].cpu_size[fixed_group];
    find_thermo
    (
        volume,
        group,
        mass,
        potential_per_atom,
        velocity_per_atom,
        virial_per_atom,
        thermo
    );

    // re-scale the velocities
    double ek[1];
    thermo.copy_to_host(ek, 1);
    int ndeg = 3 * (number_of_atoms - N_fixed);
    ek[0] *= ndeg * K_B * 0.5; // from temperature to kinetic energy
    double sigma = ndeg * K_B * temperature * 0.5;
    double factor = resamplekin(ek[0], sigma, ndeg, temperature_coupling);
    factor = sqrt(factor / ek[0]);
    scale_velocity_global(factor, velocity_per_atom);
}


// integrate by one step, with heating and cooling, using the BDP method
void Ensemble_BDP::integrate_heat_bdp_2
(
    const double time_step,
    const std::vector<Group>& group,
    const GPU_Vector<double>& mass,
    const GPU_Vector<double>& force_per_atom,
    GPU_Vector<double>& position_per_atom,
    GPU_Vector<double>& velocity_per_atom
)
{
    int label_1 = source;
    int label_2 = sink;
    int Ng = group[0].number;

    double kT1 = K_B * (temperature + delta_temperature);
    double kT2 = K_B * (temperature - delta_temperature);
    double dN1 = (double) DIM * (group[0].cpu_size[source] - 1);
    double dN2 = (double) DIM * (group[0].cpu_size[sink] - 1);
    double sigma_1 = dN1 * kT1 * 0.5;
    double sigma_2 = dN2 * kT2 * 0.5;

    // allocate some memory
    std::vector<double> ek(Ng);
    GPU_Vector<double> vcx(Ng), vcy(Ng), vcz(Ng), ke(Ng);

    velocity_verlet
    (
        false,
        time_step,
        group,
        mass,
        force_per_atom,
        position_per_atom,
        velocity_per_atom
     );

    // get center of mass velocity and relative kinetic energy
    find_vc_and_ke
    (
        group,
        mass,
        velocity_per_atom,
        vcx.data(),
        vcy.data(),
        vcz.data(),
        ke.data()
    );

    ke.copy_to_host(ek.data());
    ek[label_1] *= 0.5;
    ek[label_2] *= 0.5;

    // get the re-scaling factors
    double factor_1
        = resamplekin(ek[label_1], sigma_1, dN1, temperature_coupling);
    double factor_2
        = resamplekin(ek[label_2], sigma_2, dN2, temperature_coupling);
    factor_1 = sqrt(factor_1 / ek[label_1]);
    factor_2 = sqrt(factor_2 / ek[label_2]);

    // accumulate the energies transferred from the system to the baths
    energy_transferred[0] += ek[label_1] * (1.0 - factor_1 * factor_1);
    energy_transferred[1] += ek[label_2] * (1.0 - factor_2 * factor_2);

    scale_velocity_local
    (
        factor_1,
        factor_2,
        vcx.data(),
        vcy.data(),
        vcz.data(),
        ke.data(),
        group,
        velocity_per_atom
    );
}


void Ensemble_BDP::compute1
(
    const double time_step,
    const std::vector<Group>& group,
    const GPU_Vector<double>& mass,
    const GPU_Vector<double>& potential_per_atom,
    const GPU_Vector<double>& force_per_atom,
    const GPU_Vector<double>& virial_per_atom,
    Box& box,
    GPU_Vector<double>& position_per_atom,
    GPU_Vector<double>& velocity_per_atom,
    GPU_Vector<double>& thermo
)
{
    velocity_verlet
    (
        true,
        time_step,
        group,
        mass,
        force_per_atom,
        position_per_atom,
        velocity_per_atom
    );
}


void Ensemble_BDP::compute2
(
    const double time_step,
    const std::vector<Group>& group,
    const GPU_Vector<double>& mass,
    const GPU_Vector<double>& potential_per_atom,
    const GPU_Vector<double>& force_per_atom,
    const GPU_Vector<double>& virial_per_atom,
    Box& box,
    GPU_Vector<double>& position_per_atom,
    GPU_Vector<double>& velocity_per_atom,
    GPU_Vector<double>& thermo
)
{
    if (type == 4)
    {
        integrate_nvt_bdp_2
        (
            time_step,
            box.get_volume(),
            group,
            mass,
            potential_per_atom,
            force_per_atom,
            virial_per_atom,
            position_per_atom,
            velocity_per_atom,
            thermo
        );
    }
    else
    {
        integrate_heat_bdp_2
        (
            time_step,
            group,
            mass,
            force_per_atom,
            position_per_atom,
            velocity_per_atom
        );
    }
}


// The following functions are from Bussi's website
// https://sites.google.com/site/giovannibussi/Research/algorithms
// I have only added "static" in front of the functions, 
// without any other changes
static double resamplekin(double kk,double sigma, int ndeg, double taut){
/*
  kk:    present value of the kinetic energy of the atoms to be thermalized (in arbitrary units)
  sigma: target average value of the kinetic energy (ndeg k_b T/2)  (in the same units as kk)
  ndeg:  number of degrees of freedom of the atoms to be thermalized
  taut:  relaxation time of the thermostat, in units of 'how often this routine is called'
*/
  double factor,rr;
  if(taut>0.1){
    factor=exp(-1.0/taut);
  } else{
    factor=0.0;
  }
  rr = gasdev();
  return kk + (1.0-factor)* (sigma*(resamplekin_sumnoises(ndeg-1)+rr*rr)/ndeg-kk)
            + 2.0*rr*sqrt(kk*sigma/ndeg*(1.0-factor)*factor);
}


static double resamplekin_sumnoises(int nn){
/*
  returns the sum of n independent gaussian noises squared
   (i.e. equivalent to summing the square of the return values of nn calls to gasdev)
*/
  double rr;
  if(nn==0) {
    return 0.0;
  } else if(nn==1) {
    rr=gasdev();
    return rr*rr;
  } else if(nn%2==0) {
    return 2.0*gamdev(nn/2);
  } else {
    rr=gasdev();
    return 2.0*gamdev((nn-1)/2) + rr*rr;
  }
}


static double gamdev(const int ia)
{
	int j;
	double am,e,s,v1,v2,x,y;

	if (ia < 1) {}; // FATAL ERROR
	if (ia < 6) {
		x=1.0;
		for (j=1;j<=ia;j++) x *= ran1();
		x = -log(x);
	} else {
		do {
			do {
				do {
					v1=ran1();
					v2=2.0*ran1()-1.0;
				} while (v1*v1+v2*v2 > 1.0);
				y=v2/v1;
				am=ia-1;
				s=sqrt(2.0*am+1.0);
				x=s*y+am;
			} while (x <= 0.0);
			e=(1.0+y*y)*exp(am*log(x/am)-s*y);
		} while (ran1() > e);
	}
	return x;
}


static double gasdev()
{
	static int iset=0;
	static double gset;
	double fac,rsq,v1,v2;

	if (iset == 0) {
		do {
			v1=2.0*ran1()-1.0;
			v2=2.0*ran1()-1.0;
			rsq=v1*v1+v2*v2;
		} while (rsq >= 1.0 || rsq == 0.0);
		fac=sqrt(-2.0*log(rsq)/rsq);
		gset=v1*fac;
		iset=1;
		return v2*fac;
	} else {
		iset=0;
		return gset;
	}
}


static double ran1()
{
	const int IA=16807,IM=2147483647,IQ=127773,IR=2836,NTAB=32;
	const int NDIV=(1+(IM-1)/NTAB);
	const double EPS=3.0e-16,AM=1.0/IM,RNMX=(1.0-EPS);
	static int iy=0;
	static int iv[NTAB];
	int j,k;
	double temp;
        static int idum=0; /* ATTENTION: THE SEED IS HARDCODED */

	if (idum <= 0 || !iy) {
		if (-idum < 1) idum=1;
		else idum = -idum;
		for (j=NTAB+7;j>=0;j--) {
			k=idum/IQ;
			idum=IA*(idum-k*IQ)-IR*k;
			if (idum < 0) idum += IM;
			if (j < NTAB) iv[j] = idum;
		}
		iy=iv[0];
	}
	k=idum/IQ;
	idum=IA*(idum-k*IQ)-IR*k;
	if (idum < 0) idum += IM;
	j=iy/NDIV;
	iy=iv[j];
	iv[j] = idum;
	if ((temp=AM*iy) > RNMX) return RNMX;
	else return temp;
}


