#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#define USE_THRUST



#include "neighbor_ON1.cuh"
#include "atom.cuh"
#include "error.cuh"
#include "parameters.cuh"

#include <thrust/scan.h>
#include <thrust/execution_policy.h>


#define BLOCK_SIZE 128
#ifdef USE_DP
    #define HALF  0.5
#else
    #define HALF  0.5f
#endif




template <int pbc_x, int pbc_y, int pbc_z>
static __device__ void dev_apply_mic
(real lx, real ly, real lz, real *x12, real *y12, real *z12)
{
    if      (pbc_x == 1 && *x12 < - lx * HALF) {*x12 += lx;}
    else if (pbc_x == 1 && *x12 > + lx * HALF) {*x12 -= lx;}
    if      (pbc_y == 1 && *y12 < - ly * HALF) {*y12 += ly;}
    else if (pbc_y == 1 && *y12 > + ly * HALF) {*y12 -= ly;}
    if      (pbc_z == 1 && *z12 < - lz * HALF) {*z12 += lz;}
    else if (pbc_z == 1 && *z12 > + lz * HALF) {*z12 -= lz;}
}




// find the cell id for an atom
static __device__ void find_cell_id
(
    real x, real y, real z, real cell_size, 
    int cell_n_x, int cell_n_y, int cell_n_z, int* cell_id
)
{
    int cell_id_x = floor(x / cell_size);
    int cell_id_y = floor(y / cell_size);
    int cell_id_z = floor(z / cell_size);
    while (cell_id_x < 0)         cell_id_x += cell_n_x;
    while (cell_id_x >= cell_n_x) cell_id_x -= cell_n_x;
    while (cell_id_y < 0)         cell_id_y += cell_n_y;
    while (cell_id_y >= cell_n_y) cell_id_y -= cell_n_y;
    while (cell_id_z < 0)         cell_id_z += cell_n_z;
    while (cell_id_z >= cell_n_z) cell_id_z -= cell_n_z;
    *cell_id =  cell_id_x + cell_n_x*cell_id_y + cell_n_x*cell_n_y*cell_id_z;
}



// find the cell id for an atom
static __device__ void find_cell_id
(
    real x, real y, real z, real cell_size, 
    int cell_n_x, int cell_n_y, int cell_n_z, 
    int *cell_id_x, int *cell_id_y, int *cell_id_z, int *cell_id
)
{
    *cell_id_x = floor(x / cell_size);
    *cell_id_y = floor(y / cell_size);
    *cell_id_z = floor(z / cell_size);
    while (*cell_id_x < 0)         *cell_id_x += cell_n_x;
    while (*cell_id_x >= cell_n_x) *cell_id_x -= cell_n_x;
    while (*cell_id_y < 0)         *cell_id_y += cell_n_y;
    while (*cell_id_y >= cell_n_y) *cell_id_y -= cell_n_y;
    while (*cell_id_z < 0)         *cell_id_z += cell_n_z;
    while (*cell_id_z >= cell_n_z) *cell_id_z -= cell_n_z;
    *cell_id = (*cell_id_x) + cell_n_x * (*cell_id_y) 
             + cell_n_x * cell_n_y * (*cell_id_z);
}




// cell_count[i] = number of atoms in the i-th cell
static __global__ void find_cell_counts
(
    int N, int* cell_count, real* x, real* y,real* z, 
    int cell_n_x, int cell_n_y, int cell_n_z, real cell_size
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {
        int cell_id;
        find_cell_id
        (
            x[n1], y[n1], z[n1], cell_size, 
            cell_n_x, cell_n_y, cell_n_z, &cell_id
        );
        atomicAdd(&cell_count[cell_id], 1);
    }
}




// cell_contents[some index] = an atom index
static __global__ void find_cell_contents
(
    int N, int* cell_count, int* cell_count_sum, int* cell_contents, 
    real* x, real* y, real* z,
    int cell_n_x, int cell_n_y, int cell_n_z, real cell_size
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 < N)
    {
        int cell_id;
        find_cell_id
        (
            x[n1], y[n1], z[n1], cell_size, 
            cell_n_x, cell_n_y, cell_n_z, &cell_id
        );
        int ind = atomicAdd(&cell_count[cell_id], 1);
        cell_contents[cell_count_sum[cell_id] + ind] = n1;
    }
}




// a simple (but 100% correct) version of prefix sum (used for testing)
#ifndef USE_THRUST
static __global__ void prefix_sum
(int N_cells, int* cell_count, int* cell_count_sum)
{
    //<<< 1,1 >>>
    cell_count_sum[0] = 0;
    for (int i=1; i<N_cells; ++i) 
    cell_count_sum[i] = cell_count_sum[i-1] + cell_count[i-1];
}
#endif




// new version (faster)
template<int pbc_x, int pbc_y, int pbc_z>
static __global__ void gpu_find_neighbor_ON1
(
    int N, int* cell_counts, int* cell_count_sum, int* cell_contents, 
    int* NN, int* NL,
    real* x, real* y, real* z, int cell_n_x, int cell_n_y, int cell_n_z, 
    real *box, real cutoff, real cutoff_square
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;

    int count = 0;
    if (n1 < N)
    {
        real lx = box[0];
        real ly = box[1];
        real lz = box[2];

        real x1 = x[n1];
        real y1 = y[n1];
        real z1 = z[n1];
        
        int cell_id;
        int cell_id_x;
        int cell_id_y;
        int cell_id_z;
        find_cell_id
        (
            x1, y1, z1, cutoff, cell_n_x, cell_n_y, cell_n_z, 
            &cell_id_x, &cell_id_y, &cell_id_z, &cell_id
        );

        int klim = pbc_z ? 1 : 0;
        int jlim = pbc_y ? 1 : 0;
        int ilim = pbc_x ? 1 : 0;
       
        // loop over the neighbor cells of the central cell
        for (int k=-klim; k<klim+1; ++k)
        {
            for (int j=-jlim; j<jlim+1; ++j)
            {
                for (int i=-ilim; i<ilim+1; ++i)
                {
                    int neighbour=cell_id+k*cell_n_x*cell_n_y+j*cell_n_x+i;
                    if (cell_id_x + i < 0)
                        neighbour += cell_n_x;
                    if (cell_id_x + i >= cell_n_x) 
                        neighbour -= cell_n_x;
                    if (cell_id_y + j < 0)
                        neighbour += cell_n_y*cell_n_x;
                    if (cell_id_y + j >= cell_n_y) 
                        neighbour -= cell_n_y*cell_n_x;
                    if (cell_id_z + k < 0) 
                        neighbour += cell_n_z*cell_n_y*cell_n_x;
                    if (cell_id_z + k >= cell_n_z) 
                        neighbour -= cell_n_z*cell_n_y*cell_n_x;
                        
                    // loop over the atoms in a neighbor cell
                    for (int k = 0; k < cell_counts[neighbour]; ++k)
                    {
                        int n2 = cell_contents[cell_count_sum[neighbour] + k];
                        if (n1 == n2) continue;

                        real x12 = x[n2]-x1;
                        real y12 = y[n2]-y1;
                        real z12 = z[n2]-z1;

                        dev_apply_mic<pbc_x, pbc_y, pbc_z>
                        (lx, ly, lz, &x12, &y12, &z12);

                        real d2 = x12*x12 + y12*y12 + z12*z12;
                        if (d2 < cutoff_square)
                        {        
                            NL[count * N + n1] = n2;
                            count++;
                        }
                    }
                }
            }
        }
        NN[n1] = count;
    }
}




// a driver function
void find_neighbor_ON1
(Parameters *para, Atom *atom, int cell_n_x, int cell_n_y, int cell_n_z)
{                           
    int N = atom->N;
    int grid_size = (N - 1) / BLOCK_SIZE + 1; 
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    real rc = para->neighbor.rc;
    real rc2 = rc * rc; 
    int *NN = atom->NN;
    int *NL = atom->NL;
    real *x = atom->x;
    real *y = atom->y;
    real *z = atom->z;
    real *box = atom->box_length;

    int N_cells = cell_n_x * cell_n_y * cell_n_z;

    // some local data
    int* cell_count;
    int* cell_count_sum;
    int* cell_contents;
    CHECK(hipMalloc((void**)&cell_count, sizeof(int)*N_cells));
    CHECK(hipMemset(cell_count, 0, sizeof(int)*N_cells));
    CHECK(hipMalloc((void**)&cell_count_sum, sizeof(int)*N_cells));
    CHECK(hipMemset(cell_count_sum, 0, sizeof(int)*N_cells));
    CHECK(hipMalloc((void**)&cell_contents, sizeof(int)*N));
    CHECK(hipMemset(cell_contents, 0, sizeof(int)*N));

    // Find the number of particles in each cell
    find_cell_counts<<<grid_size, BLOCK_SIZE>>>
    (N, cell_count, x, y, z, cell_n_x, cell_n_y, cell_n_z, rc);
    #ifdef DEBUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif

#ifndef USE_THRUST
    // Simple (but 100% correct) version of prefix sum
    prefix_sum<<<1, 1>>>(N_cells, cell_count, cell_count_sum);
#else
    // use thrust to calculate the prefix sum
    thrust::exclusive_scan
    (thrust::device, cell_count, cell_count + N_cells, cell_count_sum);
#endif
    // reset to zero
    CHECK(hipMemset(cell_count, 0, sizeof(int)*N_cells));
	
    // Create particle list for each cell
    find_cell_contents<<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, 
        x, y, z, cell_n_x, cell_n_y, cell_n_z, rc
    );
    #ifdef DEBUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif

    // new version (faster)

    if (pbc_x && pbc_y && pbc_z)
    gpu_find_neighbor_ON1<1, 1, 1><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (pbc_x && pbc_y && !pbc_z)
    gpu_find_neighbor_ON1<1, 1, 0><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (pbc_x && !pbc_y && pbc_z)
    gpu_find_neighbor_ON1<1, 0, 1><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (!pbc_x && pbc_y && pbc_z)
    gpu_find_neighbor_ON1<0, 1, 1><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (pbc_x && !pbc_y && !pbc_z)
    gpu_find_neighbor_ON1<1, 0, 0><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (!pbc_x && pbc_y && !pbc_z)
    gpu_find_neighbor_ON1<0, 1, 0><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (!pbc_x && !pbc_y && pbc_z)
    gpu_find_neighbor_ON1<0, 0, 1><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );

    if (!pbc_x && !pbc_y && !pbc_z)
    gpu_find_neighbor_ON1<0, 0, 0><<<grid_size, BLOCK_SIZE>>>
    (
        N, cell_count, cell_count_sum, cell_contents, NN, NL, x, y, z, 
        cell_n_x, cell_n_y, cell_n_z, box, rc, rc2
    );


    #ifdef DEBUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif

    CHECK(hipFree(cell_count));
    CHECK(hipFree(cell_count_sum));
    CHECK(hipFree(cell_contents));
}



