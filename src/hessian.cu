#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
Then calculate the dynamical matrices with different k points.
------------------------------------------------------------------------------*/


#include "hessian.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"
#include "cusolver_wrapper.cuh"
#define BLOCK_SIZE 128


void Hessian::compute
(char* input_dir, Atom* atom, Force* force, Measure* measure)
{
    if (!yes) return;
    initialize(input_dir, atom->N);
    find_H(atom, force, measure);
    find_D(input_dir, atom);
    finalize();
}


void Hessian::read_basis(char* input_dir, int N)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/basis.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_basis);
    if (count != 1) print_error("reading error for basis.in\n");
    MY_MALLOC(basis, int, num_basis);
    MY_MALLOC(mass, real, num_basis);
    for (int m = 0; m < num_basis; ++m)
    {
        count = fscanf(fid, "%d%lf", &basis[m], &mass[m]);
        if (count != 2) print_error("reading error for basis.in\n");
    }
    MY_MALLOC(label, int, N);
    for (int n = 0; n < N; ++n)
    {
        count = fscanf(fid, "%d", &label[n]);
        if (count != 1) print_error("reading error for basis.in\n");
    }
    fclose(fid);
}


void Hessian::read_kpoints(char* input_dir)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/kpoints.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_kpoints);
    if (count != 1) print_error("reading error for kpoints.in\n");
    MY_MALLOC(kpoints, real, num_kpoints * 3);
    for (int m = 0; m < num_kpoints; ++m)
    {
        count = fscanf(fid, "%lf%lf%lf", &kpoints[m * 3 + 0],
            &kpoints[m * 3 + 1], &kpoints[m * 3 + 2]);
        if (count != 3) print_error("reading error for kpoints.in\n");
    }
    fclose(fid);
}


void Hessian::initialize(char* input_dir, int N)
{
    cutoff_square = cutoff * cutoff;
    read_basis(input_dir, N);
    read_kpoints(input_dir);
    int num_H = num_basis * N * 9;
    int num_D = num_basis * num_basis * 9;
    MY_MALLOC(H, real, num_H);
    MY_MALLOC(DR, real, num_D);
    MY_MALLOC(DI, real, num_D);
    for (int n = 0; n < num_H; ++n) { H[n] = 0; }
}


void Hessian::finalize(void)
{
    MY_FREE(basis);
    MY_FREE(label);
    MY_FREE(mass);
    MY_FREE(kpoints);
    MY_FREE(H);
    MY_FREE(DR);
    MY_FREE(DI);
}


static void apply_mic
(
    int triclinic, int pbc_x, int pbc_y, int pbc_z,
    real* h, real &x12, real &y12, real &z12
)
{
    if (triclinic == 0) // orthogonal box
    {
        if      (pbc_x == 1 && x12 < - h[0] * HALF) {x12 += h[0];}
        else if (pbc_x == 1 && x12 > + h[0] * HALF) {x12 -= h[0];}
        if      (pbc_y == 1 && y12 < - h[1] * HALF) {y12 += h[1];}
        else if (pbc_y == 1 && y12 > + h[1] * HALF) {y12 -= h[1];}
        if      (pbc_z == 1 && z12 < - h[2] * HALF) {z12 += h[2];}
        else if (pbc_z == 1 && z12 > + h[2] * HALF) {z12 -= h[2];}
    }
    else // triclinic box
    {
        real sx12 = h[9]  * x12 + h[10] * y12 + h[11] * z12;
        real sy12 = h[12] * x12 + h[13] * y12 + h[14] * z12;
        real sz12 = h[15] * x12 + h[16] * y12 + h[17] * z12;
        if (pbc_x == 1) sx12 -= nearbyint(sx12);
        if (pbc_y == 1) sy12 -= nearbyint(sy12);
        if (pbc_z == 1) sz12 -= nearbyint(sz12);
        x12 = h[0] * sx12 + h[1] * sy12 + h[2] * sz12;
        y12 = h[3] * sx12 + h[4] * sy12 + h[5] * sz12;
        z12 = h[6] * sx12 + h[7] * sy12 + h[8] * sz12;
    }
}


bool Hessian::is_too_far(int n1, int n2, Atom* atom)
{
    real x12 = atom->cpu_x[n2] - atom->cpu_x[n1];
    real y12 = atom->cpu_y[n2] - atom->cpu_y[n1];
    real z12 = atom->cpu_z[n2] - atom->cpu_z[n1];
    apply_mic
    (
        atom->box.triclinic, atom->box.pbc_x, atom->box.pbc_y,
        atom->box.pbc_z, atom->box.cpu_h, x12, y12, z12
    );
    real d12_square = x12 * x12 + y12 * y12 + z12 * z12;
    return (d12_square > cutoff_square);
}


void Hessian::find_H(Atom* atom, Force* force, Measure* measure)
{
    int N = atom->N;
    for (int nb = 0; nb < num_basis; ++nb)
    {
        int n1 = basis[nb];
        for (int n2 = 0; n2 < N; ++n2)
        {
            if(is_too_far(n1, n2, atom)) continue;
            int offset = (nb * N + n2) * 9;
            find_H12(dx, n1, n2, atom, force, measure, H + offset);
        }
    }
}


static void find_exp_ikr
(int n1, int n2, real* k, Atom* atom, real& cos_kr, real& sin_kr)
{
    real x12 = atom->cpu_x[n2] - atom->cpu_x[n1];
    real y12 = atom->cpu_y[n2] - atom->cpu_y[n1];
    real z12 = atom->cpu_z[n2] - atom->cpu_z[n1];
    apply_mic
    (
        atom->box.triclinic, atom->box.pbc_x, atom->box.pbc_y, 
        atom->box.pbc_z, atom->box.cpu_h, x12, y12, z12
    );
    real kr = k[0] * x12 + k[1] * y12 + k[2] * z12;
    cos_kr = cos(kr);
    sin_kr = sin(kr);
}


void Hessian::output_D(FILE* fid)
{
    for (int n1 = 0; n1 < num_basis * 3; ++n1)
    {
        int offset = n1 * num_basis * 3;
        for (int n2 = 0; n2 < num_basis * 3; ++n2)
        {
            fprintf(fid, "%g ", DR[offset + n2]);
        }
        if (num_kpoints > 1)
        {
            for (int n2 = 0; n2 < num_basis * 3; ++n2)
            {
                fprintf(fid, "%g ", DI[offset + n2]);
            }
        }
        fprintf(fid, "\n");
    }
}


void Hessian::find_omega(FILE* fid)
{
    int dim = num_basis * 3;
    double* W; MY_MALLOC(W, double, dim);
    eig_hermitian_Jacobi(dim, DR, DI, W);
    double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION*TIME_UNIT_CONVERSION);
    for (int n = 0; n < dim; ++n)
    {
        fprintf(fid, "%g ", W[n] * natural_to_THz);
    }
    fprintf(fid, "\n");
    MY_FREE(W);
}


void Hessian::find_D(char* input_dir, Atom* atom)
{
    char file_D[200];
    strcpy(file_D, input_dir);
    strcat(file_D, "/D.out");
    FILE *fid_D = fopen(file_D, "w");
    char file_omega2[200];
    strcpy(file_omega2, input_dir);
    strcat(file_omega2, "/omega2.out");
    FILE *fid_omega2 = fopen(file_omega2, "w");
    for (int nk = 0; nk < num_kpoints; ++nk)
    {
        for (int n = 0; n < num_basis*num_basis*9; ++n) { DR[n] = DI[n] = 0; }
        for (int nb = 0; nb < num_basis; ++nb)
        {
            int n1 = basis[nb];
            int label_1 = label[n1];
            real mass_1 = mass[label_1];
            for (int n2 = 0; n2 < atom->N; ++n2)
            {
                if(is_too_far(n1, n2, atom)) continue;
                real cos_kr, sin_kr;
                find_exp_ikr(n1, n2, kpoints + nk * 3, atom, cos_kr, sin_kr);
                int label_2 = label[n2];
                real mass_2 = mass[label_2];
                real mass_factor = 1.0 / sqrt(mass_1 * mass_2);
                real* H12 = H + (nb * atom->N + n2) * 9;
                for (int a = 0; a < 3; ++a)
                {
                    for (int b = 0; b < 3; ++b)
                    {
                        int a3b = a * 3 + b;
                        int row = label_1 * 3 + a;
                        int col = label_2 * 3 + b;
                        int index = row * num_basis * 3 + col;
                        DR[index] += H12[a3b] * cos_kr * mass_factor;
                        DI[index] += H12[a3b] * sin_kr * mass_factor;
                    }
                }
            }
        }
        output_D(fid_D);
        find_omega(fid_omega2);
    }
    fclose(fid_D);
    fclose(fid_omega2);
}


void Hessian::find_H12
(real dx, int n1, int n2, Atom *atom, Force *force, Measure* measure, real* H12)
{
    real dx2 = dx * 2;
    real f_positive[3];
    real f_negative[3];
    for (int beta = 0; beta < 3; ++beta)
    {
        get_f(-dx, n1, n2, beta, atom, force, measure, f_negative);
        get_f(dx, n1, n2, beta, atom, force, measure, f_positive);
        for (int alpha = 0; alpha < 3; ++alpha)
        {
            int index = alpha * 3 + beta;
            H12[index] = (f_negative[alpha] - f_positive[alpha]) / dx2;
        }
    }
}


void Hessian::get_f
(
    real dx, int n1, int n2, int beta, 
    Atom* atom, Force *force, Measure* measure, real* f
)
{
    shift_atom(dx, n2, beta, atom);
    force->compute(atom, measure);
    int M = sizeof(real);
    CHECK(hipMemcpy(f + 0, atom->fx + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 1, atom->fy + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 2, atom->fz + n1, M, hipMemcpyDeviceToHost));
    shift_atom(-dx, n2, beta, atom);
}


static __global__ void gpu_shift_atom(real dx, real *x)
{
    x[0] += dx;
}


void Hessian::shift_atom(real dx, int n2, int beta, Atom* atom)
{
    if (beta == 0)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->x + n2);
        CUDA_CHECK_KERNEL
    }
    else if (beta == 1)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->y + n2);
        CUDA_CHECK_KERNEL
    }
    else
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->z + n2);
        CUDA_CHECK_KERNEL
    }
}


