#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
Then calculate the dynamical matrices with different k points.
------------------------------------------------------------------------------*/


#include "hessian.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"
#include "mic.cuh"
#include "cusolver_wrapper.cuh"
#include "read_file.cuh"

#define BLOCK_SIZE 128


void Hessian::compute
(char* input_dir, Atom* atom, Force* force, Measure* measure)
{
    initialize(input_dir, atom->N);
    find_H(atom, force, measure);
    find_D(input_dir, atom);

    // currently for Alex's GKMA calculations
    if (num_kpoints == 1)
    {
        find_eigenvectors(input_dir, atom);
    }

    finalize();
}


void Hessian::read_basis(char* input_dir, int N)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/basis.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_basis);
    PRINT_SCANF_ERROR(count, 1, "Reading error for basis.in.");

    MY_MALLOC(basis, int, num_basis);
    MY_MALLOC(mass, real, num_basis);
    for (int m = 0; m < num_basis; ++m)
    {
        count = fscanf(fid, "%d%lf", &basis[m], &mass[m]);
        PRINT_SCANF_ERROR(count, 2, "Reading error for basis.in.");
    }
    MY_MALLOC(label, int, N);
    for (int n = 0; n < N; ++n)
    {
        count = fscanf(fid, "%d", &label[n]);
        PRINT_SCANF_ERROR(count, 1, "Reading error for basis.in.");
    }
    fclose(fid);
}


void Hessian::read_kpoints(char* input_dir)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/kpoints.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_kpoints);
    PRINT_SCANF_ERROR(count, 1, "Reading error for kpoints.in.");

    MY_MALLOC(kpoints, real, num_kpoints * 3);
    for (int m = 0; m < num_kpoints; ++m)
    {
        count = fscanf(fid, "%lf%lf%lf", &kpoints[m * 3 + 0],
            &kpoints[m * 3 + 1], &kpoints[m * 3 + 2]);
        PRINT_SCANF_ERROR(count, 3, "Reading error for kpoints.in.");
    }
    fclose(fid);
}


void Hessian::initialize(char* input_dir, int N)
{
    read_basis(input_dir, N);
    read_kpoints(input_dir);
    int num_H = num_basis * N * 9;
    int num_D = num_basis * num_basis * 9 * num_kpoints;
    MY_MALLOC(H, real, num_H);
    MY_MALLOC(DR, real, num_D);
    MY_MALLOC(DI, real, num_D);
    for (int n = 0; n < num_H; ++n) { H[n] = 0; }
    for (int n = 0; n < num_D; ++n) { DR[n] = DI[n] = 0; }
}


void Hessian::finalize(void)
{
    MY_FREE(basis);
    MY_FREE(label);
    MY_FREE(mass);
    MY_FREE(kpoints);
    MY_FREE(H);
    MY_FREE(DR);
    MY_FREE(DI);
}


bool Hessian::is_too_far(int n1, int n2, Atom* atom)
{
    real x12 = atom->cpu_x[n2] - atom->cpu_x[n1];
    real y12 = atom->cpu_y[n2] - atom->cpu_y[n1];
    real z12 = atom->cpu_z[n2] - atom->cpu_z[n1];
    apply_mic
    (
        atom->box.triclinic, atom->box.pbc_x, atom->box.pbc_y,
        atom->box.pbc_z, atom->box.cpu_h, x12, y12, z12
    );
    real d12_square = x12 * x12 + y12 * y12 + z12 * z12;
    return (d12_square > (cutoff * cutoff));
}


void Hessian::find_H(Atom* atom, Force* force, Measure* measure)
{
    int N = atom->N;
    for (int nb = 0; nb < num_basis; ++nb)
    {
        int n1 = basis[nb];
        for (int n2 = 0; n2 < N; ++n2)
        {
            if(is_too_far(n1, n2, atom)) continue;
            int offset = (nb * N + n2) * 9;
            find_H12(n1, n2, atom, force, measure, H + offset);
        }
    }
}


static void find_exp_ikr
(int n1, int n2, real* k, Atom* atom, real& cos_kr, real& sin_kr)
{
    real x12 = atom->cpu_x[n2] - atom->cpu_x[n1];
    real y12 = atom->cpu_y[n2] - atom->cpu_y[n1];
    real z12 = atom->cpu_z[n2] - atom->cpu_z[n1];
    apply_mic
    (
        atom->box.triclinic, atom->box.pbc_x, atom->box.pbc_y, 
        atom->box.pbc_z, atom->box.cpu_h, x12, y12, z12
    );
    real kr = k[0] * x12 + k[1] * y12 + k[2] * z12;
    cos_kr = cos(kr);
    sin_kr = sin(kr);
}


void Hessian::output_D(char* input_dir)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/D.out");
    FILE *fid = fopen(file, "w");
    for (int nk = 0; nk < num_kpoints; ++nk)
    {
        int offset = nk * num_basis * num_basis * 9;
        for (int n1 = 0; n1 < num_basis * 3; ++n1)
        {
            for (int n2 = 0; n2 < num_basis * 3; ++n2)
            {
                // cuSOLVER requires column-major
                fprintf(fid, "%g ", DR[offset + n1 + n2 * num_basis * 3]);
            }
            if (num_kpoints > 1)
            {
                for (int n2 = 0; n2 < num_basis * 3; ++n2)
                {
                    // cuSOLVER requires column-major
                    fprintf(fid, "%g ", DI[offset + n1 + n2 * num_basis * 3]);
                }
            }
            fprintf(fid, "\n");
        }
    }
    fclose(fid);
}


void Hessian::find_omega(FILE* fid, int offset)
{
    int dim = num_basis * 3;
    double* W; MY_MALLOC(W, double, dim);
    eig_hermitian_QR(dim, DR+offset, DI+offset, W);
    double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION*TIME_UNIT_CONVERSION);
    for (int n = 0; n < dim; ++n)
    {
        fprintf(fid, "%g ", W[n] * natural_to_THz);
    }
    fprintf(fid, "\n");
    MY_FREE(W);
}


void Hessian::find_omega_batch(FILE* fid)
{
    int dim = num_basis * 3;
    double* W; MY_MALLOC(W, double, dim * num_kpoints);
    eig_hermitian_Jacobi_batch(dim, num_kpoints, DR, DI, W);
    double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION*TIME_UNIT_CONVERSION);
    for (int nk = 0; nk < num_kpoints; ++nk)
    {
        int offset = nk * dim;
        for (int n = 0; n < dim; ++n)
        {
            fprintf(fid, "%g ", W[offset + n] * natural_to_THz);
        }
        fprintf(fid, "\n");
    }
    MY_FREE(W);
}


void Hessian::find_D(char* input_dir, Atom* atom)
{
    char file_omega2[200];
    strcpy(file_omega2, input_dir);
    strcat(file_omega2, "/omega2.out");
    FILE *fid_omega2 = fopen(file_omega2, "w");
    for (int nk = 0; nk < num_kpoints; ++nk)
    {
        int offset = nk * num_basis * num_basis * 9;
        for (int nb = 0; nb < num_basis; ++nb)
        {
            int n1 = basis[nb];
            int label_1 = label[n1];
            real mass_1 = mass[label_1];
            for (int n2 = 0; n2 < atom->N; ++n2)
            {
                if(is_too_far(n1, n2, atom)) continue;
                real cos_kr, sin_kr;
                find_exp_ikr(n1, n2, kpoints + nk * 3, atom, cos_kr, sin_kr);
                int label_2 = label[n2];
                real mass_2 = mass[label_2];
                real mass_factor = 1.0 / sqrt(mass_1 * mass_2);
                real* H12 = H + (nb * atom->N + n2) * 9;
                for (int a = 0; a < 3; ++a)
                {
                    for (int b = 0; b < 3; ++b)
                    {
                        int a3b = a * 3 + b;
                        int row = label_1 * 3 + a;
                        int col = label_2 * 3 + b;
                        // cuSOLVER requires column-major
                        int index = offset + col * num_basis * 3 + row;
                        DR[index] += H12[a3b] * cos_kr * mass_factor;
                        DI[index] += H12[a3b] * sin_kr * mass_factor;
                    }
                }
            }
        }
        if (num_basis > 10) { find_omega(fid_omega2, offset); } // > 32x32
    }
    output_D(input_dir);
    if (num_basis <= 10) { find_omega_batch(fid_omega2); } // <= 32x32
    fclose(fid_omega2);
}


void Hessian::find_H12
(int n1, int n2, Atom *atom, Force *force, Measure* measure, real* H12)
{
    real dx2 = displacement * 2;
    real f_positive[3];
    real f_negative[3];
    for (int beta = 0; beta < 3; ++beta)
    {
        get_f(-displacement, n1, n2, beta, atom, force, measure, f_negative);
        get_f(displacement, n1, n2, beta, atom, force, measure, f_positive);
        for (int alpha = 0; alpha < 3; ++alpha)
        {
            int index = alpha * 3 + beta;
            H12[index] = (f_negative[alpha] - f_positive[alpha]) / dx2;
        }
    }
}


void Hessian::find_eigenvectors(char* input_dir, Atom* atom)
{
    char file_eigenvectors[200];
    strcpy(file_eigenvectors, input_dir);
    strcat(file_eigenvectors, "/eigenvector.out");
    FILE *fid_eigenvectors = my_fopen(file_eigenvectors, "w");

    int dim = num_basis * 3;
    double* W; MY_MALLOC(W, double, dim);
    double* eigenvectors; MY_MALLOC(eigenvectors, double, dim * dim);
    eigenvectors_symmetric_Jacobi(dim, DR, W, eigenvectors);

    double natural_to_THz = 1.0e6 / (TIME_UNIT_CONVERSION*TIME_UNIT_CONVERSION);

    // output eigenvalues
    for(int n = 0; n < dim; n++)
    {
        fprintf(fid_eigenvectors, "%g ",  W[n] * natural_to_THz);
    }
    fprintf(fid_eigenvectors, "\n");

    // output eigenvectors
    for(int col = 0; col < dim; col++)
    {
        for (int a = 0; a < 3; a++)
        {
            for(int b = 0; b < num_basis; b++)
            {
                 int row = a + b * 3;
                 // column-major order from cuSolver
                 fprintf(fid_eigenvectors, "%g ",  eigenvectors[row+col*dim]);
            }
        }
        fprintf(fid_eigenvectors, "\n");
    }

    MY_FREE(W);
    MY_FREE(eigenvectors);
    fclose(fid_eigenvectors);
}


void Hessian::get_f
(
    real dx, int n1, int n2, int beta, 
    Atom* atom, Force *force, Measure* measure, real* f
)
{
    shift_atom(dx, n2, beta, atom);
    force->compute(atom, measure);
    int M = sizeof(real);
    CHECK(hipMemcpy(f + 0, atom->fx + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 1, atom->fy + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 2, atom->fz + n1, M, hipMemcpyDeviceToHost));
    shift_atom(-dx, n2, beta, atom);
}


static __global__ void gpu_shift_atom(real dx, real *x)
{
    x[0] += dx;
}


void Hessian::shift_atom(real dx, int n2, int beta, Atom* atom)
{
    if (beta == 0)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->x + n2);
        CUDA_CHECK_KERNEL
    }
    else if (beta == 1)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->y + n2);
        CUDA_CHECK_KERNEL
    }
    else
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->z + n2);
        CUDA_CHECK_KERNEL
    }
}


void Hessian::parse_cutoff(char **param, int num_param)
{
    if (num_param != 2)
    {
        PRINT_INPUT_ERROR("cutoff should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &cutoff))
    {
        PRINT_INPUT_ERROR("cutoff for hessian should be a number.\n");
    }
    if (cutoff <= 0)
    {
        PRINT_INPUT_ERROR("cutoff for hessian should be positive.\n");
    }
    printf("Cutoff distance for hessian = %g A.\n", cutoff);
}


void Hessian::parse_delta(char **param, int num_param)
{
    if (num_param != 2)
    {
        PRINT_INPUT_ERROR("delta should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &displacement))
    {
        PRINT_INPUT_ERROR("delta for hessian should be a number.\n");
    }
    if (displacement <= 0)
    {
        PRINT_INPUT_ERROR("delta for hessian should be positive.\n");
    }
    printf("delta for hessian = %g A.\n", displacement);
}


