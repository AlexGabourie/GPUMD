#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
------------------------------------------------------------------------------*/


#include "hessian.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"
#define BLOCK_SIZE 128


void Hessian::compute
(char* input_dir, Atom* atom, Force* force, Measure* measure)
{
    initialize(input_dir, atom->N);
    find_H(input_dir, atom, force, measure);
    find_D(input_dir, atom, force, measure);
    finalize();
}


void Hessian::read_basis(char* input_dir, int N)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/basis.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_basis);
    if (count != 1) print_error("reading error for basis.in\n");
    MY_MALLOC(basis, int, num_basis);
    MY_MALLOC(mass, real, num_basis);
    for (int m = 0; m < num_basis; ++m)
    {
        count = fscanf(fid, "%d%lf", &basis[m], &mass[m]);
        if (count != 2) print_error("reading error for basis.in\n");
    }
    MY_MALLOC(label, int, N);
    for (int n = 0; n < N; ++n)
    {
        count = fscanf(fid, "%d", &label[n]);
        if (count != 1) print_error("reading error for basis.in\n");
    }
    fclose(fid);
}


void Hessian::read_kpoints(char* input_dir)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/kpoints.in");
    FILE *fid = fopen(file, "r");
    int count;
    count = fscanf(fid, "%d", &num_kpoints);
    if (count != 1) print_error("reading error for kpoints.in\n");
    MY_MALLOC(kpoints, real, num_kpoints * 3);
    for (int m = 0; m < num_kpoints; ++m)
    {
        count = fscanf(fid, "%lf%lf%lf", &kpoints[m * 3 + 0],
            &kpoints[m * 3 + 1], &kpoints[m * 3 + 2]);
        if (count != 3) print_error("reading error for kpoints.in\n");
    }
    fclose(fid);
}


void Hessian::initialize(char* input_dir, int N)
{
    read_basis(input_dir, N);
    read_kpoints(input_dir);
    MY_MALLOC(H, real, num_basis * N * 9);
    MY_MALLOC(DR, real, num_basis * num_basis * 9);
    MY_MALLOC(DI, real, num_basis * num_basis * 9);
    for (int n = 0; n < num_basis * num_basis * 9; ++n) { DR[n] = DI[n] = 0; }
}


void Hessian::finalize(void)
{
    MY_FREE(basis);
    MY_FREE(label);
    MY_FREE(mass);
    MY_FREE(kpoints);
    MY_FREE(H);
    MY_FREE(DR);
    MY_FREE(DI);
}


void Hessian::find_H
(char* input_dir, Atom* atom, Force* force, Measure* measure)
{
    int N = atom->N;
    for (int nb = 0; nb < num_basis; ++nb)
    {
        int n1 = basis[nb];
        for (int n2 = 0; n2 < N; ++n2)
        {
            int offset = (nb * N + n2) * 9;
            find_H12(dx, n1, n2, atom, force, measure, H + offset);
        }
    }
}


static void apply_mic
(
    int triclinic, int pbc_x, int pbc_y, int pbc_z,
    real* h, real &x12, real &y12, real &z12
)
{
    if (triclinic == 0) // orthogonal box
    {
        if      (pbc_x == 1 && x12 < - h[0] * HALF) {x12 += h[0];}
        else if (pbc_x == 1 && x12 > + h[0] * HALF) {x12 -= h[0];}
        if      (pbc_y == 1 && y12 < - h[1] * HALF) {y12 += h[1];}
        else if (pbc_y == 1 && y12 > + h[1] * HALF) {y12 -= h[1];}
        if      (pbc_z == 1 && z12 < - h[2] * HALF) {z12 += h[2];}
        else if (pbc_z == 1 && z12 > + h[2] * HALF) {z12 -= h[2];}
    }
    else // triclinic box
    {
        real sx12 = h[9]  * x12 + h[10] * y12 + h[11] * z12;
        real sy12 = h[12] * x12 + h[13] * y12 + h[14] * z12;
        real sz12 = h[15] * x12 + h[16] * y12 + h[17] * z12;
        if (pbc_x == 1) sx12 -= nearbyint(sx12);
        if (pbc_y == 1) sy12 -= nearbyint(sy12);
        if (pbc_z == 1) sz12 -= nearbyint(sz12);
        x12 = h[0] * sx12 + h[1] * sy12 + h[2] * sz12;
        y12 = h[3] * sx12 + h[4] * sy12 + h[5] * sz12;
        z12 = h[6] * sx12 + h[7] * sy12 + h[8] * sz12;
    }
}


void Hessian::find_D
(char* input_dir, Atom* atom, Force* force, Measure* measure)
{
    char file[200];
    strcpy(file, input_dir);
    strcat(file, "/D.out");
    FILE *fid = fopen(file, "w");
    for (int nk = 0; nk < num_kpoints; ++nk)
    {
        for (int nb = 0; nb < num_basis; ++nb)
        {
            int n1 = basis[nb];
            int label_1 = label[n1];
            real mass_1 = mass[label_1];
            for (int n2 = 0; n2 < atom->N; ++n2)
            {
                real x12, y12, z12;
                x12 = atom->cpu_x[n2] - atom->cpu_x[n1];
                y12 = atom->cpu_y[n2] - atom->cpu_y[n1];
                z12 = atom->cpu_z[n2] - atom->cpu_z[n1];
                apply_mic
                (
                    atom->box.triclinic, atom->box.pbc_x, atom->box.pbc_y, 
                    atom->box.pbc_z, atom->box.cpu_h, x12, y12, z12
                );
                real kr = kpoints[nk * 3 + 0] * x12 
                        + kpoints[nk * 3 + 1] * y12 
                        + kpoints[nk * 3 + 2] * z12;
                int label_2 = label[n2];
                real mass_2 = mass[label_2];
                real mass_factor = 1.0 / sqrt(mass_1 * mass_2);
                real* H12 = H + (nb * atom->N + n2) * 9;
                int offset = (label_1 * num_basis + label_2) * 9;
                for (int a = 0; a < 3; ++a)
                {
                    for (int b = 0; b < 3; ++b)
                    {
                        int a3b = a * 3 + b;
                        DR[offset + a3b] += H12[a3b] * cos(kr) * mass_factor;
                        DI[offset + a3b] += H12[a3b] * sin(kr) * mass_factor;
                    }
                }
            }
        }
        for (int b1 = 0; b1 < num_basis; ++b1)
        {
            for (int b2 = 0; b2 < num_basis; ++b2)
            {
                int offset = (b1 * num_basis + b2) * 9;
                for (int k = 0; k < 9; ++k) 
                {
                    fprintf(fid, "%g ", DR[offset + k]);
                }
                if (num_kpoints > 1)
                {
                    for (int k = 0; k < 9; ++k) 
                    {
                        fprintf(fid, "%g ", DI[offset + k]);
                    }
                }
                fprintf(fid, "\n");
            }
        }
    }
    fclose(fid);
}


void Hessian::find_H12
(real dx, int n1, int n2, Atom *atom, Force *force, Measure* measure, real* H12)
{
    real dx2 = dx * 2;
    real f_positive[3];
    real f_negative[3];
    for (int beta = 0; beta < 3; ++beta)
    {
        get_f(-dx, n1, n2, beta, atom, force, measure, f_negative);
        get_f(dx, n1, n2, beta, atom, force, measure, f_positive);
        for (int alpha = 0; alpha < 3; ++alpha)
        {
            int index = alpha * 3 + beta;
            H12[index] = (f_negative[alpha] - f_positive[alpha]) / dx2;
        }
    }
}


void Hessian::get_f
(
    real dx, int n1, int n2, int beta, 
    Atom* atom, Force *force, Measure* measure, real* f
)
{
    shift_atom(dx, n2, beta, atom);
    force->compute(atom, measure);
    int M = sizeof(real);
    CHECK(hipMemcpy(f + 0, atom->fx + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 1, atom->fy + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 2, atom->fz + n1, M, hipMemcpyDeviceToHost));
    shift_atom(-dx, n2, beta, atom);
}


static __global__ void gpu_shift_atom(real dx, real *x)
{
    x[0] += dx;
}


void Hessian::shift_atom(real dx, int n2, int beta, Atom* atom)
{
    if (beta == 0)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->x + n2);
        CUDA_CHECK_KERNEL
    }
    else if (beta == 1)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->y + n2);
        CUDA_CHECK_KERNEL
    }
    else
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->z + n2);
        CUDA_CHECK_KERNEL
    }
}


