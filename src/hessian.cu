#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Use finite difference to calculate the hessian (force constants).
    H_ij^ab = [F_i^a(-) - F_i^a(+)] / [u_j^b(+) - u_j^b(-)]
------------------------------------------------------------------------------*/


#include "hessian.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "error.cuh"
#define BLOCK_SIZE 128


Hessian::Hessian
(real dx, int n1, int n2, Atom *atom, Force *force, Measure* measure, real* H12)
{
    real dx2 = dx * 2;
    real f_positive[3];
    real f_negative[3];
    for (int beta = 0; beta < 3; ++beta)
    {
        get_f(-dx, n1, n2, beta, atom, force, measure, f_negative);
        get_f(dx, n1, n2, beta, atom, force, measure, f_positive);
        for (int alpha = 0; alpha < 3; ++alpha)
        {
            int index = alpha * 3 + beta;
            H12[index] = (f_negative[alpha] - f_positive[alpha]) / dx2;
        }
    }
}


void Hessian::get_f
(
    real dx, int n1, int n2, int beta, 
    Atom* atom, Force *force, Measure* measure, real* f
)
{
    shift_atom(dx, n2, beta, atom);
    force->compute(atom, measure);
    int M = sizeof(real);
    CHECK(hipMemcpy(f + 0, atom->fx + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 1, atom->fy + n1, M, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(f + 2, atom->fz + n1, M, hipMemcpyDeviceToHost));
    shift_atom(-dx, n2, beta, atom);
}


static __global__ void gpu_shift_atom(real dx, real *x)
{
    x[0] += dx;
}


void Hessian::shift_atom(real dx, int n2, int beta, Atom* atom)
{
    if (beta == 0)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->x + n2);
        CUDA_CHECK_KERNEL
    }
    else if (beta == 1)
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->y + n2);
        CUDA_CHECK_KERNEL
    }
    else
    {
        gpu_shift_atom<<<1, 1>>>(dx, atom->z + n2);
        CUDA_CHECK_KERNEL
    }
}


