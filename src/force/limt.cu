#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
A new potential that is under development
LIMT = Lattie Inversion based Morse-Tersoff
------------------------------------------------------------------------------*/

#include "limt.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"

#define BLOCK_SIZE_FORCE 64

LIMT::LIMT(FILE* fid, int num_of_types, const Neighbor& neighbor)
{
  num_types = num_of_types;
  printf("Use LIMT (%d-element) potential.\n", num_types);
  int n_entries = 2 * num_types - 1; // 1 or 3 entries

  const char err[] = "Reading error for LIMT potential.\n";
  rc = 0.0;
  int count;
  double d0, a, r0, s, n, beta, h, r1, r2, gamma;
  for (int i = 0; i < n_entries; i++) {
    count = fscanf(
      fid, "%lf%lf%lf%lf%lf%lf%lf%lf%lf%lf", &d0, &a, &r0, &s, &n, &beta, &h, &r1, &r2, &gamma);
    PRINT_SCANF_ERROR(count, 10, err);
    if (d0 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (a <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (r0 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (n < 0.0)
      PRINT_INPUT_ERROR(err);
    if (beta < 0.0)
      PRINT_INPUT_ERROR(err);
    if (h < -1.0 || h > 1.0)
      PRINT_INPUT_ERROR(err);
    if (r1 < 0.0)
      PRINT_INPUT_ERROR(err);
    if (r2 <= 0.0)
      PRINT_INPUT_ERROR(err);
    if (r2 <= r1)
      PRINT_INPUT_ERROR(err);
    if (gamma < 0.0)
      PRINT_INPUT_ERROR(err);

    para.a[i] = d0 / (s - 1.0) * exp(sqrt(2.0 * s) * a * r0);
    para.b[i] = s * d0 / (s - 1.0) * exp(sqrt(2.0 / s) * a * r0);
    para.lambda[i] = sqrt(2.0 * s) * a;
    para.mu[i] = sqrt(2.0 / s) * a;
    para.n[i] = n;
    para.beta[i] = beta;
    para.h[i] = h;
    para.r1[i] = r1;
    para.r2[i] = r2;
    para.gamma[i] = gamma;
    para.pi_factor[i] = PI / (r2 - r1);
    para.minus_half_over_n[i] = -0.5 / n;
    rc = r2 > rc ? r2 : rc;
  }

  const int number_of_atoms = neighbor.NN.size();
  const int num_of_neighbors = min(neighbor.MN, 50) * number_of_atoms;
  LIMT_data.b.resize(num_of_neighbors);
  LIMT_data.bp.resize(num_of_neighbors);
  LIMT_data.f12x.resize(num_of_neighbors);
  LIMT_data.f12y.resize(num_of_neighbors);
  LIMT_data.f12z.resize(num_of_neighbors);
  LIMT_data.NN_short.resize(number_of_atoms);
  LIMT_data.NL_short.resize(num_of_neighbors);
}

LIMT::~LIMT(void)
{
  // nothing
}

static __device__ void find_fr_and_frp(double a, double lambda, double d12, double& fr, double& frp)
{
  fr = a * exp(-lambda * d12);
  frp = -lambda * fr;
}

static __device__ void find_fa_and_fap(double b, double mu, double d12, double& fa, double& fap)
{
  fa = b * exp(-mu * d12);
  fap = -mu * fa;
}

static __device__ void find_fa(double b, double mu, double d12, double& fa)
{
  fa = b * exp(-mu * d12);
}

static __device__ void
find_fc_and_fcp(double r1, double r2, double pi_factor, double d12, double& fc, double& fcp)
{
  if (d12 < r1) {
    fc = 1.0;
    fcp = 0.0;
  } else if (d12 < r2) {
    fc = 0.5 * cos(pi_factor * (d12 - r1)) + 0.5;
    fcp = -sin(pi_factor * (d12 - r1)) * pi_factor * 0.5;
  } else {
    fc = 0.0;
    fcp = 0.0;
  }
}

static __device__ void find_fc(double r1, double r2, double pi_factor, double d12, double& fc)
{
  if (d12 < r1) {
    fc = 1.0;
  } else if (d12 < r2) {
    fc = 0.5 * cos(pi_factor * (d12 - r1)) + 0.5;
  } else {
    fc = 0.0;
  }
}

static __device__ void find_g_and_gp(double h, double cos, double& g, double& gp)
{
  double tmp = cos - h;
  g = tmp * tmp;
  gp = 2.0 * tmp;
}

static __device__ void find_g(double h, double cos, double& g)
{
  double tmp = cos - h;
  g = tmp * tmp;
}

// 2-body part (kernel)
static __global__ void find_force_step0(
  const LIMT_Para para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  int* g_NN_short,
  int* g_NL_short,
  const int* g_type,
  const int shift,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial,
  double* g_potential)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  double s_fx = 0.0;                                   // force_x
  double s_fy = 0.0;                                   // force_y
  double s_fz = 0.0;                                   // force_z
  double s_pe = 0.0;                                   // potential energy
  double s_sxx = 0.0;                                  // virial_stress_xx
  double s_sxy = 0.0;                                  // virial_stress_xy
  double s_sxz = 0.0;                                  // virial_stress_xz
  double s_syx = 0.0;                                  // virial_stress_yx
  double s_syy = 0.0;                                  // virial_stress_yy
  double s_syz = 0.0;                                  // virial_stress_yz
  double s_szx = 0.0;                                  // virial_stress_zx
  double s_szy = 0.0;                                  // virial_stress_zy
  double s_szz = 0.0;                                  // virial_stress_zz

  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_NN[n1];
    int type1 = g_type[n1] - shift;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    int count = 0; // initialize g_NN_short[n1] to 0

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_NL[n1 + number_of_particles * i1];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      int type12 = type1 + g_type[n2] - shift;

      if (d12 < para.r1[type12]) {
        g_NL_short[n1 + number_of_particles * (count++)] = n2;
      } else {
        double fr, frp, fa, fap;
        find_fr_and_frp(para.a[type12], para.lambda[type12], d12, fr, frp);
        find_fa_and_fap(para.b[type12], para.mu[type12], d12, fa, fap);
        double f2 = (frp - fap) / d12;
        s_pe += (fr - fa) * 0.5f;
        s_fx += x12 * f2;
        s_fy += y12 * f2;
        s_fz += z12 * f2;
        s_sxx -= x12 * x12 * f2 * 0.5f;
        s_syy -= y12 * y12 * f2 * 0.5f;
        s_szz -= z12 * z12 * f2 * 0.5f;
        s_sxy -= x12 * y12 * f2 * 0.5f;
        s_sxz -= x12 * z12 * f2 * 0.5f;
        s_syz -= y12 * z12 * f2 * 0.5f;
        s_syx -= y12 * x12 * f2 * 0.5f;
        s_szx -= z12 * x12 * f2 * 0.5f;
        s_szy -= z12 * y12 * f2 * 0.5f;
      }
    }

    g_NN_short[n1] = count; // now the local neighbor list has been built

    g_fx[n1] += s_fx; // save force
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;
  }
}

// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_step1(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const int shift,
  const LIMT_Para para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_b,
  double* g_bp)
{
  // start from the N1-th atom
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  // to the (N2-1)-th atom
  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1] - shift;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type12 = type1 + g_type[n2] - shift;
      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double zeta = 0.0;
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_particles * i2];
        int type13 = type1 + g_type[n3] - shift;
        if (n3 == n2) {
          continue;
        } // ensure that n3 != n2
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12 * d13);
        double fc13, g123;
        find_fc(para.r1[type13], para.r2[type13], para.pi_factor[type13], d13, fc13);
        find_g(para.h[type12], cos123, g123);
        zeta += fc13 * g123;
      }

      double bzn, b12;
      bzn = pow(para.beta[type12] * zeta, para.n[type12]);
      b12 = pow(1.0 + bzn, para.minus_half_over_n[type12]);
      if (zeta < 1.0e-16) // avoid division by 0
      {
        g_b[i1 * number_of_particles + n1] = 1.0;
        g_bp[i1 * number_of_particles + n1] = 0.0;
      } else {
        g_b[i1 * number_of_particles + n1] = b12;
        g_bp[i1 * number_of_particles + n1] = -b12 * bzn * 0.5 / ((1.0 + bzn) * zeta);
      }
    }
  }
}

// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void __launch_bounds__(BLOCK_SIZE_FORCE, 10) find_force_step2(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int num_types,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const int shift,
  const LIMT_Para para,
  const double* __restrict__ g_b,
  const double* __restrict__ g_bp,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_potential,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  // start from the N1-th atom
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  // to the (N2-1)-th atom
  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1] - shift;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    double pot_energy = 0.0;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_particles + n1;
      int n2 = g_neighbor_list[index];
      int type12 = type1 + g_type[n2] - shift;

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
      double d12inv = 1.0 / d12;
      double fc12, fcp12, fa12, fap12, fr12, frp12;
      find_fc_and_fcp(para.r1[type12], para.r2[type12], para.pi_factor[type12], d12, fc12, fcp12);
      find_fa_and_fap(para.b[type12], para.mu[type12], d12, fa12, fap12);
      find_fr_and_frp(para.a[type12], para.lambda[type12], d12, fr12, frp12);

      // (i,j) part
      double b12 = g_b[index];
      double factor3 = (fcp12 * (fr12 - b12 * fa12) + fc12 * (frp12 - b12 * fap12)) * d12inv;
      double f12x = x12 * factor3 * 0.5;
      double f12y = y12 * factor3 * 0.5;
      double f12z = z12 * factor3 * 0.5;

      // accumulate potential energy
      pot_energy += fc12 * (fr12 - b12 * fa12) * 0.5;

      // (i,j,k) part
      double bp12 = g_bp[index];
      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int index_2 = n1 + number_of_particles * i2;
        int n3 = g_neighbor_list[index_2];
        if (n3 == n2) {
          continue;
        }
        int type13 = type1 + g_type[n3] - shift;
        double x13 = g_x[n3] - x1;
        double y13 = g_y[n3] - y1;
        double z13 = g_z[n3] - z1;
        apply_mic(box, x13, y13, z13);
        double d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
        double fc13, fa13;
        find_fc(para.r1[type13], para.r2[type13], para.pi_factor[type13], d13, fc13);
        find_fa(para.b[type13], para.mu[type13], d13, fa13);
        double bp13 = g_bp[index_2];
        double one_over_d12d13 = 1.0 / (d12 * d13);
        double cos123 = (x12 * x13 + y12 * y13 + z12 * z13) * one_over_d12d13;
        double cos123_over_d12d12 = cos123 * d12inv * d12inv;
        double g123, gp123;
        find_g_and_gp(para.h[type12], cos123, g123, gp123);
        // derivatives with cosine
        double dc = -fc12 * bp12 * fa12 * fc13 * gp123 - fc12 * bp13 * fa13 * fc13 * gp123;
        // derivatives with rij
        double dr = -fcp12 * bp13 * fa13 * g123 * fc13 * d12inv;
        double cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
        f12x += (x12 * dr + dc * cos_d) * 0.5;
        cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
        f12y += (y12 * dr + dc * cos_d) * 0.5;
        cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
        f12z += (z12 * dr + dc * cos_d) * 0.5;
      }
      g_f12x[index] = f12x;
      g_f12y[index] = f12y;
      g_f12z[index] = f12z;
    }
    // save potential
    g_potential[n1] += pot_energy;
  }
}

__global__ void
scale_force(const int N1, const int N2, const double gamma, double* fx, double* fy, double* fz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    fx[n1] *= gamma;
    fy[n1] *= gamma;
    fz[n1] *= gamma;
  }
}

void LIMT::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_FORCE + 1;

  find_force_step0<<<grid_size, BLOCK_SIZE_FORCE>>>(
    para, number_of_atoms, N1, N2, box, neighbor.NN_local.data(), neighbor.NL_local.data(),
    LIMT_data.NN_short.data(), LIMT_data.NL_short.data(), type.data(), type_shift,
    position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, force_per_atom.data(),
    force_per_atom.data() + number_of_atoms, force_per_atom.data() + 2 * number_of_atoms,
    virial_per_atom.data(), potential_per_atom.data());
  CUDA_CHECK_KERNEL

  // pre-compute the bond order functions and their derivatives
  find_force_step1<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, num_types, LIMT_data.NN_short.data(), LIMT_data.NL_short.data(),
    type.data(), type_shift, para, position_per_atom.data(),
    position_per_atom.data() + number_of_atoms, position_per_atom.data() + number_of_atoms * 2,
    LIMT_data.b.data(), LIMT_data.bp.data());
  CUDA_CHECK_KERNEL

  // pre-compute the partial forces
  find_force_step2<<<grid_size, BLOCK_SIZE_FORCE>>>(
    number_of_atoms, N1, N2, box, num_types, LIMT_data.NN_short.data(), LIMT_data.NL_short.data(),
    type.data(), type_shift, para, LIMT_data.b.data(), LIMT_data.bp.data(),
    position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(),
    LIMT_data.f12x.data(), LIMT_data.f12y.data(), LIMT_data.f12z.data());
  CUDA_CHECK_KERNEL

  // the final step: calculate force and related quantities
  find_properties_many_body(
    box, LIMT_data.NN_short.data(), LIMT_data.NL_short.data(), LIMT_data.f12x.data(),
    LIMT_data.f12y.data(), LIMT_data.f12z.data(), position_per_atom, force_per_atom,
    virial_per_atom);

  // force scaling
  scale_force<<<(N2 - N1 - 1) / 128 + 1, 128>>>(
    N1, N2, para.gamma[0], force_per_atom.data(), force_per_atom.data() + number_of_atoms,
    force_per_atom.data() + 2 * number_of_atoms);
  CUDA_CHECK_KERNEL
}
