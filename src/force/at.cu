#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "at.cuh"
#include "utilities/error.cuh"

#define BLOCK_SIZE_AT 128

/*----------------------------------------------------------------------------80
This file implements the Axilrod-Teller potential.
[1] B.M. Axilrod and E. Teller,
    Interaction of the van der Waals Type Between Three Atoms,
    J. Chem. Phys. 11, 299 (1943).
    The implementation supports up to two atom types.
------------------------------------------------------------------------------*/

AT::AT(FILE* fid, int num_of_types, const Neighbor& neighbor)
{
  if (num_of_types == 1) {
    initialize_at_1(fid);
  }
  else {
    initialize_at_2(fid);
  }

  // memory for the partial forces dU_i/dr_ij
  const int num_of_neighbors = min(neighbor.MN, 50) * neighbor.NN.size();
  at_data.f12x.resize(num_of_neighbors);
  at_data.f12y.resize(num_of_neighbors);
  at_data.f12z.resize(num_of_neighbors);
}

void AT::initialize_at_1(FILE* fid)
{
  printf("Use single-element Axilrod-Teller potential.\n");
  int count, rcnum;
  double z, rcglobal, rc3;
  count = fscanf(fid, "%lf%lf%lf%d", &z, &rcglobal, &rc3, &rcnum);
  PRINT_SCANF_ERROR(count, 4, "Reading error for AT potential.");

  at_para.z[0] = z;
  if (rcglobal <= 0.0 || rc3 <= 0.0){
    PRINT_INPUT_ERROR("AT potential error: Cutoffs must be positive.\n");
  }
    rc = rcglobal;
  at_para.rc2 = rc*rc;
  at_para.rc6 = rc3*rc3;
  if (rcnum != 2 && rcnum != 3){
    PRINT_INPUT_ERROR("AT potential error: Number of enforcable cutoffs should be 2 or 3.\n");
  }
  at_para.rcnum = rcnum;
}

void AT::initialize_at_2(FILE* fid)
{
  printf("Use two-element Axilrod-Teller potential.\n");
  int count, rcnum;
  double z[4], rcglobal, rc3;
  count = fscanf(fid, "%lf%lf%lf%lf%lf%lf%d", &z[0], &z[1], &z[2], &z[3],&rcglobal, &rc3, &rcnum);
  PRINT_SCANF_ERROR(count, 7, "Reading error for AT potential.");

  at_para.z[0] = z[0];
  at_para.z[7] = z[1];
  at_para.z[1] = at_para.z[2] = at_para.z[4] = z[2];
  at_para.z[3] = at_para.z[5] = at_para.z[6] = z[3];
  if (rcglobal <= 0.0 || rc3 <= 0.0){
      PRINT_INPUT_ERROR("AT potential error: Cutoffs must be positive.\n");
    }
  rc = rcglobal;
  at_para.rc2 = rc*rc;
  at_para.rc6 = rc3*rc3;
  if (rcnum != 2 && rcnum != 3){
      PRINT_INPUT_ERROR("AT potential error: Number of enforcable cutoffs should be 2 or 3.\n");
    }
  at_para.rcnum = rcnum;
}

AT::~AT(void)
{
  // nothing
}

static __global__ void gpu_set_f12_to_zero(
  const int N,
  const int N1,
  const int N2,
  const int* g_NN,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  if (n1 >= N1 && n1 < N2) {
    int neighbor_number = g_NN[n1];
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * N + n1;
      g_f12x[index] = 0.0;
      g_f12y[index] = 0.0;
      g_f12z[index] = 0.0;
    }
  }
}


static __global__ void gpu_find_force_at_partial(
  const int number_of_atoms,
  const int N1,
  const int N2,
  const Box box,
  const AT_Para at,
  const int* g_neighbor_number,
  const int* g_neighbor_list,
  const int* g_type,
  const int shift,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  double* g_potential,
  double* g_f12x,
  double* g_f12y,
  double* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 >= N1 && n1 < N2){
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1] - shift;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    double potential_energy = 0.0;

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = i1 * number_of_atoms + n1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2] - shift;
      int tally_12 = 1; // denotes valid pair
      double x2 = g_x[n2];
      double y2 = g_y[n2];
      double z2 = g_z[n2];
      double x12 = x2 - x1;
      double y12 = y2 - y1;
      double z12 = z2 - z1;
      apply_mic(box, x12, y12, z12);
      double d12d12 = x12*x12 + y12*y12 + z12*z12;
      if (d12d12 > at.rc2) {
        tally_12 = 0;
      }
      double d12d12inv = 1/d12d12;
      double f12x, f12y, f12z;
      f12x = f12y = f12z = 0;

      for (int i2 = 0; i2 < neighbor_number; ++i2) {
        int n3 = g_neighbor_list[n1 + number_of_atoms * i2];
        if (n3 == n2) {
          continue;
        }
        int type3 = g_type[n3] - shift;
        int tally_13 = 1;
        int tally_23 = 1;
        double x3 = g_x[n3];
        double y3 = g_y[n3];
        double z3 = g_z[n3];
        double x13 = x3 - x1;
        double y13 = y3 - y1;
        double z13 = z3 - z1;
        apply_mic(box, x13, y13, z13);
        double d13d13 = x13*x13 + y13*y13 + z13*z13;
        if (d13d13 > at.rc2) {
          tally_13 = 0;
        }
        double d13d13inv = 1/d13d13;

        double x23 = x3 - x2;
        double y23 = y3 - y2;
        double z23 = z3 - z2;
        apply_mic(box, x23, y23, z23);
        double d23d23 = x23*x23 + y23*y23 + z23*z23;
        if (d23d23 > at.rc2) {
          tally_23 = 0;
        }
        double dist2_prod = d12d12*d13d13*d23d23;
        if (dist2_prod > at.rc6){
          continue;
        }

        if (tally_12 + tally_13 + tally_23 < at.rcnum){
          continue;
        }

        double z = at.z[(type1<<2)+(type2<<1)+type3];
        double scale = z/(dist2_prod*dist2_prod*sqrt(dist2_prod));
        double d12d13 = x12*x13 + y12*y13 + z12*z13;
        double d12d23 = x12*x23 + y12*y23 + z12*z23;
        double d13d23 = x13*x23 + y13*y23 + z13*z23;
        double ddd = d12d13*d12d23*d13d23;

        potential_energy += scale*dist2_prod - 3.0*scale*ddd;

        double tmp1 = 5*ddd*d12d12inv - d12d23*d13d23 - d13d13*d23d23;
        double tmp2 = 5*ddd*d13d13inv - d12d23*d13d23 - d12d12*d23d23;
        double tmp3 = -1*(d12d13*d13d23 + d12d13*d12d23);

        f12x += scale*(tmp1*x12 + tmp2*x13 + tmp3*x23);
        f12y += scale*(tmp1*y12 + tmp2*y13 + tmp3*y23);
        f12z += scale*(tmp1*z12 + tmp2*z13 + tmp3*z23);
      }
      g_f12x[index] = f12x;
      g_f12y[index] = f12y;
      g_f12z[index] = f12z;

    }
    // save potential
    g_potential[n1] = potential_energy/6.0;

  }
}

void AT::compute(
  const int type_shift,
  const Box& box,
  const Neighbor& neighbor,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<double>& potential_per_atom,
  GPU_Vector<double>& force_per_atom,
  GPU_Vector<double>& virial_per_atom)
{
  const int number_of_atoms = type.size();
  const int grid_size = (N2 - N1 - 1) / BLOCK_SIZE_AT + 1;

  gpu_set_f12_to_zero<<<grid_size, BLOCK_SIZE_AT>>>(
    number_of_atoms, N1, N2, neighbor.NN_local.data(), at_data.f12x.data(), at_data.f12y.data(),
    at_data.f12z.data());
  CUDA_CHECK_KERNEL

  // step 1: calculate the partial forces
  gpu_find_force_at_partial<<<grid_size, BLOCK_SIZE_AT>>>(
    number_of_atoms, N1, N2, box, at_para, neighbor.NN_local.data(), neighbor.NL_local.data(),
    type.data(), type_shift, position_per_atom.data(), position_per_atom.data() + number_of_atoms,
    position_per_atom.data() + number_of_atoms * 2, potential_per_atom.data(), at_data.f12x.data(),
    at_data.f12y.data(), at_data.f12z.data());
  CUDA_CHECK_KERNEL

  // step 2: calculate force and related quantities
  find_properties_many_body(
    box, neighbor.NN_local.data(), neighbor.NL_local.data(), at_data.f12x.data(),
    at_data.f12y.data(), at_data.f12z.data(), position_per_atom, force_per_atom, virial_per_atom);
}
