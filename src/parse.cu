#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Parse the commands in run.in.
------------------------------------------------------------------------------*/


#include "run.cuh"
#include "atom.cuh"
#include "ensemble.cuh"
#include "error.cuh"
#include "force.cuh"
#include "integrate.cuh"
#include "measure.cuh"
#include "hessian.cuh"
#include "read_file.cuh"
#include "dump_xyz.cuh"

#ifdef USE_NETCDF
#include "dump_netcdf.cuh"
#endif

void parse_potential_definition
(char **param, int num_param, Atom *atom, Force *force)
{
    // 'potential_definition' must be called before all 'potential' keywords
    if (force->num_of_potentials > 0)
    {
        print_error("potential_definition must be called before all "
                "potential keywords.\n");
    }

    if (num_param != 2 && num_param != 3)
    {
        print_error("potential_definition should have only 1 or 2 "
                "parameters.\n");
    }
    if (num_param == 2)
    {
        //default is to use type, check for deviations
        if(strcmp(param[1], "group") == 0)
        {
            print_error("potential_definition must have "
                    "group_method listed.\n");
        }
        else if(strcmp(param[1], "type") != 0)
        {
            print_error("potential_definition only accepts "
                    "'type' or 'group' kind.\n");
        }
    }
    if (num_param == 3)
    {
        if(strcmp(param[1], "group") != 0)
        {
            print_error("potential_definition: kind must be 'group' if 2 "
                    "parameters are used.\n");

        }
        else if(!is_valid_int(param[2], &force->group_method))
        {
            print_error("potential_definition: group_method should be an "
                    "integer.\n");
        }
        else if(force->group_method > MAX_NUMBER_OF_GROUPS)
        {
            print_error("Specified group_method is too large (> 10).\n");
        }
    }
}

// a potential
void parse_potential(char **param, int num_param, Force *force)
{
    // check for at least the file path
    if (num_param < 3)
    {
        print_error("potential should have at least 2 parameters.\n");
    }
    strcpy(force->file_potential[force->num_of_potentials], param[1]);

    //open file to check number of types used in potential
    char potential_name[20];
    FILE *fid_potential = my_fopen(
            force->file_potential[force->num_of_potentials], "r");
    int count = fscanf(fid_potential, "%s", potential_name);
    int num_types = force->get_number_of_types(fid_potential);
    fclose(fid_potential);

    if (num_param != num_types + 2)
    {
        print_error("potential has incorrect number of types/groups defined.\n");
    }

    force->participating_kinds.resize(num_types);

    for (int i = 0; i < num_types; i++)
    {
        if(!is_valid_int(param[i+2], &force->participating_kinds[i]))
        {
            print_error("type/groups should be an integer.\n");
        }
        if (i != 0 &&
            force->participating_kinds[i] < force->participating_kinds[i-1])
        {
            print_error("potential types/groups must be listed in "
                    "ascending order.\n");
        }
    }
    force->atom_begin[force->num_of_potentials] =
            force->participating_kinds[0];
    force->atom_end[force->num_of_potentials] =
            force->participating_kinds[num_types-1];

    force->num_of_potentials++;

}


void parse_velocity(char **param, int num_param, Atom *atom)
{
    if (num_param != 2)
    {
        print_error("velocity should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &atom->initial_temperature))
    {
        print_error("initial temperature should be a real number.\n");
    }
    if (atom->initial_temperature <= 0.0)
    {
        print_error("initial temperature should be a positive number.\n");
    }
}


// coding conventions:
//0:     NVE
//1-10:  NVT
//11-20: NPT
//21-30: heat (NEMD method for heat conductivity)
void parse_ensemble 
(char **param,  int num_param, Atom *atom, Integrate *integrate)
{
    // 1. Determine the integration method
    if (strcmp(param[1], "nve") == 0)
    {
        integrate->type = 0;
        if (num_param != 2)
        {
            print_error("ensemble nve should have 0 parameter.\n");
        }
    }
    else if (strcmp(param[1], "nvt_ber") == 0)
    {
        integrate->type = 1;
        if (num_param != 5)
        {
            print_error("ensemble nvt_ber should have 3 parameters.\n");
        }
    }
    else if (strcmp(param[1], "nvt_nhc") == 0)
    {
        integrate->type = 2;
        if (num_param != 5)
        {
            print_error("ensemble nvt_nhc should have 3 parameters.\n"); 
        }
    }
    else if (strcmp(param[1], "nvt_lan") == 0)
    {
        integrate->type = 3;
        if (num_param != 5)
        {
            print_error("ensemble nvt_lan should have 3 parameters.\n"); 
        }
    }
    else if (strcmp(param[1], "nvt_bdp") == 0)
    {
        integrate->type = 4;
        if (num_param != 5)
        {
            print_error("ensemble nvt_bdp should have 3 parameters.\n"); 
        }
    }
    else if (strcmp(param[1], "npt_ber") == 0)
    {
        integrate->type = 11;
        if (num_param != 9)
        {
            print_error("ensemble npt_ber should have 7 parameters.\n"); 
        } 
    }
    else if (strcmp(param[1], "heat_nhc") == 0)
    {
        integrate->type = 21;
        if (num_param != 7)
        {
            print_error("ensemble heat_nhc should have 5 parameters.\n"); 
        }
    }
    else if (strcmp(param[1], "heat_lan") == 0)
    {
        integrate->type = 22;
        if (num_param != 7)
        {
            print_error("ensemble heat_lan should have 5 parameters.\n"); 
        }
    }
    else if (strcmp(param[1], "heat_bdp") == 0)
    {
        integrate->type = 23;
        if (num_param != 7)
        {
            print_error("ensemble heat_bdp should have 5 parameters.\n"); 
        }
    }
    else
    {
        print_error("invalid ensemble type.\n");
    }

    // 2. Temperatures and temperature_coupling (NVT and NPT)
    if (integrate->type >= 1 && integrate->type <= 20)
    {
        // initial temperature
        if (!is_valid_real(param[2], &integrate->temperature1))
        {
            print_error("ensemble temperature should be a real number.\n");
        }
        if (integrate->temperature1 <= 0.0)
        {
            print_error("ensemble temperature should be a positive number.\n");
        }

        // final temperature
        if (!is_valid_real(param[3], &integrate->temperature2))
        {
            print_error("ensemble temperature should be a real number.\n");
        }
        if (integrate->temperature2 <= 0.0)
        {
            print_error("ensemble temperature should be a positive number.\n");
        }

        integrate->temperature = integrate->temperature1;

        // temperature_coupling
        if (!is_valid_real(param[4], &integrate->temperature_coupling))
        {
            print_error("temperature_coupling should be a real number.\n");
        }
        if (integrate->temperature_coupling <= 0.0)
        {
            print_error("temperature_coupling should be a positive number.\n");
        }
    }

    // 3. Pressures and pressure_coupling (NPT)
    real pressure[3];
    if (integrate->type >= 11 && integrate->type <= 20)
    {  
        // pressures:   
        for (int i = 0; i < 3; i++)
        {
            if (!is_valid_real(param[5+i], &pressure[i]))
            {
                print_error("ensemble pressure should be a real number.\n");
            }
        }
        // Change the unit of pressure form GPa to that used in the code
        integrate->pressure_x = pressure[0] / PRESSURE_UNIT_CONVERSION;
        integrate->pressure_y = pressure[1] / PRESSURE_UNIT_CONVERSION;
        integrate->pressure_z = pressure[2] / PRESSURE_UNIT_CONVERSION;

        // pressure_coupling:
        if (!is_valid_real(param[8], &integrate->pressure_coupling))
        {
            print_error("pressure_coupling should be a real number.\n");
        }
        if (integrate->pressure_coupling <= 0.0)
        {
            print_error("pressure_coupling should be a positive number.\n");
        }
    }

    // 4. heating and cooling wiht fixed temperatures
    if (integrate->type >= 21 && integrate->type <= 30)
    {
        // temperature
        if (!is_valid_real(param[2], &integrate->temperature))
        {
            print_error("ensemble temperature should be a real number.\n");
        }
        if (integrate->temperature <= 0.0)
        {
            print_error("ensemble temperature should be a positive number.\n");
        }

        // temperature_coupling
        if (!is_valid_real(param[3], &integrate->temperature_coupling))
        {
            print_error("temperature_coupling should be a real number.\n");
        }
        if (integrate->temperature_coupling <= 0.0)
        {
            print_error("temperature_coupling should be a positive number.\n");
        }

        // temperature difference
        if (!is_valid_real(param[4], &integrate->delta_temperature))
        {
            print_error("delta_temperature should be a real number.\n");
        }

        // group labels of heat source and sink
        if (!is_valid_int(param[5], &integrate->source))
        {
            print_error("heat.source should be an integer.\n");
        }
        if (!is_valid_int(param[6], &integrate->sink))
        {
            print_error("heat.sink should be an integer.\n");
        }
    }

    switch (integrate->type)
    {
        case 0:
            printf("Use NVE ensemble for this run.\n");
            break;
        case 1:
            printf("Use NVT ensemble for this run.\n");
            printf("    choose the Berendsen method.\n"); 
            printf("    initial temperature is %g K.\n", integrate->temperature1);
            printf("    final temperature is %g K.\n", integrate->temperature2);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            break;
        case 2:
            printf("Use NVT ensemble for this run.\n");
            printf("    choose the Nose-Hoover chain method.\n"); 
            printf("    initial temperature is %g K.\n", integrate->temperature1);
            printf("    final temperature is %g K.\n", integrate->temperature2);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            break;
        case 3:
            printf("Use NVT ensemble for this run.\n");
            printf("    choose the Langevin method.\n"); 
            printf("    initial temperature is %g K.\n", integrate->temperature1);
            printf("    final temperature is %g K.\n", integrate->temperature2);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            break;
        case 4:
            printf("Use NVT ensemble for this run.\n");
            printf("    choose the Bussi-Donadio-Parrinello method.\n"); 
            printf("    initial temperature is %g K.\n", integrate->temperature1);
            printf("    final temperature is %g K.\n", integrate->temperature2);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            break;
        case 11:
            printf("Use NPT ensemble for this run.\n");
            printf("    choose the Berendsen method.\n");      
            printf("    initial temperature is %g K.\n", integrate->temperature1);
            printf("    final temperature is %g K.\n", integrate->temperature2);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            printf("    pressure_x is %g GPa.\n", pressure[0]);
            printf("    pressure_y is %g GPa.\n", pressure[1]);
            printf("    pressure_z is %g GPa.\n", pressure[2]);
            printf("    p_coupling is %g.\n", integrate->pressure_coupling);
            break;
        case 21:
            printf("Integrate with heating and cooling for this run.\n");
            printf("    choose the Nose-Hoover chain method.\n"); 
            printf("    temperature is %g K.\n", integrate->temperature);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            printf("    delta_T is %g K.\n", integrate->delta_temperature);
            printf("    heat source is group %d.\n", integrate->source);
            printf("    heat sink is group %d.\n", integrate->sink);
            break; 
        case 22:
            printf("Integrate with heating and cooling for this run.\n");
            printf("    choose the Langevin method.\n"); 
            printf("    temperature is %g K.\n", integrate->temperature);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            printf("    delta_T is %g K.\n", integrate->delta_temperature);
            printf("    heat source is group %d.\n", integrate->source);
            printf("    heat sink is group %d.\n", integrate->sink);
            break;
        case 23:
            printf("Integrate with heating and cooling for this run.\n");
            printf("    choose the Bussi-Donadio-Parrinello method.\n"); 
            printf("    temperature is %g K.\n", integrate->temperature);
            printf("    T_coupling is %g.\n", integrate->temperature_coupling);
            printf("    delta_T is %g K.\n", integrate->delta_temperature);
            printf("    heat source is group %d.\n", integrate->source);
            printf("    heat sink is group %d.\n", integrate->sink);
            break;
        default:
            print_error("invalid ensemble type.\n");
            break; 
    }
}


void parse_time_step (char **param,  int num_param, Atom* atom)
{
    if (num_param != 2)
    {
        print_error("time_step should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &atom->time_step))
    {
        print_error("time_step should be a real number.\n");
    }
    printf("Time step for this run is %g fs.\n", atom->time_step);
    atom->time_step /= TIME_UNIT_CONVERSION;
}


void parse_neighbor
(char **param,  int num_param, Atom* atom, Force *force)
{
    atom->neighbor.update = 1;

    if (num_param != 2)
    {
        print_error("neighbor should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &atom->neighbor.skin))
    {
        print_error("neighbor list skin should be a number.\n");
    }
    printf("Build neighbor list with a skin of %g A.\n", atom->neighbor.skin);

    // change the cutoff
    atom->neighbor.rc = force->rc_max + atom->neighbor.skin;
}


void parse_dump_thermo(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_thermo should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_thermo))
    {
        print_error("thermo dump interval should be an integer number.\n");
    }
    measure->dump_thermo = 1;
    printf("Dump thermo every %d steps.\n", measure->sample_interval_thermo);
}


void parse_dump_position(char **param,  int num_param, Measure *measure,
		Atom *atom)
{
	int interval;

    if (num_param < 2)
    {
        print_error("dump_position should have at least 1 parameter.\n");
    }
    if (num_param > 6)
    {
    	print_error("dump_position has too many parameters.\n");
    }

    // sample interval
    if (!is_valid_int(param[1], &interval))
    {
        print_error("position dump interval should be an integer number.\n");
    }

    int format = 0; // default xyz
    int precision = 0; // default normal (unlesss netCDF -> 64 bit)
    // Process optional arguments
    for (int k = 2; k < num_param; k++)
    {
    	// format check
    	if (strcmp(param[k], "format") == 0)
    	{
    		// check if there are enough inputs
    		if (k + 2 > num_param)
    		{
    			print_error("Not enough arguments for optional "
    					" 'format' dump_position command.\n");
    		}
    		if ((strcmp(param[k+1], "xyz") != 0) &&
				(strcmp(param[k+1], "netcdf") != 0))
    		{
    			print_error("Invalid format for dump_position command.\n");
    		}
    		else if(strcmp(param[k+1], "netcdf") == 0)
    		{
    			format = 1;
    			k++;
    		}
    	}
    	// precision check
    	else if(strcmp(param[k], "precision") == 0)
    	{
    		// check for enough inputs
    		if (k + 2 > num_param)
			{
				print_error("Not enough arguments for optional "
						" 'precision' dump_position command.\n");
			}
    		if ((strcmp(param[k+1], "single") != 0) &&
				(strcmp(param[k+1], "double") != 0))
			{
				print_error("Invalid precision for dump_position command.\n");
			}
			else
			{
				if(strcmp(param[k+1], "single") == 0)
				{
					precision = 1;
				}
				else if(strcmp(param[k+1], "double") == 0)
                {
                    precision = 2;
                }
				k++;
			}
    	}
    }

    if (format == 1) // netcdf output
    {
#ifdef USE_NETCDF
    	DUMP_NETCDF *dump_netcdf = new DUMP_NETCDF(atom->N, atom->global_time);
    	measure->dump_pos = dump_netcdf;
    	if (!precision) precision = 2; // double precision default
#else
    	print_error("USE_NETCDF flag is not set. NetCDF output not available.\n");
#endif
    }
    else // xyz default output
    {
    	DUMP_XYZ *dump_xyz = new DUMP_XYZ();
    	measure->dump_pos = dump_xyz;
    }
    measure->dump_pos->interval = interval;
    measure->dump_pos->precision = precision;


    if (precision == 1 && format)
    {
    	printf("Note: Single precision netCDF output does not follow AMBER conventions.\n"
    	       "      However, it will still work for many readers.\n");
    }

    printf("Dump position every %d steps.\n",
        measure->dump_pos->interval);
}


void parse_dump_restart(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_restart should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_restart))
    {
        print_error("restart dump interval should be an integer number.\n");
    }
    measure->dump_restart = 1;
    printf("Dump restart every %d steps.\n", measure->sample_interval_restart);
}


void parse_dump_velocity(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_velocity should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_velocity))
    {
        print_error("velocity dump interval should be an integer number.\n");
    }
    measure->dump_velocity = 1;
    printf("Dump velocity every %d steps.\n",
        measure->sample_interval_velocity);
}


void parse_dump_force(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_force should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_force))
    {
        print_error("force dump interval should be an integer number.\n");
    }
    measure->dump_force = 1;
    printf("Dump force every %d steps.\n", measure->sample_interval_force);
}


void parse_dump_potential(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_potential should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_potential))
    {
        print_error("potential dump interval should be an integer number.\n");
    }
    measure->dump_potential = 1;
    printf("Dump potential every %d steps.\n",
        measure->sample_interval_potential);
}


void parse_dump_virial(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_virial should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_virial))
    {
        print_error("virial dump interval should be an integer number.\n");
    }
    measure->dump_virial = 1;
    printf("Dump virial every %d steps.\n",
        measure->sample_interval_virial);
}


void parse_dump_heat(char **param,  int num_param, Measure *measure)
{
    if (num_param != 2)
    {
        print_error("dump_heat should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &measure->sample_interval_heat))
    {
        print_error("heat dump interval should be an integer number.\n");
    }
    measure->dump_heat = 1;
    printf("Dump heat every %d steps.\n", measure->sample_interval_heat);
}

// Helper functions for parse_compute_dos
void parse_group(char **param, Measure *measure, int *k, Group *group)
{
	// grouping_method
	if (!is_valid_int(param[*k+1], &measure->vac.grouping_method))
	{
		print_error("grouping method for VAC should be an integer number.\n");
	}
	if (measure->vac.grouping_method < 0 || measure->vac.grouping_method > 2)
	{
		print_error("grouping method for VAC should be 0 <= x <= 2.\n");
	}
	// group
	if (!is_valid_int(param[*k+2], &measure->vac.group))
	{
		print_error("group for VAC should be an integer number.\n");
	}
	if (measure->vac.group < 0 ||
			measure->vac.group > group[measure->vac.grouping_method].number)
	{
		print_error("group for VAC must be >= 0 and < number of groups.\n");
	}
	*k += 2; // update index for next command
}

void parse_num_dos_points(char **param, Measure *measure, int *k)
{
	// number of DOS points
	if (!is_valid_int(param[*k+1], &measure->dos.num_dos_points))
	{
		print_error("number of DOS points for VAC should be an integer "
				"number.\n");
	}
	if (measure->dos.num_dos_points < 1)
	{
		print_error("number of DOS points for DOS must be > 0.\n");
	}
	*k += 1; //
}

void parse_compute_dos(char **param,  int num_param, Measure *measure,
		Group *group)
{
    printf("Compute phonon DOS.\n");
    measure->vac.compute_dos = 1;

    if (num_param < 4)
    {
        print_error("compute_dos should have at least 3 parameters.\n");
    }
    if (num_param > 9)
	{
		print_error("compute_dos has too many parameters.\n");
	}

    // sample interval
    if (!is_valid_int(param[1], &measure->vac.sample_interval))
    {
        print_error("sample interval for VAC should be an integer number.\n");
    }
    if (measure->vac.sample_interval <= 0)
    {
        print_error("sample interval for VAC should be positive.\n");
    }
    printf("    sample interval is %d.\n", measure->vac.sample_interval);

    // number of correlation steps
    if (!is_valid_int(param[2], &measure->vac.Nc))
    {
        print_error("Nc for VAC should be an integer number.\n");
    }
    if (measure->vac.Nc <= 0)
    {
        print_error("Nc for VAC should be positive.\n");
    }
    printf("    Nc is %d.\n", measure->vac.Nc);

    // maximal omega
    if (!is_valid_real(param[3], &measure->dos.omega_max))
    {
        print_error("omega_max should be a real number.\n");
    }
    if (measure->dos.omega_max <= 0)
    {
        print_error("omega_max should be positive.\n");
    }
    printf("    omega_max is %g THz.\n", measure->dos.omega_max);

    // Process optional arguments
    for (int k = 4; k < num_param; k++)
    {
    	if (strcmp(param[k], "group") == 0)
    	{
    		// check if there are enough inputs
    		if (k + 3 > num_param)
    		{
    			print_error("Not enough arguments for optional "
    					"'group' DOS command.\n");
    		}
    		parse_group(param, measure,  &k, group);
    		printf("    grouping_method is %d and group is %d.\n",
    				measure->vac.grouping_method, measure->vac.group);
    	}
    	else if (strcmp(param[k], "num_dos_points") == 0)
    	{
    		// check if there are enough inputs
    		if (k + 2 > num_param)
    		{
    			print_error("Not enough arguments for optional "
						"'group' dos command.\n");
    		}
    		parse_num_dos_points(param, measure, &k);
    		printf("    num_dos_points is %d.\n",measure->dos.num_dos_points);
    	}
    	else
    	{
    		print_error("Unrecognized argument in compute_dos.\n");
    	}
    }
}

void parse_compute_sdc(char **param,  int num_param, Measure *measure,
		Group *group)
{
    printf("Compute SDC.\n");
    measure->vac.compute_sdc = 1;

    if (num_param < 3)
    {
        print_error("compute_sdc should have at least 2 parameters.\n");
    }
    if (num_param > 6)
    {
    	print_error("compute_sdc has too many parameters.\n");
    }

    // sample interval
    if (!is_valid_int(param[1], &measure->vac.sample_interval))
    {
        print_error("sample interval for VAC should be an integer number.\n");
    }
    if (measure->vac.sample_interval <= 0)
    {
        print_error("sample interval for VAC should be positive.\n");
    }
    printf("    sample interval is %d.\n", measure->vac.sample_interval);

    // number of correlation steps
    if (!is_valid_int(param[2], &measure->vac.Nc))
    {
        print_error("Nc for VAC should be an integer number.\n");
    }
    if (measure->vac.Nc <= 0)
    {
        print_error("Nc for VAC should be positive.\n");
    }
    printf("    Nc is %d.\n", measure->vac.Nc);

    // Process optional arguments
	for (int k = 3; k < num_param; k++)
	{
		if (strcmp(param[k], "group") == 0)
		{
			// check if there are enough inputs
			if (k + 3 > num_param)
			{
				print_error("Not enough arguments for optional "
						"'group' SDC command.\n");
			}
			parse_group(param, measure,  &k, group);
			printf("    grouping_method is %d and group is %d.\n",
					measure->vac.grouping_method, measure->vac.group);
		}
		else
		{
			print_error("Unrecognized argument in compute_sdc.\n");
		}
	}
}


void parse_compute_hac(char **param,  int num_param, Measure* measure)
{
    measure->hac.compute = 1;

    printf("Compute HAC.\n");

    if (num_param != 4)
    {
        print_error("compute_hac should have 3 parameters.\n");
    }

    if (!is_valid_int(param[1], &measure->hac.sample_interval))
    {
        print_error("sample interval for HAC should be an integer number.\n");
    }
    printf("    sample interval is %d.\n", measure->hac.sample_interval);

    if (!is_valid_int(param[2], &measure->hac.Nc))
    {
        print_error("Nc for HAC should be an integer number.\n");
    }
    printf("    Nc is %d\n", measure->hac.Nc);

    if (!is_valid_int(param[3], &measure->hac.output_interval))
    {
        print_error("output_interval for HAC should be an integer number.\n");
    }
    printf("    output_interval is %d\n", measure->hac.output_interval);
}

void parse_compute_gkma(char **param, int num_param, Measure* measure, Atom* atom)
{
    measure->gkma.compute = 1;

    printf("Compute modal heat current using GKMA method.\n");

    /*
     * There is a hidden feature that allows for specification of atom
     * types to included (must be contiguously defined like potentials)
     * -- Works for types only, not groups --
     */

    if (num_param != 6 && num_param != 9)
    {
        print_error("compute_gkma should have 5 parameters.\n");
    }
    if (!is_valid_int(param[1], &measure->gkma.sample_interval) ||
        !is_valid_int(param[2], &measure->gkma.first_mode)      ||
        !is_valid_int(param[3], &measure->gkma.last_mode)       )
    {
        print_error("A parameter for GKMA should be an integer.\n");
    }

    if (strcmp(param[4], "bin_size") == 0)
    {
        measure->gkma.f_flag = 0;
        if(!is_valid_int(param[5], &measure->gkma.bin_size))
        {
            print_error("GKMA bin_size must be an integer.\n");
        }
    }
    else if (strcmp(param[4], "f_bin_size") == 0)
    {
        measure->gkma.f_flag = 1;
        if(!is_valid_real(param[5], &measure->gkma.f_bin_size))
        {
            print_error("GKMA f_bin_size must be a real number.\n");
        }
    }
    else
    {
        print_error("Invalid binning keyword for compute_gkma.\n");
    }

    GKMA *g = &measure->gkma;
    // Parameter checking
    if (g->sample_interval < 1  || g->first_mode < 1 || g->last_mode < 1)
        print_error("compute_gkma parameters must be positive integers.\n");
    if (g->first_mode > g->last_mode)
        print_error("first_mode <= last_mode required.\n");

    printf("    sample_interval is %d.\n"
           "    first_mode is %d.\n"
           "    last_mode is %d.\n",
          g->sample_interval, g->first_mode, g->last_mode);

    if (g->f_flag)
    {
        if (g->f_bin_size <= 0.0)
        {
            print_error("bin_size must be greater than zero.\n");
        }
        printf("    Bin by frequency.\n"
               "    f_bin_size is %f THz.\n", g->f_bin_size);
    }
    else
    {
        if (g->bin_size < 1)
        {
            print_error("compute_gkma parameters must be positive integers.\n");
        }
        int num_modes = g->last_mode - g->first_mode + 1;
        if (num_modes % g->bin_size != 0)
            print_error("number of modes must be divisible by bin_size.\n");
        printf("    Bin by modes.\n"
               "    bin_size is %d THz.\n", g->bin_size);
    }


    // Hidden feature implementation
    if (num_param == 9)
    {
        if (strcmp(param[6], "atom_range") == 0)
        {
            if(!is_valid_int(param[7], &measure->gkma.atom_begin) ||
               !is_valid_int(param[8], &measure->gkma.atom_end))
            {
                print_error("GKMA atom_begin & atom_end must be integers.\n");
            }
            if (measure->gkma.atom_begin > measure->gkma.atom_end)
            {
                print_error("atom_begin must be less than atom_end.\n");
            }
            if (measure->gkma.atom_begin < 0)
            {
                print_error("atom_begin must be greater than 0.\n");
            }
            if (measure->gkma.atom_end >= atom->number_of_types)
            {
                print_error("atom_end must be greater than 0.\n");
            }
        }
        else
        {
            print_error("Invalid GKMA keyword.\n");
        }
        printf("    Use select atom range.\n"
               "    Atom types %d to %d.\n",
               measure->gkma.atom_begin, measure->gkma.atom_end);
    }
    else // default behavior
    {
        measure->gkma.atom_begin = 0;
        measure->gkma.atom_end = atom->number_of_types - 1;
    }

}

void parse_compute_hnema(char **param, int num_param, Measure* measure, Atom* atom)
{
    measure->hnema.compute = 1;

    printf("Compute modal thermal conductivity using HNEMA method.\n");

    /*
     * There is a hidden feature that allows for specification of atom
     * types to included (must be contiguously defined like potentials)
     * -- Works for types only, not groups --
     */

    if (num_param != 10 && num_param != 13)
    {
        print_error("compute_hnema should have 9 parameters.\n");
    }
    if (!is_valid_int(param[1], &measure->hnema.sample_interval) ||
        !is_valid_int(param[2], &measure->hnema.output_interval) ||
        !is_valid_int(param[6], &measure->hnema.first_mode)      ||
        !is_valid_int(param[7], &measure->hnema.last_mode)       )
    {
        print_error("A parameter for HNEMA should be an integer.\n");
    }

    // HNEMD driving force parameters -> Use HNEMD object
    if (!is_valid_real(param[3], &measure->hnemd.fe_x))
    {
        print_error("fe_x for HNEMD should be a real number.\n");
    }
    printf("    fe_x = %g /A\n", measure->hnemd.fe_x);
    if (!is_valid_real(param[4], &measure->hnemd.fe_y))
    {
        print_error("fe_y for HNEMD should be a real number.\n");
    }
    printf("    fe_y = %g /A\n", measure->hnemd.fe_y);
    if (!is_valid_real(param[5], &measure->hnemd.fe_z))
    {
        print_error("fe_z for HNEMD should be a real number.\n");
    }
    printf("    fe_z = %g /A\n", measure->hnemd.fe_z);
    // magnitude of the vector
    measure->hnemd.fe  = measure->hnemd.fe_x * measure->hnemd.fe_x;
    measure->hnemd.fe += measure->hnemd.fe_y * measure->hnemd.fe_y;
    measure->hnemd.fe += measure->hnemd.fe_z * measure->hnemd.fe_z;
    measure->hnemd.fe  = sqrt(measure->hnemd.fe);


    if (strcmp(param[8], "bin_size") == 0)
    {
        measure->hnema.f_flag = 0;
        if(!is_valid_int(param[9], &measure->hnema.bin_size))
        {
            print_error("HNEMA bin_size must be an integer.\n");
        }
    }
    else if (strcmp(param[8], "f_bin_size") == 0)
    {
        measure->hnema.f_flag = 1;
        if(!is_valid_real(param[9], &measure->hnema.f_bin_size))
        {
            print_error("HNEMA f_bin_size must be a real number.\n");
        }
    }
    else
    {
        print_error("Invalid binning keyword for compute_hnema.\n");
    }

    HNEMA *h = &measure->hnema;
    // Parameter checking
    if (h->sample_interval < 1  || h->output_interval < 1 ||
            h->first_mode < 1 || h->last_mode < 1)
        print_error("compute_hnema parameters must be positive integers.\n");
    if (h->first_mode > h->last_mode)
        print_error("first_mode <= last_mode required.\n");
    if (h->output_interval % h->sample_interval != 0)
            print_error("sample_interval must divide output_interval an integer\n"
                    " number of times.\n");

    printf("    sample_interval is %d.\n"
           "    output_interval is %d.\n"
           "    first_mode is %d.\n"
           "    last_mode is %d.\n",
          h->sample_interval, h->output_interval, h->first_mode, h->last_mode);

    if (h->f_flag)
    {
        if (h->f_bin_size <= 0.0)
        {
            print_error("bin_size must be greater than zero.\n");
        }
        printf("    Bin by frequency.\n"
               "    f_bin_size is %f THz.\n", h->f_bin_size);
    }
    else
    {
        if (h->bin_size < 1)
        {
            print_error("compute_hnema parameters must be positive integers.\n");
        }
        int num_modes = h->last_mode - h->first_mode + 1;
        if (num_modes % h->bin_size != 0)
            print_error("number of modes must be divisible by bin_size.\n");
        printf("    Bin by modes.\n"
               "    bin_size is %d THz.\n", h->bin_size);
    }


    // Hidden feature implementation
    if (num_param == 13)
    {
        if (strcmp(param[10], "atom_range") == 0)
        {
            if(!is_valid_int(param[11], &measure->hnema.atom_begin) ||
               !is_valid_int(param[12], &measure->hnema.atom_end))
            {
                print_error("HNEMA atom_begin & atom_end must be integers.\n");
            }
            if (measure->hnema.atom_begin > measure->hnema.atom_end)
            {
                print_error("atom_begin must be less than atom_end.\n");
            }
            if (measure->hnema.atom_begin < 0)
            {
                print_error("atom_begin must be greater than 0.\n");
            }
            if (measure->hnema.atom_end >= atom->number_of_types)
            {
                print_error("atom_end must be greater than 0.\n");
            }
        }
        else
        {
            print_error("Invalid HNEMA keyword.\n");
        }
        printf("    Use select atom range.\n"
               "    Atom types %d to %d.\n",
               measure->hnema.atom_begin, measure->hnema.atom_end);
    }
    else // default behavior
    {
        measure->hnema.atom_begin = 0;
        measure->hnema.atom_end = atom->number_of_types - 1;
    }

}

void parse_compute_hnemd(char **param, int num_param, Measure* measure)
{
    measure->hnemd.compute = 1;

    printf("Compute thermal conductivity using the HNEMD method.\n");

    if (num_param != 5)
    {
        print_error("compute_hnemd should have 4 parameters.\n");
    }

    if (!is_valid_int(param[1], &measure->hnemd.output_interval))
    {
        print_error("output_interval for HNEMD should be an integer number.\n");
    }
    printf("    output_interval = %d\n", measure->hnemd.output_interval);
    if (measure->hnemd.output_interval < 1)
    {
        print_error("output_interval for HNEMD should be larger than 0.\n");
    }
    if (!is_valid_real(param[2], &measure->hnemd.fe_x))
    {
        print_error("fe_x for HNEMD should be a real number.\n");
    }
    printf("    fe_x = %g /A\n", measure->hnemd.fe_x);
    if (!is_valid_real(param[3], &measure->hnemd.fe_y))
    {
        print_error("fe_y for HNEMD should be a real number.\n");
    }
    printf("    fe_y = %g /A\n", measure->hnemd.fe_y);
    if (!is_valid_real(param[4], &measure->hnemd.fe_z))
    {
        print_error("fe_z for HNEMD should be a real number.\n");
    }
    printf("    fe_z = %g /A\n", measure->hnemd.fe_z);

    // magnitude of the vector
    measure->hnemd.fe  = measure->hnemd.fe_x * measure->hnemd.fe_x;
    measure->hnemd.fe += measure->hnemd.fe_y * measure->hnemd.fe_y;
    measure->hnemd.fe += measure->hnemd.fe_z * measure->hnemd.fe_z;
    measure->hnemd.fe  = sqrt(measure->hnemd.fe);
}


void parse_compute_shc(char **param,  int num_param, Measure* measure)
{
    printf("Compute SHC.\n");
    measure->shc.compute = 1;

    // check the number of parameters
    if ((num_param != 4) && (num_param != 5) && (num_param != 6))
    {
        print_error("compute_shc should have 3 or 4 or 5 parameters.\n");
    }

    // group method and group id
    int offset = 0;
    if (num_param == 4)
    {
        measure->shc.group_method = -1;
        printf("    for the whole system.\n");
    }
    else if (num_param == 5)
    {
        offset = 1;
        measure->shc.group_method = 0;
        if (!is_valid_int(param[1], &measure->shc.group_id))
        {
            print_error("grouping id should be an integer.\n");
        }
        printf("    for atoms in group %d.\n", measure->shc.group_id);
        printf("    using group method 0.\n");
    }
    else
    {
        offset = 2;
        if (!is_valid_int(param[1], &measure->shc.group_method))
        {
            print_error("group method should be an integer.\n");
        }
        if (!is_valid_int(param[2], &measure->shc.group_id))
        {
            print_error("grouping id should be an integer.\n");
        }
        printf("    for atoms in group %d.\n", measure->shc.group_id);
        printf("    using group method %d.\n", measure->shc.group_method);
    }

    // sample interval 
    if (!is_valid_int(param[1+offset], &measure->shc.sample_interval))
    {
        print_error("shc.sample_interval should be an integer.\n");
    }
    printf
    ("    sample interval for SHC is %d.\n", measure->shc.sample_interval);

    // number of correlation data
    if (!is_valid_int(param[2+offset], &measure->shc.Nc))
    {
        print_error("Nc for SHC should be an integer.\n");
    }
    printf("    number of correlation data is %d.\n", measure->shc.Nc);

    // transport direction
    if (!is_valid_int(param[3+offset], &measure->shc.direction))
    {
        print_error("direction for SHC should be an integer.\n");
    }
    printf("    transport direction is %d.\n", measure->shc.direction);
}


void parse_deform(char **param,  int num_param, Integrate* integrate)
{
    printf("Deform the box.\n");

    if (num_param != 5)
    {
        print_error("deform should have 4 parameters.\n");
    }

    // strain rate
    if (!is_valid_real(param[1], &integrate->deform_rate))
    {
        print_error("defrom rate should be a number.\n");
    }
    printf("    strain rate is %g A / step.\n",
        integrate->deform_rate);

    // direction
    if (!is_valid_int(param[2], &integrate->deform_x))
    {
        print_error("deform_x should be integer.\n");
    }
    if (!is_valid_int(param[3], &integrate->deform_y))
    {
        print_error("deform_y should be integer.\n");
    }
    if (!is_valid_int(param[4], &integrate->deform_z))
    {
        print_error("deform_z should be integer.\n");
    }

    if (integrate->deform_x)
    {
        printf("    apply strain in x direction.\n");
    }
    if (integrate->deform_y)
    {
        printf("    apply strain in y direction.\n");
    }
    if (integrate->deform_z)
    {
        printf("    apply strain in z direction.\n");
    }
}


void parse_compute(char **param,  int num_param, Measure* measure)
{
    printf("Compute group average of:\n");
    if (num_param < 5)
        print_error("compute should have at least 4 parameters.\n");
    if (!is_valid_int(param[1], &measure->compute.grouping_method))
    {
        print_error("grouping method of compute should be integer.\n");
    }
    if (!is_valid_int(param[2], &measure->compute.sample_interval))
    {
        print_error("sampling interval of compute should be integer.\n");
    }
    if (!is_valid_int(param[3], &measure->compute.output_interval))
    {
        print_error("output interval of compute should be integer.\n");
    }
    for (int k = 0; k < num_param - 4; ++k)
    {
        if (strcmp(param[k + 4], "temperature") == 0)
        {
            measure->compute.compute_temperature = 1;
            printf("    temperature\n");
        }
        else if (strcmp(param[k + 4], "potential") == 0)
        {
            measure->compute.compute_potential = 1;
            printf("    potential energy\n");
        }
        else if (strcmp(param[k + 4], "force") == 0)
        {
            measure->compute.compute_force = 1;
            printf("    force\n");
        }
        else if (strcmp(param[k + 4], "virial") == 0)
        {
            measure->compute.compute_virial = 1;
            printf("    virial\n");
        }
        else if (strcmp(param[k + 4], "jp") == 0)
        {
            measure->compute.compute_jp = 1;
            printf("    potential part of heat current\n");
        }
        else if (strcmp(param[k + 4], "jk") == 0)
        {
            measure->compute.compute_jk = 1;
            printf("    kinetic part of heat current\n");
        }
    }
    printf("    using grouping method %d.\n",
        measure->compute.grouping_method);
    printf("    with sampling interval %d.\n",
        measure->compute.sample_interval);
    printf("    and output interval %d.\n",
        measure->compute.output_interval);
}


void parse_fix(char **param, int num_param, Integrate *integrate)
{
    if (num_param != 2)
    {
        print_error("fix should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &integrate->fixed_group))
    {
        print_error("fixed_group should be an integer.\n");
    }
    printf("Group %d will be fixed.\n", integrate->fixed_group);
}


void parse_run(char **param,  int num_param, Atom* atom)
{
    if (num_param != 2)
    {
        print_error("run should have 1 parameter.\n");
    }
    if (!is_valid_int(param[1], &atom->number_of_steps))
    {
        print_error("number of steps should be an integer.\n");
    }
    printf("Run %d steps.\n", atom->number_of_steps);
}


void parse_cutoff(char **param, int num_param, Hessian* hessian)
{
    if (num_param != 2)
    {
        print_error("cutoff should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &hessian->cutoff))
    {
        print_error("cutoff for hessian should be a number.\n");
    }
    if (hessian->cutoff <= 0)
    {
        print_error("cutoff for hessian should be positive.\n");
    }
    printf("Cutoff distance for hessian = %g A.\n", hessian->cutoff);
}


void parse_delta(char **param, int num_param, Hessian* hessian)
{
    if (num_param != 2)
    {
        print_error("compute_hessian should have 1 parameter.\n");
    }
    if (!is_valid_real(param[1], &hessian->dx))
    {
        print_error("displacement for hessian should be a number.\n");
    }
    if (hessian->dx <= 0)
    {
        print_error("displacement for hessian should be positive.\n");
    }
    printf("Displacement for hessian = %g A.\n", hessian->dx);
}


