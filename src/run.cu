#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




/*----------------------------------------------------------------------------80
Run simulation according to the inputs in the run.in file.
------------------------------------------------------------------------------*/




#include "gpumd.cuh"

#include "force.cuh"
#include "validate.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"




// run a number of steps for a given set of inputs
static void process_run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    integrate->initialize(atom);
    measure->initialize(input_dir, atom);

    clock_t time_begin = clock();
    for (int step = 0; step < atom->number_of_steps; ++step)
    {
        if (atom->neighbor.update) { atom->find_neighbor(0); }

        // set the current temperature;
        if (integrate->ensemble->type >= 1 && integrate->ensemble->type <= 20)
        {
            integrate->ensemble->temperature = atom->temperature1 
                + (atom->temperature2 - atom->temperature1)
                * real(step) / atom->number_of_steps;   
        }

        integrate->compute(atom, force, measure);
        measure->process(input_dir, atom, integrate, step);

        if (atom->number_of_steps >= 10)
        {
            if ((step + 1) % (atom->number_of_steps / 10) == 0)
            {
                printf("    %d steps completed.\n", step + 1);
            }
        }
    }

    print_line_1();
    clock_t time_finish = clock();
    real time_used = (time_finish - time_begin) / (real) CLOCKS_PER_SEC;
    printf("Time used for this run = %g s.\n", time_used);
    real run_speed = atom->N * (atom->number_of_steps / time_used);
    printf("Speed of this run = %g atom*step/second.\n", run_speed);
    print_line_2();

    measure->finalize(input_dir, atom, integrate);
    integrate->finalize();
}




#ifdef FORCE
static void print_initial_force(char* input_dir, Atom* atom)
{

    int m = sizeof(real) * atom->N;
    real *cpu_fx; MY_MALLOC(cpu_fx, real, atom->N);
    real *cpu_fy; MY_MALLOC(cpu_fy, real, atom->N);
    real *cpu_fz; MY_MALLOC(cpu_fz, real, atom->N);
    CHECK(hipMemcpy(cpu_fx, atom->fx, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_fy, atom->fy, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_fz, atom->fz, m, hipMemcpyDeviceToHost));
    char file_force[200];
    strcpy(file_force, input_dir);
    strcat(file_force, "/f.out");
    FILE *fid_force = my_fopen(file_force, "w");
    for (int n = 0; n < atom->N; n++)
    {
        fprintf(fid_force, "%20.10e%20.10e%20.10e\n", 
            cpu_fx[n], cpu_fy[n], cpu_fz[n]);
    }
    fflush(fid_force);
    fclose(fid_force);
    MY_FREE(cpu_fx);
    MY_FREE(cpu_fy);
    MY_FREE(cpu_fz);
}
#endif




// Read and process the inputs from the "run.in" file
void GPUMD::run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    char file_run[200];
    strcpy(file_run, input_dir);
    strcat(file_run, "/run.in");
    char *input = get_file_contents(file_run);
    char *input_ptr = input; // Keep the pointer in order to free later

    // Iterate the rows
    const int max_num_param = 10; // never use more than 9 parameters
    int num_param;
    char *param[max_num_param];

    initialize_run(atom, measure); // set some default values

    print_line_1();
    printf("Started executing the commands in run.in.\n");
    print_line_2();

    while (input_ptr)
    {
        // get one line from the input file
        input_ptr = row_find_param(input_ptr, param, &num_param);
        if (num_param == 0) { continue; } 

        // set default values
        int is_potential = 0;
        int is_velocity = 0;
        int is_run = 0;

        // parse a line of the input file 
        parse(param, num_param, atom, force, integrate, measure,
            &is_potential, &is_velocity, &is_run);

        // check for some special keywords
        if (is_potential)
        {
            force->initialize(input_dir, atom);
            force->compute(atom, measure);
#ifdef FORCE
            print_initial_force(input_dir, atom);
#endif
        }
        if (is_velocity) { atom->initialize_velocity(); }
        if (is_run)
        {
            process_run(input_dir, atom, force, integrate, measure);
            initialize_run(atom, measure); // change back to the default
        }
    }

    print_line_1();
    printf("Finished executing the commands in run.in.\n");
    print_line_2();

    MY_FREE(input); // Free the input file contents
}




