#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "atom.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128
#ifdef USE_DP
    #define HALF  0.5
#else
    #define HALF  0.5f
#endif




static __device__ void dev_apply_mic
(
    int pbc_x, int pbc_y, int pbc_z, real &x12, real &y12, real &z12, 
    real lx, real ly, real lz
)
{
    if      (pbc_x == 1 && x12 < - lx * HALF) {x12 += lx;}
    else if (pbc_x == 1 && x12 > + lx * HALF) {x12 -= lx;}
    if      (pbc_y == 1 && y12 < - ly * HALF) {y12 += ly;}
    else if (pbc_y == 1 && y12 > + ly * HALF) {y12 -= ly;}
    if      (pbc_z == 1 && z12 < - lz * HALF) {z12 += lz;}
    else if (pbc_z == 1 && z12 > + lz * HALF) {z12 -= lz;}
}




// a simple O(N^2) version of neighbor list construction
static __global__ void gpu_find_neighbor_ON2
(
    int pbc_x, int pbc_y, int pbc_z,
    int N, real cutoff_square, 
    real *box,
    int *NN, int *NL, real *x, real *y, real *z
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    int count = 0;
    if (n1 < N)
    {  
        real x1 = x[n1];   
        real y1 = y[n1];
        real z1 = z[n1];  
        for (int n2 = 0; n2 < N; ++n2)
        { 
            if (n2 == n1) { continue; }
            real x12  = x[n2] - x1;  
            real y12  = y[n2] - y1;
            real z12  = z[n2] - z1;

            dev_apply_mic
            (pbc_x, pbc_y, pbc_z, x12, y12, z12, box[0], box[1], box[2]);

            real distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < cutoff_square)
            {        
                NL[count * N + n1] = n2;
                ++count;
            }
        }
        NN[n1] = count;
    }
}




// a driver function
void Atom::find_neighbor_ON2(void)
{
    int grid_size = (N - 1) / BLOCK_SIZE + 1; 
    real rc = neighbor.rc;
    real rc2 = rc * rc; 
    real *box = box_length;

    // Find neighbours
    gpu_find_neighbor_ON2<<<grid_size, BLOCK_SIZE>>>
    (pbc_x, pbc_y, pbc_z, N, rc2, box, NN, NL, x, y, z);
    CUDA_CHECK_KERNEL
}



