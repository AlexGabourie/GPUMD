#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.cuh"
#include "mic.cuh"
#include "neighbor_ON2.cuh"



// a simple O(N^2) version of neighbor list construction
static __global__ void gpu_find_neighbor_ON2
(
    int pbc_x, int pbc_y, int pbc_z,
    int N, real cutoff_square, 
    real *box, 
    #ifdef TRICLINIC
    real *box_inv,
    #endif
    int *NN, int *NL, real *x, real *y, real *z
)
{
    //<<<(N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    int count = 0;
    if (n1 < N)
    {  
        real x1 = x[n1];   
        real y1 = y[n1];
        real z1 = z[n1];  
        for (int n2 = 0; n2 < N; ++n2)
        { 
            if (n2 == n1) { continue; }
            real x12  = x[n2] - x1;  
            real y12  = y[n2] - y1;
            real z12  = z[n2] - z1;
            
            #ifdef TRICLINIC
            apply_mic(pbc_x, pbc_y, pbc_z, box, box_inv, x12, y12, z12);
            #else
            dev_apply_mic
            (pbc_x, pbc_y, pbc_z, x12, y12, z12, box[0], box[1], box[2]);
            #endif

            real distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < cutoff_square)
            {        
                NL[count * N + n1] = n2;
                ++count;
            }
        }
        NN[n1] = count;
    }
}




// a driver function
void find_neighbor_ON2(Parameters *para, GPU_Data *gpu_data)
{                           
    int N = para->N;
    int grid_size = (N - 1) / BLOCK_SIZE + 1; 
    int pbc_x = para->pbc_x;
    int pbc_y = para->pbc_y;
    int pbc_z = para->pbc_z;
    real rc = para->neighbor.rc;
    real rc2 = rc * rc; 
    int *NN = gpu_data->NN;
    int *NL = gpu_data->NL;
    real *x = gpu_data->x;
    real *y = gpu_data->y;
    real *z = gpu_data->z;
    #if TRICLINIC
    real *box = gpu_data->box_matrix;
    real *box_inv = gpu_data->box_matrix_inv;
    #else
    real *box = gpu_data->box_length;
    #endif
    
    // Find neighbours
    #if TRICLINIC
    gpu_find_neighbor_ON2<<<grid_size, BLOCK_SIZE>>>
    (pbc_x, pbc_y, pbc_z, N, rc2, box, box_inv, NN, NL, x, y, z);
    #else
    gpu_find_neighbor_ON2<<<grid_size, BLOCK_SIZE>>>
    (pbc_x, pbc_y, pbc_z, N, rc2, box, NN, NL, x, y, z);
    #endif

    #ifdef DEBUG
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
    #endif
}



