#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Use finite difference to validate the analytical force calculations.
------------------------------------------------------------------------------*/


#include "validate.cuh"
#include "force.cuh"
#include "atom.cuh"
#include "gpu_vector.cuh"
#include "error.cuh"

#define BLOCK_SIZE 128

// This choice gives optimal accuracy for finite-difference calculations
#define DX1 1.0e-7
#define DX2 2.0e-7


// move one atom left or right
static __global__ void shift_atom
(
    const int d,
    const int n,
    const int direction,
    const double *x0,
    const double *y0,
    const double *z0,
    double *x,
    double *y,
    double *z
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (n1 == n)
    {
        if (d == 0)
        {
            if (direction == 1)
            {
                x[n] = x0[n] - DX1;
            }
            else
            {
                x[n] = x0[n] + DX1;
            }
        }
        else if (d == 1)
        {
            if (direction == 1)
            {
                y[n] = y0[n] - DX1;
            }
            else
            {
                y[n] = y0[n] + DX1;
            }
        }
        else
        {
            if (direction == 1)
            {
                z[n] = z0[n] - DX1;
            }
            else
            {
                z[n] = z0[n] + DX1;
            }
        } 
    }
}


// get the total potential form the per-atom potentials
static __global__ void sum_potential
(
    const int N,
    const int m,
    const double *p,
    double *p_sum
)
{
    int tid = threadIdx.x;
    int number_of_patches = (N - 1) / 1024 + 1; 
    
    __shared__ double s_sum[1024];
    s_sum[tid] = 0;
    
    for (int patch = 0; patch < number_of_patches; ++patch)
    { 
        int n = tid + patch * 1024;
        if (n < N)
        {        
            s_sum[tid] += p[n];
        }
    }
    
    __syncthreads();
    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset) { s_sum[tid] += s_sum[tid + offset]; }
        __syncthreads();
    } 

    if (tid ==  0) 
    {
        p_sum[m] = s_sum[0]; 
    }
}


// get the forces from the potential energies using finite difference
static __global__ void find_force_from_potential
(
    const int N,
    const double *p1,
    const double *p2,
    double *fx,
    double *fy,
    double *fz
)
{
    int n1 = blockIdx.x * blockDim.x + threadIdx.x;
    int m;
    if (n1 < N)
    {
        m = n1; fx[n1] = (p1[m] - p2[m]) / DX2;
        m += N; fy[n1] = (p1[m] - p2[m]) / DX2;
        m += N; fz[n1] = (p1[m] - p2[m]) / DX2; 
    }
}


void validate_force
(
    Force *force,
    Atom *atom
)
{
    int N = atom->N;
    int grid_size = (N - 1) / BLOCK_SIZE + 1; 
    std::vector<double> cpu_force(N * 3);

    // first calculate the forces directly:
    force->compute(atom);

    // make a copy of the positions
    GPU_Vector<double> r0(N * 3);
    r0.copy_from_device(atom->position_per_atom.data());

    // get the potentials
    GPU_Vector<double> p1(N * 3), p2(N * 3);
    for (int d = 0; d < 3; ++d)
    {
        for (int n = 0; n < N; ++n)
        {
            int m = d * N + n;

            // shift one atom to the left by a small amount
            shift_atom<<<grid_size, BLOCK_SIZE>>>
            (
                d,
                n,
                1,
                r0.data(),
                r0.data() + N,
                r0.data() + N * 2,
                atom->position_per_atom.data(),
                atom->position_per_atom.data() + N,
                atom->position_per_atom.data() + N * 2
            );
            CUDA_CHECK_KERNEL

            // get the potential energy
            force->compute(atom);

            // sum up the potential energy
            sum_potential<<<1, 1024>>>
            (
                N,
                m,
                atom->potential_per_atom.data(),
                p1.data()
            );
            CUDA_CHECK_KERNEL

            // shift one atom to the right by a small amount
            shift_atom<<<grid_size, BLOCK_SIZE>>>
            (
                d,
                n,
                2,
                r0.data(),
                r0.data() + N,
                r0.data() + N * 2,
                atom->position_per_atom.data(),
                atom->position_per_atom.data() + N,
                atom->position_per_atom.data() + N * 2
            );
            CUDA_CHECK_KERNEL

            // get the potential energy
            force->compute(atom);

            // sum up the potential energy
            sum_potential<<<1, 1024>>>
            (
                N,
                m,
                atom->potential_per_atom.data(),
                p2.data()
            );
            CUDA_CHECK_KERNEL
        }
    }

    // copy the positions back (as if nothing happens)
    r0.copy_to_device(atom->position_per_atom.data());

    // get the forces from the potential energies using finite difference
    GPU_Vector<double> force_compare(N * 3);
    find_force_from_potential<<<grid_size, BLOCK_SIZE>>>
    (
        N,
        p1.data(),
        p2.data(),
        force_compare.data(),
        force_compare.data() + N,
        force_compare.data() + N * 2
    );
    CUDA_CHECK_KERNEL

    // open file
    FILE *fid = my_fopen("f_compare.out", "w");
    
    // output the forces from direct calculations
    atom->force_per_atom.copy_to_host(cpu_force.data());
    for (int n = 0; n < N; n++)
    {
        fprintf
        (
            fid, "%25.15e%25.15e%25.15e\n",
            cpu_force[n], cpu_force[n + N], cpu_force[n + N * 2]
        );
    }
 
    // output the forces from finite difference
    force_compare.copy_to_host(cpu_force.data());
    for (int n = 0; n < N; n++)
    {
        fprintf
        (
            fid, "%25.15e%25.15e%25.15e\n",
            cpu_force[n], cpu_force[n + N], cpu_force[n + N * 2]
        );
    }
    
    // close file
    fflush(fid);
    fclose(fid); 
}


