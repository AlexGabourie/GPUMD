#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, Mikko Ervasti, and Ari Harju
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "gpumd.cuh"

#include "force.cuh"
#include "validate.cuh"
#include "integrate.cuh"
#include "ensemble.cuh"
#include "measure.cuh"
#include "atom.cuh"
#include "error.cuh"

#include <ctype.h>




GPUMD::GPUMD(char *input_dir)
{
    Atom        atom(input_dir);
    Force       force;
    Integrate   integrate;
    Measure     measure(input_dir);

    check_velocity_and_potential
    (input_dir, &atom, &force, &integrate, &measure);

    run(input_dir, &atom, &force, &integrate, &measure);
}




GPUMD::~GPUMD(void)
{
    // nothing
}




// run a number of steps for a given set of inputs
static void process_run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    integrate->initialize(atom);
    measure->initialize(input_dir, atom);

    clock_t time_begin = clock();
    for (int step = 0; step < atom->number_of_steps; ++step)
    {
        if (atom->neighbor.update) { atom->find_neighbor(0); }

        // set the current temperature;
        if (integrate->ensemble->type >= 1 && integrate->ensemble->type <= 20)
        {
            integrate->ensemble->temperature = atom->temperature1 
                + (atom->temperature2 - atom->temperature1)
                * real(step) / atom->number_of_steps;   
        }

        integrate->compute(atom, force, measure);
        measure->compute(input_dir, atom, integrate, step);

        if (atom->number_of_steps >= 10)
        {
            if ((step + 1) % (atom->number_of_steps / 10) == 0)
            {
                printf("    %d steps completed.\n", step + 1);
            }
        }
    }

    print_line_1();
    clock_t time_finish = clock();
    real time_used = (time_finish - time_begin) / (real) CLOCKS_PER_SEC;
    printf("Time used for this run = %g s.\n", time_used);
    real run_speed = atom->N * (atom->number_of_steps / time_used);
    printf("Speed of this run = %g atom*step/second.\n", run_speed);
    print_line_2();

    measure->finalize(input_dir, atom, integrate);
    integrate->finalize();
}




// set some default values after each run
static void initialize_run(Atom* atom, Measure* measure)
{
    // modify
    atom->neighbor.update = 0;
    atom->fixed_group     = -1; // no group has an index of -1

    // measure
    measure->heat.sample    = 0;
    measure->shc.compute    = 0;
    measure->vac.compute    = 0;
    measure->hac.compute    = 0;
    measure->hnemd.compute  = 0;
    measure->dump_thermo    = 0;
    measure->dump_position  = 0;
    measure->dump_velocity  = 0;
    measure->dump_force     = 0;
    measure->dump_potential = 0;
    measure->dump_virial    = 0;
    measure->dump_heat      = 0;
}




// Read the input file to memory
static char *get_file_contents (char *filename)
{
    char *contents;
    int contents_size;
    FILE *in = my_fopen(filename, "r");

    // Find file size
    fseek(in, 0, SEEK_END);
    contents_size = ftell(in);
    rewind(in);

    MY_MALLOC(contents, char, contents_size + 1);
    int size_read_in = fread(contents, sizeof(char), contents_size, in);
    if (size_read_in != contents_size)
    {
        print_error ("File size mismatch.");
    }

    fclose(in);
    contents[contents_size] = '\0'; // Assures proper null termination

    return contents;
}




// Parse a single row
static char *row_find_param (char *s, char *param[], int *num_param)
{
    *num_param = 0;
    int start_new_word = 1, comment_found = 0;
    if (s == NULL) return NULL;

    while(*s)
    {
        if(*s == '\n')
        {
            *s = '\0';
            return s + sizeof(char);
        }
        else if (comment_found)
        {
            // Do nothing
        }
        else if (*s == '#')
        {
            *s = '\0';
            comment_found = 1;
        }
        else if(isspace(*s))
        {
            *s = '\0';
            start_new_word = 1;
        }
        else if (start_new_word)
        {
            param[*num_param] = s;
            ++(*num_param);
            start_new_word = 0;
        }
        ++s;
    }
    return NULL;
}




#ifdef FORCE
static void print_initial_force(char* input_dir, Atom* atom)
{

    int m = sizeof(real) * atom->N;
    real *cpu_fx; MY_MALLOC(cpu_fx, real, atom->N);
    real *cpu_fy; MY_MALLOC(cpu_fy, real, atom->N);
    real *cpu_fz; MY_MALLOC(cpu_fz, real, atom->N);
    CHECK(hipMemcpy(cpu_fx, atom->fx, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_fy, atom->fy, m, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(cpu_fz, atom->fz, m, hipMemcpyDeviceToHost));
    char file_force[FILE_NAME_LENGTH];
    strcpy(file_force, input_dir);
    strcat(file_force, "/f.out");
    FILE *fid_force = my_fopen(file_force, "w");
    for (int n = 0; n < atom->N; n++)
    {
        fprintf(fid_force, "%20.10e%20.10e%20.10e\n", 
            cpu_fx[n], cpu_fy[n], cpu_fz[n]);
    }
    fflush(fid_force);
    fclose(fid_force);
    MY_FREE(cpu_fx);
    MY_FREE(cpu_fy);
    MY_FREE(cpu_fz);
}
#endif




static void print_velocity_and_potential_error_1
(int number_of_times_potential, int number_of_times_velocity)
{
    if (0 == number_of_times_potential)
    {
        print_error("No 'potential(s)' keyword before run.\n");
    }
    else if (1 < number_of_times_potential)
    {
        print_error("Multiple 'potential(s)' keywords before run.\n");
    }

    if (0 == number_of_times_velocity)
    {
        print_error("No 'velocity' keyword before run.\n");
    }
    else if (1 < number_of_times_velocity)
    {
        print_error("Multiple 'velocity' keywords before run.\n");
    }
}




static void print_velocity_and_potential_error_2
(int number_of_times_potential, int number_of_times_velocity)
{
    if (1 < number_of_times_potential)
    {
        print_error("Multiple 'potential(s)' keywords.\n");
    }
    if (1 < number_of_times_velocity)
    {
        print_error("Multiple 'velocity' keywords.\n");
    }
}




void GPUMD::check_velocity_and_potential
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    char file_run[FILE_NAME_LENGTH];
    strcpy(file_run, input_dir);
    strcat(file_run, "/run.in");
    char *input = get_file_contents(file_run);
    char *input_ptr = input; // Keep the pointer in order to free later

    const int max_num_param = 10; // never use more than 9 parameters
    int num_param;
    char *param[max_num_param];

    initialize_run(atom, measure); // set some default values

    print_line_1();
    printf("Started checking the inputs in run.in.\n");
    print_line_2();

    while (input_ptr)
    {
        // get one line from the input file
        input_ptr = row_find_param(input_ptr, param, &num_param);
        if (num_param == 0) { continue; } 

        // set default values
        int is_potential = 0;
        int is_velocity = 0;
        int is_run = 0;

        // parse a line of the input file 
        parse(param, num_param, atom, force, integrate, measure,
            &is_potential, &is_velocity, &is_run);

        // check for some special keywords
        if (is_potential) { number_of_times_potential++; }
        if (is_velocity) { number_of_times_velocity++; }
        if (is_run)
        {
            print_velocity_and_potential_error_1
            (number_of_times_potential, number_of_times_velocity);
            initialize_run(atom, measure); // change back to the default
        }
    }
    print_velocity_and_potential_error_2
    (number_of_times_potential, number_of_times_velocity);

    print_line_1();
    printf("Finished checking the inputs in run.in.\n");
    print_line_2();

    MY_FREE(input); // Free the input file contents
}




// Read and process the inputs from the "run.in" file
void GPUMD::run
(
    char *input_dir, Atom *atom, Force *force, Integrate *integrate,
    Measure *measure
)
{
    char file_run[FILE_NAME_LENGTH];
    strcpy(file_run, input_dir);
    strcat(file_run, "/run.in");
    char *input = get_file_contents(file_run);
    char *input_ptr = input; // Keep the pointer in order to free later

    // Iterate the rows
    const int max_num_param = 10; // never use more than 9 parameters
    int num_param;
    char *param[max_num_param];

    initialize_run(atom, measure); // set some default values

    print_line_1();
    printf("Started executing the commands in run.in.\n");
    print_line_2();

    while (input_ptr)
    {
        // get one line from the input file
        input_ptr = row_find_param(input_ptr, param, &num_param);
        if (num_param == 0) { continue; } 

        // set default values
        int is_potential = 0;
        int is_velocity = 0;
        int is_run = 0;

        // parse a line of the input file 
        parse(param, num_param, atom, force, integrate, measure,
            &is_potential, &is_velocity, &is_run);

        // check for some special keywords
        if (is_potential)
        {
            force->initialize(input_dir, atom);
            force->compute(atom, measure);
#ifdef FORCE
            print_initial_force(input_dir, atom);
#endif
        }
        if (is_velocity) { atom->initialize_velocity(); }
        if (is_run)
        {
            process_run(input_dir, atom, force, integrate, measure);
            initialize_run(atom, measure); // change back to the default
        }
    }

    print_line_1();
    printf("Finished executing the commands in run.in.\n");
    print_line_2();

    MY_FREE(input); // Free the input file contents
}




